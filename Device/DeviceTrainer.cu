#include "hip/hip_runtime.h"
/*
 * DeviceTrainer.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include "DeviceTrainer.h"
#include "Splitter/Initiator.h"
#include "Splitter/DeviceSplitter.h"
#include "Memory/gbdtGPUMemManager.h"
#include "Bagging/BagManager.h"
#include "FindSplit/IndexComputer.h"
#include "CSR/CsrCompressor.h"
#include "../SharedUtility/CudaMacro.h"

/**
 * @brief: initialise tree
 */
void DeviceTrainer::InitTree(RegTree &tree, void *pStream, int bagId)
{
	total_find_fea_t = 0;
	total_split_t = 0;
	total_prune_t = 0;

	//#### initial root node in GPU has been moved to grow tree.

	//all instances belong to the root node
	BagManager bagManager;
	hipMemsetAsync(bagManager.m_pInsIdToNodeIdEachBag + bagId * bagManager.m_numIns, 0, sizeof(int) * bagManager.m_numIns, (*(hipStream_t*)pStream));
}

/**
 * @brief: release memory used by trees
 */
void DeviceTrainer::ReleaseTree(vector<RegTree> &v_Tree)
{
	int nNumofTree = v_Tree.size();
	for(int i = 0; i < nNumofTree; i++)
	{
		int nNumofNodes = v_Tree[i].nodes.size();
		delete[] v_Tree[i].nodes[0];
	}
}

/**
 * @brief: grow the tree by splitting nodes to the full extend
 */
void DeviceTrainer::GrowTree(RegTree &tree, void *pStream, int bagId)
{
	int nNumofSplittableNode = 0;

	clock_t init_start = clock();
	//copy the root node to GPU
	BagManager bagManager;
	GBDTGPUMemManager manager;
	InitRootNode<<<1, 1, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pNodeTreeOnTrainingEachBag + bagId * bagManager.m_maxNumNode,
									bagManager.m_pCurNumofNodeTreeOnTrainingEachBag_d + bagId, bagManager.m_numIns);

	manager.MemcpyDeviceToDeviceAsync(bagManager.m_pNodeTreeOnTrainingEachBag + bagId * bagManager.m_maxNumNode,
								  	  bagManager.m_pSplittableNodeEachBag + bagId * bagManager.m_maxNumSplittable,
								  	  sizeof(TreeNode), pStream);
	clock_t init_end = clock();
	total_init_t += (init_end - init_start);

	nNumofSplittableNode++;
	//manager.m_curNumofSplitable = 1;
	bagManager.m_curNumofSplitableEachBag_h[bagId] = 1;

	vector<TreeNode*> splittableNode;

	//split node(s)
	int nCurDepth = 0;
	DeviceSplitter *pDSpliter = (DeviceSplitter*)splitter;
#ifdef _DEBUG
	pDSpliter->total_scan_t = 0;
	pDSpliter->total_com_gain_t = 0;
	pDSpliter->total_fill_gd_t = 0;
	pDSpliter->total_search_t = 0;
	pDSpliter->total_fix_gain_t = 0;
	pDSpliter->total_com_idx_t = 0;
	pDSpliter->total_csr_len_t = 0;
	pDSpliter->total_weight_t = 0;
	pDSpliter->total_create_node_t = 0;
	pDSpliter->total_unique_id_t = 0;
	pDSpliter->total_ins2node_t = 0;
	pDSpliter->total_ins2default_t = 0;
	pDSpliter->total_update_new_splittable_t = 0;
#endif
	while(bagManager.m_curNumofSplitableEachBag_h[bagId] > 0 && nCurDepth <= m_nMaxDepth)
	{
		pDSpliter->m_nCurDept = nCurDepth;
//		cout << "splitting " << nCurDepth << " level..." << endl;

		vector<SplitPoint> vBest;
		vector<nodeStat> rchildStat, lchildStat;
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t begin_find_fea = clock();

		if(nCurDepth < m_nMaxDepth){//don't need to find split for the last level
			if(CsrCompressor::bUseCsr == true)
				pDSpliter->FeaFinderAllNode2(pStream, bagId);
			else
				pDSpliter->FeaFinderAllNode(pStream, bagId);
		}

		clock_t end_find_fea = clock();
		total_find_fea_t += (double(end_find_fea - begin_find_fea) / CLOCKS_PER_SEC);

		//split all the splittable nodes
		clock_t start_split_t = clock();
		bool bLastLevel = false;
		if(nCurDepth == m_nMaxDepth)
			bLastLevel = true;

		int curNumofNode = -1;//this is fine even though bagging is used, as each bag is handled by a host thread.
		manager.MemcpyDeviceToHostAsync(bagManager.m_pCurNumofNodeTreeOnTrainingEachBag_d + bagId, &curNumofNode, sizeof(int), pStream);
		PROCESS_ERROR(curNumofNode > 0);
		pDSpliter->SplitAll(curNumofNode, bLastLevel, pStream, bagId);

		manager.MemcpyDeviceToHostAsync(bagManager.m_pNumofNewNodeTreeOnTrainingEachBag + bagId, bagManager.m_curNumofSplitableEachBag_h + bagId,
								   sizeof(int), pStream);
		clock_t end_split_t = clock();
		total_split_t += (double(end_split_t - start_split_t) / CLOCKS_PER_SEC);
		nCurDepth++;
	}

	//copy tree nodes back to host
	clock_t begin_prune = clock();
	int numofNode = 0;
	manager.MemcpyDeviceToHostAsync(bagManager.m_pCurNumofNodeTreeOnTrainingEachBag_d + bagId, &numofNode,
									sizeof(int), pStream);
	cout << "number of nodes " << numofNode << endl;
	TreeNode *pAllNode = new TreeNode[numofNode];
	manager.MemcpyDeviceToHostAsync(bagManager.m_pNodeTreeOnTrainingEachBag + bagId * bagManager.m_maxNumNode,
									pAllNode, sizeof(TreeNode) * numofNode, pStream);
	TreeNode **ypAllNode = new TreeNode*[numofNode];
	PROCESS_ERROR(tree.nodes.size() == 0);
	int nDefault2right = 0;
	for(int n = 0; n < numofNode; n++){
		ypAllNode[n] = &pAllNode[n];
		tree.nodes.push_back(&pAllNode[n]);//for getting features of trees
		if(pAllNode[n].m_bDefault2Right == true)
			nDefault2right++;
	}
	printf("default to right %d\n", nDefault2right);
	pruner.pruneLeaf(ypAllNode, numofNode);
	delete []ypAllNode;
	//########### can be improved by storing only the valid nodes afterwards

	StoreFinalTree(pAllNode, numofNode, pStream, bagId);

#ifdef _DEBUG
	clock_t end_prune = clock();
	total_prune_t += (double(end_prune - begin_prune) / CLOCKS_PER_SEC);

	double total_scan = pDSpliter->total_scan_t;
	double total_gain = pDSpliter->total_com_gain_t;
	double total_fill = pDSpliter->total_fill_gd_t;
	double total_search = pDSpliter->total_search_t;
	double total_fix = pDSpliter->total_fix_gain_t;
	double total_com_idx = pDSpliter->total_com_idx_t;
	cout << "com idx " << total_com_idx/CLOCKS_PER_SEC
		 << "; com csr len " << pDSpliter->total_csr_len_t/CLOCKS_PER_SEC
		 << "; scan takes " << total_scan/CLOCKS_PER_SEC << "; comp gain takes " << total_gain/CLOCKS_PER_SEC
		 << "; fix gain takes " << total_fix / CLOCKS_PER_SEC
		 << "; fill gd takes " << total_fill/CLOCKS_PER_SEC << "; search takes " << total_search/CLOCKS_PER_SEC << endl;

	//split
	double total_weight = pDSpliter->total_weight_t;
	double total_create_node = pDSpliter->total_create_node_t;
	double total_unique_id = pDSpliter->total_unique_id_t;
	double total_ins2node = pDSpliter->total_ins2node_t;
	double total_ins2default = pDSpliter->total_ins2default_t;
	double total_update_new_sp = pDSpliter->total_update_new_splittable_t;
	cout << "comp weight " << total_weight/CLOCKS_PER_SEC
		 << "; create node " << total_create_node/CLOCKS_PER_SEC
		 << "; unique id " << total_unique_id/CLOCKS_PER_SEC
		 << "; ins2node " << total_ins2node/CLOCKS_PER_SEC
		 << "; ins2default " << total_ins2default/CLOCKS_PER_SEC
		 << "; update new splittable " << total_update_new_sp/CLOCKS_PER_SEC << endl;
#endif
}

/**
 * @brief: store the tree learned at this round to GPU memory
 */
void DeviceTrainer::StoreFinalTree(TreeNode *pAllNode, int numofNode, void *pStream, int bagId)
{
	BagManager bagManager;
	GBDTGPUMemManager manager;
	//copy the final tree to GPU memory
	manager.MemcpyHostToDeviceAsync(pAllNode, bagManager.m_pNodeTreeOnTrainingEachBag + bagId * bagManager.m_maxNumNode,
									sizeof(TreeNode) * numofNode, pStream);

	//copy the final tree for ensembling
	int numofTreeLearnt = manager.m_pNumofTreeLearntEachBag_h[bagId];
	int curLearningTreeId = numofTreeLearnt;
	manager.MemcpyHostToDeviceAsync(&numofNode, manager.m_pNumofNodeEachTreeEachBag + bagId * bagManager.m_numTreeEachBag + curLearningTreeId,
									sizeof(int), pStream);
	int numofNodePreviousTree = 0;
	int previousTreeStartPosInBag = bagId * bagManager.m_numTreeEachBag * bagManager.m_maxNumNode;
	if(numofTreeLearnt > 0)
	{
		int lastLearntTreeId = numofTreeLearnt - 1;
		manager.MemcpyDeviceToHostAsync(manager.m_pNumofNodeEachTreeEachBag + bagId * bagManager.m_numTreeEachBag + lastLearntTreeId,
										&numofNodePreviousTree, sizeof(int), pStream);
		manager.MemcpyDeviceToHostAsync(manager.m_pStartPosOfEachTreeEachBag + bagId * bagManager.m_numTreeEachBag + lastLearntTreeId,
										&previousTreeStartPosInBag, sizeof(int), pStream);
	}
	int treeStartPos = previousTreeStartPosInBag + numofNodePreviousTree;
	manager.MemcpyHostToDeviceAsync(&treeStartPos, manager.m_pStartPosOfEachTreeEachBag + bagId * bagManager.m_numTreeEachBag + curLearningTreeId,
									sizeof(int), pStream);
	manager.MemcpyDeviceToDeviceAsync(bagManager.m_pNodeTreeOnTrainingEachBag + bagId * bagManager.m_maxNumNode,
										manager.m_pAllTreeEachBag + treeStartPos, sizeof(TreeNode) * numofNode, pStream);
	manager.m_pNumofTreeLearntEachBag_h[bagId]++;
}
