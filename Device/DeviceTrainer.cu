#include "hip/hip_runtime.h"
/*
 * DeviceTrainer.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include "DeviceTrainer.h"
#include "Splitter/Initiator.h"
#include "Splitter/DeviceSplitter.h"
#include "Memory/gbdtGPUMemManager.h"
#include "Memory/SplitNodeMemManager.h"
#include "Memory/dtMemManager.h"

/**
 * @brief: initialise tree
 */
void DeviceTrainer::InitTree(RegTree &tree)
{
	#ifdef _COMPARE_HOST
	TreeNode *root = new TreeNode[1];
	m_nNumofNode = 1;
	root->nodeId = 0;
	root->level = 0;

	tree.nodes.push_back(root);

	//all instances are under node 0
	splitter->m_nodeIds.clear();
	for(int i = 0; i < m_vvInsSparse.size(); i++)
	{
		splitter->m_nodeIds.push_back(0);
	}
	#endif

	total_find_fea_t = 0;
	total_split_t = 0;
	total_prune_t = 0;

	//#### initial root node in GPU has been moved to grow tree.

	//all instances belong to the root node
	GBDTGPUMemManager manager;
	hipMemset(manager.m_pInsIdToNodeId, 0, sizeof(int) * manager.m_numofIns);
}

/**
 * @brief: release memory used by trees
 */
void DeviceTrainer::ReleaseTree(vector<RegTree> &v_Tree)
{
	int nNumofTree = v_Tree.size();
	for(int i = 0; i < nNumofTree; i++)
	{
		int nNumofNodes = v_Tree[i].nodes.size();
		delete[] v_Tree[i].nodes[0];
	}
}

/**
 * @brief: grow the tree by splitting nodes to the full extend
 */
void DeviceTrainer::GrowTree(RegTree &tree)
{
	int nNumofSplittableNode = 0;

	//copy the root node to GPU
	GBDTGPUMemManager manager;
	SNGPUManager snManager;
	snManager.resetForNextTree();//reset tree nodes to default value

	InitRootNode<<<1, 1>>>(snManager.m_pTreeNode, snManager.m_pCurNumofNode);

	manager.MemcpyDeviceToDevice(snManager.m_pTreeNode, manager.m_pSplittableNode, sizeof(TreeNode));

	nNumofSplittableNode++;
	manager.m_curNumofSplitable = 1;

	vector<TreeNode*> splittableNode;

	//split node(s)
	int nCurDepth = 0;
	while(manager.m_curNumofSplitable > 0 && nCurDepth <= m_nMaxDepth)
	{
		splitter->m_nCurDept = nCurDepth;
//		cout << "splitting " << nCurDepth << " level..." << endl;

		vector<SplitPoint> vBest;
		vector<nodeStat> rchildStat, lchildStat;
		clock_t begin_find_fea = clock();

		splitter->FeaFinderAllNode(vBest, rchildStat, lchildStat);

		clock_t end_find_fea = clock();
		total_find_fea_t += (double(end_find_fea - begin_find_fea) / CLOCKS_PER_SEC);

		//split all the splittable nodes
		clock_t start_split_t = clock();
		bool bLastLevel = false;
		if(nCurDepth == m_nMaxDepth)
			bLastLevel = true;

		int curNumofNode = -1;
		manager.MemcpyDeviceToHost(snManager.m_pCurNumofNode, &curNumofNode, sizeof(int));
		PROCESS_ERROR(curNumofNode > 0);
		splitter->SplitAll(splittableNode, vBest, tree, curNumofNode, rchildStat, lchildStat, bLastLevel);
		clock_t end_split_t = clock();
		total_split_t += (double(end_split_t - start_split_t) / CLOCKS_PER_SEC);

		manager.MemcpyDeviceToHost(snManager.m_pNumofNewNode, &manager.m_curNumofSplitable, sizeof(int));
//		cout << "number of new/splittable nodes is " << manager.m_curNumofSplitable << endl;

		nCurDepth++;
	}

	//copy tree nodes back to host
	clock_t begin_prune = clock();
	int numofNode = 0;
	manager.MemcpyDeviceToHost(snManager.m_pCurNumofNode, &numofNode, sizeof(int));
	cout << "number of nodes " << numofNode << endl;
	TreeNode *pAllNode = new TreeNode[numofNode];
	manager.MemcpyDeviceToHost(snManager.m_pTreeNode, pAllNode, sizeof(TreeNode) * numofNode);
	TreeNode **ypAllNode = new TreeNode*[numofNode];
	for(int n = 0; n < numofNode; n++)
	{
		ypAllNode[n] = &pAllNode[n];
		tree.nodes.push_back(&pAllNode[n]);//for getting features of trees
	}
	pruner.pruneLeaf(ypAllNode, numofNode);
	delete []ypAllNode;
	//########### can be improved by storing only the valid nodes afterwards

	StoreFinalTree(pAllNode, numofNode);

	clock_t end_prune = clock();
	total_prune_t += (double(end_prune - begin_prune) / CLOCKS_PER_SEC);
}

/**
 * @brief: store the tree learned at this round to GPU memory
 */
void DeviceTrainer::StoreFinalTree(TreeNode *pAllNode, int numofNode)
{
	GBDTGPUMemManager manager;
	SNGPUManager snManager;
	//copy the final tree to GPU memory
	manager.MemcpyHostToDevice(pAllNode, snManager.m_pTreeNode, sizeof(TreeNode) * numofNode);

	//copy the final tree for ensembling
	DTGPUMemManager treeManager;
	int numofTreeLearnt = treeManager.m_numofTreeLearnt;
	int curLearningTreeId = numofTreeLearnt;
	manager.MemcpyHostToDevice(&numofNode, treeManager.m_pNumofNodeEachTree + curLearningTreeId, sizeof(int));
	int numofNodePreviousTree = 0;
	int previousStartPos = 0;
	if(numofTreeLearnt > 0)
	{
		int lastLearntTreeId = numofTreeLearnt - 1;
		manager.MemcpyDeviceToHost(treeManager.m_pNumofNodeEachTree + lastLearntTreeId, &numofNodePreviousTree, sizeof(int));
		manager.MemcpyDeviceToHost(treeManager.m_pStartPosOfEachTree + lastLearntTreeId, &previousStartPos, sizeof(int));
	}
	int treeStartPos = previousStartPos + numofNodePreviousTree;
	manager.MemcpyHostToDevice(&treeStartPos, treeManager.m_pStartPosOfEachTree + curLearningTreeId, sizeof(int));
	manager.MemcpyDeviceToDevice(snManager.m_pTreeNode, treeManager.m_pAllTree + treeStartPos, sizeof(TreeNode) * numofNode);
	treeManager.m_numofTreeLearnt++;
}
