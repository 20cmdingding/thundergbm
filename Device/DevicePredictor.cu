/*
 * DevicePrediction.cu
 *
 *  Created on: 23 Jun 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <stdio.h>
#include "Hashing.h"
#include "KernelConf.h"
#include "DevicePredictor.h"
#include "DevicePredictorHelper.h"
#include "Memory/gbdtGPUMemManager.h"
#include "Memory/dtMemManager.h"
#include "../DeviceHost/DefineConst.h"
#include "../DeviceHost/TreeNode.h"
#include "../DeviceHost/SparsePred/DenseInstance.h"
#include "../DeviceHost/MyAssert.h"


/**
 * @brief: prediction function for sparse instances
 */
void DevicePredictor::PredictSparseIns(vector<vector<KeyValue> > &v_vInstance, vector<RegTree> &vTree, vector<double> &v_fPredValue)
{
	GBDTGPUMemManager manager;
	DTGPUMemManager treeManager;
	DenseInsConverter denseInsConverter(vTree);
	int numofUsedFea = denseInsConverter.usedFeaSet.size();

	if(manager.m_maxUsedFeaInTrees < numofUsedFea)
	{
		cout << "numofUsedFea=" << numofUsedFea << " v.s. maxUsedFeaInTrees " << manager.m_maxUsedFeaInTrees << endl;
		exit(0);
	}

	//build the hash table for feature id and position id
	int *pHashUsedFea = NULL;
	int *pSortedUsedFea = NULL;
	GetUsedFeature(denseInsConverter.usedFeaSet, pHashUsedFea, pSortedUsedFea);

	//for each tree
	int nNumofIns = v_vInstance.size();
	int nNumofTree = treeManager.m_numofTreeLearnt;
	PROCESS_ERROR(treeManager.m_numofTree == treeManager.m_numofTreeLearnt);
	PROCESS_ERROR(nNumofTree > 0);

	//start prediction
	checkCudaErrors(hipMemset(manager.m_pTargetValue, 0, sizeof(float_point) * nNumofIns));

	long long startPos = 0;
	int startInsId = 0;
	long long *pInsStartPos = manager.m_pInsStartPos + startInsId;
	manager.MemcpyDeviceToHost(pInsStartPos, &startPos, sizeof(long long));
//			cout << "start pos ins" << insId << "=" << startPos << endl;
	float_point *pDevInsValue = manager.m_pdDInsValue + startPos;
	int *pDevFeaId = manager.m_pDFeaId + startPos;
	int *pNumofFea = manager.m_pDNumofFea + startInsId;
	int numofInsToFill = nNumofIns;
	KernelConf conf;
	int threadPerBlock;
	dim3 dimNumofBlock;
	conf.ConfKernel(numofInsToFill, threadPerBlock, dimNumofBlock);

	FillMultiDense<<<dimNumofBlock, threadPerBlock>>>(
										  pDevInsValue, pInsStartPos, pDevFeaId, pNumofFea, manager.m_pdDenseIns,
										  manager.m_pSortedUsedFeaId, manager.m_pHashFeaIdToDenseInsPos,
										  numofUsedFea, startInsId, numofInsToFill);

#if testing
		if(hipGetLastError() != hipSuccess)
		{
			cout << "error in FillMultiDense" << endl;
			exit(0);
		}
#endif


//		FillDenseIns(i, numofUsedFea);
		//prediction using the last tree
		for(int t = 0; t < nNumofTree; t++)
		{
			int numofNodeOfTheTree = 0;
			TreeNode *pTree = NULL;

			int treeId = t;
			GetTreeInfo(pTree, numofNodeOfTheTree, treeId);
			PROCESS_ERROR(pTree != NULL);
			PredMultiTarget<<<dimNumofBlock, threadPerBlock>>>(
														manager.m_pTargetValue, numofInsToFill, pTree,
														manager.m_pdDenseIns, numofUsedFea,
														manager.m_pHashFeaIdToDenseInsPos, treeManager.m_maxTreeDepth);
			hipDeviceSynchronize();
		}

	for(int i = 0; i < nNumofIns; i++)
	{
		float_point fTarget = 0;
		manager.MemcpyDeviceToHost(manager.m_pTargetValue + i, &fTarget, sizeof(float_point));

		v_fPredValue.push_back(fTarget);
	}
}

/**
 * @brief: get the feature value.
 */
void DevicePredictor::GetUsedFeature(vector<int> &v_usedFeaSortedId, int *&pHashUsedFea, int *&pSortedUsedFea)
{
	int numofUsedFea = v_usedFeaSortedId.size();
	if(numofUsedFea == 0)
		return;

	pHashUsedFea = new int[numofUsedFea];
	memset(pHashUsedFea, -1, sizeof(int) * numofUsedFea);
	for(int uf = 0; uf < numofUsedFea; uf++)
	{
		bool bIsNewHashValue = false;
		int hashValue = Hashing::HostAssignHashValue(pHashUsedFea, v_usedFeaSortedId[uf], numofUsedFea, bIsNewHashValue);
//			cout << "hash value of " << denseInsConverter.usedFeaSet[uf] << " is " << hashValue << endl;
	}

	pSortedUsedFea = new int[numofUsedFea];
	for(int uf = 0; uf < numofUsedFea; uf++)
		pSortedUsedFea[uf] = v_usedFeaSortedId[uf];

	//copy hash map to gpu memory
	GBDTGPUMemManager manager;
	checkCudaErrors(hipMemset(manager.m_pHashFeaIdToDenseInsPos, -1, sizeof(int) * manager.m_maxUsedFeaInTrees));
	checkCudaErrors(hipMemset(manager.m_pSortedUsedFeaId, -1, sizeof(int) * manager.m_maxUsedFeaInTrees));

	manager.MemcpyHostToDevice(pHashUsedFea, manager.m_pHashFeaIdToDenseInsPos, sizeof(int) * numofUsedFea);
	manager.MemcpyHostToDevice(pSortedUsedFea, manager.m_pSortedUsedFeaId, sizeof(int) * numofUsedFea);

}

/**
 * @brief: get the pointer to the tree and its number of nodes
 */
void DevicePredictor::GetTreeInfo(TreeNode *&pTree, int &numofNodeOfTheTree, int treeId)
{
	if(treeId < 0)
		return;
	DTGPUMemManager treeManager;
	GBDTGPUMemManager manager;
	manager.MemcpyDeviceToHost(treeManager.m_pNumofNodeEachTree + treeId, &numofNodeOfTheTree, sizeof(int));
	int startPosOfLastTree = -1;
	manager.MemcpyDeviceToHost(treeManager.m_pStartPosOfEachTree + treeId, &startPosOfLastTree, sizeof(int));
	pTree = treeManager.m_pAllTree + startPosOfLastTree;

}

/**
 * @brief: construct a dense instance
 */
void DevicePredictor::FillDenseIns(int insId, int numofUsedFea)
{
	GBDTGPUMemManager manager;
	long long startPos = -1;
	long long *pInsStartPos = manager.m_pInsStartPos + (long long)insId;
	manager.MemcpyDeviceToHost(pInsStartPos, &startPos, sizeof(long long));
//			cout << "start pos ins" << insId << "=" << startPos << endl;
	float_point *pDevInsValue = manager.m_pdDInsValue + startPos;
	int *pDevFeaId = manager.m_pDFeaId + startPos;
	int numofFeaValue = -1;
	int *pNumofFea = manager.m_pDNumofFea + insId;
	manager.MemcpyDeviceToHost(pNumofFea, &numofFeaValue, sizeof(int));

	checkCudaErrors(hipMemset(manager.m_pdDenseIns, 0, sizeof(float_point) * numofUsedFea));
	FillDense<<<1, 1>>>(pDevInsValue, pDevFeaId, numofFeaValue, manager.m_pdDenseIns,
						manager.m_pSortedUsedFeaId, manager.m_pHashFeaIdToDenseInsPos, numofUsedFea);
}
