#include "hip/hip_runtime.h"
/*
 * DeviceSplitterKernel.cu
 *
 *  Created on: 10 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <stdio.h>

#include "FindFeaKernel.h"
#include "../Splitter/DeviceSplitter.h"
#include "../DeviceHashing.h"
#include "../prefix-sum/prefixSum.h"

const float rt_2eps = 2.0 * DeviceSplitter::rt_eps;

//helper functions on device
__device__ float_point CalGain(const nodeStat &parent, const nodeStat &r_child,
						  const float_point &l_child_GD, const float_point &l_child_Hess, const float_point &lambda);

__device__ bool UpdateSplitPoint(SplitPoint &curBest, float_point fGain, float_point fSplitValue, int nFeatureId);

__device__ void UpdateLRStat(nodeStat &RChildStat, nodeStat &LChildStat,
							 const nodeStat &TempRChildStat, const float_point &grad, const float_point &hess);
__device__ bool NeedUpdate(float_point &RChildHess, float_point &LChildHess);
__device__ void UpdateSplitInfo(const nodeStat &snStat, SplitPoint &bestSP, nodeStat &RChildStat, nodeStat &LChildStat,
								const nodeStat &TempRChildStat, const float_point &tempGD, const float_point &temHess,
								const float_point &lambda, const float_point &sv, const int &featureId);

/**
 * @brief: each thread processes one feature
 */
__global__ void FindFeaSplitValue(const int *pnNumofKeyValues, const long long *pnFeaStartPos, const int *pInsId, const float_point *pFeaValue,
								  const int *pInsIdToNodeId, const float_point *pGD, const float_point *pHess,
								  nodeStat *pTempRChildStatPerThread, float_point *pLastValuePerThread,
								  const nodeStat *pSNodeStatPerThread, SplitPoint *pBestSplitPointPerThread,
								  nodeStat *pRChildStatPerThread, nodeStat *pLChildStatPerThread,
								  const int *pSNIdToBuffId, int maxNumofSplittable, const int *pBuffId, int numofSNode,
								  float_point lambda, int numofFea)
{
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	int feaId = nGlobalThreadId;
	if(feaId >= numofFea)
	{
		return;
	}

	//addresses of instance ids and key-value pairs
		//compute start position key-value pairs of the current feature
	long long startPosOfPrevFea = 0;
	int numofPreFeaKeyValues = 0;
	if(feaId > 0)
	{
		//number of key values of the previous feature
		numofPreFeaKeyValues = pnNumofKeyValues[feaId - 1];
		//copy value of the start position of the previous feature
		startPosOfPrevFea = pnFeaStartPos[feaId - 1];
	}
	long long startPosOfCurFea = startPosOfPrevFea + numofPreFeaKeyValues;
	const int *InsIdStartAddress = pInsId + startPosOfCurFea;
	const float_point *pInsValueStartAddress = pFeaValue + startPosOfCurFea;

    for(int i = 0; i < pnNumofKeyValues[nGlobalThreadId]; i++)
    {
    	int insId = InsIdStartAddress[i];
    	int nid = pInsIdToNodeId[insId];
		if(nid < -1)
		{
			printf("Error: nid=%d\n", nid);
			return;
		}
		if(nid == -1)
			continue;

		// start working
		float_point fvalue = pInsValueStartAddress[i];

		// get the buffer id of node nid
		int hashValue = GetBufferId(pSNIdToBuffId, nid, maxNumofSplittable);
		int bufferPos = hashValue + feaId * maxNumofSplittable;

		if(pTempRChildStatPerThread[bufferPos].sum_hess == 0.0)//equivalent to IsEmpty()
		{
			pTempRChildStatPerThread[bufferPos].sum_gd += pGD[insId];
			pTempRChildStatPerThread[bufferPos].sum_hess += pHess[insId];
			pLastValuePerThread[bufferPos] = fvalue;
		}
		else
		{
			// try to find a split
			if(fabs(fvalue - pLastValuePerThread[bufferPos]) > rt_2eps)
			{
				//SNodeStatPerThread is the same for all the features, so using hashValue is fine and can save memory
				float_point tempGD = pSNodeStatPerThread[hashValue].sum_gd - pTempRChildStatPerThread[bufferPos].sum_gd;
				float_point tempHess = pSNodeStatPerThread[hashValue].sum_hess - pTempRChildStatPerThread[bufferPos].sum_hess;
				bool needUpdate = NeedUpdate(pTempRChildStatPerThread[bufferPos].sum_hess, tempHess);
				if(needUpdate == true)
				{
					float_point sv = (fvalue + pLastValuePerThread[bufferPos]) * 0.5f;
					if(hashValue == 1)
					{
//						float_point loss_chg = CalGain(pSNodeStatPerThread[bufferPos], pTempRChildStatPerThread[bufferPos], tempGD, tempHess, lambda);
//						printf("nid=%d, sv=%f, gain=%f\n", nid, sv, loss_chg);
					}

		            UpdateSplitInfo(pSNodeStatPerThread[hashValue], pBestSplitPointPerThread[bufferPos], pRChildStatPerThread[bufferPos],
		            							  pLChildStatPerThread[bufferPos], pTempRChildStatPerThread[bufferPos], tempGD, tempHess,
		            							  lambda, sv, feaId);
				}
			}
			//update the statistics
			pTempRChildStatPerThread[bufferPos].sum_gd += pGD[insId];
			pTempRChildStatPerThread[bufferPos].sum_hess += pHess[insId];
			pLastValuePerThread[bufferPos] = fvalue;
		}
	}


    // finish updating all statistics, check if it is possible to include all sum statistics
    for(int i = 0; i < numofSNode; i++)
    {
    	if(pBuffId[i] < 0)
    		printf("Error in buffer id %d, i=%d, numofSN=%d\n", pBuffId[i], i, numofSNode);

    	int hashVaue = pBuffId[i];
    	int buffId = hashVaue + feaId * maxNumofSplittable;//an id in the buffer
    	float_point tempGD = pSNodeStatPerThread[hashVaue].sum_gd - pTempRChildStatPerThread[buffId].sum_gd;
    	float_point tempHess = pSNodeStatPerThread[hashVaue].sum_hess - pTempRChildStatPerThread[buffId].sum_hess;
    	bool needUpdate = NeedUpdate(pTempRChildStatPerThread[buffId].sum_hess, tempHess);
        if(needUpdate == true)
        {
            const float delta = fabs(pLastValuePerThread[buffId]) + DeviceSplitter::rt_eps;
            float_point sv = pLastValuePerThread[buffId] + delta;

            UpdateSplitInfo(pSNodeStatPerThread[hashVaue], pBestSplitPointPerThread[buffId], pRChildStatPerThread[buffId], pLChildStatPerThread[buffId],
            							  pTempRChildStatPerThread[buffId], tempGD, tempHess, lambda, sv, feaId);
        }
    }
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void ObtainGDEachNode(const int *pnNumofKeyValues, const long long *pnFeaStartPos, const int *pInsId, const float_point *pFeaValue,
		  const int *pInsIdToNodeId, const float_point *pGD, const float_point *pHess,  int numofSNode, int smallestFeaId, int totalNumofFea, int feaBatch,
		  float_point *pGDOnEachFeaValue, float_point *pHessOnEachFeaValue, float_point *pValueOneEachFeaValue)
{
	//blockIdx.x corresponds to a feature which has multiple values
	//blockIdx.y corresponds to a feature id
	//blockIdx.z corresponds to a splittable node id
	//## global id looks ok, but need to be careful
	int nGlobalThreadId = (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	int snId = blockIdx.z;
	if(snId >= numofSNode)
		printf("# of block groups is larger than # of splittable nodes: %d v.s. %d\n", snId, numofSNode);
	int feaId = blockIdx.y + smallestFeaId;//###### need to add a shift here to process only part of the features
	int numofValuePerFea = gridDim.x * blockDim.x;
	if(feaId >= totalNumofFea)
		printf("# of block groups is larger than # of features: %d v.s. %d\n", feaId, totalNumofFea);

	//addresses of instance ids and key-value pairs
		//compute start position key-value pairs of the current feature
	long long startPosOfPrevFea = 0;
	int numofPreFeaKeyValues = 0;

	if(feaId > 0)
	{
		//number of key values of the previous feature
		numofPreFeaKeyValues = pnNumofKeyValues[feaId - 1];
		//copy value of the start position of the previous feature
		startPosOfPrevFea = pnFeaStartPos[feaId - 1];
	}
	int tidForEachFeaValue = blockIdx.x * blockDim.x + threadIdx.x;
	if(tidForEachFeaValue >= numofValuePerFea)
	{
		printf("numofValuePerFea is smaller than the numof threads!\n");
	}

	long long startPosOfCurFea = startPosOfPrevFea + numofPreFeaKeyValues;
	const int *InsIdStartAddress = pInsId + startPosOfCurFea;
	const float_point *pInsValueStartAddress = pFeaValue + startPosOfCurFea;

	int insId = InsIdStartAddress[tidForEachFeaValue];
	int nid = pInsIdToNodeId[insId];
	if(nid < -1)
	{
		printf("Error: nid=%d\n", nid);
		return;
	}
	if(nid == -1)
	{//some leave nodes
	}
	else
	{//some splittable nodes
		int bufferPos = snId * numofValuePerFea * feaBatch + feaId * numofValuePerFea + tidForEachFeaValue;
		if(pGDOnEachFeaValue[bufferPos] != 0 || pHessOnEachFeaValue[bufferPos]!= 0 || pValueOneEachFeaValue[bufferPos] != 0)
			printf("default value of gd/hess/fvalue is incorrect in ObtainGDEachNode.\n");

		//GD/Hess of the same node is stored consecutively.
		pGDOnEachFeaValue[bufferPos] = pGD[insId];
		pHessOnEachFeaValue[bufferPos] = pHess[insId];
		pValueOneEachFeaValue[bufferPos] = pInsValueStartAddress[tidForEachFeaValue];
	}
}

/**
 * @brief: compute the prefix sum for gd and hess
 */
void PrefixSumForEachNode(int feaBatch, float_point *pGDOnEachFeaValue_d, float_point *pHessOnEachFeaValue_d,
						  const int *pnStartPosEachFeaInBatch, const int *pnEachFeaLen)
{
	prefixsumForDeviceArray(pGDOnEachFeaValue_d, pnStartPosEachFeaInBatch, pnEachFeaLen, feaBatch);
	prefixsumForDeviceArray(pHessOnEachFeaValue_d, pnStartPosEachFeaInBatch, pnEachFeaLen, feaBatch);
}

__global__ void ComputeGain(const nodeStat *pSNodeStatPerThread, int feaBatch, float_point *pGDOnEachFeaValue_d,
							float_point *pHessOnEachFeaValue_d, const int *pnStartPosEachFeaInBatch, const int *pnEachFeaLen)
{
	int nGlobalThreadId = (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
}


__device__ float_point CalGain(const nodeStat &parent, const nodeStat &r_child,
						  const float_point &l_child_GD, const float_point &l_child_Hess,
						  const float_point &lambda)
{
//	PROCESS_ERROR(abs(parent.sum_gd - l_child_GD - r_child.sum_gd) < 0.0001);
//	PROCESS_ERROR(parent.sum_hess == l_child_Hess + r_child.sum_hess);

//	printf("lgd=%f, lhe=%f, rgd=%f, rhe=%f, pgd=%f, phe=%f, lamb=%f\n", l_child_GD, l_child_Hess,
//			r_child.sum_gd, r_child.sum_hess, parent.sum_gd, parent.sum_hess, lambda);

	//compute the gain
	float_point fGain = (l_child_GD * l_child_GD)/(l_child_Hess + lambda) +
				   (r_child.sum_gd * r_child.sum_gd)/(r_child.sum_hess + lambda) -
				   (parent.sum_gd * parent.sum_gd)/(parent.sum_hess + lambda);
//	if(fGain > -10)
//	{
//		printf("gain=%f, lgd=%f, lhe=%f, rgd=%f, rhe=%f, pgd=%f, phe=%f, lamb=%f\n", fGain, l_child_GD, l_child_Hess,
//				r_child.sum_gd, r_child.sum_hess, parent.sum_gd, parent.sum_hess, lambda);
//	}


	return fGain;
}


 __device__ bool UpdateSplitPoint(SplitPoint &curBest, float_point fGain, float_point fSplitValue, int nFeatureId)
{
	if(fGain > curBest.m_fGain )//|| (fGain == m_fGain && nFeatureId == m_nFeatureId) NOT USE (second condition is for updating to a new split value)
	{
		curBest.m_fGain = fGain;
		curBest.m_fSplitValue = fSplitValue;
		curBest.m_nFeatureId = nFeatureId;
		return true;
	}
	return false;
}

__device__ void UpdateLRStat(nodeStat &RChildStat, nodeStat &LChildStat,
							 const nodeStat &TempRChildStat, const float_point &grad, const float_point &hess)
{
	LChildStat.sum_gd = grad;
	LChildStat.sum_hess = hess;
	RChildStat = TempRChildStat;
}

__device__ bool NeedUpdate(float_point &RChildHess, float_point &LChildHess)
{
	if(LChildHess >= DeviceSplitter::min_child_weight && RChildHess >= DeviceSplitter::min_child_weight)
		return true;
	return false;
}

__device__ void UpdateSplitInfo(const nodeStat &snStat, SplitPoint &bestSP, nodeStat &RChildStat, nodeStat &LChildStat,
								const nodeStat &TempRChildStat, const float_point &tempGD, const float_point &tempHess,
								const float_point &lambda, const float_point &sv, const int &featureId)
{
	float_point loss_chg = CalGain(snStat, TempRChildStat, tempGD, tempHess, lambda);
    bool bUpdated = UpdateSplitPoint(bestSP, loss_chg, sv, featureId);
	if(bUpdated == true)
	{
		UpdateLRStat(RChildStat, LChildStat, TempRChildStat, tempGD, tempHess);
	}
}

