#include "hip/hip_runtime.h"
/*
 * ComputeGainDense.cu
 *
 *  Created on: 22 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: kernels gain computing using dense arrays
 */

#include <stdio.h>
#include <float.h>
#include <limits>
#include "FindFeaKernel.h"
#include "../Splitter/DeviceSplitter.h"
#include "../../DeviceHost/svm-shared/DeviceUtility.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/getMin.h"
#include "../../SharedUtility/binarySearch.h"

const float rt_2eps = 2.0 * DeviceSplitter::rt_eps;

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalueRoot(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//store GD and Hess.
	pGDEachFeaValue[gTid] = pInsGD[insId];
	pHessEachFeaValue[gTid] = pInsHess[insId];
	pDenseFeaValue[gTid] = pAllFeaValue[gTid];
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalue(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, const unsigned int *pDstIndexEachFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	CONCHECKER(idx >= 0);
	CONCHECKER(idx < numFeaValue);

	//scatter: store GD, Hess and the feature value.
	pGDEachFeaValue[idx] = pInsGD[insId];
	pHessEachFeaValue[idx] = pInsHess[insId];
	pDenseFeaValue[idx] = pAllFeaValue[gTid];
}

/**
 * @brief: compute the gain in parallel, each gain is computed by a thread.
 */
__global__ void ComputeGainDense(const nodeStat *pSNodeStat, const int *pId2SNPos, real lambda,
							const double *pGDPrefixSumOnEachFeaValue, const real *pHessPrefixSumOnEachFeaValue,
							const real *pDenseFeaValue, int numofDenseValue,
							const uint *pEachFeaStartEachNode, const int *pEachFeaLenEachNode,
							const uint *pnKey, int numFea, real *pGainOnEachFeaValue, bool *pDefault2Right)
{
	//one thread loads one value
	uint gTid = GLOBAL_TID();
	if(gTid >= numofDenseValue)//the thread has no gain to compute, i.e. a thread per gain
		return;

	uint segId = pnKey[gTid];
	uint pid = segId / numFea;

	int snPos = pId2SNPos[pid];
	ECHECKER(snPos);

	if(gTid == 0)
	{
		//assign gain to 0 to the first feature value
    	pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	//if the previous fea value is the same as the current fea value, gain is 0 for the current fea value.
	real preFvalue = pDenseFeaValue[gTid - 1], curFvalue = pDenseFeaValue[gTid];
	if(preFvalue - curFvalue <= rt_2eps && preFvalue - curFvalue >= -rt_2eps)//############## backwards is not considered!
	{//avoid same feature value different gain issue
		pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	int exclusiveSumPos = gTid - 1;//following xgboost using exclusive sum on gd and hess

	//forward consideration (fvalues are sorted descendingly)
	double rChildGD = pGDPrefixSumOnEachFeaValue[exclusiveSumPos];
	double rChildHess = pHessPrefixSumOnEachFeaValue[exclusiveSumPos];
	double parentGD = pSNodeStat[snPos].sum_gd;
	double parentHess = pSNodeStat[snPos].sum_hess;
	double tempGD = parentGD - rChildGD;
	double tempHess = parentHess - rChildHess;
	bool needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true)//need to compute the gain
    {
		double tempGain = (tempGD * tempGD)/(tempHess + lambda) +
						  	   (rChildGD * rChildGD)/(rChildHess + lambda) -
	  						   (parentGD * parentGD)/(parentHess + lambda);
    	pGainOnEachFeaValue[gTid] = tempGain; 
    }
    else{
    	//assign gain to 0
    	pGainOnEachFeaValue[gTid] = 0;
    }

    //backward consideration
    int segLen = pEachFeaLenEachNode[segId];
    uint segStartPos = pEachFeaStartEachNode[segId];
    uint lastFvaluePos = segStartPos + segLen - 1;
    double totalMissingGD = parentGD - pGDPrefixSumOnEachFeaValue[lastFvaluePos];
    double totalMissingHess = parentHess - pHessPrefixSumOnEachFeaValue[lastFvaluePos];
    if(totalMissingHess < 1)//there is no instance with missing values
    	return;
    //missing values to the right child
    rChildGD += totalMissingGD;
    rChildHess += totalMissingHess;
    tempGD = parentGD - rChildGD;
    tempHess = parentHess - rChildHess;
    needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true){
    	double tempGain = (tempGD * tempGD)/(tempHess + lambda) +
			  	   	    (rChildGD * rChildGD)/(rChildHess + lambda) -
			  	   	    (parentGD * parentGD)/(parentHess + lambda);

    	if(tempGain > 0 && tempGain - pGainOnEachFeaValue[gTid] > 0.1){
    		pGainOnEachFeaValue[gTid] = tempGain;
    		pDefault2Right[gTid] = true;
    	}
    }
}

/**
 * @brief: change the gain of the first value of each feature to 0
 */
__global__ void FirstFeaGain(const unsigned int *pEachFeaStartPosEachNode, int numFeaStartPos, real *pGainOnEachFeaValue, uint numFeaValue)
{
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaStartPos)//no gain to fix
		return;
	unsigned int gainPos = pEachFeaStartPosEachNode[gTid];
	if(gainPos >= numFeaValue)
		return;//there may be some ending 0s (e.g. the last node has some features with any values).
	pGainOnEachFeaValue[gainPos] = 0;
}
