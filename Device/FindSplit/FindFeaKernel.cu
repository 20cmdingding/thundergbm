#include "hip/hip_runtime.h"
/*
 * ComputeGainDense.cu
 *
 *  Created on: 22 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: kernels gain computing using dense arrays
 */

#include <stdio.h>
#include <float.h>
#include <limits>
#include "FindFeaKernel.h"
#include "../Splitter/DeviceSplitter.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/binarySearch.h"

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalueRoot(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//store GD and Hess.
	pGDEachFeaValue[gTid] = pInsGD[insId];
	pHessEachFeaValue[gTid] = pInsHess[insId];
	pDenseFeaValue[gTid] = pAllFeaValue[gTid];
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalue(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, const unsigned int *pDstIndexEachFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//index for scatter
	uint idx = pDstIndexEachFeaValue[gTid];
	if(idx == LARGE_4B_UINT)//instance is in a leaf node
		return;

	CONCHECKER(idx < numFeaValue);

	//scatter: store GD, Hess and the feature value.
	pGDEachFeaValue[idx] = pInsGD[insId];
	pHessEachFeaValue[idx] = pInsHess[insId];
	pDenseFeaValue[idx] = pAllFeaValue[gTid];
}

/**
 * @brief: change the gain of the first value of each feature to 0
 */
__global__ void FirstFeaGain(const unsigned int *pEachFeaStartPosEachNode, int numFeaStartPos, real *pGainOnEachFeaValue, uint numFeaValue)
{
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaStartPos)//no gain to fix
		return;
	unsigned int gainPos = pEachFeaStartPosEachNode[gTid];
	if(gainPos >= numFeaValue)
		return;//there may be some ending 0s (e.g. the last node has some features with any values).
	pGainOnEachFeaValue[gainPos] = 0;
}
