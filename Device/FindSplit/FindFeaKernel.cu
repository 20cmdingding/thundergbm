#include "hip/hip_runtime.h"
/*
 * ComputeGainDense.cu
 *
 *  Created on: 22 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: kernels gain computing using dense arrays
 */

#include <stdio.h>
#include <float.h>
#include <limits>
#include "FindFeaKernel.h"
#include "../Splitter/DeviceSplitter.h"
#include "../../DeviceHost/svm-shared/DeviceUtility.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/getMin.h"

const float rt_2eps = 2.0 * DeviceSplitter::rt_eps;

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalueRoot(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//store GD and Hess.
	pGDEachFeaValue[gTid] = pInsGD[insId];
	pHessEachFeaValue[gTid] = pInsHess[insId];
	pDenseFeaValue[gTid] = pAllFeaValue[gTid];
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalue(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, const unsigned int *pDstIndexEachFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	CONCHECKER(idx >= 0);
	CONCHECKER(idx < numFeaValue);

	//scatter: store GD, Hess and the feature value.
	pGDEachFeaValue[idx] = pInsGD[insId];
	pHessEachFeaValue[idx] = pInsHess[insId];
	pDenseFeaValue[idx] = pAllFeaValue[gTid];
}

/**
 * @brief: compute the gain in parallel, each gain is computed by a thread.
 */
__global__ void ComputeGainDense(const nodeStat *pSNodeStat, const int *pId2SNPos, real lambda,
							const double *pGDPrefixSumOnEachFeaValue, const real *pHessPrefixSumOnEachFeaValue,
							const real *pDenseFeaValue, int numofDenseValue, const unsigned int *pnLastFvalueOfThisFvalue,
							const uint *pnKey, int numFea, real *pGainOnEachFeaValue, bool *pDefault2Right)
{
	//one thread loads one value
	uint gTid = GLOBAL_TID();
	if(gTid >= numofDenseValue)//the thread has no gain to compute, i.e. a thread per gain
		return;

	uint segId = pnKey[gTid];
	uint pid = segId / numFea;

	int snPos = pId2SNPos[pid];
	ECHECKER(snPos);

	if(gTid == 0)
	{
		//assign gain to 0 to the first feature value
    	pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	//if the previous fea value is the same as the current fea value, gain is 0 for the current fea value.
	real preFvalue = pDenseFeaValue[gTid - 1], curFvalue = pDenseFeaValue[gTid];
	if(preFvalue - curFvalue <= rt_2eps && preFvalue - curFvalue >= -rt_2eps)
	{//avoid same feature value different gain issue
		pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	int exclusiveSumPos = gTid - 1;//following xgboost using exclusive sum on gd and hess

	//forward consideration (fvalues are sorted descendingly)
	double rChildGD = pGDPrefixSumOnEachFeaValue[exclusiveSumPos];
	real rChildHess = pHessPrefixSumOnEachFeaValue[exclusiveSumPos];
	real parentGD = pSNodeStat[snPos].sum_gd;
	real parentHess = pSNodeStat[snPos].sum_hess;
	real tempGD = parentGD - rChildGD;
	real tempHess = parentHess - rChildHess;
	bool needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true)//need to compute the gain
    {
		real tempGain = (tempGD * tempGD)/(tempHess + lambda) + 
						  	   (rChildGD * rChildGD)/(rChildHess + lambda) -
	  						   (parentGD * parentGD)/(parentHess + lambda);
    	pGainOnEachFeaValue[gTid] = tempGain; 
//    	if(snPos == 1 && tempGain > 7848)
//    		printf("forwards: gain=%f, gTid=%d\n", tempGain, gTid);
    }
    else{
    	//assign gain to 0
    	pGainOnEachFeaValue[gTid] = 0;
    }

    //backward consideration
    unsigned int lastFvaluePos = pnLastFvalueOfThisFvalue[gTid];
    real totalMissingGD = parentGD - pGDPrefixSumOnEachFeaValue[lastFvaluePos];
    real totalMissingHess = parentHess - pHessPrefixSumOnEachFeaValue[lastFvaluePos];
    if(totalMissingHess < 1)//there is no instance with missing values
    	return;
    //missing values to the right child
    rChildGD += totalMissingGD;
    rChildHess += totalMissingHess;
    tempGD = parentGD - rChildGD;
    tempHess = parentHess - rChildHess;
    needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true){
    	real tempGain = (tempGD * tempGD)/(tempHess + lambda) +
			  	   	    (rChildGD * rChildGD)/(rChildHess + lambda) -
			  	   	    (parentGD * parentGD)/(parentHess + lambda);
//    	if(snPos == 1 && tempGain > 7848)
//    		printf("backwards: gain=%f, gTid=%d\n", tempGain, gTid);
    	if(tempGain > 0 && tempGain - pGainOnEachFeaValue[gTid] > 0.1){
    		pGainOnEachFeaValue[gTid] = tempGain;
    		pDefault2Right[gTid] = true;
    	}
    }
}

/**
 * @brief: change the gain of the first value of each feature to 0
 */
__global__ void FirstFeaGain(const unsigned int *pEachFeaStartPosEachNode, int numFeaStartPos, real *pGainOnEachFeaValue, long long numFeaValue)
{
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaStartPos)//no gain to fix
		return;
	unsigned int gainPos = pEachFeaStartPosEachNode[gTid];
	if(gainPos >= numFeaValue)
		return;//there may be some ending 0s (e.g. the last node has some features with any values).
	pGainOnEachFeaValue[gainPos] = 0;
//	if(gTid == 0){
//		printf("pEachFeaStartPosEachNode[8]=%f\n", pEachFeaStartPosEachNode[8]);
//	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 * Each block.y processes one node, a thread processes a reduction.
 */
__global__ void PickLocalBestSplitEachNode(const uint *pnNumFeaValueEachNode, const uint *pFeaStartPosEachNode,
										   const real *pGainOnEachFeaValue,
								   	   	   real *pfLocalBestGain, int *pnLocalBestGainKey)
{
	//best gain of each node is search by a few blocks
	//blockIdx.z corresponds to a splittable node id
	int snId = blockIdx.z;
	uint numValueThisNode = pnNumFeaValueEachNode[snId];//get the number of feature value of this node
	int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
	uint tid0 = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
	if(tid0 >= numValueThisNode){
		pfLocalBestGain[blockId] = 0;
		pnLocalBestGainKey[blockId] = tid0;
		return;
	}

	__shared__ real pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;
	if(localTid == 0){//initialise local best value
		pfLocalBestGain[blockId] = FLT_MAX;
		pnLocalBestGainKey[blockId] = -1;
	}

	uint tidForEachNode = tid0 + threadIdx.x;
	uint nPos = pFeaStartPosEachNode[snId] + tidForEachNode;//feature value gain position


	if(tidForEachNode >= numValueThisNode){//no gain to load
		pfGain[localTid] = 0;
		pnBetterGainKey[localTid] = INT_MAX;
	}
	else{
		pfGain[localTid] = -pGainOnEachFeaValue[nPos];//change to find min of -gain
		pnBetterGainKey[localTid] = nPos;
	}
	__syncthreads();

	//find the local best split point
	GetMinValueOriginal(pfGain, pnBetterGainKey);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		pfLocalBestGain[blockId] = pfGain[0];
		pnLocalBestGainKey[blockId] = pnBetterGainKey[0];

		ECHECKER(pnBetterGainKey[0]);
		//if(pnBetterGainKey[0] < 0)
		//	printf("negative key: snId=%d, blockId=%d, gain=%f, key=%d\n", snId, blockId, pfGain[0], pnBetterGainKey[0]);
	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 */
__global__ void PickGlobalBestSplitEachNode(const real *pfLocalBestGain, const int *pnLocalBestGainKey,
								   	   	    real *pfGlobalBestGain, int *pnGlobalBestGainKey,
								   	   	    int numBlockPerNode, int numofSNode)
{
	//a block for finding the best gain of a node
	int blockId = blockIdx.x;

	int snId = blockId;
	CONCHECKER(blockIdx.y <= 1);
	CONCHECKER(snId < numofSNode);

	__shared__ real pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;

	if(localTid >= numBlockPerNode)//number of threads is larger than the number of blocks
	{
		return;
	}

	int curFeaLocalBestStartPos = snId * numBlockPerNode;

	LoadToSharedMem(numBlockPerNode, curFeaLocalBestStartPos, pfLocalBestGain, pnLocalBestGainKey, pfGain, pnBetterGainKey);
	 __syncthreads();	//wait until the thread within the block

	//find the local best split point
	GetMinValueOriginal(pfGain, pnBetterGainKey);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		pfGlobalBestGain[snId] = -pfGain[0];//make the gain back to its original sign
		pnGlobalBestGainKey[snId] = pnBetterGainKey[0];
		ECHECKER(pnBetterGainKey[0]);
		if(pnBetterGainKey[0] < 0)
			printf("negative key: snId=%d, gain=%f, key=%d, blockDim.x=%d, blockSize=%d, blockpPerNode=%d, numSN=%d\n",
			snId, pfGain[0], pnBetterGainKey[0], blockDim.x, BLOCK_SIZE, numBlockPerNode, numofSNode);
	}
}

/**
 * @brief: find split points
 */
__global__ void FindSplitInfo(const unsigned int *pEachFeaStartPosEachNode, const int *pEachFeaLenEachNode,
							  const real *pDenseFeaValue, const real *pfGlobalBestGain, const int *pnGlobalBestGainKey,
							  const int *pPartitionId2SNPos, const int numFea,
							  const nodeStat *snNodeStat, const double *pPrefixSumGD, const real *pPrefixSumHess,
							  const bool *pDefault2Right, const unsigned int *pnLastFvalueOfThisFvalue,
							  SplitPoint *pBestSplitPoint, nodeStat *pRChildStat, nodeStat *pLChildStat)
{
	//a thread for constructing a split point
	int snId = threadIdx.x;//position in the dense array of nodes
	int key = pnGlobalBestGainKey[snId];//position in the dense array

	//find best feature id
	int bestFeaId = -1;
	for(int f = 0; f < numFea; f++)
	{
		int feaPos = f + snId * numFea;
		int numofFValue = pEachFeaLenEachNode[feaPos];
		if(pEachFeaStartPosEachNode[feaPos] + numofFValue < key)//####### key should be represented using long long
			continue;
		else//key is in the range of values of f
		{
			bestFeaId = f;
			break;
		}
	}

	CONCHECKER(bestFeaId != -1);

	int snPos = pPartitionId2SNPos[snId];//snId to buffer id (i.e. hash value)

	pBestSplitPoint[snPos].m_fGain = pfGlobalBestGain[snId];//change the gain back to positive
	if(pfGlobalBestGain[snId] <= 0){//no gain
		return;
	}

	pBestSplitPoint[snPos].m_nFeatureId = bestFeaId;
	ECHECKER(key);
	pBestSplitPoint[snPos].m_fSplitValue = 0.5f * (pDenseFeaValue[key] + pDenseFeaValue[key - 1]);
	pBestSplitPoint[snPos].m_bDefault2Right = false;

	//child node stat
	int idxPreSum = key - 1;//follow xgboost using exclusive
	if(pDefault2Right[key] == false){
		pLChildStat[snPos].sum_gd = snNodeStat[snPos].sum_gd - pPrefixSumGD[idxPreSum];
		pLChildStat[snPos].sum_hess = snNodeStat[snPos].sum_hess - pPrefixSumHess[idxPreSum];
		pRChildStat[snPos].sum_gd = pPrefixSumGD[idxPreSum];
		pRChildStat[snPos].sum_hess = pPrefixSumHess[idxPreSum];
	}
	else{
		pBestSplitPoint[snPos].m_bDefault2Right = true;

		real parentGD = snNodeStat[snPos].sum_gd;
		real parentHess = snNodeStat[snPos].sum_hess;
		unsigned int lastFvaluePos = pnLastFvalueOfThisFvalue[key];
		real totalMissingGD = parentGD - pPrefixSumGD[lastFvaluePos];
		real totalMissingHess = parentHess - pPrefixSumHess[lastFvaluePos];

		double rChildGD = totalMissingGD + pPrefixSumGD[idxPreSum];
		real rChildHess = totalMissingHess + pPrefixSumHess[idxPreSum];
		real lChildGD = parentGD - rChildGD;
		real lChildHess = parentHess - rChildHess;

		pRChildStat[snPos].sum_gd = rChildGD;
		pRChildStat[snPos].sum_hess = rChildHess;
		pLChildStat[snPos].sum_gd = lChildGD;
		pLChildStat[snPos].sum_hess = lChildHess;
	}
	ECHECKER(pLChildStat[snPos].sum_hess);
	ECHECKER(pRChildStat[snPos].sum_hess);
//	printf("split: f=%d, value=%f, gain=%f, gd=%f v.s. %f, hess=%f v.s. %f, buffId=%d, key=%d\n", bestFeaId, pBestSplitPoint[snPos].m_fSplitValue,
//			pBestSplitPoint[snPos].m_fGain, pLChildStat[snPos].sum_gd, pRChildStat[snPos].sum_gd, pLChildStat[snPos].sum_hess, pRChildStat[snPos].sum_hess, snPos, key);
}

__device__ bool NeedUpdate(real &RChildHess, real &LChildHess)
{
	if(LChildHess >= DeviceSplitter::min_child_weight && RChildHess >= DeviceSplitter::min_child_weight)
		return true;
	return false;
}

