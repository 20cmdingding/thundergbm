#include "hip/hip_runtime.h"
/*
 * ComputeGainDense.cu
 *
 *  Created on: 22 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: kernels gain computing using dense arrays
 */

#include <stdio.h>
#include <float.h>
#include <limits>
#include "FindFeaKernel.h"
#include "../Splitter/DeviceSplitter.h"
#include "../../DeviceHost/svm-shared/DeviceUtility.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/getMin.h"
#include "../../SharedUtility/binarySearch.h"

const float rt_2eps = 2.0 * DeviceSplitter::rt_eps;

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalueRoot(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//store GD and Hess.
	pGDEachFeaValue[gTid] = pInsGD[insId];
	pHessEachFeaValue[gTid] = pInsHess[insId];
	pDenseFeaValue[gTid] = pAllFeaValue[gTid];
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalue(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, const unsigned int *pDstIndexEachFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	CONCHECKER(idx >= 0);
	CONCHECKER(idx < numFeaValue);

	//scatter: store GD, Hess and the feature value.
	pGDEachFeaValue[idx] = pInsGD[insId];
	pHessEachFeaValue[idx] = pInsHess[insId];
	pDenseFeaValue[idx] = pAllFeaValue[gTid];
}

/**
 * @brief: compute the gain in parallel, each gain is computed by a thread.
 */
__global__ void ComputeGainDense(const nodeStat *pSNodeStat, const int *pId2SNPos, real lambda,
							const double *pGDPrefixSumOnEachFeaValue, const real *pHessPrefixSumOnEachFeaValue,
							const real *pDenseFeaValue, int numofDenseValue,
							const uint *pEachFeaStartEachNode, const int *pEachFeaLenEachNode,
							const uint *pnKey, int numFea, real *pGainOnEachFeaValue, bool *pDefault2Right)
{
	//one thread loads one value
	uint gTid = GLOBAL_TID();
	if(gTid >= numofDenseValue)//the thread has no gain to compute, i.e. a thread per gain
		return;

	uint segId = pnKey[gTid];
	uint pid = segId / numFea;

	int snPos = pId2SNPos[pid];
	ECHECKER(snPos);

	if(gTid == 0)
	{
		//assign gain to 0 to the first feature value
    	pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	//if the previous fea value is the same as the current fea value, gain is 0 for the current fea value.
	real preFvalue = pDenseFeaValue[gTid - 1], curFvalue = pDenseFeaValue[gTid];
	if(preFvalue - curFvalue <= rt_2eps && preFvalue - curFvalue >= -rt_2eps)//############## backwards is not considered!
	{//avoid same feature value different gain issue
		pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	int exclusiveSumPos = gTid - 1;//following xgboost using exclusive sum on gd and hess

	//forward consideration (fvalues are sorted descendingly)
	double rChildGD = pGDPrefixSumOnEachFeaValue[exclusiveSumPos];
	double rChildHess = pHessPrefixSumOnEachFeaValue[exclusiveSumPos];
	double parentGD = pSNodeStat[snPos].sum_gd;
	double parentHess = pSNodeStat[snPos].sum_hess;
	double tempGD = parentGD - rChildGD;
	double tempHess = parentHess - rChildHess;
	bool needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true)//need to compute the gain
    {
		double tempGain = (tempGD * tempGD)/(tempHess + lambda) +
						  	   (rChildGD * rChildGD)/(rChildHess + lambda) -
	  						   (parentGD * parentGD)/(parentHess + lambda);
    	pGainOnEachFeaValue[gTid] = tempGain; 
    }
    else{
    	//assign gain to 0
    	pGainOnEachFeaValue[gTid] = 0;
    }

    //backward consideration
    int segLen = pEachFeaLenEachNode[segId];
    uint segStartPos = pEachFeaStartEachNode[segId];
    uint lastFvaluePos = segStartPos + segLen - 1;
    double totalMissingGD = parentGD - pGDPrefixSumOnEachFeaValue[lastFvaluePos];
    double totalMissingHess = parentHess - pHessPrefixSumOnEachFeaValue[lastFvaluePos];
    if(totalMissingHess < 1)//there is no instance with missing values
    	return;
    //missing values to the right child
    rChildGD += totalMissingGD;
    rChildHess += totalMissingHess;
    tempGD = parentGD - rChildGD;
    tempHess = parentHess - rChildHess;
    needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true){
    	double tempGain = (tempGD * tempGD)/(tempHess + lambda) +
			  	   	    (rChildGD * rChildGD)/(rChildHess + lambda) -
			  	   	    (parentGD * parentGD)/(parentHess + lambda);

    	if(tempGain > 0 && tempGain - pGainOnEachFeaValue[gTid] > 0.1){
    		pGainOnEachFeaValue[gTid] = tempGain;
    		pDefault2Right[gTid] = true;
    	}
    }
}

/**
 * @brief: change the gain of the first value of each feature to 0
 */
__global__ void FirstFeaGain(const unsigned int *pEachFeaStartPosEachNode, int numFeaStartPos, real *pGainOnEachFeaValue, uint numFeaValue)
{
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaStartPos)//no gain to fix
		return;
	unsigned int gainPos = pEachFeaStartPosEachNode[gTid];
	if(gainPos >= numFeaValue)
		return;//there may be some ending 0s (e.g. the last node has some features with any values).
	pGainOnEachFeaValue[gainPos] = 0;
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 * Each block.y processes one node, a thread processes a reduction.
 */
__global__ void PickLocalBestSplitEachNode(const uint *pnNumFeaValueEachNode, const uint *pFeaStartPosEachNode,
										   const real *pGainOnEachFeaValue,
								   	   	   real *pfLocalBestGain, int *pnLocalBestGainKey)
{
	//best gain of each node is search by a few blocks
	//blockIdx.z corresponds to a splittable node id
	int snId = blockIdx.z;
	uint numValueThisNode = pnNumFeaValueEachNode[snId];//get the number of feature value of this node
	int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
	uint tid0 = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
	if(tid0 >= numValueThisNode){
		pfLocalBestGain[blockId] = 0;
		pnLocalBestGainKey[blockId] = tid0;
		return;
	}

	__shared__ real pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;
	if(localTid == 0){//initialise local best value
		pfLocalBestGain[blockId] = FLT_MAX;
		pnLocalBestGainKey[blockId] = -1;
	}

	uint tidForEachNode = tid0 + threadIdx.x;
	uint nPos = pFeaStartPosEachNode[snId] + tidForEachNode;//feature value gain position


	if(tidForEachNode >= numValueThisNode){//no gain to load
		pfGain[localTid] = 0;
		pnBetterGainKey[localTid] = INT_MAX;
	}
	else{
		pfGain[localTid] = -pGainOnEachFeaValue[nPos];//change to find min of -gain
		pnBetterGainKey[localTid] = nPos;
	}
	__syncthreads();

	//find the local best split point
	GetMinValueOriginal(pfGain, pnBetterGainKey);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		pfLocalBestGain[blockId] = pfGain[0];
		pnLocalBestGainKey[blockId] = pnBetterGainKey[0];

		ECHECKER(pnBetterGainKey[0]);
		//if(pnBetterGainKey[0] < 0)
		//	printf("negative key: snId=%d, blockId=%d, gain=%f, key=%d\n", snId, blockId, pfGain[0], pnBetterGainKey[0]);
	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 */
__global__ void PickGlobalBestSplitEachNode(const real *pfLocalBestGain, const int *pnLocalBestGainKey,
								   	   	    real *pfGlobalBestGain, int *pnGlobalBestGainKey,
								   	   	    int numBlockPerNode, int numofSNode)
{
	//a block for finding the best gain of a node
	int blockId = blockIdx.x;

	int snId = blockId;
	CONCHECKER(blockIdx.y <= 1);
	CONCHECKER(snId < numofSNode);

	__shared__ real pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;

	if(localTid >= numBlockPerNode)//number of threads is larger than the number of blocks
	{
		return;
	}

	int curFeaLocalBestStartPos = snId * numBlockPerNode;

	LoadToSharedMem(numBlockPerNode, curFeaLocalBestStartPos, pfLocalBestGain, pnLocalBestGainKey, pfGain, pnBetterGainKey);
	 __syncthreads();	//wait until the thread within the block

	//find the local best split point
	GetMinValueOriginal(pfGain, pnBetterGainKey);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		pfGlobalBestGain[snId] = -pfGain[0];//make the gain back to its original sign
		pnGlobalBestGainKey[snId] = pnBetterGainKey[0];
		ECHECKER(pnBetterGainKey[0]);
		if(pnBetterGainKey[0] < 0)
			printf("negative key: snId=%d, gain=%f, key=%d, blockDim.x=%d, blockSize=%d, blockpPerNode=%d, numSN=%d\n",
			snId, pfGain[0], pnBetterGainKey[0], blockDim.x, BLOCK_SIZE, numBlockPerNode, numofSNode);
	}
}
