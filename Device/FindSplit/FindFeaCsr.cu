#include "hip/hip_runtime.h"
/*
 * FindFeaCsr.cu
 *
 *  Created on: Jul 28, 2017
 *      Author: zeyi
 */

#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <unistd.h>
#include <set>
#include <fstream>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Bagging/BagManager.h"
#include "../CSR/BagCsrManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/segmentedMax.h"
#include "../../SharedUtility/segmentedSum.h"
#include "../../SharedUtility/setSegmentKey.h"

#include "../CSR/CsrSplit.h"
#include "../CSR/CsrCompressor.h"
#include "../../syncarray.h"
using std::set;


void CsrCompression(int numofSNode, uint &totalNumCsrFvalue, uint *eachCompressedFeaStartPos, uint *eachCompressedFeaLen,
		uint *eachNodeSizeInCsr, uint *eachCsrNodeStartPos, real *csrFvalue, double *csrGD_h, real *csrHess_h, uint *eachCsrLen){
	BagManager bagManager;
	real *fvalue_h = new real[bagManager.m_numFeaValue];
	uint *eachFeaLenEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	uint *eachFeaStartPosEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	checkCudaErrors(hipMemcpy(fvalue_h, fvalue_d, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(eachFeaLenEachNode_h, bagManager.m_pEachFeaLenEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(eachFeaStartPosEachNode_h, bagManager.m_pEachFeaStartPosEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));

	uint csrId = 0, curFvalueToCompress = 0;
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		eachCompressedFeaLen[i] = 0;
		uint feaLen = eachFeaLenEachNode_h[i];
		uint feaStart = eachFeaStartPosEachNode_h[i];
		if(feaLen == 0)continue;
		csrFvalue[csrId] = fvalue_h[feaStart];
		eachCsrLen[csrId] = 1;
		eachCompressedFeaLen[i] = 1;
		for(int l = 1; l < feaLen; l++){
			curFvalueToCompress++;
			if(fabs(fvalue_h[feaStart + l] - csrFvalue[csrId]) > DeviceSplitter::rt_eps){
				eachCompressedFeaLen[i]++;
				csrId++;
				csrFvalue[csrId] = fvalue_h[feaStart + l];
				eachCsrLen[csrId] = 1;
			}
			else
				eachCsrLen[csrId]++;
		}
		csrId++;
		curFvalueToCompress++;
	}
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		uint prefix = 0;
		for(int l = 0; l < i; l++)
			prefix += eachCompressedFeaLen[l];
		eachCompressedFeaStartPos[i] = prefix;
	}

	for(int i = 0; i < numofSNode; i++){
		int posOfLastFeaThisNode = (i + 1) * bagManager.m_numFea - 1;
		int posOfFirstFeaThisNode = i * bagManager.m_numFea;
		eachNodeSizeInCsr[i] = eachCompressedFeaStartPos[posOfLastFeaThisNode] - eachCompressedFeaStartPos[posOfFirstFeaThisNode];
		eachNodeSizeInCsr[i] += eachCompressedFeaLen[posOfLastFeaThisNode];
		eachCsrNodeStartPos[i] = eachCompressedFeaStartPos[posOfFirstFeaThisNode];
//		printf("node %d starts %u, len=%u\n", i, eachCsrNodeStartPos[i], eachNodeSizeInCsr[i]);
	}

	totalNumCsrFvalue = csrId;
//	printf("csrLen=%u, totalLen=%u, numofFeaValue=%u\n", csrId, totalLen, bagManager.m_numFeaValue);
	PROCESS_ERROR(totalNumCsrFvalue < bagManager.m_numFeaValue);
	//compute csr gd and hess
	double *gd_h = new double[bagManager.m_numFeaValue];
	real *hess_h = new real[bagManager.m_numFeaValue];
	checkCudaErrors(hipMemcpy(gd_h, fgd_d, sizeof(double) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hess_h, fhess_d, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));

	uint globalPos = 0;
	for(int i = 0; i < csrId; i++){
		csrGD_h[i] = 0;
		csrHess_h[i] = 0;
		uint len = eachCsrLen[i];
		for(int v = 0; v < len; v++){
			csrGD_h[i] += gd_h[globalPos];
			csrHess_h[i] += hess_h[globalPos];
			globalPos++;
		}
	}

	printf("org=%u v.s. csr=%u\n", bagManager.m_numFeaValue, totalNumCsrFvalue);

	delete[] fvalue_h;
	delete[] eachFeaLenEachNode_h;
	delete[] eachFeaStartPosEachNode_h;
	delete[] gd_h;
	delete[] hess_h;
}

/**
 * @brief: efficient best feature finder
 */
__global__ void LoadFvalueInsId(const int *pOrgFvalueInsId, int *pNewFvalueInsId, const unsigned int *pDstIndexEachFeaValue, int numFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	CONCHECKER(idx >= 0);
	CONCHECKER(idx < numFeaValue);

	//scatter: store GD, Hess and the feature value.
	pNewFvalueInsId[idx] = pOrgFvalueInsId[gTid];
}


uint numofDenseValue_previous;
bool firstTime = true;

void AfterCompression(GBDTGPUMemManager &manager, BagCsrManager &csrManager, BagManager &bagManager, IndexComputer &indexComp, KernelConf &conf, void *pStream, double &total_scan_t,
					  int &maxNumFeaValueOneNode){
	//cout << "prefix sum" << endl;
	printf("prefix sum\n");
	clock_t start_scan = clock();
	int nNumofFeature = manager.m_numofFea;
	double *pGD_d = (double*)indexComp.histogram_d.addr;//reuse memory; must be here, as curNumCsr may change in different level.
	real *pHess_d = (real*)(((uint*)indexComp.histogram_d.addr) + csrManager.curNumCsr * 2);//reuse memory
	real *pGain_d = (real*)(((uint*)indexComp.histogram_d.addr) + csrManager.curNumCsr * (sizeof(real)/sizeof(uint) + 2));

	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[0];
	int numSeg = bagManager.m_numFea * numofSNode;
	//construct keys for exclusive scan
	checkCudaErrors(hipMemset(csrManager.getMutableCsrKey(), -1, sizeof(uint) * csrManager.curNumCsr));
//	checkCudaErrors(hipMemset(pCSRMultableKey, -1, sizeof(uint) * csrManager.curNumCsr));
	printf("done constructing key... number of segments is %d\n", numSeg);

	//set keys by GPU
	uint maxSegLen = 0;
	uint *pMaxLen = thrust::max_element(thrust::device, csrManager.pEachCsrFeaLen, csrManager.pEachCsrFeaLen + numSeg);
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(&maxSegLen, pMaxLen, sizeof(uint), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = numSeg;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	if(optimiseSetKey == false)
		SetKey<<<numSeg, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, csrManager.getMutableCsrKey());
//		(csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, pCSRMultableKey);
	else{
		if(numSeg < 1000000)
			SetKey<<<numSeg, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
				(csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, csrManager.getMutableCsrKey());
//			(csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, pCSRMultableKey);
		else{
			int numSegEachBlk = numSeg/10000;
			int numofBlkSetKey = (numSeg + numSegEachBlk - 1) / numSegEachBlk;
			SetKey<<<numofBlkSetKey, blockSize, 0, (*(hipStream_t*)pStream)>>>(csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen,
					numSegEachBlk, numSeg, csrManager.getMutableCsrKey());
//					numSegEachBlk, numSeg, pCSRMultableKey);
		}
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));
	hipDeviceSynchronize();


	//compute prefix sum for gd and hess (more than one arrays)
	thrust::inclusive_scan_by_key(thrust::device, csrManager.getCsrKey(), csrManager.getCsrKey() + csrManager.curNumCsr, pGD_d, pGD_d);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::device, csrManager.getCsrKey(), csrManager.getCsrKey() + csrManager.curNumCsr, pHess_d, pHess_d);

	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//compute gain; default to left or right
	bool *default2Right = (bool*)indexComp.partitionMarker.addr;
	checkCudaErrors(hipMemset(default2Right, 0, sizeof(bool) * csrManager.curNumCsr));//this is important (i.e. initialisation)
	checkCudaErrors(hipMemset(pGain_d, 0, sizeof(real) * csrManager.curNumCsr));

//	cout << "compute gain" << endl;
	printf("compute gain\n");
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(csrManager.curNumCsr, blockSizeComGain, dimNumofBlockToComGain);
	hipDeviceSynchronize();
	GETERROR("before ComputeGainDense");

	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag,
											bagManager.m_pPartitionId2SNPosEachBag,
											DeviceSplitter::m_lambda, pGD_d, pHess_d, csrManager.getCsrFvalue(),
											csrManager.curNumCsr, csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, csrManager.getCsrKey(), bagManager.m_numFea,
//											csrManager.curNumCsr, csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, pCSRKey, bagManager.m_numFea,
											pGain_d, default2Right);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");

	//change the gain of the first feature value to 0
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numSeg, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
											csrManager.pEachCsrFeaStartPos, numSeg, pGain_d, csrManager.curNumCsr);

	//	cout << "searching" << endl;
	hipDeviceSynchronize();
	clock_t start_search = clock();
	real *pMaxGain_d;
	uint *pMaxGainKey_d;
	checkCudaErrors(hipMalloc((void**)&pMaxGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pMaxGainKey_d, sizeof(uint) * numofSNode));
	checkCudaErrors(hipMemset(pMaxGainKey_d, -1, sizeof(uint) * numofSNode));
	//compute # of blocks for each node
	uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, csrManager.pEachNodeSizeInCsr, csrManager.pEachNodeSizeInCsr + numofSNode);
	checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
	printf("max fvalue one node=%d\n", maxNumFeaValueOneNode);
	SegmentedMax(maxNumFeaValueOneNode, numofSNode, csrManager.pEachNodeSizeInCsr, csrManager.pEachCsrNodeStartPos,
				 pGain_d, pStream, pMaxGain_d, pMaxGainKey_d);

	printf("finding split info\n");
	//find the split value and feature
	FindSplitInfo<<<1, numofSNode, 0>>>(
										 csrManager.pEachCsrFeaStartPos,
										 csrManager.pEachCsrFeaLen,
										 csrManager.getCsrFvalue(),
										 pMaxGain_d, pMaxGainKey_d,
										 bagManager.m_pPartitionId2SNPosEachBag, nNumofFeature,
					  	  	  	  	  	 bagManager.m_pSNodeStatEachBag,
					  	  	  	  	  	 pGD_d, pHess_d,
					  	  	  	  	  	 default2Right, csrManager.getCsrKey(),
//					  	  	  	  	  	 default2Right, pCSRKey,
					  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag,
					  	  	  	  	  	 bagManager.m_pRChildStatEachBag,
					  	  	  	  	  	 bagManager.m_pLChildStatEachBag);
	hipDeviceSynchronize();
	checkCudaErrors(hipFree(pMaxGain_d));
	checkCudaErrors(hipFree(pMaxGainKey_d));
}

void AllNode2CompGD(GBDTGPUMemManager &manager, BagCsrManager &csrManager, BagManager &bagManager, IndexComputer &indexComp, KernelConf &conf, void *pStream,
		int &curNumofNode, double &total_com_idx_t, double &total_fill_gd_t, int &maxNumFeaValueOneNode, double &total_csr_len_t){
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
	int nNumofFeature = manager.m_numofFea;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[0];
	PROCESS_ERROR(nNumofFeature > 0);
	if(curNumofNode == 1){
		checkCudaErrors(hipMemcpy(csrManager.preFvalueInsId, manager.m_pDInsId, sizeof(int) * bagManager.m_numFeaValue, hipMemcpyDeviceToDevice));
		numofDenseValue_previous = bagManager.m_numFeaValue;//initialise dense value length
	}

	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	clock_t csr_len_t = clock();
	if(numofSNode > 1)
	{
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, 0);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d;
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		hipDeviceSynchronize();
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		PROCESS_ERROR(bagManager.m_numFeaValue >= csrManager.curNumCsr);
		//split nodes
		csr_len_t = clock();

		if(indexComp.partitionMarker.reservedSize < csrManager.curNumCsr * 8){//make sure enough memory for reuse
			printf("reallocate memory for marker (sn=%d): %u v.s. %u.......\n", numofSNode, indexComp.partitionMarker.reservedSize/8, csrManager.curNumCsr);
			indexComp.partitionMarker.reserveSpace(csrManager.curNumCsr * 8, sizeof(unsigned char));
		}
		uint *pOldCsrLen_d = (uint*)indexComp.partitionMarker.addr;
		unsigned char *pCsrId2Pid = (unsigned char*)(((uint*)indexComp.partitionMarker.addr) + csrManager.curNumCsr);
		checkCudaErrors(hipMemcpy(pOldCsrLen_d, csrManager.getCsrLen(), sizeof(uint) * csrManager.curNumCsr, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemset(pCsrId2Pid, (int)-1, sizeof(char) * csrManager.curNumCsr));

		thrust::exclusive_scan(thrust::device, csrManager.getCsrLen(), csrManager.getCsrLen() + csrManager.curNumCsr, csrManager.getMutableCsrStart());

uint *pCsrNewLen_d;// = (uint*)(indexComp.histogram_d.addr);
//uint *pCsrNewLen_d = (uint*)(indexComp.histogram_d.addr);
checkCudaErrors(hipHostMalloc((void**)&pCsrNewLen_d, sizeof(uint) * csrManager.curNumCsr * 2));
		checkCudaErrors(hipMemset(pCsrNewLen_d, 0, sizeof(uint) * csrManager.curNumCsr * 2));
		checkCudaErrors(hipMemset(csrManager.pEachCsrFeaLen, 0, sizeof(uint) * bagManager.m_numFea * numofSNode));
		dim3 dimNumofBlockToCsrLen;
		uint blockSizeCsrLen = 128;

hipDeviceSynchronize();
		dimNumofBlockToCsrLen.x = (numofDenseValue_previous + blockSizeCsrLen - 1) / blockSizeCsrLen;
		newCsrLenFvalue<<<dimNumofBlockToCsrLen, blockSizeCsrLen, blockSizeCsrLen * sizeof(uint)>>>(
				csrManager.preFvalueInsId, numofDenseValue_previous,
				bagManager.m_pInsIdToNodeIdEachBag,
				bagManager.m_pPreMaxNid_h[0], csrManager.getCsrStart(),
				csrManager.getCsrFvalue(), csrManager.curNumCsr,
				csrManager.pEachCsrFeaStartPos, bagManager.m_pPreNumSN_h[0],
				bagManager.m_numFea, csrManager.getCsrKey(), pCsrNewLen_d, pCsrId2Pid);
//				bagManager.m_numFea, pCSRKey, pCsrNewLen_d, pCsrId2Pid);

		GETERROR("after newCsrLenFvalue");
		LoadFvalueInsId<<<dimNumofBlockToLoadGD, blockSizeLoadGD>>>(
						bagManager.m_numIns, manager.m_pDInsId, csrManager.preFvalueInsId, bagManager.m_pIndicesEachBag_d, bagManager.m_numFeaValue);
		GETERROR("after LoadFvalueInsId");

		printf("filling fvalue\n");
		hipDeviceSynchronize();

		real *pCsrFvalueSpare = (real*)(((int*)indexComp.histogram_d.addr) + csrManager.curNumCsr * 2);//reuse memory

		int blockSizeFillFvalue;
		dim3 dimNumBlockToFillFvalue;
		conf.ConfKernel(csrManager.curNumCsr, blockSizeFillFvalue, dimNumBlockToFillFvalue);
//fid hess sum
uint *hess_cnt_d;
checkCudaErrors(hipMalloc((void**)&hess_cnt_d, sizeof(uint) * bagManager.m_numFea));
checkCudaErrors(hipMemset(hess_cnt_d, 0, sizeof(uint) * bagManager.m_numFea));
		fillFvalue<<<dimNumBlockToFillFvalue, blockSizeFillFvalue>>>(csrManager.getCsrFvalue(), csrManager.curNumCsr, csrManager.pEachCsrFeaStartPos,
				   bagManager.m_pPreNumSN_h[0], bagManager.m_numFea, csrManager.getCsrKey(), pOldCsrLen_d, pCsrId2Pid,
//				   bagManager.m_pPreNumSN_h[bagId], bagManager.m_numFea, pCSRKey, pOldCsrLen_d, pCsrId2Pid,
				   pCsrFvalueSpare, pCsrNewLen_d, csrManager.pEachCsrFeaLen);
		GETERROR("after fillFvalue");
		hipDeviceSynchronize();

		//compute number of CSR in each node
		checkCudaErrors(hipMemset(csrManager.pEachNodeSizeInCsr, 0, sizeof(uint) * bagManager.m_maxNumSplittable));
		printf("done filling\n");
		dim3 dimNumSeg;
		dimNumSeg.x = numofSNode;
		uint blockSize = 128;
		segmentedSum<<<dimNumSeg, blockSize, blockSize * sizeof(uint)>>>(csrManager.pEachCsrFeaLen, bagManager.m_numFea, csrManager.pEachNodeSizeInCsr);
		GETERROR("after segmentedSum");

		int blockSizeLoadCsrLen;
		dim3 dimNumofBlockToLoadCsrLen;
		conf.ConfKernel(csrManager.curNumCsr * 2, blockSizeLoadCsrLen, dimNumofBlockToLoadCsrLen);
		//uint *pCsrMarker = (uint*)indexComp.partitionMarker.addr;
uint *pCsrMarker;
checkCudaErrors(hipMalloc((void**)&pCsrMarker, sizeof(uint) * csrManager.curNumCsr * 2));
		checkCudaErrors(hipMemset(pCsrMarker, 0, sizeof(uint) * csrManager.curNumCsr * 2));
		map2One<<<dimNumofBlockToLoadCsrLen, blockSizeLoadCsrLen>>>(pCsrNewLen_d, csrManager.curNumCsr * 2, pCsrMarker);
		GETERROR("after map2One");
		hipDeviceSynchronize();
		thrust::inclusive_scan(thrust::device, pCsrMarker, pCsrMarker + csrManager.curNumCsr * 2, pCsrMarker);
		hipDeviceSynchronize();
		uint previousNumCsr = csrManager.curNumCsr;
		checkCudaErrors(hipMemcpy(&csrManager.curNumCsr, pCsrMarker + csrManager.curNumCsr * 2 - 1, sizeof(uint), hipMemcpyDefault));

		checkCudaErrors(hipMemset(csrManager.getMutableCsrLen(), 0, sizeof(uint) * csrManager.curNumCsr));
hipDeviceSynchronize();
		loadDenseCsr<<<dimNumofBlockToLoadCsrLen, blockSizeLoadCsrLen>>>(pCsrFvalueSpare, pCsrNewLen_d,
				previousNumCsr * 2, csrManager.curNumCsr, pCsrMarker,
				csrManager.getMutableCsrFvalue(), csrManager.getMutableCsrLen());
		GETERROR("after loadDenseCsr");
		printf("done load dense csr: number of csr is %d\n", csrManager.curNumCsr);
		thrust::exclusive_scan(thrust::device, csrManager.pEachCsrFeaLen, csrManager.pEachCsrFeaLen + numofSNode * bagManager.m_numFea, csrManager.pEachCsrFeaStartPos);
hipDeviceSynchronize();


		thrust::exclusive_scan(thrust::device, csrManager.pEachNodeSizeInCsr, csrManager.pEachNodeSizeInCsr + numofSNode, csrManager.pEachCsrNodeStartPos);
		numofDenseValue_previous = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);//number of dense fvalues.
		uint *pCsrStartCurRound = (uint*)indexComp.partitionMarker.addr;
		thrust::exclusive_scan(thrust::device, csrManager.getCsrLen(), csrManager.getCsrLen() + csrManager.curNumCsr, pCsrStartCurRound);
		PROCESS_ERROR(csrManager.curNumCsr <= bagManager.m_numFeaValue);
		hipDeviceSynchronize();
checkCudaErrors(hipFree(pCsrMarker));
		printf("exit if\n");
	}
	else
	{
		clock_t start_gd = clock();
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d,
									    sizeof(int) * nNumofFeature, pStream);

		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
		//###### compress
		hipDeviceSynchronize();
		CsrCompressor compressor;
		csrManager.curNumCsr = compressor.totalOrgNumCsr;
		compressor.CsrCompression(csrManager.curNumCsr, csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen,
								  csrManager.pEachNodeSizeInCsr, csrManager.pEachCsrNodeStartPos, csrManager.getMutableCsrFvalue(), csrManager.getMutableCsrLen());
	}
	//need to compute for every new tree
	printf("reserve memory\n");
	if(indexComp.histogram_d.reservedSize < csrManager.curNumCsr * (2 + 2 * sizeof(real)/sizeof(uint))){//make sure enough memory for reuse
		printf("reallocate memory for histogram (sn=%u): %u v.s. %u.......\n", numofSNode, indexComp.histogram_d.reservedSize,
				csrManager.curNumCsr * (2 + 2 * sizeof(real)/sizeof(uint)));
		indexComp.histogram_d.reserveSpace(csrManager.curNumCsr * (2 + 2 * sizeof(real)/sizeof(uint)), sizeof(uint));
	}
	hipDeviceSynchronize();
	double *pGD_d = (double*)indexComp.histogram_d.addr;//reuse memory; must be here, as curNumCsr may change in different level.
	real *pHess_d = (real*)(((uint*)indexComp.histogram_d.addr) + csrManager.curNumCsr * 2);//reuse memory
	checkCudaErrors(hipMemset(pGD_d, 0, sizeof(double) * csrManager.curNumCsr));
	checkCudaErrors(hipMemset(pHess_d, 0, sizeof(real) * csrManager.curNumCsr));
	dim3 dimNumofBlockForGD;
	dimNumofBlockForGD.x = csrManager.curNumCsr;
	uint blockSizeForGD = 64;
	uint sharedMemSizeForGD = blockSizeForGD * (sizeof(double) + sizeof(double));
	const uint *pCsrStartPos_d;
	if(numofSNode == 1)
		pCsrStartPos_d = CsrCompressor::pCsrStart_d;
	else
		pCsrStartPos_d = (uint*)indexComp.partitionMarker.addr;
	printf("comp gd and hess\n");


	ComputeGDHess<<<dimNumofBlockForGD, blockSizeForGD, sharedMemSizeForGD>>>(csrManager.getCsrLen(), pCsrStartPos_d,
			bagManager.m_pInsGradEachBag,
			bagManager.m_pInsHessEachBag,
			csrManager.preFvalueInsId, pGD_d, pHess_d);
	hipDeviceSynchronize();
	GETERROR("after ComputeGD");
	clock_t csr_len_end = clock();
	total_csr_len_t += (csr_len_end - csr_len_t);
}

void DeviceSplitter::FeaFinderAllNode2(void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	BagCsrManager csrManager(bagManager.m_numFea, bagManager.m_maxNumSplittable, bagManager.m_numFeaValue);

	IndexComputer indexComp;
	indexComp.AllocMem(bagManager.m_numFea, numofSNode, bagManager.m_maxNumSplittable);
	KernelConf conf;
	hipDeviceSynchronize();

	int curNumofNode;
	manager.MemcpyDeviceToHostAsync(bagManager.m_pCurNumofNodeTreeOnTrainingEachBag_d + bagId, &curNumofNode, sizeof(int), pStream);
	int maxNumFeaValueOneNode = -1;
	AllNode2CompGD(manager, csrManager, bagManager, indexComp, conf, pStream, curNumofNode, total_com_idx_t, total_fill_gd_t, maxNumFeaValueOneNode, total_csr_len_t);

	AfterCompression(manager, csrManager, bagManager, indexComp, conf, pStream, total_scan_t, maxNumFeaValueOneNode);
}


int *preFvalueInsId = NULL;
uint totalNumCsrFvalue_merge;
uint *eachCompressedFeaStartPos_merge;
uint *eachCompressedFeaLen_merge;
double *csrGD_h_merge;
real *csrHess_h_merge;
uint *eachNodeSizeInCsr_merge;
uint *eachCsrNodeStartPos_merge;
real *csrFvalue_merge;
uint *eachCsrLen_merge;
uint *eachNewCompressedFeaLen_merge;
uint *eachNewCompressedFeaStart_merge;

void AllNode3CompGD(GBDTGPUMemManager &manager, BagCsrManager &csrManager, BagManager &bagManager, IndexComputer &indexComp, KernelConf &conf, void *pStream,
					int &curNumofNode, double &total_com_idx_t, double &total_fill_gd_t, int &maxNumFeaValueOneNode){
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[0];
	int nNumofFeature = manager.m_numofFea;
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
	vector<vector<real> > newCsrFvalue(numofSNode * bagManager.m_numFea, vector<real>());
		vector<vector<uint> > eachNewCsrLen(numofSNode * bagManager.m_numFea, vector<uint>());

		if(preFvalueInsId == NULL || curNumofNode == 1){
			eachNewCompressedFeaLen_merge = new uint[bagManager.m_numFea * bagManager.m_maxNumSplittable];
			eachNewCompressedFeaStart_merge = new uint[bagManager.m_numFea * bagManager.m_maxNumSplittable];
			eachCompressedFeaStartPos_merge = new uint[bagManager.m_numFea * bagManager.m_maxNumSplittable];
			eachCompressedFeaLen_merge = new uint[bagManager.m_numFea * bagManager.m_maxNumSplittable];
			csrGD_h_merge = new double[bagManager.m_numFeaValue];
			csrHess_h_merge = new real[bagManager.m_numFeaValue];
			eachNodeSizeInCsr_merge = new uint[bagManager.m_maxNumSplittable];
			eachCsrNodeStartPos_merge = new uint[bagManager.m_maxNumSplittable];
			csrFvalue_merge = new real[bagManager.m_numFeaValue];
			eachCsrLen_merge = new uint[bagManager.m_numFeaValue];
			checkCudaErrors(hipHostMalloc((void**)&preFvalueInsId, sizeof(int) * bagManager.m_numFeaValue));
			checkCudaErrors(hipMemcpy(preFvalueInsId, manager.m_pDInsId, sizeof(int) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
		}
		//split nodes
		int *pInsId2Nid = new int[bagManager.m_numIns];//ins id to node id
		checkCudaErrors(hipMemcpy(pInsId2Nid, bagManager.m_pInsIdToNodeIdEachBag, sizeof(int) * bagManager.m_numIns, hipMemcpyDeviceToHost));
		//################3

		//reset memory for this bag
		{
			manager.MemsetAsync(fgd_d, 0, sizeof(double) * bagManager.m_numFeaValue, pStream);
			manager.MemsetAsync(fhess_d, 0, sizeof(real) * bagManager.m_numFeaValue, pStream);
			manager.MemsetAsync(fgain_d, 0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		}
		hipStreamSynchronize((*(hipStream_t*)pStream));

		//compute index for each feature value
		int blockSizeLoadGD;
		dim3 dimNumofBlockToLoadGD;
		conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
		//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
		int numofDenseValue = -1;
		if(numofSNode > 1)
		{
			//####################
			printf("total csr fvalue=%u\n", totalNumCsrFvalue_merge);/**/
			//split nodes
			PROCESS_ERROR(bagManager.m_numFeaValue >= totalNumCsrFvalue_merge);
			memset(eachNewCompressedFeaLen_merge, 0, sizeof(uint) * bagManager.m_numFea * numofSNode);
			uint globalFvalueId = 0;
			clock_t extra_start = clock();
			for(int csrId = 0; csrId < totalNumCsrFvalue_merge; csrId++){
				uint csrLen = eachCsrLen_merge[csrId];
				//fid of this csr
				int fid = -1;
				for(int segId = 0; segId < bagManager.m_numFea * numofSNode; segId++){
					uint segStart = eachCompressedFeaStartPos_merge[segId];
					uint feaLen = eachCompressedFeaLen_merge[segId];
					if(csrId >= segStart && csrId < segStart + feaLen){
						fid = segId % bagManager.m_numFea;//may have problem here??
						break;
					}
				}
				PROCESS_ERROR(fid != -1 && fid < bagManager.m_numFea);

				//decompressed
				for(int i = 0; i < csrLen; i++){
					int insId = preFvalueInsId[globalFvalueId];
					globalFvalueId++;
					PROCESS_ERROR(insId >= 0);
					int pid = pInsId2Nid[insId] - bagManager.m_pPreMaxNid_h[0] - 1;//mapping to new node
					if(csrFvalue_merge[csrId] == 0.369250 && i == 0){
						printf("pid=%d, insId=%d, old csrLen=%d\n", pid, insId, csrLen);
					}
					if(pid < 0)
						continue;//############## this way okay?
					PROCESS_ERROR(pid >= 0 && pid < numofSNode);
					if(i == 0 || newCsrFvalue[pid * bagManager.m_numFea + fid].empty() ||
					   fabs(newCsrFvalue[pid * bagManager.m_numFea + fid].back() - csrFvalue_merge[csrId]) > DeviceSplitter::rt_eps){
						newCsrFvalue[pid * bagManager.m_numFea + fid].push_back(csrFvalue_merge[csrId]);
						eachNewCsrLen[pid * bagManager.m_numFea + fid].push_back(1);
						eachNewCompressedFeaLen_merge[pid * bagManager.m_numFea + fid]++;
					}
					else
						eachNewCsrLen[pid * bagManager.m_numFea + fid].back()++;
				}
			}
			clock_t extra_end = clock();
			total_extra_time += (double(extra_end - extra_start)/CLOCKS_PER_SEC);

			uint totalNewCsr = 0;
			for(int i = 0; i < numofSNode * bagManager.m_numFea; i++)
				totalNewCsr += eachNewCsrLen[i].size();
			printf("hello world org=%u v.s. csr=%u\n", bagManager.m_numFeaValue, totalNewCsr);
			thrust::exclusive_scan(thrust::host, eachNewCompressedFeaLen_merge, eachNewCompressedFeaLen_merge + numofSNode * bagManager.m_numFea, eachNewCompressedFeaStart_merge);
			delete[] pInsId2Nid;
			//###############################
			PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
			clock_t comIdx_start = clock();
			//compute gather index via GPUs
			indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, 0);
			clock_t comIdx_end = clock();
			total_com_idx_t += (comIdx_end - comIdx_start);

			//copy # of feature values of each node
			uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d;

			clock_t start_gd = clock();
			//scatter operation
			//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
			numofDenseValue = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
			LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag,
																   bagManager.m_pInsHessEachBag,
																   bagManager.m_numIns, manager.m_pDInsId, fvalue_org_d,
																   bagManager.m_pIndicesEachBag_d, numofDenseValue,
																   fgd_d, fhess_d, fvalue_d);
			hipStreamSynchronize((*(hipStream_t*)pStream));
			clock_t end_gd = clock();
			total_fill_gd_t += (end_gd - start_gd);
			uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
			checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
			//###########
			LoadFvalueInsId<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(
							manager.m_pDInsId, preFvalueInsId, bagManager.m_pIndicesEachBag_d, bagManager.m_numFeaValue);
			hipStreamSynchronize((*(hipStream_t*)pStream));
			//##############
		}
		else
		{
			clock_t start_gd = clock();
			LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag,
																   	   	bagManager.m_pInsHessEachBag, bagManager.m_numIns,
																   	   	manager.m_pDInsId, bagManager.m_numFeaValue,
																   		fgd_d, fhess_d);
			checkCudaErrors(hipMemcpy(fvalue_d, fvalue_org_d, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDefault));
			hipStreamSynchronize((*(hipStream_t*)pStream));
			clock_t end_gd = clock();
			total_fill_gd_t += (end_gd - start_gd);

			clock_t comIdx_start = clock();
			//copy # of feature values of a node
			manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d, sizeof(uint), pStream);
			//copy feature value start position of each node
			manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d, sizeof(uint), pStream);
			//copy each feature start position in each node
			manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d,
											sizeof(uint) * nNumofFeature, pStream);
			//copy # of feature values of each feature in each node
			manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d,
										    sizeof(int) * nNumofFeature, pStream);

			numofDenseValue = manager.m_numFeaValue;//for computing gain of each fvalue
			maxNumFeaValueOneNode = manager.m_numFeaValue;
			clock_t comIdx_end = clock();
			total_com_idx_t += (comIdx_end - comIdx_start);
		}

		//###### compress
		CsrCompression(numofSNode, totalNumCsrFvalue_merge, eachCompressedFeaStartPos_merge, eachCompressedFeaLen_merge,
					   eachNodeSizeInCsr_merge, eachCsrNodeStartPos_merge, csrFvalue_merge, csrGD_h_merge, csrHess_h_merge, eachCsrLen_merge);
		printf("total csr fvalue=%u\n", totalNumCsrFvalue_merge);

//test two ways of computing csr
		int segIdCnt = 0, innerCsrIdCnt = 0;
		if(numofSNode > 1)
		for(int csrId = 0; csrId < totalNumCsrFvalue_merge; csrId++){
			if(eachCsrLen_merge[csrId] != eachNewCsrLen[segIdCnt][innerCsrIdCnt]){
				printf("csrId=%d, segIdCnt=%d, innerCsrIdCnt=%d, segSize=%d\n", csrId, segIdCnt, innerCsrIdCnt, eachNewCsrLen[segIdCnt].size());
				printf("right csr len=%d v.s. wrong len=%d\n", eachCsrLen_merge[csrId], eachNewCsrLen[segIdCnt][innerCsrIdCnt]);
				printf("right fvalue=%f v.s. wrong fvalue=%f\n", csrFvalue_merge[csrId], newCsrFvalue[segIdCnt][innerCsrIdCnt]);
				printf("previous right fvalue=%f v.s. wrong fvalue=%f; right csr len=%d v.s. wrong len=%d\n", csrFvalue_merge[csrId - 1],
						newCsrFvalue[segIdCnt][innerCsrIdCnt - 1], eachCsrLen_merge[csrId - 1], eachNewCsrLen[segIdCnt][innerCsrIdCnt - 1]);
				printf("next right fvalue=%f v.s. wrong fvalue=%f; right csr len=%d v.s. wrong len=%d\n", csrFvalue_merge[csrId + 1],
						newCsrFvalue[segIdCnt][innerCsrIdCnt + 1], eachCsrLen_merge[csrId + 1], eachNewCsrLen[segIdCnt][innerCsrIdCnt + 1]);
				//exit(0);
			}
			innerCsrIdCnt++;
			if(eachNewCsrLen[segIdCnt].size() == innerCsrIdCnt){
				segIdCnt++;
				innerCsrIdCnt = 0;
			}
		}
//end test

		//	cout << "prefix sum" << endl;
		int numSeg = bagManager.m_numFea * numofSNode;

		csrManager.curNumCsr = totalNumCsrFvalue_merge;
		checkCudaErrors(hipMemcpy(csrManager.pEachCsrFeaStartPos, eachCompressedFeaStartPos_merge, sizeof(uint) * numSeg, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(csrManager.pEachCsrFeaLen, eachCompressedFeaLen_merge, sizeof(uint) * numSeg, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(csrManager.pCsrFvalue, csrFvalue_merge, sizeof(real) * totalNumCsrFvalue_merge, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(csrManager.getMutableCsrLen(), eachCsrLen_merge, sizeof(uint) * totalNumCsrFvalue_merge, hipMemcpyHostToDevice));
		if(indexComp.partitionMarker.reservedSize < csrManager.curNumCsr){//make sure enough memory for reuse
			indexComp.partitionMarker.reserveSpace(csrManager.curNumCsr, sizeof(bool));
		}
		if(indexComp.histogram_d.reservedSize < csrManager.curNumCsr * 4){//make sure enough memory for reuse
			indexComp.histogram_d.reserveSpace(csrManager.curNumCsr * 4, sizeof(uint));
		}
		double *pGD_d = (double*)indexComp.histogram_d.addr;//reuse memory; must be here, as curNumCsr may change in different level.
		real *pHess_d = (real*)(((uint*)indexComp.histogram_d.addr) + csrManager.curNumCsr * 2);//reuse memory

		checkCudaErrors(hipMemcpy(pHess_d, csrHess_h_merge, sizeof(real) * totalNumCsrFvalue_merge, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(pGD_d, csrGD_h_merge, sizeof(double) * totalNumCsrFvalue_merge, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(csrManager.pEachNodeSizeInCsr, eachNodeSizeInCsr_merge, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(csrManager.pEachCsrNodeStartPos, eachCsrNodeStartPos_merge, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));
		//compute the feature with the maximum number of values
		hipDeviceSynchronize();//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled
}

void DeviceSplitter::FeaFinderAllNode3(void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	BagCsrManager csrManager(bagManager.m_numFea, bagManager.m_maxNumSplittable, bagManager.m_numFeaValue);

	IndexComputer indexComp;
	indexComp.AllocMem(bagManager.m_numFea, numofSNode, bagManager.m_maxNumSplittable);
	KernelConf conf;
	hipDeviceSynchronize();

//	cout << bagManager.m_maxNumSplittable << endl;
	int curNumofNode;
	manager.MemcpyDeviceToHostAsync(bagManager.m_pCurNumofNodeTreeOnTrainingEachBag_d + bagId, &curNumofNode, sizeof(int), pStream);
	int maxNumFeaValueOneNode = -1;

	AllNode2CompGD(manager, csrManager, bagManager, indexComp, conf, pStream, curNumofNode, total_com_idx_t, total_fill_gd_t, maxNumFeaValueOneNode, total_csr_len_t);
SyncArray<real> csrFvalue(csrManager.curNumCsr);
csrFvalue.set_device_data(csrManager.getMutableCsrFvalue());
real *csrFvalue_h = csrFvalue.host_data();
SyncArray<uint> csrLen(csrManager.curNumCsr);
csrLen.set_device_data(csrManager.getMutableCsrLen());
uint *csrLen_h = csrLen.host_data();

	printf("done with fast csr\n");
	AllNode3CompGD(manager, csrManager, bagManager, indexComp, conf, pStream, curNumofNode, total_com_idx_t, total_fill_gd_t, maxNumFeaValueOneNode);

SyncArray<real> csrFvalue3(csrManager.curNumCsr);
csrFvalue3.set_device_data(csrManager.getMutableCsrFvalue());
real *csrFvalue_h3 = csrFvalue3.host_data();
SyncArray<uint> csrLen3(csrManager.curNumCsr);
csrLen3.set_device_data(csrManager.getMutableCsrLen());
uint *csrLen_h3 = csrLen3.host_data();
	printf("done with naive csr\n");

//compare two compression methods
	for(int i = 0; i < csrManager.curNumCsr; i++){
		if(csrFvalue_h[i] == 0.369250){
			printf("%f v.s. %f, id=%d; len: %d v.s. %d\n", csrFvalue_h[i], csrFvalue_h3[i], i, csrLen_h[i], csrLen_h3[i]);
		}
	}
	for(int i = 0; i < csrManager.curNumCsr; i++){
		if(csrFvalue_h3[i] == 0.369250){
			printf("%f v.s. %f, id=%d; len: %d v.s. %d\n", csrFvalue_h[i], csrFvalue_h3[i], i, csrLen_h[i], csrLen_h3[i]);
		}
	}

	int cnt = 0;
	for(int i = 0; i < csrManager.curNumCsr; i++){
		if(csrFvalue_h[i] != csrFvalue_h3[i]){
			printf("%f v.s. %f, id=%d; len: %d v.s. %d\n", csrFvalue_h[i], csrFvalue_h3[i], i, csrLen_h[i], csrLen_h3[i]);
			if(cnt == 0)
				printf("previous %f v.s. %f, id=%d; len: %d v.s. %d\n", csrFvalue_h[i-1], csrFvalue_h3[i-1], i-1, csrLen_h[i-1], csrLen_h3[i-1]);
			cnt++;
		}
		if(csrLen_h[i] != csrLen_h3[i]){
			printf("%f v.s. %f, id=%d; len: %d v.s. %d\n", csrFvalue_h[i], csrFvalue_h3[i], i, csrLen_h[i], csrLen_h3[i]);
			if(cnt == 0)
				printf("previous %f v.s. %f, id=%d; len: %d v.s. %d\n", csrFvalue_h[i-1], csrFvalue_h3[i-1], i-1, csrLen_h[i-1], csrLen_h3[i-1]);
			cnt++;
		}

		if(cnt == 10)
			exit(0);
	}
//end comparison

	AfterCompression(manager, csrManager, bagManager, indexComp, conf, pStream, total_scan_t, maxNumFeaValueOneNode);
}


