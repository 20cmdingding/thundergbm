#include "hip/hip_runtime.h"
/*
 * FindFeaCsr.cu
 *
 *  Created on: Jul 28, 2017
 *      Author: zeyi
 */

#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Bagging/BagManager.h"
#include "../CSR/BagCsrManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/segmentedMax.h"
#include "../../SharedUtility/segmentedSum.h"
#include "../../SharedUtility/setSegmentKey.h"

#include "../CSR/CsrSplit.h"
#include "../CSR/CsrCompressor.h"
uint numofDenseValue_previous;
void DeviceSplitter::FeaFinderAllNode2(void *pStream, int bagId)
{
	hipDeviceSynchronize();
	GBDTGPUMemManager manager;
	BagManager bagManager;
	BagCsrManager csrManager(bagManager.m_numFea, bagManager.m_maxNumSplittable, bagManager.m_numFeaValue);
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];

	IndexComputer indexComp;
	indexComp.AllocMem(bagManager.m_numFea, numofSNode, bagManager.m_maxNumSplittable);
	
	double *pGD_d;
	real *pHess_d;//reuse memory

	int maxNumofSplittable = bagManager.m_maxNumSplittable;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);
	int curNumofNode;
	manager.MemcpyDeviceToHostAsync(bagManager.m_pCurNumofNodeTreeOnTrainingEachBag_d + bagId, &curNumofNode, sizeof(int), pStream);

	if(curNumofNode == 1){
		checkCudaErrors(hipMemcpy(csrManager.preFvalueInsId, manager.m_pDInsId, sizeof(int) * bagManager.m_numFeaValue, hipMemcpyDeviceToDevice));
		numofDenseValue_previous = bagManager.m_numFeaValue;//initialise dense value length
	}

	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	int maxNumFeaValueOneNode = -1;
	clock_t csr_len_t;
	if(numofSNode > 1)
	{
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		hipDeviceSynchronize();
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		indexComp.FreeMem();
		PROCESS_ERROR(bagManager.m_numFeaValue >= csrManager.curNumCsr);
		//split nodes
		csr_len_t = clock();

		//for testing; need to optimise later
		checkCudaErrors(hipMemcpy(csrManager.getMutableCsrOldLen(), csrManager.getCsrLen(), sizeof(uint) * csrManager.curNumCsr, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemset(csrManager.getMutableCsrId2Pid(), (int)-1, sizeof(char) * csrManager.curNumCsr));

		thrust::exclusive_scan(thrust::device, csrManager.getCsrLen(), csrManager.getCsrLen() + csrManager.curNumCsr, csrManager.getMutableCsrStart());

		uint *pCsrNewLen_d = indexComp.m_pHistogram_d;
		checkCudaErrors(hipMemset(pCsrNewLen_d, 0, sizeof(uint) * csrManager.curNumCsr * 2));
		checkCudaErrors(hipMemset(csrManager.pEachCsrFeaLen, 0, sizeof(uint) * bagManager.m_numFea * numofSNode));
		dim3 dimNumofBlockToCsrLen;
		uint blockSizeCsrLen = 128;
		dimNumofBlockToCsrLen.x = (numofDenseValue_previous + blockSizeCsrLen - 1) / blockSizeCsrLen;
		newCsrLenFvalue<<<dimNumofBlockToCsrLen, blockSizeCsrLen, blockSizeCsrLen * sizeof(uint)>>>(
				csrManager.preFvalueInsId, numofDenseValue_previous,
				bagManager.m_pInsIdToNodeIdEachBag + bagId * bagManager.m_numIns,
				bagManager.m_pPreMaxNid_h[bagId], csrManager.getCsrStart(),
				csrManager.getCsrFvalue(), csrManager.curNumCsr,
				csrManager.pEachCsrFeaStartPos, bagManager.m_pPreNumSN_h[bagId],
				bagManager.m_numFea, csrManager.getCsrKey(), pCsrNewLen_d, csrManager.getMutableCsrId2Pid());

		GETERROR("after newCsrLenFvalue");
		LoadFvalueInsId<<<dimNumofBlockToLoadGD, blockSizeLoadGD>>>(
						bagManager.m_numIns, manager.m_pDInsId, csrManager.preFvalueInsId, bagManager.m_pIndicesEachBag_d, bagManager.m_numFeaValue);
		GETERROR("after LoadFvalueInsId");

		real *pCsrFvalueSpare = (real*)(indexComp.m_pHistogram_d + csrManager.curNumCsr * 2);//reuse memory

		int blockSizeFillFvalue;
		dim3 dimNumBlockToFillFvalue;
		conf.ConfKernel(csrManager.curNumCsr, blockSizeFillFvalue, dimNumBlockToFillFvalue);
		fillFvalue<<<dimNumBlockToFillFvalue, blockSizeFillFvalue>>>(csrManager.getCsrFvalue(), csrManager.curNumCsr, csrManager.pEachCsrFeaStartPos,
				   bagManager.m_pPreNumSN_h[bagId], bagManager.m_numFea, csrManager.getCsrKey(), csrManager.getCsrOldLen(), csrManager.getCsrId2Pid(),
				   pCsrFvalueSpare, pCsrNewLen_d, csrManager.pEachCsrFeaLen);
		GETERROR("after fillFvalue");
		//compute number of CSR in each node
		checkCudaErrors(hipMemset(csrManager.pEachNodeSizeInCsr, 0, sizeof(uint) * bagManager.m_maxNumSplittable));
		dim3 dimNumSeg;
		dimNumSeg.x = numofSNode;
		uint blockSize = 128;
		segmentedSum<<<dimNumSeg, blockSize, blockSize * sizeof(uint)>>>(csrManager.pEachCsrFeaLen, bagManager.m_numFea, csrManager.pEachNodeSizeInCsr);
		GETERROR("after segmentedSum");

		int blockSizeLoadCsrLen;
		dim3 dimNumofBlockToLoadCsrLen;
		conf.ConfKernel(csrManager.curNumCsr * 2, blockSizeLoadCsrLen, dimNumofBlockToLoadCsrLen);
		checkCudaErrors(hipMemset(csrManager.getMutableCsrMarker(), 0, sizeof(uint) * csrManager.curNumCsr * 2));
		map2One<<<dimNumofBlockToLoadCsrLen, blockSizeLoadCsrLen>>>(pCsrNewLen_d, csrManager.curNumCsr * 2, csrManager.getMutableCsrMarker());
		GETERROR("after map2One");
		thrust::inclusive_scan(thrust::device, csrManager.getMutableCsrMarker(), csrManager.getMutableCsrMarker() + csrManager.curNumCsr * 2, csrManager.getMutableCsrMarker());
		hipDeviceSynchronize();
		uint previousNumCsr = csrManager.curNumCsr;
		checkCudaErrors(hipMemcpy(&csrManager.curNumCsr, csrManager.getCsrMarker() + csrManager.curNumCsr * 2 - 1, sizeof(uint), hipMemcpyDefault));

		checkCudaErrors(hipMemset(csrManager.getMutableCsrLen(), 0, sizeof(uint) * csrManager.curNumCsr));
		loadDenseCsr<<<dimNumofBlockToLoadCsrLen, blockSizeLoadCsrLen>>>(pCsrFvalueSpare, pCsrNewLen_d,
				previousNumCsr * 2, csrManager.curNumCsr, csrManager.getCsrMarker(),
				csrManager.getMutableCsrFvalue(), csrManager.getMutableCsrLen());
		GETERROR("after loadDenseCsr");
		thrust::exclusive_scan(thrust::device, csrManager.pEachCsrFeaLen, csrManager.pEachCsrFeaLen + numofSNode * bagManager.m_numFea, csrManager.pEachCsrFeaStartPos);

		thrust::exclusive_scan(thrust::device, csrManager.pEachNodeSizeInCsr, csrManager.pEachNodeSizeInCsr + numofSNode, csrManager.pEachCsrNodeStartPos);
		numofDenseValue_previous = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);//number of dense fvalues.
		thrust::exclusive_scan(thrust::device, csrManager.getCsrLen(), csrManager.getCsrLen() + csrManager.curNumCsr, csrManager.getMutableCsrStartCurRound());
		PROCESS_ERROR(csrManager.curNumCsr <= bagManager.m_numFeaValue);
		hipDeviceSynchronize();
	}
	else
	{
		clock_t start_gd = clock();
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
		//###### compress
		hipDeviceSynchronize();
		CsrCompressor compressor;
		compressor.CsrCompression(csrManager.curNumCsr, csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen,
								  csrManager.pEachNodeSizeInCsr, csrManager.pEachCsrNodeStartPos);
	}
	//need to compute for every new tree
	if(indexComp.numIntMem < csrManager.curNumCsr * 4){//make sure enough memory for reuse
		checkCudaErrors(hipFree(indexComp.m_pHistogram_d));
		printf("reallocate memory for histogram (sn=1): %u v.s. %u.......\n", indexComp.numIntMem, csrManager.curNumCsr * 6);
		indexComp.numIntMem = csrManager.curNumCsr * 4 * 1.5;
		checkCudaErrors(hipMalloc((void**)&indexComp.m_pHistogram_d, sizeof(uint) * indexComp.numIntMem));
	}
	hipDeviceSynchronize();
	pGD_d = (double*)indexComp.m_pHistogram_d;//reuse memory; must be here, as curNumCsr may change in different level.
	pHess_d = (real*)(indexComp.m_pHistogram_d + csrManager.curNumCsr * 2);//reuse memory
	checkCudaErrors(hipMemset(pGD_d, 0, sizeof(double) * csrManager.curNumCsr));
	checkCudaErrors(hipMemset(pHess_d, 0, sizeof(real) * csrManager.curNumCsr));
	dim3 dimNumofBlockForGD;
	dimNumofBlockForGD.x = csrManager.curNumCsr;
	uint blockSizeForGD = 64;
	uint sharedMemSizeForGD = blockSizeForGD * (sizeof(double) + sizeof(real));
	const uint *pCsrStartPos_d;
	if(numofSNode == 1)
		pCsrStartPos_d = CsrCompressor::pCsrStart_d;
	else
		pCsrStartPos_d = csrManager.getCsrStartCurRound();
	ComputeGDHess<<<dimNumofBlockForGD, blockSizeForGD, sharedMemSizeForGD>>>(csrManager.getCsrLen(), pCsrStartPos_d,
			bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
			bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns,
			csrManager.preFvalueInsId, pGD_d, pHess_d);
	hipDeviceSynchronize();
	GETERROR("after ComputeGD");
	clock_t csr_len_end = clock();
	total_csr_len_t += (csr_len_end - csr_len_t);

	//cout << "prefix sum" << endl;
	int numSeg = bagManager.m_numFea * numofSNode;
	clock_t start_scan = clock();

	//construct keys for exclusive scan
	checkCudaErrors(hipMemset(csrManager.getMutableCsrKey(), -1, sizeof(uint) * csrManager.curNumCsr));

	//set keys by GPU
	uint maxSegLen = 0;
	uint *pMaxLen = thrust::max_element(thrust::device, csrManager.pEachCsrFeaLen, csrManager.pEachCsrFeaLen + numSeg);
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpyAsync(&maxSegLen, pMaxLen, sizeof(uint), hipMemcpyDeviceToHost, (*(hipStream_t*)pStream)));
	hipStreamSynchronize((*(hipStream_t*)pStream));

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = numSeg;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<numSeg, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, csrManager.getMutableCsrKey());
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	thrust::inclusive_scan_by_key(thrust::device, csrManager.getCsrKey(), csrManager.getCsrKey() + csrManager.curNumCsr,
								  pGD_d, pGD_d);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::device, csrManager.getCsrKey(), csrManager.getCsrKey() + csrManager.curNumCsr,
			pHess_d, pHess_d);
	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//compute gain; default to left or right
	checkCudaErrors(hipMemset(csrManager.getMutableDefault2Right(), 0, sizeof(bool) * csrManager.curNumCsr));//this is important (i.e. initialisation)
	checkCudaErrors(hipMemset(csrManager.getMutableCsrGain(), 0, sizeof(real) * csrManager.curNumCsr));

//	cout << "compute gain" << endl;
	uint test = thrust::reduce(thrust::device, csrManager.pEachCsrFeaLen, csrManager.pEachCsrFeaLen + numSeg);
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(csrManager.curNumCsr, blockSizeComGain, dimNumofBlockToComGain);
	hipDeviceSynchronize();
	GETERROR("before ComputeGainDense");
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, pGD_d, pHess_d, csrManager.getCsrFvalue(),
											csrManager.curNumCsr, csrManager.pEachCsrFeaStartPos, csrManager.pEachCsrFeaLen, csrManager.getCsrKey(), bagManager.m_numFea,
											csrManager.getMutableCsrGain(), csrManager.getMutableDefault2Right());
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");

	//change the gain of the first feature value to 0
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numSeg, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
			csrManager.pEachCsrFeaStartPos, numSeg, csrManager.getMutableCsrGain(), csrManager.curNumCsr);

	//	cout << "searching" << endl;
	hipDeviceSynchronize();
	clock_t start_search = clock();
	real *pMaxGain_d;
	uint *pMaxGainKey_d;
	checkCudaErrors(hipMalloc((void**)&pMaxGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pMaxGainKey_d, sizeof(uint) * numofSNode));
	checkCudaErrors(hipMemset(pMaxGainKey_d, -1, sizeof(uint) * numofSNode));
	//compute # of blocks for each node
	uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, csrManager.pEachNodeSizeInCsr, csrManager.pEachNodeSizeInCsr + numofSNode);
	checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
	SegmentedMax(maxNumFeaValueOneNode, numofSNode, csrManager.pEachNodeSizeInCsr, csrManager.pEachCsrNodeStartPos,
				 csrManager.getCsrGain(), pStream, pMaxGain_d, pMaxGainKey_d);

	//find the split value and feature
	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
										 csrManager.pEachCsrFeaStartPos,
										 csrManager.pEachCsrFeaLen,
										 csrManager.getCsrFvalue(),
										 pMaxGain_d, pMaxGainKey_d,
										 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
					  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 pGD_d,
					  	  	  	  	pHess_d,
					  	  	  	  	  	 csrManager.getDefault2Right(), csrManager.getCsrKey(),
					  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pMaxGain_d));
	checkCudaErrors(hipFree(pMaxGainKey_d));
}
