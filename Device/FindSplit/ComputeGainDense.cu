#include "hip/hip_runtime.h"
/*
 * ComputeGainDense.cu
 *
 *  Created on: 22 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: kernels gain computing using dense arrays
 */

#include <stdio.h>
#include <float.h>
#include <limits>
#include "FindFeaKernel.h"
#include "../KernelConst.h"
#include "../../DeviceHost/svm-shared/DeviceUtility.h"
#include "../Splitter/DeviceSplitter.h"

const float rt_2eps = 2.0 * DeviceSplitter::rt_eps;

#define testing


__global__ void ComputeIndex(int *pDstIndexEachFeaValue, long long totalFeaValue)
{
	long long gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(gTid >= totalFeaValue)
		return;
	pDstIndexEachFeaValue[gTid] = gTid;
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalueRoot(const float_point *pInsGD, const float_point *pInsHess, int numIns,
						   const int *pInsId, const float_point *pAllFeaValue, int numFeaValue,
						   float_point *pGDEachFeaValue, float_point *pHessEachFeaValue, float_point *pDenseFeaValue)
{
	//one thread loads one value
	//## global id looks ok, but need to be careful
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

#ifdef testing
	if(insId >= numIns)
		printf("Instance id is larger than the number of instances!\n");
#endif

	//store GD and Hess.
	pGDEachFeaValue[gTid] = pInsGD[insId];
	pHessEachFeaValue[gTid] = pInsHess[insId];
	pDenseFeaValue[gTid] = pAllFeaValue[gTid];
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalue(const float_point *pInsGD, const float_point *pInsHess, int numIns,
						   const int *pInsId, const float_point *pAllFeaValue, const int *pDstIndexEachFeaValue, int numFeaValue,
						   float_point *pGDEachFeaValue, float_point *pHessEachFeaValue, float_point *pDenseFeaValue)
{
	//one thread loads one value
	//## global id looks ok, but need to be careful
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

#ifdef testing
	if(insId >= numIns)
		printf("Instance id is larger than the number of instances!\n");
#endif

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

#ifdef testing
	if(idx < 0)
		printf("index to out array is negative!\n");
	if(idx >= numFeaValue)
		printf("index to out array is too large: %d. numFvalue=%d!\n", idx, numFeaValue);
#endif

	//scatter: store GD, Hess and the feature value.
	pGDEachFeaValue[idx] = pInsGD[insId];
	pHessEachFeaValue[idx] = pInsHess[insId];
	pDenseFeaValue[idx] = pAllFeaValue[gTid];
}

/**
 * @brief: compute the gain in parallel, each gain is computed by a thread.
 */
__global__ void ComputeGainDense(const nodeStat *pSNodeStat, const long long *pFeaValueStartPosEachNode, int numSN,
							const int *pBuffId, float_point lambda,
							const float_point *pGDPrefixSumOnEachFeaValue, const float_point *pHessPrefixSumOnEachFeaValue,
							const float_point *pDenseFeaValue, int numofDenseValue, float_point *pGainOnEachFeaValue)
{
	//one thread loads one value
	long long gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	//compute node id
	int snId = -1;
	for(int i = 0; i < numSN; i++)
	{
		if(i == numSN - 1)
		{
			snId = i;
			break;
		}
		else if(gTid >= pFeaValueStartPosEachNode[i] && gTid < pFeaValueStartPosEachNode[i + 1])
		{
			snId = i;
			break;
		}
	}
	int hashVaue = pBuffId[snId];
	if(hashVaue < 0)
		printf("Error in ComputeGain: buffer id %d, i=%d\n", hashVaue, snId);

	if(gTid >= numofDenseValue)//the thread has no gain to compute, i.e. a thread per gain
	{
		return;
	}

	if(gTid == 0)
	{
		//assign gain to 0 to the first feature value
    	pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	//if the previous fea value is the same as the current fea value, gain is 0 for the current fea value.
	if(fabs(pDenseFeaValue[gTid - 1] - pDenseFeaValue[gTid]) <= rt_2eps)
	{//avoid same feature value different gain issue
		pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	int exclusiveSumPos = gTid - 1;//following xgboost using exclusive sum on gd and hess

	float_point rChildGD = pGDPrefixSumOnEachFeaValue[exclusiveSumPos];
	float_point rChildHess = pHessPrefixSumOnEachFeaValue[exclusiveSumPos];
	float_point parentGD = pSNodeStat[hashVaue].sum_gd;
	float_point parentHess = pSNodeStat[hashVaue].sum_hess;
	float_point tempGD = parentGD - rChildGD;
	float_point tempHess = parentHess - rChildHess;
	bool needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true)//need to compute the gain
    {
		float_point tempGain = (tempGD * tempGD)/(tempHess + lambda) + 
						  	   (rChildGD * rChildGD)/(rChildHess + lambda) -
	  						   (parentGD * parentGD)/(parentHess + lambda);
    	pGainOnEachFeaValue[gTid] = tempGain; 
//    	if(pGainOnEachFeaValue[gTid] > 0 && ((rChildHess == 463714 && tempHess == 1) || (rChildHess == 1 && tempHess == 463714)))
 //   		printf("gain=%f, gid=%d, rhess=%f, lhess=%f\n", pGainOnEachFeaValue[gTid], gTid, rChildHess, tempHess);
    }
    else
    {
    	//assign gain to 0
    	pGainOnEachFeaValue[gTid] = 0;
    }
}

/**
 * @brief: change the gain of the first value of each feature to 0
 */
__global__ void FirstFeaGain(const long long *pEachFeaStartPosEachNode, int numFeaStartPos, float_point *pGainOnEachFeaValue)
{
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if(gTid >= numFeaStartPos)//no gain to fix
	{
		return;
	}
	long long gainPos = pEachFeaStartPosEachNode[gTid];
//	printf("gTid=%d, gainPos=%ld\n", gTid, gainPos);
//	printf("change %f to 0 pos at %d, gainPos=%ld\n", pGainOnEachFeaValue[gainPos], pEachFeaStartPosEachNode[gTid], gainPos);
	pGainOnEachFeaValue[gainPos] = 0;
//	if(gTid == 0){
//		printf("pEachFeaStartPosEachNode[8]=%f\n", pEachFeaStartPosEachNode[8]);
//	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 * Each block.y processes one node, a thread processes a reduction.
 */
__global__ void PickLocalBestSplitEachNode(const long long *pnNumFeaValueEachNode, const long long *pFeaStartPosEachNode,
										   const float_point *pGainOnEachFeaValue,
								   	   	   float_point *pfLocalBestGain, int *pnLocalBestGainKey)
{
	//best gain of each node is search by a few blocks
	//blockIdx.z corresponds to a splittable node id
	int snId = blockIdx.z;

	__shared__ float_point pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;
	if(localTid == 0){//initialise local best value
		int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
		pfLocalBestGain[blockId] = FLT_MAX;
		pnLocalBestGainKey[blockId] = -1;
	}

	long long numValueThisNode = pnNumFeaValueEachNode[snId];//get the number of feature value of this node
	long long tidForEachNode = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	long long nPos = pFeaStartPosEachNode[snId] + tidForEachNode;//feature value gain position
	if(nPos < 0)
		printf("sp pos is nagative! %d\n", nPos);
	if(tidForEachNode >= numValueThisNode){//no gain to load
		return;
	}
	pfGain[localTid] = -pGainOnEachFeaValue[nPos];//change to find min of -gain
	pnBetterGainKey[localTid] = nPos;//############ need to be long long
	__syncthreads();

	//find the local best split point
	GetMinValue(pfGain, pnBetterGainKey, blockDim.x);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
		pfLocalBestGain[blockId] = pfGain[0];
		pnLocalBestGainKey[blockId] = pnBetterGainKey[0];
		if(pnBetterGainKey[0] < 0)
			printf("negative key: snId=%d, blockId=%d, gain=%f, key=%d\n", snId, blockId, pfGain[0], pnBetterGainKey[0]);
	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 */
__global__ void PickGlobalBestSplitEachNode(const float_point *pfLocalBestGain, const int *pnLocalBestGainKey,
								   	   	    float_point *pfGlobalBestGain, int *pnGlobalBestGainKey,
								   	   	    int numBlockPerNode, int numofSNode)
{
	//a block for finding the best gain of a node
	int blockId = blockIdx.x;

	int snId = blockId;
	if(blockIdx.y > 1)
		printf("One block is not enough to find global best split.\n");

	if(snId >= numofSNode)
		printf("Error in PickBestFea: kernel split %d nods, but only %d splittable nodes\n", snId, numofSNode);

	__shared__ float_point pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;

	if(localTid >= numBlockPerNode)//number of threads is larger than the number of blocks
	{
		return;
	}

	int curFeaLocalBestStartPos = snId * numBlockPerNode;

	LoadToSharedMem(numBlockPerNode, curFeaLocalBestStartPos, pfLocalBestGain, pnLocalBestGainKey, pfGain, pnBetterGainKey);
	 __syncthreads();	//wait until the thread within the block

	//find the local best split point
	GetMinValue(pfGain, pnBetterGainKey, blockDim.x);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		pfGlobalBestGain[snId] = -pfGain[0];//make the gain back to its original sign
		pnGlobalBestGainKey[snId] = pnBetterGainKey[0];
		if(pnBetterGainKey[0] < 0)
			printf("negative key: snId=%d, gain=%f, key=%d, blockDim.x=%d, blockSize=%d, blockpPerNode=%d\n", snId, pfGain[0], pnBetterGainKey[0], blockDim.x, BLOCK_SIZE, numBlockPerNode);
	}
}

/**
 * @brief: find split points
 */
__global__ void FindSplitInfo(const long long *pEachFeaStartPosEachNode, const int *pEachFeaLenEachNode,
							  const float_point *pDenseFeaValue, const float_point *pfGlobalBestGain, const int *pnGlobalBestGainKey,
							  const int *pPosToBuffId, const int numFea,
							  const nodeStat *snNodeStat, const float_point *pPrefixSumGD, const float_point *pPrefixSumHess,
							  SplitPoint *pBestSplitPoint, nodeStat *pRChildStat, nodeStat *pLChildStat)
{
	//a thread for constructing a split point
	int snId = threadIdx.x;//position in the dense array of nodes
	int key = pnGlobalBestGainKey[snId];//position in the dense array

	//find best feature id
	int bestFeaId = -1;
	for(int f = 0; f < numFea; f++)
	{
		int feaPos = f + snId * numFea;
		int numofFValue = pEachFeaLenEachNode[feaPos];
		if(pEachFeaStartPosEachNode[feaPos] + numofFValue < key)//####### key should be represented using long long
			continue;
		else//key is in the range of values of f
		{
			bestFeaId = f;
			break;
		}
	}

	if(bestFeaId == -1)
		printf("Error: bestFeaId=%d\n", bestFeaId);

	int buffId = pPosToBuffId[snId];//snId to buffer id (i.e. hash value)

	pBestSplitPoint[buffId].m_fGain = pfGlobalBestGain[snId];//change the gain back to positive
	if(pfGlobalBestGain[snId] <= 0){//no gain
		return;
	}

	pBestSplitPoint[buffId].m_nFeatureId = bestFeaId;
	if(key < 1)
		printf("Error: best key=%d, is < 1\n", key);
	pBestSplitPoint[buffId].m_fSplitValue = 0.5f * (pDenseFeaValue[key] + pDenseFeaValue[key - 1]);

	//child node stat
	int idxPreSum = key - 1;//follow xgboost using exclusive
	pLChildStat[buffId].sum_gd = snNodeStat[buffId].sum_gd - pPrefixSumGD[idxPreSum];
	pLChildStat[buffId].sum_hess = snNodeStat[buffId].sum_hess - pPrefixSumHess[idxPreSum];
//	if(pLChildStat[buffId].sum_hess == 1)
//		printf("Have a look at here\n");
	pRChildStat[buffId].sum_gd = pPrefixSumGD[idxPreSum];
	pRChildStat[buffId].sum_hess = pPrefixSumHess[idxPreSum];
	if(pLChildStat[buffId].sum_hess < 0 || pRChildStat[buffId].sum_hess < 0)
		printf("Error: hess is negative l hess=%d, r hess=%d\n", pLChildStat[buffId].sum_hess, pRChildStat[buffId].sum_hess);
//	printf("split: f=%d, value=%f, gain=%f, gd=%f v.s. %f, hess=%f v.s. %f, buffId=%d, key=%d\n", bestFeaId, pBestSplitPoint[buffId].m_fSplitValue,
//			pBestSplitPoint[buffId].m_fGain, pLChildStat[buffId].sum_gd, pRChildStat[buffId].sum_gd, pLChildStat[buffId].sum_hess, pRChildStat[buffId].sum_hess, buffId, key);
}
