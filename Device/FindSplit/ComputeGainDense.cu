#include "hip/hip_runtime.h"
/*
 * ComputeGainDense.cu
 *
 *  Created on: 22 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: kernels gain computing using dense arrays
 */

#include <stdio.h>
#include <float.h>
#include "FindFeaKernel.h"
#include "../KernelConst.h"
#include "../svm-shared/DeviceUtility.h"

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHess(const float_point *pInsGD, const float_point *pInsHess, int numIns,
						   const int *pInsId, const int *pDstIndexEachFeaValue, int numFeaValue,
						   float_point *pGDEachFeaValue, float_point *pHessEachFeaValue)
{
	//one thread loads one value
	//## global id looks ok, but need to be careful
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id
	if(insId >= numIns)
		printf("Instance id is larger than the number of instances!\n");

	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	if(idx < 0)
		printf("index to out array is negative!\n");
	if(idx >= numFeaValue)
		printf("index to out array is too large!\n");

	//store GD and Hess.
	pGDEachFeaValue[idx] = pInsGD[insId];
	pHessEachFeaValue[idx] = pInsHess[insId];
}

/**
 * @brief: compute the gain in parallel, each gain is computed by a thread.
 */
__global__ void ComputeGainDense(const nodeStat *pSNodeStat, const int *pFeaValueStartPosEachNode, int numSN,
							const int *pBuffId,	float_point lambda,
							const float_point *pGDPrefixSumOnEachFeaValue, const float_point *pHessPrefixSumOnEachFeaValue,
							int numofDenseValue, float_point *pGainOnEachFeaValue)
{
	//one thread loads one value
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	//compute node id
	int snId = -1;
	for(int i = 0; i < numSN; i++)
	{
		if(i == numSN - 1)
		{
			snId = i;
			break;
		}
		else if(gTid >= pFeaValueStartPosEachNode[i] && gTid < pFeaValueStartPosEachNode[i + 1])
		{
			snId = i;
			break;
		}
	}

	int hashVaue = pBuffId[snId];
	if(hashVaue < 0)
		printf("Error in ComputeGain: buffer id %d, i=%d\n", hashVaue, snId);

	if(gTid >= numofDenseValue)//the thread has no gain to compute
	{
		return;
	}

	if(gTid == 0)
	{
		//assign gain to 0
    	pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	int exclusiveSumPos = gTid - 1;//following xgboost using exclusive sum on gd and hess

	float_point rChildGD = pGDPrefixSumOnEachFeaValue[exclusiveSumPos];
	float_point rChildHess = pHessPrefixSumOnEachFeaValue[exclusiveSumPos];
	float_point snGD = pSNodeStat[hashVaue].sum_gd;
	float_point snHess = pSNodeStat[hashVaue].sum_hess;
	float_point tempGD = snGD - rChildGD;
	float_point tempHess = snHess - rChildHess;
	bool needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true)//need to compute the gain
    {
    	pGainOnEachFeaValue[gTid] = (tempGD * tempGD)/(tempHess + lambda) +
    									 	 (rChildGD * rChildGD)/(rChildHess + lambda) -
    									 	 (snGD * snGD)/(snHess + lambda);
    }
    else
    {
    	//assign gain to 0
    	pGainOnEachFeaValue[gTid] = 0;
    }

}

/**
 * @brief: change the gain of the first value of each feature to 0
 */
__global__ void FirstFeaGain(const int *pEachFeaStartPosEachNode, int numFeaStartPos, float_point *pGainOnEachFeaValue)
{
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if(gTid >= numFeaStartPos)//no gain to fix
	{
		return;
	}
	int gainPos = pEachFeaStartPosEachNode[gTid];
	pGainOnEachFeaValue[gainPos] = 0;
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 * Each block.y processes one node, a thread processes a reduction.
 */
__global__ void PickLocalBestSplitEachNode(const int *pnNumFeaValueEachNode, const int *pFeaStartPosEachNode,
										   const float_point *pGainOnEachFeaValue,
								   	   	   float_point *pfLocalBestGain, int *pnLocalBestGainKey)
{
	//best gain of each node is search by a few blocks
	//blockIdx.z corresponds to a splittable node id
	int snId = blockIdx.z;

	__shared__ float_point pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;
	if(localTid == 0)
	{//initialise local best value
		int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
		pfLocalBestGain[blockId] = FLT_MAX;
		pnLocalBestGainKey[blockId] = -1;
	}

	int numValueThisNode = pnNumFeaValueEachNode[snId];//get the number of feature value of this node
	int tidForEachNode = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if(tidForEachNode >= numValueThisNode)//no gain to load
	{
		return;
	}

	int nPos = pFeaStartPosEachNode[snId] + tidForEachNode;//feature value gain position
	if(nPos < 0)
		printf("sp pos is nagative! %d\n", nPos);

	pfGain[localTid] = -pGainOnEachFeaValue[nPos];//change to find min of -gain
	pnBetterGainKey[localTid] = nPos;//############ need to be the key in the whole fea value array
	__syncthreads();

	//find the local best split point
	GetMinValue(pfGain, pnBetterGainKey, blockDim.x);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
		pfLocalBestGain[blockId] = pfGain[0];
		pnLocalBestGainKey[blockId] = pnBetterGainKey[0];
		if(pnBetterGainKey[0] < 0)
			printf("negative key: snId=%d, blockId=%d, gain=%f, key=%d\n", snId, blockId, pfGain[0], pnBetterGainKey[0]);
	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 */
__global__ void PickGlobalBestSplitEachNode(const float_point *pfLocalBestGain, const int *pnLocalBestGainKey,
								   	   	    float_point *pfGlobalBestGain, int *pnGlobalBestGainKey,
								   	   	    int numBlockPerNode, int numofSNode)
{
	//a block for finding the best gain of a node
	int blockId = blockIdx.x;

	int snId = blockId;
	if(blockIdx.y > 1)
		printf("One block is not enough to find global best split.\n");

	if(snId >= numofSNode)
		printf("Error in PickBestFea: kernel split %d nods, but only %d splittable nodes\n", snId, numofSNode);

	__shared__ float_point pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;

	if(localTid >= numBlockPerNode)//number of threads is larger than the number of blocks
	{
		return;
	}

	int curFeaLocalBestStartPos = snId * numBlockPerNode;

	LoadToSharedMem(numBlockPerNode, curFeaLocalBestStartPos, pfLocalBestGain, pnLocalBestGainKey, pfGain, pnBetterGainKey);
	 __syncthreads();	//wait until the thread within the block

	//find the local best split point
	GetMinValue(pfGain, pnBetterGainKey, blockDim.x);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		pfGlobalBestGain[snId] = -pfGain[0];//make the gain back to its original sign
		pnGlobalBestGainKey[snId] = pnBetterGainKey[0];
		if(pnBetterGainKey[0] < 0)
			printf("negative key: snId=%d, gain=%f, key=%d\n", snId, pfGain[0], pnBetterGainKey[0]);
	}
}
