#include "hip/hip_runtime.h"
/*
 * ComputeGainDense.cu
 *
 *  Created on: 22 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: kernels gain computing using dense arrays
 */

#include <stdio.h>
#include <float.h>
#include <limits>
#include "FindFeaKernel.h"
#include "../Splitter/DeviceSplitter.h"
#include "../../DeviceHost/svm-shared/DeviceUtility.h"
#include "../../SharedUtility/CudaMacro.h"

const float rt_2eps = 2.0 * DeviceSplitter::rt_eps;

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalueRoot(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//store GD and Hess.
	pGDEachFeaValue[gTid] = pInsGD[insId];
	pHessEachFeaValue[gTid] = pInsHess[insId];
	pDenseFeaValue[gTid] = pAllFeaValue[gTid];
}

/**
 * @brief: copy the gd, hess and feaValue for each node based on some features on similar number of values
 */
__global__ void LoadGDHessFvalue(const real *pInsGD, const real *pInsHess, int numIns,
						   const int *pInsId, const real *pAllFeaValue, const unsigned int *pDstIndexEachFeaValue, int numFeaValue,
						   double *pGDEachFeaValue, real *pHessEachFeaValue, real *pDenseFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	int insId = pInsId[gTid];//instance id

	CONCHECKER(insId < numIns);

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	CONCHECKER(idx >= 0);
	CONCHECKER(idx < numFeaValue);

	//scatter: store GD, Hess and the feature value.
	pGDEachFeaValue[idx] = pInsGD[insId];
	pHessEachFeaValue[idx] = pInsHess[insId];
	pDenseFeaValue[idx] = pAllFeaValue[gTid];
}

/**
 * @brief: compute the gain in parallel, each gain is computed by a thread.
 */
__global__ void ComputeGainDense(const nodeStat *pSNodeStat, const unsigned int *pFeaValueStartPosEachNode, int numSN,
							const int *pBuffId, real lambda,
							const double *pGDPrefixSumOnEachFeaValue, const real *pHessPrefixSumOnEachFeaValue,
							const real *pDenseFeaValue, int numofDenseValue, real *pGainOnEachFeaValue)
{
	//one thread loads one value
	long long gTid = GLOBAL_TID();

	//compute node id
	int snId = -1;
	for(int i = 0; i < numSN; i++)
	{
		if(i == numSN - 1)
		{
			snId = i;
			break;
		}
		else if(gTid >= pFeaValueStartPosEachNode[i] && gTid < pFeaValueStartPosEachNode[i + 1])
		{
			snId = i;
			break;
		}
	}
	int hashVaue = pBuffId[snId];
	ECHECKER(hashVaue);

	if(gTid >= numofDenseValue)//the thread has no gain to compute, i.e. a thread per gain
	{
		return;
	}

	if(gTid == 0)
	{
		//assign gain to 0 to the first feature value
    	pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	//if the previous fea value is the same as the current fea value, gain is 0 for the current fea value.
	if(fabs(pDenseFeaValue[gTid - 1] - pDenseFeaValue[gTid]) <= rt_2eps)
	{//avoid same feature value different gain issue
		pGainOnEachFeaValue[gTid] = 0;
		return;
	}

	int exclusiveSumPos = gTid - 1;//following xgboost using exclusive sum on gd and hess

	double rChildGD = pGDPrefixSumOnEachFeaValue[exclusiveSumPos];
	real rChildHess = pHessPrefixSumOnEachFeaValue[exclusiveSumPos];
	real parentGD = pSNodeStat[hashVaue].sum_gd;
	real parentHess = pSNodeStat[hashVaue].sum_hess;
	real tempGD = parentGD - rChildGD;
	real tempHess = parentHess - rChildHess;
	bool needUpdate = NeedUpdate(rChildHess, tempHess);
    if(needUpdate == true)//need to compute the gain
    {
		real tempGain = (tempGD * tempGD)/(tempHess + lambda) + 
						  	   (rChildGD * rChildGD)/(rChildHess + lambda) -
	  						   (parentGD * parentGD)/(parentHess + lambda);
    	pGainOnEachFeaValue[gTid] = tempGain; 
//    	if(pGainOnEachFeaValue[gTid] > 0 && ((rChildHess == 463714 && tempHess == 1) || (rChildHess == 1 && tempHess == 463714)))
 //   		printf("gain=%f, gid=%d, rhess=%f, lhess=%f\n", pGainOnEachFeaValue[gTid], gTid, rChildHess, tempHess);
    }
    else
    {
    	//assign gain to 0
    	pGainOnEachFeaValue[gTid] = 0;
    }
}

/**
 * @brief: change the gain of the first value of each feature to 0
 */
__global__ void FirstFeaGain(const unsigned int *pEachFeaStartPosEachNode, int numFeaStartPos, real *pGainOnEachFeaValue, long long numFeaValue)
{
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaStartPos)//no gain to fix
	{
		return;
	}
	unsigned int gainPos = pEachFeaStartPosEachNode[gTid];
	if(gainPos >= numFeaValue){
		return;//there may be some ending 0s (e.g. the last node has some features with any values).
	}
	pGainOnEachFeaValue[gainPos] = 0;
//	if(gTid == 0){
//		printf("pEachFeaStartPosEachNode[8]=%f\n", pEachFeaStartPosEachNode[8]);
//	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 * Each block.y processes one node, a thread processes a reduction.
 */
__global__ void PickLocalBestSplitEachNode(const unsigned int *pnNumFeaValueEachNode, const unsigned int *pFeaStartPosEachNode,
										   const real *pGainOnEachFeaValue,
								   	   	   real *pfLocalBestGain, int *pnLocalBestGainKey)
{
	//best gain of each node is search by a few blocks
	//blockIdx.z corresponds to a splittable node id
	int snId = blockIdx.z;

	__shared__ real pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;
	if(localTid == 0){//initialise local best value
		int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
		pfLocalBestGain[blockId] = FLT_MAX;
		pnLocalBestGainKey[blockId] = -1;
	}

	unsigned int numValueThisNode = pnNumFeaValueEachNode[snId];//get the number of feature value of this node
	long long tidForEachNode = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	unsigned int nPos = pFeaStartPosEachNode[snId] + tidForEachNode;//feature value gain position

	if(tidForEachNode >= numValueThisNode){//no gain to load
		pfGain[localTid] = 0;
		pnBetterGainKey[localTid] = INT_MAX;
	}
	else{
		pfGain[localTid] = -pGainOnEachFeaValue[nPos];//change to find min of -gain
		pnBetterGainKey[localTid] = nPos;
	}
	__syncthreads();

	//find the local best split point
	GetMinValue(pfGain, pnBetterGainKey, blockDim.x);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		int blockId = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;
		pfLocalBestGain[blockId] = pfGain[0];
		pnLocalBestGainKey[blockId] = pnBetterGainKey[0];

		ECHECKER(pnBetterGainKey[0]);
		//if(pnBetterGainKey[0] < 0)
		//	printf("negative key: snId=%d, blockId=%d, gain=%f, key=%d\n", snId, blockId, pfGain[0], pnBetterGainKey[0]);
	}
}

/**
 * @brief: pick best feature of this batch for all the splittable nodes
 */
__global__ void PickGlobalBestSplitEachNode(const real *pfLocalBestGain, const int *pnLocalBestGainKey,
								   	   	    real *pfGlobalBestGain, int *pnGlobalBestGainKey,
								   	   	    int numBlockPerNode, int numofSNode)
{
	//a block for finding the best gain of a node
	int blockId = blockIdx.x;

	int snId = blockId;
	CONCHECKER(blockIdx.y <= 1);
	CONCHECKER(snId < numofSNode);

	__shared__ real pfGain[BLOCK_SIZE];
	__shared__ int pnBetterGainKey[BLOCK_SIZE];
	int localTid = threadIdx.x;
	pfGain[localTid] = FLT_MAX;//initialise to a large positive number
	pnBetterGainKey[localTid] = -1;

	if(localTid >= numBlockPerNode)//number of threads is larger than the number of blocks
	{
		return;
	}

	int curFeaLocalBestStartPos = snId * numBlockPerNode;

	LoadToSharedMem(numBlockPerNode, curFeaLocalBestStartPos, pfLocalBestGain, pnLocalBestGainKey, pfGain, pnBetterGainKey);
	 __syncthreads();	//wait until the thread within the block

	//find the local best split point
	GetMinValue(pfGain, pnBetterGainKey, blockDim.x);
	__syncthreads();
	if(localTid == 0)//copy the best gain to global memory
	{
		pfGlobalBestGain[snId] = -pfGain[0];//make the gain back to its original sign
		pnGlobalBestGainKey[snId] = pnBetterGainKey[0];
		ECHECKER(pnBetterGainKey[0]);
		if(pnBetterGainKey[0] < 0)
			printf("negative key: snId=%d, gain=%f, key=%d, blockDim.x=%d, blockSize=%d, blockpPerNode=%d, numSN=%d\n",
			snId, pfGain[0], pnBetterGainKey[0], blockDim.x, BLOCK_SIZE, numBlockPerNode, numofSNode);
	}
}

/**
 * @brief: find split points
 */
__global__ void FindSplitInfo(const unsigned int *pEachFeaStartPosEachNode, const int *pEachFeaLenEachNode,
							  const real *pDenseFeaValue, const real *pfGlobalBestGain, const int *pnGlobalBestGainKey,
							  const int *pPartitionId2SNPos, const int numFea,
							  const nodeStat *snNodeStat, const double *pPrefixSumGD, const real *pPrefixSumHess,
							  SplitPoint *pBestSplitPoint, nodeStat *pRChildStat, nodeStat *pLChildStat)
{
	//a thread for constructing a split point
	int snId = threadIdx.x;//position in the dense array of nodes
	int key = pnGlobalBestGainKey[snId];//position in the dense array

	//find best feature id
	int bestFeaId = -1;
	for(int f = 0; f < numFea; f++)
	{
		int feaPos = f + snId * numFea;
		int numofFValue = pEachFeaLenEachNode[feaPos];
		if(pEachFeaStartPosEachNode[feaPos] + numofFValue < key)//####### key should be represented using long long
			continue;
		else//key is in the range of values of f
		{
			bestFeaId = f;
			break;
		}
	}

	CONCHECKER(bestFeaId != -1);

	int buffId = pPartitionId2SNPos[snId];//snId to buffer id (i.e. hash value)

	pBestSplitPoint[buffId].m_fGain = pfGlobalBestGain[snId];//change the gain back to positive
	if(pfGlobalBestGain[snId] <= 0){//no gain
		return;
	}

	pBestSplitPoint[buffId].m_nFeatureId = bestFeaId;
	ECHECKER(key);
	pBestSplitPoint[buffId].m_fSplitValue = 0.5f * (pDenseFeaValue[key] + pDenseFeaValue[key - 1]);

	//child node stat
	int idxPreSum = key - 1;//follow xgboost using exclusive
	pLChildStat[buffId].sum_gd = snNodeStat[buffId].sum_gd - pPrefixSumGD[idxPreSum];
	pLChildStat[buffId].sum_hess = snNodeStat[buffId].sum_hess - pPrefixSumHess[idxPreSum];
	pRChildStat[buffId].sum_gd = pPrefixSumGD[idxPreSum];
	pRChildStat[buffId].sum_hess = pPrefixSumHess[idxPreSum];
	ECHECKER(pLChildStat[buffId].sum_hess);
	ECHECKER(pRChildStat[buffId].sum_hess);
	printf("split: f=%d, value=%f, gain=%f, gd=%f v.s. %f, hess=%f v.s. %f, buffId=%d, key=%d\n", bestFeaId, pBestSplitPoint[buffId].m_fSplitValue,
			pBestSplitPoint[buffId].m_fGain, pLChildStat[buffId].sum_gd, pRChildStat[buffId].sum_gd, pLChildStat[buffId].sum_hess, pRChildStat[buffId].sum_hess, buffId, key);
}
