#include "hip/hip_runtime.h"
/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Bagging/BagManager.h"
#include "../Bagging/BagOrgManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/segmentedMax.h"

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	BagOrgManager orgManager(bagManager.m_numFeaValue, bagManager.m_numBag);
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	//reset memory for this bag
	{
		manager.MemsetAsync(orgManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);

		manager.MemsetAsync(orgManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(double) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(orgManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(orgManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = -1, maxNumFeaValueOneNode = -1;
	if(numofSNode > 1)
	{
		IndexComputer indexComp;
		indexComp.AllocMem(bagManager.m_numFea, numofSNode);
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
	
		clock_t start_gd = clock();
		//scatter operation
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		numofDenseValue = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		//printf("# of useful fvalue=%d\n", numofDenseValue);
		LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_numIns, manager.m_pDInsId, orgManager.m_pdDFeaValue,
															   bagManager.m_pIndicesEachBag_d, bagManager.m_numFeaValue,
															   orgManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   orgManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   orgManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		indexComp.FreeMem();
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   	   	manager.m_pDInsId, bagManager.m_numFeaValue,
															   		orgManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	orgManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		checkCudaErrors(hipMemcpy(orgManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue, orgManager.m_pdDFeaValue, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDefault));
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		numofDenseValue = manager.m_numFeaValue;//for computing gain of each fvalue
		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
	}

	//cout << "prefix sum" << endl;
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	int totalNumArray = bagManager.m_numFea * numofSNode;
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled

	//construct keys for exclusive scan
	uint *pnKey_d;
	checkCudaErrors(hipMalloc((void**)&pnKey_d, bagManager.m_numFeaValue * sizeof(uint)));
	uint *pTempEachFeaStartEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;

	//set keys by GPU
	int maxSegLen = 0;
	int *pTempEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	int *pMaxLen = thrust::max_element(thrust::device, pTempEachFeaLenEachNode, pTempEachFeaLenEachNode + totalNumArray);
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpyAsync(&maxSegLen, pMaxLen, sizeof(int), hipMemcpyDeviceToHost, (*(hipStream_t*)pStream)));
	hipStreamSynchronize((*(hipStream_t*)pStream));

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = totalNumArray;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<totalNumArray, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	double *pTempGDSum = orgManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	real *pTempHessSum = orgManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempGDSum, pTempGDSum);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempHessSum, pTempHessSum);

	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//default to left or right
	bool *pDefault2Right;
	checkCudaErrors(hipMalloc((void**)&pDefault2Right, sizeof(bool) * bagManager.m_numFeaValue));
	checkCudaErrors(hipMemset(pDefault2Right, 0, sizeof(bool) * bagManager.m_numFeaValue));

	//cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(numofDenseValue, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, orgManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											orgManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											orgManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
											numofDenseValue, pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d, bagManager.m_numFea,
											orgManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
											pDefault2Right);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");
	
	//change the gain of the first feature value to 0
	int numFeaStartPos = bagManager.m_numFea * numofSNode;
//	printf("num fea start pos=%d (%d * %d)\n", numFeaStartPos, bagManager.m_numFea, numofSNode);
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numFeaStartPos, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
																bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
																numFeaStartPos, orgManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
																bagManager.m_numFeaValue);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after FirstFeaGain");

	clock_t end_comp_gain = clock();
	total_com_gain_t += (end_comp_gain - start_comp_gain);

	//cout << "searching" << endl;
	clock_t start_search = clock();
	real *pfGlobalBestGain_d;
	int *pnGlobalBestGainKey_d;
	checkCudaErrors(hipMalloc((void**)&pfGlobalBestGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pnGlobalBestGainKey_d, sizeof(int) * numofSNode));

	SegmentedMax(maxNumFeaValueOneNode, numofSNode, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
			bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
			orgManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue, pStream, pfGlobalBestGain_d, pnGlobalBestGainKey_d);

	hipStreamSynchronize((*(hipStream_t*)pStream));
	clock_t end_search = clock();
	total_search_t += end_search - start_search;

	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
									 bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 orgManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
									 pfGlobalBestGain_d, pnGlobalBestGainKey_d,
				  	  	  	  	  	 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
				  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 orgManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 orgManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 pDefault2Right, pnKey_d,
				  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pnKey_d));
	checkCudaErrors(hipFree(pDefault2Right));
	checkCudaErrors(hipFree(pfGlobalBestGain_d));
	checkCudaErrors(hipFree(pnGlobalBestGainKey_d));
}
