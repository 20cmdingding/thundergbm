#include "hip/hip_runtime.h"
/*
 * CsrSplit.cu
 *
 *  Created on: Jul 11, 2017
 *      Author: zeyi
 */

#include "CsrSplit.h"
#include "../Bagging/BagManager.h"
#include "../Bagging/BagCsrManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/binarySearch.h"

void CsrCompression(int numofSNode, uint &totalNumCsrFvalue, uint *eachCompressedFeaStartPos_d, uint *eachCompressedFeaLen_d,
		uint *eachNodeSizeInCsr_d, uint *eachCsrNodeStartPos_d){
	BagManager bagManager;
	BagCsrManager csrManager(bagManager.m_numFea, bagManager.m_maxNumSplittable, bagManager.m_numFeaValue);
	real *fvalue_h = new real[bagManager.m_numFeaValue];
	uint *eachFeaLenEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	uint *eachFeaStartPosEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	uint *eachCsrFeaStartPos_h = new uint[bagManager.m_numFea * numofSNode];
	uint *eachCompressedFeaLen_h = new uint[bagManager.m_numFea * numofSNode];
	uint *eachCsrLen_h = new uint[bagManager.m_numFeaValue];
	uint *eachCsrNodeStartPos_h = new uint[numofSNode];
	double *csrGD_h = new double[bagManager.m_numFeaValue];
	real *csrHess_h = new real[bagManager.m_numFeaValue];
	uint *eachNodeSizeInCsr_h = new uint[numofSNode];
	real *csrFvalue_h = new real[bagManager.m_numFeaValue];
	checkCudaErrors(hipMemcpy(fvalue_h, bagManager.m_pDenseFValueEachBag, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(eachFeaLenEachNode_h, bagManager.m_pEachFeaLenEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(eachFeaStartPosEachNode_h, bagManager.m_pEachFeaStartPosEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));

	uint csrId = 0, curFvalueToCompress = 0;
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		eachCompressedFeaLen_h[i] = 0;
		uint feaLen = eachFeaLenEachNode_h[i];
		uint feaStart = eachFeaStartPosEachNode_h[i];
		if(feaLen == 0)continue;
		csrFvalue_h[csrId] = fvalue_h[feaStart];
		eachCsrLen_h[csrId] = 1;
		eachCompressedFeaLen_h[i] = 1;
		for(int l = 1; l < feaLen; l++){
			curFvalueToCompress++;
			if(fabs(fvalue_h[feaStart + l] - csrFvalue_h[csrId]) > DeviceSplitter::rt_eps){
				eachCompressedFeaLen_h[i]++;
				csrId++;
				csrFvalue_h[csrId] = fvalue_h[feaStart + l];
				eachCsrLen_h[csrId] = 1;
			}
			else
				eachCsrLen_h[csrId]++;
		}
		csrId++;
		curFvalueToCompress++;
	}
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		uint prefix = 0;
		for(int l = 0; l < i; l++)
			prefix += eachCompressedFeaLen_h[l];
		eachCsrFeaStartPos_h[i] = prefix;
	}

	for(int i = 0; i < numofSNode; i++){
		int posOfLastFeaThisNode = (i + 1) * bagManager.m_numFea - 1;
		int posOfFirstFeaThisNode = i * bagManager.m_numFea;
		eachNodeSizeInCsr_h[i] = eachCsrFeaStartPos_h[posOfLastFeaThisNode] - eachCsrFeaStartPos_h[posOfFirstFeaThisNode];
		eachNodeSizeInCsr_h[i] += eachCompressedFeaLen_h[posOfLastFeaThisNode];
		eachCsrNodeStartPos_h[i] = eachCsrFeaStartPos_h[posOfFirstFeaThisNode];
//		printf("node %d starts %u, len=%u\n", i, eachCsrNodeStartPos[i], eachNodeSizeInCsr[i]);
	}

	totalNumCsrFvalue = csrId;
//	printf("csrLen=%u, totalLen=%u, numofFeaValue=%u\n", csrId, totalLen, bagManager.m_numFeaValue);
	PROCESS_ERROR(totalNumCsrFvalue < bagManager.m_numFeaValue);
	//compute csr gd and hess
	double *gd_h = new double[bagManager.m_numFeaValue];
	real *hess_h = new real[bagManager.m_numFeaValue];
	checkCudaErrors(hipMemcpy(gd_h, bagManager.m_pdGDPrefixSumEachBag, sizeof(double) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hess_h, bagManager.m_pHessPrefixSumEachBag, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));

	uint globalPos = 0;
	for(int i = 0; i < csrId; i++){
		csrGD_h[i] = 0;
		csrHess_h[i] = 0;
		uint len = eachCsrLen_h[i];
		for(int v = 0; v < len; v++){
			csrGD_h[i] += gd_h[globalPos];
			csrHess_h[i] += hess_h[globalPos];
			globalPos++;
		}
	}
	checkCudaErrors(hipMemcpy(eachCompressedFeaStartPos_d, eachCsrFeaStartPos_h, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(eachCompressedFeaLen_d, eachCompressedFeaLen_h, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(eachCsrNodeStartPos_d, eachCsrNodeStartPos_h, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(eachNodeSizeInCsr_d, eachNodeSizeInCsr_h, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(csrManager.getMutableCsrLen(), eachCsrLen_h, sizeof(uint) * totalNumCsrFvalue, hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(csrManager.getMutableCsrGD(), csrGD_h, sizeof(double) * totalNumCsrFvalue, hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(csrManager.getMutableCsrHess(), csrHess_h, sizeof(real) * totalNumCsrFvalue, hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(csrManager.getMutableCsrFvalue(), csrFvalue_h, sizeof(real) * totalNumCsrFvalue, hipMemcpyDefault));

	printf("org=%u v.s. csr=%u\n", bagManager.m_numFeaValue, totalNumCsrFvalue);

	delete[] fvalue_h;
	delete[] eachFeaLenEachNode_h;
	delete[] eachFeaStartPosEachNode_h;
	delete[] gd_h;
	delete[] hess_h;
	delete[] eachCsrFeaStartPos_h;
	delete[] eachCompressedFeaLen_h;
	delete[] eachCsrLen_h;
	delete[] eachCsrNodeStartPos_h;
	delete[] csrGD_h;
	delete[] csrHess_h;
	delete[] eachNodeSizeInCsr_h;
	delete[] csrFvalue_h;
}

/**
 * @brief: efficient best feature finder
 */
__global__ void LoadFvalueInsId(int numIns, const int *pOrgFvalueInsId, int *pNewFvalueInsId, const unsigned int *pDstIndexEachFeaValue, int numFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load; note that "numFeaValue" needs to be the length of whole dataset
		return;

	//index for scatter
	uint idx = pDstIndexEachFeaValue[gTid];
	if(idx == LARGE_4B_UINT)//instance is in a leaf node
		return;

	CONCHECKER(idx < numFeaValue);

	//scatter: store the feature value ins id.
	CONCHECKER(numIns >= pOrgFvalueInsId[gTid] && pOrgFvalueInsId[gTid] >= 0);
	pNewFvalueInsId[idx] = pOrgFvalueInsId[gTid];
}

__global__ void newCsrLenFvalue(const int *preFvalueInsId, int numFeaValue, const int *pInsId2Nid, int maxNid,
						  const uint *eachCsrStart, const real *csrFvalue, uint numCsr, const uint *preRoundSegStartPos, const uint preRoundNumSN, int numFea,
						  real *eachCsrFvalueSparse, uint *csrNewLen, uint *eachNewSegLen, uint *eachNodeSizeInCsr, int numSN, uint *eachNodeFvalue){
	//one thread for one fvalue
	uint gTid = GLOBAL_TID();
	if(gTid >= numFeaValue)//thread has nothing to do
		return;

	int insId = preFvalueInsId[gTid];//insId is not -1, as preFvalueInsId is dense.
	if(pInsId2Nid[insId] <= maxNid)//leaf node
		return;
	int pid = pInsId2Nid[insId] - maxNid - 1;//mapping to new node
	atomicAdd(eachNodeFvalue + pid, 1);
	CONCHECKER(pid < numSN && pid >= 0);
	uint csrId = numCsr;
	RangeBinarySearch(gTid, eachCsrStart, numCsr, csrId);
	CONCHECKER(csrId < numCsr);
	uint segId = numFea * preRoundNumSN;
	RangeBinarySearch(csrId, preRoundSegStartPos, numFea * preRoundNumSN, segId);
	uint prePid = segId / numFea;
	uint prePartStartPos = preRoundSegStartPos[prePid * numFea];
	uint numCsrPrePartsAhead = prePartStartPos;
	uint numCsrCurPart;
	if(prePid == preRoundNumSN - 1)
		numCsrCurPart = numCsr - prePartStartPos;
	else
		numCsrCurPart = preRoundSegStartPos[(prePid + 1) * numFea] - prePartStartPos;
	uint posInPart = csrId - numCsrPrePartsAhead;//id in the partition
	uint orgValue;
	//compute len of each csr
	if(pid % 2 == 1){
		uint testLen1 = csrNewLen[numCsrPrePartsAhead * 2 + numCsrCurPart + posInPart];
		orgValue = atomicAdd(csrNewLen + numCsrPrePartsAhead * 2 + numCsrCurPart + posInPart, 1);
		if(orgValue == 0)
			eachCsrFvalueSparse[numCsrPrePartsAhead * 2 + numCsrCurPart + posInPart] = csrFvalue[csrId];
	}
	else{
		uint testLen2 = csrNewLen[numCsrPrePartsAhead * 2 + posInPart];
		orgValue = atomicAdd(csrNewLen + numCsrPrePartsAhead * 2 + posInPart, 1);
		if(orgValue == 0)
			eachCsrFvalueSparse[numCsrPrePartsAhead * 2 + posInPart] = csrFvalue[csrId];
	}

	//compute len of each segment
	if(orgValue == 0){
		uint feaId = segId % numFea;
		CONCHECKER(feaId < numFea);
		uint testLen3 = eachNewSegLen[pid * numFea + feaId];
		uint tempLen = atomicAdd(eachNewSegLen + pid * numFea + feaId, 1);
		uint testLen4 = eachNodeSizeInCsr[pid];
		atomicAdd(eachNodeSizeInCsr + pid, 1);
	}
}

__global__ void map2One(const uint *eachCsrLen, uint numCsr, uint *csrMarker){
	uint gTid = GLOBAL_TID();
	if(gTid >= numCsr)
		return;
	if(eachCsrLen[gTid] > 0)
		csrMarker[gTid] = 1;
	else
		csrMarker[gTid] = 0;
}

__global__ void loadDenseCsr(const real *eachCsrFvalueSparse, const uint *eachCsrFeaLen, uint numCsr, uint numCsrThisRound,
							 const uint *csrIdx, real *eachCsrFvalueDense, uint *eachCsrFeaLenDense){
	uint gTid = GLOBAL_TID();
	if(gTid >= numCsr)
		return;
	if(eachCsrFeaLen[gTid] != 0){
		uint idx = csrIdx[gTid] - 1;//inclusive scan is used to compute indices.
		CONCHECKER(csrIdx[gTid] <= numCsrThisRound);
		eachCsrFeaLenDense[idx] = eachCsrFeaLen[gTid];
		eachCsrFvalueDense[idx] = eachCsrFvalueSparse[gTid];
	}
}

__global__ void compCsrGDHess(const int *preFvalueInsId, uint numUsefulFvalue, const uint *eachCsrStart, uint numCsr,
							  const real *pInsGrad, const real *pInsHess, int numIns,
							  double *csrGD, real *csrHess){
	uint gTid = GLOBAL_TID();
	if(gTid >= numUsefulFvalue)
		return;
	uint csrId = numCsr;
	RangeBinarySearch(gTid, eachCsrStart, numCsr, csrId);
	CONCHECKER(csrId < numCsr);
	int insId = preFvalueInsId[gTid];
	CONCHECKER(insId >= 0 && insId < numIns);
	double temp = pInsGrad[insId];
	atomicAdd(csrGD + csrId, temp);
	atomicAdd(csrHess + csrId, pInsHess[insId]);
}
