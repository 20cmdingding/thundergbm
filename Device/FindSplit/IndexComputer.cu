#include "hip/hip_runtime.h"
/*
 * IndexComputer.cpp
 *
 *  Created on: 21 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: compute index for each feature value in the feature lists
 */

#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include "IndexComputer.h"
#include "../Hashing.h"
#include "../Bagging/BagManager.h"
#include "../Bagging/BagOrgManager.h"
#include "../CSR/CsrCompressor.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/powerOfTwo.h"
#include "../../SharedUtility/HostUtility.h"
#include "../../SharedUtility/binarySearch.h"
#include "../../SharedUtility/setSegmentKey.h"
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>

const int BLOCK_SIZE_ = 512;

const int NUM_BLOCKS = 32 * 56;

#define KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)
using std::vector;

int IndexComputer::m_totalFeaValue = -1;//total number of feature values in the whole dataset
int IndexComputer::m_numFea = -1;	//number of features
int IndexComputer::m_maxNumofSN = -1;
long long IndexComputer::m_total_copy = -1;
uint IndexComputer::numIntMem = 0;
uint IndexComputer::numCharMem = 0;

MemVector IndexComputer::partitionMarker;
uint *IndexComputer::m_pnKey = NULL;

//histogram based partitioning
MemVector IndexComputer::histogram_d;
uint IndexComputer::m_numElementEachThd = LARGE_4B_UINT;
uint IndexComputer::m_totalNumEffectiveThd = LARGE_4B_UINT;
uint *IndexComputer::m_pEachNodeStartPos_d;

__global__ void compute_id(const unsigned char *pid_map, uint *fid_map, int num_f, uint *pf_id, int n_id){
    KERNEL_LOOP(i, n_id){
        pf_id[i] = pid_map[i] * num_f + fid_map[i];
    }
}
/**
  *@brief: mark feature values beloning to node with id=snId by 1
  */
__global__ void MarkPartition(int preMaxNid, int *pFvToInsId, int *pInsIdToNodeId,
							int totalNumFv, unsigned char *pParitionMarker){
	int gTid = GLOBAL_TID();
	if(gTid >= totalNumFv)//thread has nothing to mark; note that "totalNumFv" will not decrease!
		return;

	uint insId = pFvToInsId[gTid];
	int nid = pInsIdToNodeId[insId];
	if(nid <= preMaxNid){//instance in leaf node
		pParitionMarker[gTid] = 0xff;//can only support 8 level trees
		return;
	}
	int partitionId = nid - preMaxNid - 1;
	ECHECKER(partitionId);
	pParitionMarker[gTid] = partitionId;
}

/**
 * @brief: count number of elements in each segment in the partition marker
 */
__global__ void PartitionHistogram(unsigned char *pPartitionMarker, uint markerLen, uint numParition,
								   uint numEleEachThd, uint totalNumThd, uint *pHistogram_d){
	extern __shared__ uint counters[];
	int gTid = GLOBAL_TID();
	uint tid = threadIdx.x;
	for(int p = 0; p < numParition; p++){
		counters[tid * numParition + p] = 0;
	}
	if(gTid >= totalNumThd){//thread has nothing to do
		return;
	}
	for(int i = 0; i < numEleEachThd; i++){
		if(gTid * numEleEachThd + i >= markerLen){//no element to process
			break;
		}
		int pid = pPartitionMarker[gTid * numEleEachThd + i];
		if(pid >= numParition)//this is possible, because some elements are "marked" as leaves.
			continue;//skip this element
		counters[tid * numParition + pid]++;
	}
	//store counters to global memory
	for(int p = 0; p < numParition; p++){
		//counters of the same partition are consecutive
		pHistogram_d[p * totalNumThd + gTid] = counters[tid * numParition + p];
	}
}

__global__ void ComputeNumFvalueEachNode(const uint *pHistogram_d, uint totalNumThd, uint *pNumFeaValueEachSN){
	//update number of feature values of each new node
	pNumFeaValueEachSN[threadIdx.x] = pHistogram_d[threadIdx.x * totalNumThd + totalNumThd - 1];
}

/**
  * @brief: store gather indices
  */
__global__ void CollectGatherIdx(const unsigned char *pPartitionMarker, uint markerLen,
								 const uint *pHistogram_d, uint *pEachNodeStartPos_d, uint numParition,
								 uint numEleEachThd, uint totalNumThd, uint *pGatherIdx){
	int gTid = GLOBAL_TID();
	if(gTid >= totalNumThd)//thread has nothing to collect
		return;

	uint tid = threadIdx.x;
	extern __shared__ uint eleDst[];//effectively, 4 counters for each thread

	//write start pos of each thread
	for(int p = 0; p < numParition; p++){
		uint thdCounterPos = p * totalNumThd + gTid;
		uint partitionStartPos = pEachNodeStartPos_d[p];//partition start pos
		if(gTid > 0)
			partitionStartPos += pHistogram_d[thdCounterPos - 1];

		eleDst[tid * numParition + p] = partitionStartPos;
	}

	for(int i = 0; i < numEleEachThd; i++){
		uint elePos = gTid * numEleEachThd + i;
		if(elePos >= markerLen)//no element to process
			return;
		int pid = pPartitionMarker[elePos];
		if(pid >= numParition){
			pGatherIdx[elePos] = LARGE_4B_UINT;
			continue;//skip this element, as element is marked as leaf.
		}
		uint writeIdx = tid * numParition + pid;
		pGatherIdx[elePos] = eleDst[writeIdx];//element destination ###### can be improved by shared memory
		eleDst[writeIdx]++;
	}
}

/**
  * @brief: store gather indices
  */
__global__ void EachFeaLenEachNodeCSR(const unsigned char *pPartitionMarker, uint markerLen,
								 int *pEachFeaLenEachNode, uint numFea,
								 uint numParition, uint *pEachFeaStart){
	int gTid = GLOBAL_TID();
	if(gTid >= markerLen)//thread has nothing to collect
		return;

	int pid = pPartitionMarker[gTid];
	if(pid >= numParition){
		return;//skip this element, as element is marked as leaf.
	}
	uint feaId;
	RangeBinarySearch(gTid, pEachFeaStart, numFea, feaId);
	atomicAdd(&pEachFeaLenEachNode[pid * numFea + feaId], 1);
}

/**
  * @brief: store gather indices
  */
__global__ void EachFeaLenEachNodeOrg(const unsigned char *pPartitionMarker, uint markerLen,
								 int *pEachFeaLenEachNode, const uint *tid2Fid, uint numFea,
								 uint numParition, uint *pEachFeaStart){
	int gTid = GLOBAL_TID();
	if(gTid >= markerLen)//thread has nothing to collect
		return;

	int pid = pPartitionMarker[gTid];
	if(pid >= numParition){
		return;//skip this element, as element is marked as leaf.
	}
	uint feaId = tid2Fid[gTid];
	//RangeBinarySearch(gTid, pEachFeaStart, numFea, feaId);
	atomicAdd(&pEachFeaLenEachNode[pid * numFea + feaId], 1);
}


/**
  * @brief: compute gether index by GPUs
  */
void IndexComputer::ComputeIdxGPU(int numSNode, int maxNumSN, int bagId){
	PROCESS_ERROR(m_totalFeaValue > 0 && numSNode > 0 && maxNumSN >= 0);
	
	m_pnKey = ((uint*)histogram_d.addr) + m_maxNumofSN * m_totalNumEffectiveThd;//this is important, as address of pHistogram may change.

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.y = numSNode;
	uint blockSize = 128;
	dimNumofBlockToSetKey.x = (m_totalNumEffectiveThd + blockSize - 1) / blockSize;
	SetKey<<<dimNumofBlockToSetKey, blockSize>>>(m_totalNumEffectiveThd, m_pnKey);
	GETERROR("after set key in computeIdxGPU");

	BagManager bagManager;
	GBDTGPUMemManager manager;
	KernelConf conf;
	int blockSizeForFvalue;
	dim3 dimNumofBlockForFvalue;
	conf.ConfKernel(m_totalFeaValue, blockSizeForFvalue, dimNumofBlockForFvalue);

	int *pTmpInsIdToNodeId = bagManager.m_pInsIdToNodeIdEachBag + bagId * bagManager.m_numIns;
	MarkPartition<<<dimNumofBlockForFvalue, blockSizeForFvalue>>>(bagManager.m_pPreMaxNid_h[bagId], manager.m_pDInsId, pTmpInsIdToNodeId,
																  m_totalFeaValue, (unsigned char*)partitionMarker.addr);
	GETERROR("after MarkPartition");

	dim3 numBlkDim;
	int numThdPerBlk;
	conf.ConfKernel(m_totalNumEffectiveThd, numThdPerBlk, numBlkDim);
	PartitionHistogram<<<numBlkDim, numThdPerBlk, numSNode * numThdPerBlk * sizeof(uint)>>>((unsigned char*)partitionMarker.addr, m_totalFeaValue, numSNode,
																	     	 m_numElementEachThd, m_totalNumEffectiveThd, (uint*)histogram_d.addr);
	GETERROR("after PartitionHistogram");
	//compute prefix sum for one array
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, m_pnKey, m_pnKey + m_totalNumEffectiveThd * numSNode,
								 (uint*)histogram_d.addr, (uint*)histogram_d.addr);//in place prefix sum

	//get number of fvalue in each partition (i.e. each new node)
	uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
	ComputeNumFvalueEachNode<<<1, numSNode>>>((uint*)histogram_d.addr, m_totalNumEffectiveThd, pTempNumFvalueEachNode);
	hipDeviceSynchronize();//this is very important

	checkCudaErrors(hipMemcpy(m_pEachNodeStartPos_d, pTempNumFvalueEachNode, sizeof(uint) * numSNode, hipMemcpyDeviceToDevice));
	thrust::exclusive_scan(thrust::system::cuda::par, m_pEachNodeStartPos_d, m_pEachNodeStartPos_d + numSNode, m_pEachNodeStartPos_d);

	//write to gather index
	uint *pTmpGatherIdx = bagManager.m_pIndicesEachBag_d + bagId * bagManager.m_numFeaValue;
	int flags = -1;//all bits are 1
	checkCudaErrors(hipMemset(pTmpGatherIdx, flags, sizeof(uint) * m_totalFeaValue));//when leaves appear, this is effective.
	CollectGatherIdx<<<numBlkDim, numThdPerBlk, numSNode * numThdPerBlk * sizeof(uint)>>>((unsigned char*)partitionMarker.addr, m_totalFeaValue,
												  (uint*)histogram_d.addr, m_pEachNodeStartPos_d, numSNode,
												  m_numElementEachThd, m_totalNumEffectiveThd, pTmpGatherIdx);
	GETERROR("after CollectGatherIdx");

	uint *pTmpFvalueStartPosEachNode = bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
	//compute each feature length and start position in each node
	int *pTmpEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d +
								  bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	uint * pTmpEachFeaStartPosEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d +
											  bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;

	checkCudaErrors(hipMemset(pTmpEachFeaLenEachNode, 0, sizeof(int) * bagManager.m_maxNumSplittable * m_numFea));
//	if(CsrCompressor::bUseCsr == true)
//		EachFeaLenEachNodeCSR<<<dimNumofBlockForFvalue, blockSizeForFvalue>>>((unsigned char*)partitionMarker.addr, m_totalFeaValue, pTmpEachFeaLenEachNode,
//																	   m_numFea, numSNode, manager.m_pFeaStartPos);
//	else {
//		EachFeaLenEachNodeOrg << < dimNumofBlockForFvalue, blockSizeForFvalue >> >
//														   ((unsigned char *) partitionMarker.addr, m_totalFeaValue, pTmpEachFeaLenEachNode,
//																   BagOrgManager::m_pnTid2Fid, m_numFea, numSNode, manager.m_pFeaStartPos);
//	}
    uint *pf_id;
        hipMalloc((void**)&pf_id, sizeof(uint) * m_totalFeaValue);
        compute_id<<<NUM_BLOCKS, BLOCK_SIZE_>>>((const unsigned char *)partitionMarker.addr, BagOrgManager::m_pnTid2Fid, m_numFea, pf_id, m_totalFeaValue);
        thrust::sort(thrust::cuda::par, pf_id, pf_id + m_totalFeaValue);
        thrust::counting_iterator<int> search_begin(0);
        thrust::upper_bound(thrust::cuda::par, pf_id, pf_id + m_totalFeaValue, search_begin, search_begin + m_numFea * numSNode, pTmpEachFeaLenEachNode);
        thrust::adjacent_difference(thrust::cuda::par, pTmpEachFeaLenEachNode, pTmpEachFeaLenEachNode + m_numFea * numSNode, pTmpEachFeaLenEachNode);
        hipFree(pf_id);

	thrust::exclusive_scan(thrust::system::cuda::par, pTmpEachFeaLenEachNode, pTmpEachFeaLenEachNode + m_numFea * numSNode, pTmpEachFeaStartPosEachNode);

	//get feature values start position of each new node
	checkCudaErrors(hipMemcpy(pTmpFvalueStartPosEachNode, m_pEachNodeStartPos_d, sizeof(uint) * numSNode, hipMemcpyDeviceToDevice));
}

/**
 * @brief: allocate reusable memory
 */
void IndexComputer::AllocMem(int nNumofFeatures, int curNumSN, int maxNumSN)
{
	m_numFea = nNumofFeatures;
	m_maxNumofSN = maxNumSN;
	if(m_pnKey == NULL){
		//histogram based partitioning
		m_numElementEachThd = 16;
		if(m_totalFeaValue >= 500000000)
			m_numElementEachThd = 4096;
		if(m_maxNumofSN > m_numElementEachThd)
			m_numElementEachThd = m_maxNumofSN;//make sure the memory usage is the same as the training data set
		m_totalNumEffectiveThd = Ceil(m_totalFeaValue, m_numElementEachThd);

		numCharMem = m_totalFeaValue;
		numIntMem =  m_maxNumofSN * m_totalNumEffectiveThd * 2;
		printf("index comp requires %f GB\n", (numIntMem * 4 + m_totalFeaValue)/(1024.0*1024.0*1024.0));

		partitionMarker.reserveSpace(m_totalFeaValue, 1);
		histogram_d.reserveSpace(numIntMem, sizeof(uint));
		checkCudaErrors(hipMalloc((void**)&m_pEachNodeStartPos_d, sizeof(uint) * m_maxNumofSN));
		m_pnKey = ((uint*)histogram_d.addr) + m_maxNumofSN * m_totalNumEffectiveThd;//this is important, as address of pHistogram may change.
	}
}

//free memory
void IndexComputer::FreeMem()
{
//	checkCudaErrors(hipFree(pPartitionMarker));
//	//histogram based partitioning
//	checkCudaErrors(hipFree(m_pHistogram_d));
//	checkCudaErrors(hipFree(m_pnKey));
//	checkCudaErrors(hipFree(m_pEachNodeStartPos_d));
}
