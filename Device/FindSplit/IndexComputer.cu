#include "hip/hip_runtime.h"
/*
 * IndexComputer.cpp
 *
 *  Created on: 21 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: compute index for each feature value in the feature lists
 */

#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "IndexComputer.h"
#include "../../DeviceHost/MyAssert.h"
#include "../Bagging/BagManager.h"
#include "../Hashing.h"
#include "../KernelConf.h"
#include "../prefix-sum/prefixSum.h"
#include "../../GetCudaError.h"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

using std::vector;

int *IndexComputer::m_pInsId = NULL;	//instance id for each feature value in the feature lists
int IndexComputer::m_totalFeaValue = -1;//total number of feature values in the whole dataset
long long *IndexComputer::m_pFeaStartPos = NULL;//each feature start position
int IndexComputer::m_numFea = -1;	//number of features
int IndexComputer::m_maxNumofSN = -1;
long long IndexComputer::m_total_copy = -1;

int *IndexComputer::m_insIdToNodeId_dh = NULL;//instance id to node id
int *IndexComputer::m_pIndices_dh = NULL;	//index for each node
long long *IndexComputer::m_pNumFeaValueEachNode_dh = NULL;	//# of feature values of each node
long long *IndexComputer::m_pFeaValueStartPosEachNode_dh = NULL;//start positions to feature value of each node
long long *IndexComputer::m_pEachFeaStartPosEachNode_dh = NULL;//each feature start position in each node
int *IndexComputer::m_pEachFeaLenEachNode_dh = NULL;//each feature value length in each node
int *IndexComputer::m_pBuffIdToPos_dh = NULL;//map buff id to dense pos id; not all elements in this array are used, due to not continuous buffid.

unsigned int *IndexComputer::m_pEachFeaStartPos_dh = NULL;
unsigned int *IndexComputer::m_pnGatherIdx = NULL;
unsigned int *IndexComputer::m_pFvToInsId = NULL;

/**
  *@brief: mark feature values beloning to node with id=snId by 1
  */
__global__ void ArrayMarker(int *pBuffVec_d, unsigned int *pFvToInsId, int *pInsIdToNodeId, int totalNumFv, int maxNumSN, unsigned int *pSparseGatherIdx){
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(gTid >= totalNumFv)//thread has nothing to mark 
		return;

	unsigned int insId = pFvToInsId[gTid];
	int nid = pInsIdToNodeId[insId];
	if(nid < 0)
		return;
	int buffId = nid % maxNumSN;
	unsigned int arrayId = blockIdx.z; //each arrayId corresponds to a prefix sum later
	int snId = pBuffVec_d[arrayId];
	if(snId == buffId){
		pSparseGatherIdx[gTid + arrayId * totalNumFv] = 1;
	}
}

/**
  *@brief: compute length for each feature value of each node 
  */
void UpdateEachFeaLenEachNode(unsigned int *pEachFeaStartPos, int snId, int numFea, int totalFvalue, unsigned int *pSparseGatherIdx, int *pEachFeaLenEachNode){
	for(int f = 0; f < numFea; f++){
		unsigned int posOfLastFvalue;
		if(f < numFea - 1){
			PROCESS_ERROR(pEachFeaStartPos[f + 1] > 0);
			posOfLastFvalue = pEachFeaStartPos[f + 1] - 1;
		}
		else
			posOfLastFvalue = totalFvalue - 1;

		unsigned int startPos = pEachFeaStartPos[f];//start position of the feature f.
		unsigned int lenPreviousFvalue = 0;
		if(f > 0){
			lenPreviousFvalue = pSparseGatherIdx[startPos - 1];
		}
		pEachFeaLenEachNode[snId * numFea + f] = pSparseGatherIdx[posOfLastFvalue] - lenPreviousFvalue;
	}
}

/**
  *@brief: compute start position of each feature in each node
  */
void ComputeEachFeaStartPosEachNode(int numFea, int snId, unsigned int collectedGatherIdx, int *pEachFeaLenEachNode, long long *pEachFeaStartPosEachNode){
	//start pos for first feature
	pEachFeaStartPosEachNode[snId * numFea] = collectedGatherIdx;
	//start pos for other feature
	for(int f = 1; f < numFea; f++){
		unsigned int feaPos = snId * numFea + f;
		pEachFeaStartPosEachNode[feaPos] = pEachFeaStartPosEachNode[feaPos - 1] + pEachFeaLenEachNode[feaPos];
	}
}

/**
  *@brief: compute length and start position of each feature in each node
  */
__global__ void ComputeEachFeaInfo(unsigned int *pEachFeaStartPos, int numFea, int totalFvalue, unsigned int *pSparseGatherIdx, int *pEachFeaLenEachNode,
								   long long *pEachFeaStartPosEachNode, long long *pFeaValueStartPosEachNode, long long *pNumFeaValueEachNode){
	unsigned int arrayId = blockIdx.x; //each arrayId corresponds to a prefix sum
	int snId = arrayId;

	for(int f = 0; f < numFea; f++){
		unsigned int posOfLastFvalue;
		if(f < numFea - 1){
			PROCESS_ERROR(pEachFeaStartPos[f + 1] > 0);
			posOfLastFvalue = snId * totalFvalue + pEachFeaStartPos[f + 1] - 1;
		}
		else
			posOfLastFvalue = snId * totalFvalue + totalFvalue - 1;

		unsigned int startPos = pEachFeaStartPos[f];//start position of the feature f.
		unsigned int lenPreviousFvalue = 0;
		if(f > 0){
			lenPreviousFvalue = pSparseGatherIdx[snId * totalFvalue + startPos - 1];
		}
		pEachFeaLenEachNode[snId * numFea + f] = pSparseGatherIdx[posOfLastFvalue] - lenPreviousFvalue;
	}

	unsigned int arrayStartPos = 0;//start position of this array (i.e. node)
	for(int i = 1; i < arrayId; i++){//will improve it later
		arrayStartPos += pSparseGatherIdx[i * totalFvalue - 1];	
	}
	//start pos for first feature
	pEachFeaStartPosEachNode[snId * numFea] = arrayStartPos;
	//start pos for other feature
	for(int f = 1; f < numFea; f++){
		unsigned int feaPos = snId * numFea + f;
		pEachFeaStartPosEachNode[feaPos] = pEachFeaStartPosEachNode[feaPos - 1] + pEachFeaLenEachNode[feaPos];
	}

	//feature value start position of each node 
	pFeaValueStartPosEachNode[snId] = arrayStartPos;

	//number of feature values of this node
	pNumFeaValueEachNode[snId] = pSparseGatherIdx[snId * totalFvalue + totalFvalue - 1];
}

/**
  * @brief: store gather indices
  */
__global__ void CollectGatherIdx(unsigned int *pSparseGatherIdx, unsigned int totalNumFv, unsigned int *pGatherIdx){
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(gTid >= totalNumFv)//thread has nothing to mark 
		return;

	unsigned int arrayId = blockIdx.z; //each arrayId corresponds to a prefix sum 
	unsigned int arrayStartPos = 0;//start position of this array (i.e. node)
	for(int i = 1; i < arrayId; i++){//will improve it later
		arrayStartPos += pSparseGatherIdx[i * totalNumFv - 1];	
	}
	unsigned int idx = pSparseGatherIdx[gTid + arrayId * totalNumFv];
	if(gTid == 0){
		if(idx == 1)//store the first element
			pGatherIdx[gTid] = arrayStartPos + idx - 1;//set destination for element at gTid
		if(idx > 1 || idx < 0)
			printf("error in CollectGatherIdx\n");
	}
	else{
		if(idx == pSparseGatherIdx[gTid - 1])//repeated element due to prefix sum
			return;
		pGatherIdx[gTid] = arrayStartPos + idx - 1;//set destimation for element at gTid
	}
}

__global__ void FloatToUnsignedInt(float_point *pfSparseGatherIdx, unsigned int totalNumFv, unsigned int *pnSparseGatherIdx){
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(gTid >= totalNumFv)//thread has nothing to mark 
		return;
	pnSparseGatherIdx[gTid] = pfSparseGatherIdx[gTid];
}



/**
  * @brief: compute gether index by GPUs
  */
void IndexComputer::ComputeIdxGPU(int numSNode, int maxNumSN, const int *pBuffVec){
	PROCESS_ERROR(m_pInsId != NULL && m_totalFeaValue > 0 && m_insIdToNodeId_dh != NULL);
	PROCESS_ERROR(numSNode > 0 && m_pIndices_dh != NULL);
	PROCESS_ERROR(maxNumSN >= 0);
	PROCESS_ERROR(maxNumSN == m_maxNumofSN);
	
	unsigned int *pnKey;
	unsigned int *pnSparseGatherIdx;
	checkCudaErrors(hipMalloc((void**)&pnSparseGatherIdx, sizeof(unsigned int) * m_totalFeaValue * numSNode));
	checkCudaErrors(hipMalloc((void**)&pnKey, sizeof(unsigned int) * m_totalFeaValue * numSNode));
	for(int i = 0; i < numSNode; i++){
		int flag = (i % 2 == 0 ? 0:-1);
		checkCudaErrors(hipMemset(pnKey + i * m_totalFeaValue, flag, sizeof(unsigned int) * m_totalFeaValue));
	}
	checkCudaErrors(hipMemset(pnSparseGatherIdx, 0, sizeof(unsigned int) * m_totalFeaValue * numSNode));

	unsigned int flags = -1;//all bits are 1
	checkCudaErrors(hipMemset(m_pnGatherIdx, flags, sizeof(unsigned int) * m_totalFeaValue));//when leaves appear, this is effective.

	//memset for debuging; this should be removed to develop more reliable program
	memset(m_pEachFeaLenEachNode_dh, 0, sizeof(int) * maxNumSN * m_numFea);
	memset(m_pNumFeaValueEachNode_dh, 0, sizeof(long long) * maxNumSN);
	memset(m_pEachFeaStartPosEachNode_dh, 0, sizeof(long long) * m_numFea * maxNumSN);
	memset(m_pFeaValueStartPosEachNode_dh, 0, sizeof(long long) * m_numFea * maxNumSN);
	GETERROR("after memset for idx comp");
	KernelConf conf;
	int blockSizeForFvalue;
	dim3 dimNumofBlockForFvalue;
	conf.ConfKernel(m_totalFeaValue, blockSizeForFvalue, dimNumofBlockForFvalue);
	if(dimNumofBlockForFvalue.z > 1){
		printf("invalid kernel configuration!\n");
		exit(0);
	}
	dimNumofBlockForFvalue.z = numSNode;//each z value for a prefix sum.

	checkCudaErrors(hipMemcpy(m_pFvToInsId, m_pInsId, sizeof(unsigned int) * m_totalFeaValue, hipMemcpyHostToDevice));
	int *pBuffVec_d;
	checkCudaErrors(hipMalloc((void**)pBuffVec_d, sizeof(int) * numSNode));
	checkCudaErrors(hipMemcpy(pBuffVec_d, pBuffVec, sizeof(int) * numSNode, hipMemcpyHostToDevice));

	ArrayMarker<<<dimNumofBlockForFvalue, blockSizeForFvalue>>>(pBuffVec_d, m_pFvToInsId, m_insIdToNodeId_dh, m_totalFeaValue, maxNumSN, pnSparseGatherIdx);
	GETERROR("after ArrayMarker");

	//compute prefix sum for one array
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey, pnKey + m_totalFeaValue * numSNode, pnSparseGatherIdx, pnSparseGatherIdx);//in place prefix sum
	//write to gether index
	CollectGatherIdx<<<dimNumofBlockForFvalue, blockSizeForFvalue>>>(pnSparseGatherIdx, m_totalFeaValue, m_pnGatherIdx);

	//compute each feature length and start position in each node
	ComputeEachFeaInfo<<<numSNode, 1>>>(m_pEachFeaStartPos_dh, m_numFea, m_totalFeaValue, pnSparseGatherIdx, m_pEachFeaLenEachNode_dh, m_pEachFeaStartPosEachNode_dh,
										m_pFeaValueStartPosEachNode_dh, m_pNumFeaValueEachNode_dh);

	checkCudaErrors(hipFree(pnSparseGatherIdx));
}


__global__ void llToUint(long long *pLlArray, unsigned int *pUintArray, int numEle){
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(gTid >= numEle)//thread has nothing to mark 
		return;
	pUintArray[gTid] = pLlArray[gTid];

}
/**
 * @brief: convert copy long long to unsigned int
 */
void IndexComputer::LonglongToUnsignedInt(long long *pFeaStartPos, unsigned int *pEachFeaStartPos_dh, int numEle){
	KernelConf conf;
	int blockSize;
	dim3 dimNumofBlock;
	conf.ConfKernel(numEle, blockSize, dimNumofBlock);

	llToUint<<<dimNumofBlock, blockSize>>>(pFeaStartPos, pEachFeaStartPos_dh, numEle);
}


/**
 * @brief: allocate reusable memory
 */
void IndexComputer::AllocMem(int nNumofExamples, int nNumofFeatures, int maxNumofSplittableNode)
{
	m_numFea = nNumofFeatures;
	m_maxNumofSN = maxNumofSplittableNode;

	checkCudaErrors(hipHostMalloc((void**)&m_pIndices_dh, sizeof(int) * m_totalFeaValue));
	checkCudaErrors(hipHostMalloc((void**)&m_insIdToNodeId_dh, sizeof(int) * nNumofExamples));
	checkCudaErrors(hipHostMalloc((void**)&m_pNumFeaValueEachNode_dh, sizeof(long long) * m_maxNumofSN));
	checkCudaErrors(hipHostMalloc((void**)&m_pBuffIdToPos_dh, sizeof(int) * m_maxNumofSN));
	checkCudaErrors(hipHostMalloc((void**)&m_pFeaValueStartPosEachNode_dh, sizeof(long long) * m_numFea * m_maxNumofSN));
	checkCudaErrors(hipHostMalloc((void**)&m_pEachFeaStartPosEachNode_dh, sizeof(long long) * m_maxNumofSN * m_numFea));
	checkCudaErrors(hipHostMalloc((void**)&m_pEachFeaLenEachNode_dh, sizeof(int) * m_maxNumofSN * m_numFea));

	checkCudaErrors(hipHostMalloc((void**)&m_pEachFeaStartPos_dh, sizeof(unsigned int) * m_numFea));
	checkCudaErrors(hipMalloc((void**)&m_pnGatherIdx, sizeof(unsigned int) * m_totalFeaValue));
	checkCudaErrors(hipMalloc((void**)&m_pFvToInsId, sizeof(unsigned int) * m_totalFeaValue));
}
