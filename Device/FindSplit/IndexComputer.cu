#include "hip/hip_runtime.h"
/*
 * IndexComputer.cpp
 *
 *  Created on: 21 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: compute index for each feature value in the feature lists
 */

#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include "IndexComputer.h"
#include "../Hashing.h"
#include "../Bagging/BagManager.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../DeviceHost/MyAssert.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/powerOfTwo.h"
#include "../../SharedUtility/HostUtility.h"

using std::vector;

int IndexComputer::m_totalFeaValue = -1;//total number of feature values in the whole dataset
int IndexComputer::m_numFea = -1;	//number of features
int IndexComputer::m_maxNumofSN = -1;
long long IndexComputer::m_total_copy = -1;

long long *IndexComputer::m_pNumFeaValueEachNode_dh = NULL;	//# of feature values of each node
unsigned int *IndexComputer::pPartitionMarker = NULL;
unsigned int *IndexComputer::m_pnKey = NULL;

int* IndexComputer::m_pArrangedInsId_d = NULL;
float_point* IndexComputer::m_pArrangedFvalue_d = NULL;

//histogram based partitioning
unsigned int *IndexComputer::m_pHistogram_d = NULL;
unsigned int IndexComputer::m_numElementEachThd = 0xffff;
unsigned int IndexComputer::m_totalNumEffectiveThd = 0xffff;
unsigned int *IndexComputer::m_pEachNodeStartPos_d;

/**
  *@brief: mark feature values beloning to node with id=snId by 1
  */
__global__ void MarkPartition(int preMaxNid, int *pFvToInsId, int *pInsIdToNodeId,
							int totalNumFv,	int maxNumSN, unsigned int *pParitionMarker){
	int gTid = GLOBAL_TID();
	if(gTid >= totalNumFv)//thread has nothing to mark 
		return;

	unsigned int insId = pFvToInsId[gTid];
	int nid = pInsIdToNodeId[insId];
	if(nid < 0)
		return;
	int partitionId = nid - preMaxNid - 1;
	ECHECKER(partitionId);
	pParitionMarker[gTid] = partitionId;
}

/**
 * @brief: count number of elements in each segment in the partition marker
 */
__global__ void PartitionHistogram(unsigned int *pPartitionMarker, unsigned int markerLen, unsigned int numParition,
								   unsigned int numEleEachThd, unsigned int totalNumThd, unsigned int *pHistogram_d){
	extern __shared__ unsigned int counters[];
	int gTid = GLOBAL_TID();
	unsigned int tid = threadIdx.x;
	for(int p = 0; p < numParition; p++){
		counters[tid * numParition + p] = 0;
	}
	if(gTid >= totalNumThd){//thread has nothing to do
		return;
	}
	for(int i = 0; i < numEleEachThd; i++){
		if(gTid * numEleEachThd + i >= markerLen){//no element to process
			break;
		}
		int pid = pPartitionMarker[gTid * numEleEachThd + i];
		counters[tid * numParition + pid]++;
	}
	//store counters to global memory
	for(int p = 0; p < numParition; p++){
		//counters of the same partition are consecutive
		pHistogram_d[p * totalNumThd + gTid] = counters[tid * numParition + p];
	}
}

__global__ void ComputeNumFvalueEachNode(const unsigned int *pHistogram_d, unsigned int totalNumThd, long long *pNumFeaValueEachSN){
	//update number of feature values of each new node
	pNumFeaValueEachSN[threadIdx.x] = pHistogram_d[threadIdx.x * totalNumThd + totalNumThd - 1];
}

/**
  * @brief: store gather indices
  */
__global__ void CollectGatherIdx(const unsigned int *pPartitionMarker, unsigned int markerLen,
								 const unsigned int *pHistogram_d, unsigned int *pEachNodeStartPos_d, unsigned int numParition,
								 unsigned int numEleEachThd, unsigned int totalNumThd, unsigned int *pGatherIdx){
	int gTid = GLOBAL_TID();
	if(gTid >= totalNumThd)//thread has nothing to collect
		return;

	unsigned int tid = threadIdx.x;
	extern __shared__ unsigned int eleDst[];//effectively, 4 counters for each thread

	//write start pos of each thread
	for(int p = 0; p < numParition; p++){
		unsigned int thdCounterPos = p * totalNumThd + gTid;
		unsigned int partitionStartPos = pEachNodeStartPos_d[p];//partition start pos
		if(gTid > 0)
			partitionStartPos += pHistogram_d[thdCounterPos - 1];

		eleDst[tid * numParition + p] = partitionStartPos;
	}

	for(int i = 0; i < numEleEachThd; i++){
		unsigned int elePos = gTid * numEleEachThd + i;
		if(elePos >= markerLen)//no element to process
			return;
		int pid = pPartitionMarker[elePos];
		unsigned int writeIdx = tid * numParition + pid;
		pGatherIdx[elePos] = eleDst[writeIdx];//element destination ###### can be improved by shared memory
		eleDst[writeIdx]++;
	}
}

/**
  *@brief: compute length and start position of each feature in each node
  */
__global__ void ComputeEachFeaInfo(const unsigned int *pPartitionMarker, const unsigned int *pGatherIdx, int totalNumFvalue,
								   const unsigned int *pFvalueStartPosEachSN, int numFea,
								   const unsigned int *pHistogram_d, int totalNumThd,
								   int *pEachFeaLenEachNode, unsigned int *pEachFeaStartPosEachNode){
	int previousPid = threadIdx.x; //each thread corresponds to a splittable node
	extern __shared__ unsigned int eachFeaLenEachNewNode[];

	//get pids for this node
	int start = pFvalueStartPosEachSN[previousPid];
	int pid1 = pPartitionMarker[start];
	int pid2 = -1;
	//the difference between pid1 and pid2 is always 1, as they are from the same parent node
	if(pid1 % 2 == 0)
		pid2 = pid1 + 1;
	else
		pid2 = pid1 - 1;

	//get pid1 and pid2 start position
	unsigned int startPosPartition1 = 0;
	unsigned int startPosPartition2 = 0;
	//partition pid1 start pos (i.e. prefix sum)
	for(int p = 0; p < pid1; p++){
		startPosPartition1 += pHistogram_d[p * totalNumThd + totalNumThd - 1];
	}
	//partition pid2 start pos (i.e. prefix sum)
	if(pid1 > pid2)
		startPosPartition2 = startPosPartition1 - pHistogram_d[pid2 * totalNumThd + totalNumThd - 1];
	else
		startPosPartition2 = startPosPartition1 + pHistogram_d[pid1 * totalNumThd + totalNumThd - 1];

	unsigned int startPosofCurFeaPid1 = startPosPartition1;
	unsigned int startPosofCurFeaPid2 = startPosPartition2;
	for(int f = 0; f < numFea; f++){
		//get lengths for f in the two partitions
		unsigned int feaPos = previousPid * numFea + f;
		unsigned int numFvalueThisSN = pEachFeaLenEachNode[feaPos];
		unsigned int posOfLastFValue = pEachFeaStartPosEachNode[feaPos] + numFvalueThisSN - 1;
		int lastFvaluePid = pPartitionMarker[posOfLastFValue];

		//get length of f in partition that contains last fvalue
		unsigned int dstPos = pGatherIdx[posOfLastFValue];
		unsigned int startPosofCurFea = 0;
		if(lastFvaluePid == pid1)
			startPosofCurFea = startPosofCurFeaPid1;
		else
			startPosofCurFea = startPosofCurFeaPid2;

		unsigned int numThisFeaValue = dstPos - startPosofCurFea + 1;

		//start position for the next feature
		if(lastFvaluePid == pid1){
			startPosofCurFeaPid1 += numThisFeaValue;
			startPosofCurFeaPid2 += (numFvalueThisSN - numThisFeaValue);
		}
		else{
			startPosofCurFeaPid2 += numThisFeaValue;
			startPosofCurFeaPid1 += (numFvalueThisSN - numThisFeaValue);
		}

		//temporarily store each feature length in shared memory
		if(lastFvaluePid == pid1){
			eachFeaLenEachNewNode[pid1 * numFea + f] = numThisFeaValue;
			eachFeaLenEachNewNode[pid2 * numFea + f] = (numFvalueThisSN - numThisFeaValue);
		}
		else{
			eachFeaLenEachNewNode[pid2 * numFea + f] = numThisFeaValue;
			eachFeaLenEachNewNode[pid1 * numFea + f] = (numFvalueThisSN - numThisFeaValue);
		}
	}

	__syncthreads();
	//update each fea len
	for(int f = 0; f < numFea; f++){
		pEachFeaLenEachNode[pid1 * numFea + f] = eachFeaLenEachNewNode[pid1 * numFea + f];
		pEachFeaLenEachNode[pid2 * numFea + f] = eachFeaLenEachNewNode[pid2 * numFea + f];
	}
	//start pos for first feature
	pEachFeaStartPosEachNode[pid1 * numFea] = startPosPartition1;
	pEachFeaStartPosEachNode[pid2 * numFea] = startPosPartition2;
	//start pos for other feature
	for(int f = 1; f < numFea; f++){
		unsigned int feaPosPid1 = pid1 * numFea + f;
		unsigned int feaPosPid2 = pid2 * numFea + f;
		pEachFeaStartPosEachNode[feaPosPid1] = pEachFeaStartPosEachNode[feaPosPid1 - 1] + pEachFeaLenEachNode[feaPosPid1];
		pEachFeaStartPosEachNode[feaPosPid2] = pEachFeaStartPosEachNode[feaPosPid2 - 1] + pEachFeaLenEachNode[feaPosPid2];
	}
}

/**
  * @brief: compute gether index by GPUs
  */
void IndexComputer::ComputeIdxGPU(int numSNode, int maxNumSN, int bagId){
	PROCESS_ERROR(m_totalFeaValue > 0 && numSNode > 0 && maxNumSN >= 0 && maxNumSN == m_maxNumofSN);
	
	int flags = -1;//all bits are 1
	BagManager bagManager;

	KernelConf conf;
	int blockSizeForFvalue;
	dim3 dimNumofBlockForFvalue;
	conf.ConfKernel(m_totalFeaValue, blockSizeForFvalue, dimNumofBlockForFvalue);

	int *pTmpInsIdToNodeId = bagManager.m_pInsIdToNodeIdEachBag + bagId * bagManager.m_numIns;
	MarkPartition<<<dimNumofBlockForFvalue, blockSizeForFvalue>>>(bagManager.m_pPreMaxNid_h[bagId], m_pArrangedInsId_d, pTmpInsIdToNodeId,
																  m_totalFeaValue, maxNumSN, pPartitionMarker);
	GETERROR("after MarkPartition");

	dim3 numBlkDim;
	int numThdPerBlk;
	conf.ConfKernel(m_totalNumEffectiveThd, numThdPerBlk, numBlkDim);
	PartitionHistogram<<<numBlkDim, numThdPerBlk, numSNode * numThdPerBlk * sizeof(unsigned int)>>>(pPartitionMarker, m_totalFeaValue, numSNode,
																	     	 m_numElementEachThd, m_totalNumEffectiveThd, m_pHistogram_d);
	GETERROR("after PartitionHistogram");
	for(int i = 0; i < numSNode; i++){
		int flag = (i % 2 == 0 ? 0:(-1));
		checkCudaErrors(hipMemset(m_pnKey + i * m_totalNumEffectiveThd, flag, sizeof(unsigned int) * m_totalNumEffectiveThd));
	}
	//compute prefix sum for one array
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, m_pnKey, m_pnKey + m_totalNumEffectiveThd * numSNode,
								  m_pHistogram_d, m_pHistogram_d);//in place prefix sum

	//get number of fvalue in each partition (i.e. each new node)
	ComputeNumFvalueEachNode<<<1, numSNode>>>(m_pHistogram_d, m_totalNumEffectiveThd, m_pNumFeaValueEachNode_dh);
	hipDeviceSynchronize();//this is very important
	unsigned int *temp4Debugging = new unsigned int[numSNode];
	for(int i = 0; i < numSNode; i++){
		temp4Debugging[i] = m_pNumFeaValueEachNode_dh[i];
	}

	checkCudaErrors(hipMemcpy(m_pEachNodeStartPos_d, temp4Debugging, sizeof(unsigned int) * numSNode, hipMemcpyHostToDevice));
	thrust::exclusive_scan(thrust::system::cuda::par, m_pEachNodeStartPos_d, m_pEachNodeStartPos_d + numSNode, m_pEachNodeStartPos_d);

	//write to gather index
	unsigned int *pTmpGatherIdx = bagManager.m_pIndicesEachBag_d + bagId * bagManager.m_numFeaValue;
	checkCudaErrors(hipMemset(pTmpGatherIdx, flags, sizeof(unsigned int) * m_totalFeaValue));//when leaves appear, this is effective.
	CollectGatherIdx<<<numBlkDim, numThdPerBlk, numSNode * numThdPerBlk * sizeof(unsigned int)>>>(pPartitionMarker, m_totalFeaValue,
												  m_pHistogram_d, m_pEachNodeStartPos_d, numSNode,
												  m_numElementEachThd, m_totalNumEffectiveThd, pTmpGatherIdx);
	GETERROR("after CollectGatherIdx");

	unsigned int *pTmpFvalueStartPosEachNode = bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
	//compute each feature length and start position in each node
	int *pTmpEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d +
								  bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	unsigned int * pTmpEachFeaStartPosEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d +
											  bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	int numThd = Ceil(numSNode, 2);
	ComputeEachFeaInfo<<<1, numThd, numSNode * m_numFea * sizeof(unsigned int)>>>(pPartitionMarker, pTmpGatherIdx, m_totalFeaValue,
										pTmpFvalueStartPosEachNode, m_numFea,
										m_pHistogram_d, m_totalNumEffectiveThd,
										pTmpEachFeaLenEachNode, pTmpEachFeaStartPosEachNode);
	GETERROR("after ComputeEachFeaInfo");

	//get feature values start position of each new node
	checkCudaErrors(hipMemcpy(pTmpFvalueStartPosEachNode, m_pEachNodeStartPos_d, sizeof(unsigned int) * numSNode, hipMemcpyDeviceToDevice));
	delete[] temp4Debugging;
}

/**
 * @brief: allocate reusable memory
 */
void IndexComputer::AllocMem(int nNumofExamples, int nNumofFeatures, int maxNumofSplittableNode)
{
	m_numFea = nNumofFeatures;
	m_maxNumofSN = maxNumofSplittableNode;

	checkCudaErrors(hipHostMalloc((void**)&m_pNumFeaValueEachNode_dh, sizeof(long long) * m_maxNumofSN));

	checkCudaErrors(hipMalloc((void**)&pPartitionMarker, sizeof(unsigned int) * m_totalFeaValue));

	checkCudaErrors(hipMalloc((void**)&m_pArrangedInsId_d, sizeof(int) * m_totalFeaValue));
	checkCudaErrors(hipMalloc((void**)&m_pArrangedFvalue_d, sizeof(float_point) * m_totalFeaValue));

	//histogram based partitioning
	m_numElementEachThd = 16;
	m_totalNumEffectiveThd = Ceil(m_totalFeaValue, m_numElementEachThd);
	checkCudaErrors(hipMalloc((void**)&m_pHistogram_d, sizeof(unsigned int) * m_maxNumofSN * m_totalNumEffectiveThd));
	checkCudaErrors(hipMalloc((void**)&m_pnKey, sizeof(unsigned int) * m_maxNumofSN * m_totalNumEffectiveThd));

	checkCudaErrors(hipMalloc((void**)&m_pEachNodeStartPos_d, sizeof(unsigned int) * m_maxNumofSN));
}
