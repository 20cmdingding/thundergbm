#include "hip/hip_runtime.h"
/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Hashing.h"
#include "../Bagging/BagManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/HostUtility.h"

using std::cout;
using std::endl;
using std::make_pair;
using std::cerr;

__global__ void SetKey(uint *pSegStart, int *pSegLen, uint *pnKey, uint *pnPosOfLastValueOfThisSeg){
	uint segmentId = blockIdx.x;//use one x covering multiple ys, because the maximum number of x-dimension is larger.
	__shared__ uint segmentLen, segmentStartPos;
	if(threadIdx.x == 0){//the first thread loads the segment length
		segmentLen = pSegLen[segmentId];
		segmentStartPos = pSegStart[segmentId];
	}
	__syncthreads();

	uint tid0 = blockIdx.y * blockDim.x;
	uint segmentThreadId = tid0 + threadIdx.x;
	if(tid0 >= segmentLen || segmentThreadId >= segmentLen)
		return;

	uint pos = segmentThreadId;
	while(pos < segmentLen){
		pnKey[pos + segmentStartPos] = segmentId;
		pnPosOfLastValueOfThisSeg[pos + segmentStartPos] = segmentStartPos + segmentLen - 1;
		pos += blockDim.x;
	}
}

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(vector<SplitPoint> &vBest, vector<nodeStat> &rchildStat, vector<nodeStat> &lchildStat, void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	//reset memory for this bag
	{
		manager.MemsetAsync(bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);

		manager.MemsetAsync(bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(double) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	IndexComputer indexComp;
	unsigned int *pFeaValueStartPosEachNode_h = new unsigned int[bagManager.m_maxNumSplittable];

	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(indexComp.m_totalFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	if(numofSNode > 1)
	{
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		unsigned int *pTmpFvalueStartPosEachNode = bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
		checkCudaErrors(hipMemcpy(pFeaValueStartPosEachNode_h, pTmpFvalueStartPosEachNode,
								   sizeof(unsigned int) * bagManager.m_maxNumSplittable, hipMemcpyDeviceToHost));
	
		//copy # of feature values of each node
		manager.MemcpyHostToDeviceAsync(indexComp.m_pNumFeaValueEachNode_dh, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(unsigned int) * bagManager.m_maxNumSplittable, pStream);
		//copy feature value start position of each node
		//copy (in pinned mem) of feature values for each feature in each node
		manager.MemcpyHostToDeviceAsync(bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										bagManager.m_pEachFeaLenEachNodeEachBag_dh + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(int) * bagManager.m_maxNumSplittable * bagManager.m_numFea, pStream);
	
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t start_gd = clock();
		//scatter operation
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		int numFvToLoad = pFeaValueStartPosEachNode_h[numofSNode - 1] + indexComp.m_pNumFeaValueEachNode_dh[numofSNode - 1];
		LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_numIns, manager.m_pDInsId, manager.m_pdDFeaValue,
															   bagManager.m_pIndicesEachBag_d, numFvToLoad,
															   bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   	   	manager.m_pDInsId, manager.m_pdDFeaValue, indexComp.m_totalFeaValue,
															   		bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(unsigned int), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(unsigned int), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(unsigned int) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		//copy # (in pinned mem) of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_dh + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(int) * nNumofFeature, pStream);

		//set indexComp
		pFeaValueStartPosEachNode_h[0] = 0;
		indexComp.m_pNumFeaValueEachNode_dh[0] = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
	}

//	cout << "prefix sum" << endl;
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	int totalNumArray = indexComp.m_numFea * numofSNode;
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled

	//construct keys for exclusive scan
	uint *pnKey_d;
	unsigned int *pnLastFvalueOfThisFvalue_d, *pnLastFvalueOfThisFvalue_h = new unsigned int[bagManager.m_numFeaValue];
	checkCudaErrors(hipMalloc((void**)&pnKey_d, bagManager.m_numFeaValue * sizeof(uint)));
	checkCudaErrors(hipMalloc((void**)&pnLastFvalueOfThisFvalue_d, bagManager.m_numFeaValue * sizeof(unsigned int)));
	unsigned int *pTempEachFeaStartEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;

	//set keys by GPU
	int maxSegLen = 0;
	int *pTempEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	int *pMaxLen = thrust::max_element(thrust::device, pTempEachFeaLenEachNode, pTempEachFeaLenEachNode + totalNumArray);
	checkCudaErrors(hipMemcpy(&maxSegLen, pMaxLen, sizeof(int), hipMemcpyDeviceToHost));
	printf("numIns=%d, actual max=%d\n", bagManager.m_numIns, maxSegLen);

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = totalNumArray;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<totalNumArray, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d, pnLastFvalueOfThisFvalue_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	double *pTempGDSum = bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	real *pTempHessSum = bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempGDSum, pTempGDSum);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempHessSum, pTempHessSum);


	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//default to left or right
	bool *pDefault2Right;
	checkCudaErrors(hipMalloc((void**)&pDefault2Right, sizeof(bool) * bagManager.m_numFeaValue));
	checkCudaErrors(hipMemset(pDefault2Right, 0, sizeof(bool) * bagManager.m_numFeaValue));

	//cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = pFeaValueStartPosEachNode_h[numofSNode - 1] + indexComp.m_pNumFeaValueEachNode_dh[numofSNode - 1];

	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(numofDenseValue, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
											numofDenseValue, pnLastFvalueOfThisFvalue_d, pnKey_d, bagManager.m_numFea,
											bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
											pDefault2Right);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");
	
	delete[] pnLastFvalueOfThisFvalue_h;
	checkCudaErrors(hipFree(pnKey_d));
	hipDeviceSynchronize();

	//for testing gain before fixing
	real *pGain = new real[bagManager.m_numFeaValue];
	hipMemcpy(pGain, bagManager.m_pGainEachFvalueEachBag, bagManager.m_numFeaValue * sizeof(real), hipMemcpyDeviceToHost);

	//change the gain of the first feature value to 0
	int numFeaStartPos = bagManager.m_numFea * numofSNode;
//	printf("num fea start pos=%d (%d * %d)\n", numFeaStartPos, bagManager.m_numFea, numofSNode);
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numFeaStartPos, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
																bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
																numFeaStartPos, bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
																bagManager.m_numFeaValue);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after FirstFeaGain");

	clock_t end_comp_gain = clock();
	total_com_gain_t += (end_comp_gain - start_comp_gain);

//	cout << "searching" << endl;
	clock_t start_search = clock();
	//compute # of blocks for each node
	unsigned int maxNumFeaValueOneNode = 0;
	unsigned int testTotalFeaValue = 0;
	for(int n = 0; n < numofSNode; n++)
	{//find the node with the max number of element
		if(maxNumFeaValueOneNode < indexComp.m_pNumFeaValueEachNode_dh[n])
			maxNumFeaValueOneNode = indexComp.m_pNumFeaValueEachNode_dh[n];
		testTotalFeaValue += indexComp.m_pNumFeaValueEachNode_dh[n];
//		printf("fv start=%u, len=%d\t", pFeaValueStartPosEachNode_h[n], indexComp.m_pNumFeaValueEachNode_dh[n]);
	}
//	printf("monitored total fvalue: %u\n", testTotalFeaValue);

	delete []pFeaValueStartPosEachNode_h;

	PROCESS_ERROR(maxNumFeaValueOneNode > 0);
	int blockSizeLocalBestGain;
	dim3 dimNumofBlockLocalBestGain;
	conf.ConfKernel(maxNumFeaValueOneNode, blockSizeLocalBestGain, dimNumofBlockLocalBestGain);
	PROCESS_ERROR(dimNumofBlockLocalBestGain.z == 1);
	dimNumofBlockLocalBestGain.z = numofSNode;//each node per super block
	int numBlockPerNode = dimNumofBlockLocalBestGain.x * dimNumofBlockLocalBestGain.y;
	//find the block level best gain for each node
	PickLocalBestSplitEachNode<<<dimNumofBlockLocalBestGain, blockSizeLocalBestGain, 0, (*(hipStream_t*)pStream)>>>(
								bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
								bagManager.m_pfLocalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode,
								bagManager.m_pnLocalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after PickLocalBestSplitEachNode");

	//find the global best gain for each node
	if(numBlockPerNode > 1){
		int blockSizeBestGain;
		dim3 dimNumofBlockDummy;
		conf.ConfKernel(numBlockPerNode, blockSizeBestGain, dimNumofBlockDummy);
		PickGlobalBestSplitEachNode<<<numofSNode, blockSizeBestGain, 0, (*(hipStream_t*)pStream)>>>(
									bagManager.m_pfLocalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode,
									bagManager.m_pnLocalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode,
									bagManager.m_pfGlobalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
									bagManager.m_pnGlobalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
								    numBlockPerNode, numofSNode);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		GETERROR("after PickGlobalBestSplitEachNode");
	}
	else{//local best fea is the global best fea
		manager.MemcpyDeviceToDeviceAsync(bagManager.m_pfLocalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
										bagManager.m_pfGlobalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(real) * numofSNode, pStream);
		manager.MemcpyDeviceToDeviceAsync(bagManager.m_pnLocalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pnGlobalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
											sizeof(int) * numofSNode, pStream);
	}

	hipStreamSynchronize((*(hipStream_t*)pStream));
	clock_t end_search = clock();
	total_search_t += end_search - start_search;

	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
									 bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
									 bagManager.m_pfGlobalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 bagManager.m_pnGlobalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
				  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 pDefault2Right, pnLastFvalueOfThisFvalue_d,
				  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	hipFree(pnLastFvalueOfThisFvalue_d);
	hipFree(pDefault2Right);
}
