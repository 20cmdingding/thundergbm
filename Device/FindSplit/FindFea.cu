#include "hip/hip_runtime.h"
/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Hashing.h"
#include "../Bagging/BagManager.h"
#include "../Memory/SNMemManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/findFeaMemManager.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../DeviceHost/MyAssert.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/HostUtility.h"
#include "../../SharedUtility/CudaMacro.h"

using std::cout;
using std::endl;
using std::make_pair;
using std::cerr;

/**
 * @brief: rearrange marker for computing feature length and start pos of each node
 */
__global__ void RearrangeData(const int *pOldInsId, const float_point *pOldFvalue, const unsigned int *pDstIndexEachFeaValue,
							  int numFeaValue, int *pNewInsId, float_point *pNewFvalue)
{
	//one thread loads one value
	//## global id looks ok, but need to be careful
	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	//scatter: store GD, Hess and the feature value.
	pNewInsId[idx] = pOldInsId[gTid];
	pNewFvalue[idx] = pOldFvalue[gTid];
}

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(vector<SplitPoint> &vBest, vector<nodeStat> &rchildStat, vector<nodeStat> &lchildStat, void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	int maxNumofSplittable = manager.m_maxNumofSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	//reset memory for this bag
	{
		manager.MemsetAsync(bagManager.m_pGDEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(float_point) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(float_point) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(float_point) * bagManager.m_numFeaValue, pStream);

		manager.MemsetAsync(bagManager.m_pGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(float_point) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(float_point) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(float_point) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	IndexComputer indexComp;
	unsigned int *pFeaValueStartPosEachNode_h = new unsigned int[bagManager.m_maxNumSplittable];

	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(indexComp.m_totalFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	if(numofSNode > 1)
	{
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		unsigned int *pTmpFvalueStartPosEachNode = bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
		checkCudaErrors(hipMemcpy(pFeaValueStartPosEachNode_h, pTmpFvalueStartPosEachNode,
								   sizeof(unsigned int) * bagManager.m_maxNumSplittable, hipMemcpyDeviceToHost));
	
		//copy # of feature values of each node
		manager.MemcpyHostToDeviceAsync(indexComp.m_pNumFeaValueEachNode_dh, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(unsigned int) * bagManager.m_maxNumSplittable, pStream);
		//copy feature value start position of each node
		//copy (in pinned mem) of feature values for each feature in each node
		manager.MemcpyHostToDeviceAsync(bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										bagManager.m_pEachFeaLenEachNodeEachBag_dh + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(int) * bagManager.m_maxNumSplittable * bagManager.m_numFea, pStream);
	
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t start_gd = clock();
		//scatter operation
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		int numFvToLoad = pFeaValueStartPosEachNode_h[numofSNode - 1] + indexComp.m_pNumFeaValueEachNode_dh[numofSNode - 1];
		LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_numIns, indexComp.m_pArrangedInsId_d, indexComp.m_pArrangedFvalue_d,
															   bagManager.m_pIndicesEachBag_d, numFvToLoad,
															   bagManager.m_pGDEachFvalueEachBag + bagId * bagManager.m_numFeaValue, 
															   bagManager.m_pHessEachFvalueEachBag + bagId * bagManager.m_numFeaValue, 
															   bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
		int *pNewInsId;
		float_point *pNewFvalue;
		checkCudaErrors(hipMalloc((void**)&pNewInsId, sizeof(int) * numFvToLoad));
		checkCudaErrors(hipMalloc((void**)&pNewFvalue, sizeof(float_point) * numFvToLoad));
		RearrangeData<<<dimNumofBlockToLoadGD, blockSizeLoadGD>>>(indexComp.m_pArrangedInsId_d, indexComp.m_pArrangedFvalue_d,
																  bagManager.m_pIndicesEachBag_d, numFvToLoad,
																  pNewInsId, pNewFvalue);
		checkCudaErrors(hipMemcpy(indexComp.m_pArrangedInsId_d, pNewInsId, sizeof(int) * numFvToLoad, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(indexComp.m_pArrangedFvalue_d, pNewFvalue, sizeof(float_point) * numFvToLoad, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipFree(pNewInsId));
		checkCudaErrors(hipFree(pNewFvalue));
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   		indexComp.m_pArrangedInsId_d, indexComp.m_pArrangedFvalue_d, indexComp.m_totalFeaValue,
															   		bagManager.m_pGDEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pHessEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_totalNumofValues, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(unsigned int), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(unsigned int), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(unsigned int) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		//copy # (in pinned mem) of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_dh + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(int) * nNumofFeature, pStream);

		//set indexComp
		pFeaValueStartPosEachNode_h[0] = 0;
		indexComp.m_pNumFeaValueEachNode_dh[0] = manager.m_totalNumofValues;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
	}

	//initialise values for gd and hess prefix sum computing
	manager.MemcpyDeviceToDeviceAsync(bagManager.m_pGDEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
									bagManager.m_pGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
									sizeof(float_point) * manager.m_totalNumofValues, pStream);
	manager.MemcpyDeviceToDeviceAsync(bagManager.m_pHessEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
									bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
									sizeof(float_point) * manager.m_totalNumofValues, pStream);

//	cout << "prefix sum" << endl;
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	int totalNumArray = indexComp.m_numFea * numofSNode;
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled
	ComputeMaxNumValuePerFea(bagManager.m_pEachFeaLenEachNodeEachBag_dh + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea, totalNumArray, bagId);
	//cout << "max # of values per fea is " << bagManager.m_pMaxNumValuePerFeaEachBag[bagId] <<"; # of arrays is " << totalNumArray << endl;
	hipDeviceSynchronize();

	//construct keys for exclusive scan
	int *pnKey_d;
	int keyFlag = 0;
	checkCudaErrors(hipMalloc((void**)&pnKey_d, bagManager.m_numFeaValue * sizeof(int)));
	unsigned int *pTempEachFeaStartEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	unsigned int *pTempEachFeaStartEachNode_h = new unsigned int[totalNumArray];
	checkCudaErrors(hipMemcpy(pTempEachFeaStartEachNode_h, pTempEachFeaStartEachNode, sizeof(unsigned int) * totalNumArray, hipMemcpyDeviceToHost));
	for(int m = 0; m < totalNumArray; m++){
		unsigned int arrayLen = bagManager.m_pEachFeaLenEachNodeEachBag_dh[m];
		unsigned int arrayStartPos = pTempEachFeaStartEachNode_h[m];
		checkCudaErrors(hipMemset(pnKey_d + arrayStartPos, keyFlag, sizeof(int) * arrayLen));
		if(keyFlag == 0)
			keyFlag = -1;
		else 
			keyFlag = 0;
	}
	delete[] pTempEachFeaStartEachNode_h;

	//compute prefix sum for gd and hess (more than one arrays)
	float_point *pTempGDSum = bagManager.m_pGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	float_point *pTempHessSum = bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempGDSum, pTempGDSum);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempHessSum, pTempHessSum);

	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pnKey_d));
	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

//	cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = pFeaValueStartPosEachNode_h[numofSNode - 1] + indexComp.m_pNumFeaValueEachNode_dh[numofSNode - 1];
	delete []pFeaValueStartPosEachNode_h;
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(numofDenseValue, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
											numofSNode,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, bagManager.m_pGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue, numofDenseValue,
											bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");

	//change the gain of the first feature value to 0
	int numFeaStartPos = indexComp.m_numFea * numofSNode;
//	printf("num of feature start positions=%d\n", numFeaStartPos);
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numFeaStartPos, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
																bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
																numFeaStartPos, bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after FirstFeaGain");

	clock_t end_comp_gain = clock();
	total_com_gain_t += (end_comp_gain - start_comp_gain);

//	cout << "searching" << endl;
	clock_t start_search = clock();
	//compute # of blocks for each node
	unsigned int maxNumFeaValueOneNode = 0;
	for(int n = 0; n < numofSNode; n++)
	{//find the node with the max number of element
		if(maxNumFeaValueOneNode < indexComp.m_pNumFeaValueEachNode_dh[n])
			maxNumFeaValueOneNode = indexComp.m_pNumFeaValueEachNode_dh[n];
	}
	PROCESS_ERROR(maxNumFeaValueOneNode > 0);
	int blockSizeLocalBestGain;
	dim3 dimNumofBlockLocalBestGain;
	conf.ConfKernel(maxNumFeaValueOneNode, blockSizeLocalBestGain, dimNumofBlockLocalBestGain);
	PROCESS_ERROR(dimNumofBlockLocalBestGain.z == 1);
	dimNumofBlockLocalBestGain.z = numofSNode;//each node per super block
	int numBlockPerNode = dimNumofBlockLocalBestGain.x * dimNumofBlockLocalBestGain.y;
	//find the block level best gain for each node
	PickLocalBestSplitEachNode<<<dimNumofBlockLocalBestGain, blockSizeLocalBestGain, 0, (*(hipStream_t*)pStream)>>>(
								bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
								bagManager.m_pfLocalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode,
								bagManager.m_pnLocalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after PickLocalBestSplitEachNode");

	//find the global best gain for each node
	if(numBlockPerNode > 1){
		int blockSizeBestGain;
		dim3 dimNumofBlockDummy;
		conf.ConfKernel(numBlockPerNode, blockSizeBestGain, dimNumofBlockDummy);
		PickGlobalBestSplitEachNode<<<numofSNode, blockSizeBestGain, 0, (*(hipStream_t*)pStream)>>>(
									bagManager.m_pfLocalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode,
									bagManager.m_pnLocalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_maxNumofBlockPerNode,
									bagManager.m_pfGlobalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
									bagManager.m_pnGlobalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
								    numBlockPerNode, numofSNode);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		GETERROR("after PickGlobalBestSplitEachNode");
	}
	else{//local best fea is the global best fea
		manager.MemcpyDeviceToDeviceAsync(bagManager.m_pfLocalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
										bagManager.m_pfGlobalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(float_point) * numofSNode, pStream);
		manager.MemcpyDeviceToDeviceAsync(bagManager.m_pnLocalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pnGlobalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
											sizeof(int) * numofSNode, pStream);
	}

	hipStreamSynchronize((*(hipStream_t*)pStream));
	clock_t end_search = clock();
	total_search_t += end_search - start_search;

//	cout << "construct split point" << endl;
	//construct split points; memset for split points
	manager.MemcpyHostToDeviceAsync(manager.m_pBestPointHost, bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
									sizeof(SplitPoint) * bagManager.m_maxNumSplittable, pStream);
	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
									 bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
									 bagManager.m_pfGlobalBestGainEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 bagManager.m_pnGlobalBestGainKeyEachBag_d + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
				  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
//	cout << "Done find split" << endl;
}
