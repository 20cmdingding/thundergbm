#include "hip/hip_runtime.h"
/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>

#include "FindFeaKernel.h"
#include "../KernelConf.h"
#include "../Hashing.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../Memory/findFeaMemManager.h"
#include "../../DeviceHost/MyAssert.h"

using std::cout;
using std::endl;
using std::make_pair;
using std::cerr;

#ifdef testing
//#undef testing
#endif

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(vector<SplitPoint> &vBest, vector<nodeStat> &rchildStat, vector<nodeStat> &lchildStat)
{
	GBDTGPUMemManager manager;
	int numofSNode = manager.m_curNumofSplitable;

	FFMemManager ffManager;
	ffManager.resetMemForFindFea();

#if testing
	int tempSN = 0;
	manager.MemcpyDeviceToHost(manager.m_pNumofBuffId, &tempSN, sizeof(int));
	PROCESS_ERROR(numofSNode == tempSN);
#endif

	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	int maxNumofSplittable = manager.m_maxNumofSplittable;

	//set memory
	int numofElement = nNumofFeature * manager.m_maxNumofSplittable;
	checkCudaErrors(hipMemset(manager.m_pTempRChildStatPerThread, 0, sizeof(nodeStat) * numofElement));
	manager.MemcpyHostToDevice(manager.m_pBestPointHostPerThread, manager.m_pBestSplitPointPerThread, sizeof(SplitPoint) * numofElement);
		//optional memory set
	checkCudaErrors(hipMemset(manager.m_pRChildStatPerThread, 0, sizeof(nodeStat) * numofElement));
	checkCudaErrors(hipMemset(manager.m_pLChildStatPerThread, 0, sizeof(nodeStat) * numofElement));
	checkCudaErrors(hipMemset(manager.m_pLastValuePerThread, -1, sizeof(float_point) * numofElement));

//######## testing code
	int smallestFeaId = 0;//######### need to change to handle datasets with a large number of features
	int feaBatch = nNumofFeature;//find best splits for a subset of features
	int maxNumofValuePerFea = manager.m_numofIns;//maximum number of instances that have non-zero value at the feature
	long long totalEleInWholeBatch = manager.m_totalNumofValues * numofSNode;//######### use all the features

	//kernel configuration
	int blockSizeFillGD;
	dim3 dimNumofBlockToFillGD;
	KernelConf conf;
	conf.ConfKernel(maxNumofValuePerFea, blockSizeFillGD, dimNumofBlockToFillGD);
	PROCESS_ERROR(dimNumofBlockToFillGD.y == 1 && dimNumofBlockToFillGD.z == 1);//must be one dimensional block
	int numofBlockFillGD = dimNumofBlockToFillGD.x;
	dim3 dimGrid(numofBlockFillGD, feaBatch, numofSNode);
	dim3 dimBlock(blockSizeFillGD, 1, 1);

	//get gd for every splittable node
	ObtainGDEachNode<<<dimGrid, dimBlock>>>(manager.m_pDNumofKeyValue, manager.m_pFeaStartPos, manager.m_pDInsId,
											manager.m_pdDFeaValue, manager.m_pInsIdToNodeId,
											manager.m_pGrad, manager.m_pHess, manager.m_pBuffIdVec, manager.m_pSNIdToBuffId,
											maxNumofSplittable, numofSNode, smallestFeaId, nNumofFeature, feaBatch,
											ffManager.m_pGDOnEachFeaValue_d, ffManager.m_pHessOnEachFeaValue_d, ffManager.m_pValueOnEachFeaValue_d);
	hipDeviceSynchronize();

#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in ObtainGDEachNode" << endl;
		exit(0);
	}

	//nid, buffer id, snid relationships
	int *pBuffIdVec_h = new int[numofSNode];
	int *pSNIdToBuffId_h = new int[maxNumofSplittable];
	int *pInsToNodeId_h = new int[manager.m_numofIns];
	manager.MemcpyDeviceToHost(manager.m_pBuffIdVec, pBuffIdVec_h, sizeof(int) * numofSNode);
	manager.MemcpyDeviceToHost(manager.m_pSNIdToBuffId, pSNIdToBuffId_h, sizeof(int) * maxNumofSplittable);
	manager.MemcpyDeviceToHost(manager.m_pInsIdToNodeId, pInsToNodeId_h, sizeof(int) * manager.m_numofIns);

	//gd/hess for each fea value
	float_point *pGDOnEachFeaVaue_h = new float_point[totalEleInWholeBatch];
	float_point *pHessOnEachFeaValue_h = new float_point[totalEleInWholeBatch];
	manager.MemcpyDeviceToHost(ffManager.m_pGDOnEachFeaValue_d, pGDOnEachFeaVaue_h, sizeof(float_point) * totalEleInWholeBatch);
	manager.MemcpyDeviceToHost(ffManager.m_pHessOnEachFeaValue_d, pHessOnEachFeaValue_h, sizeof(float_point) * totalEleInWholeBatch);

	int *pnKeyValue = new int[nNumofFeature];
	long long *plFeaStartPos = new long long[nNumofFeature];
	int *pnInsId = new int[manager.m_totalNumofValues];
	float_point *pGrad = new float_point[manager.m_numofIns];

	manager.MemcpyDeviceToHost(manager.m_pDNumofKeyValue, pnKeyValue, sizeof(int) * nNumofFeature);
	manager.MemcpyDeviceToHost(manager.m_pFeaStartPos, plFeaStartPos, sizeof(long long) * nNumofFeature);
	manager.MemcpyDeviceToHost(manager.m_pDInsId, pnInsId, sizeof(int) * manager.m_totalNumofValues);
	manager.MemcpyDeviceToHost(manager.m_pGrad, pGrad, sizeof(float_point) * manager.m_numofIns);

	//get current fea batch size
	long long startPosOfSmallest = plFeaStartPos[smallestFeaId];
	int largestFeaId = smallestFeaId + feaBatch - 1;
	long long startPosOfLargest = plFeaStartPos[largestFeaId];
	int batchSize = startPosOfLargest - startPosOfSmallest + pnKeyValue[largestFeaId];

	int e = 0;
	for(int n = 0; n < numofSNode; n++)//for each splittable node
	{
		int hashValue = pBuffIdVec_h[n];
		e = 0;
		for(int f = 0; f < nNumofFeature; f++)
		{
			int numofKeyValue = pnKeyValue[f];
			for(int i = 0; i < numofKeyValue; i++)
			{
				int insId = pnInsId[e];
				float_point gd = pGrad[insId];
				int nid = pInsToNodeId_h[insId];
				if(nid == -1)
					continue;

				int hashValue_buffer = Hashing::HostGetBufferId(pSNIdToBuffId_h, nid, maxNumofSplittable);
				if(pHessOnEachFeaValue_h[e + n * batchSize] != 0 && pHessOnEachFeaValue_h[e + n * batchSize] != 1)
				{
					cerr << "hess should be 1 or 0: " << pHessOnEachFeaValue_h[e + n * batchSize] << endl;
				}
				if(hashValue == hashValue_buffer)
				{
					if(pGDOnEachFeaVaue_h[e + n * batchSize] != gd || pHessOnEachFeaValue_h[e + n * batchSize] != 1)
						cerr << "hessian != 1: "<< pHessOnEachFeaValue_h[e + n * batchSize] << "; gd diff: "
							 << gd << " v.s. " << pGDOnEachFeaVaue_h[e + n * batchSize] << endl;
				}
				else
					if(pGDOnEachFeaVaue_h[e + n * batchSize] != 0 || pHessOnEachFeaValue_h[e + n * batchSize] != 0)
						cerr << "hessian != 0: "<< pHessOnEachFeaValue_h[e + n * batchSize] << "; gd diff: "
							 << gd << " v.s. " << pGDOnEachFeaVaue_h[e + n * batchSize] << endl;
				e++;
			}
		}
	}

	delete []pGrad;
#endif

	//each splittable node has its own copy of feature start pos and fea value length info, for calling the API of prefix sum
	int blockSizePosEachFeaInBatch;
	dim3 dimNumofBlockFindPosEachFeaInBatch;
	conf.ConfKernel(feaBatch, blockSizePosEachFeaInBatch, dimNumofBlockFindPosEachFeaInBatch);
	PROCESS_ERROR(dimNumofBlockFindPosEachFeaInBatch.z == 1 && dimNumofBlockFindPosEachFeaInBatch.y == 1);
	GetInfoEachFeaInBatch<<<dimNumofBlockFindPosEachFeaInBatch, blockSizePosEachFeaInBatch>>>(
												manager.m_pDNumofKeyValue, manager.m_pFeaStartPos, smallestFeaId, nNumofFeature,
											    feaBatch, numofSNode, ffManager.m_pStartPosEachFeaInBatch_d, ffManager.m_pFeaLenInBatch_d);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in GetInfoEachFeaInBatch" << endl;
		exit(0);
	}
	int *pStartPosEachFeaInBatch_h = new int[feaBatch * numofSNode];
	int *pFeaLenInBatch_h = new int[feaBatch * numofSNode];

	manager.MemcpyDeviceToHost(ffManager.m_pStartPosEachFeaInBatch_d, pStartPosEachFeaInBatch_h, sizeof(int) * feaBatch * numofSNode);
	manager.MemcpyDeviceToHost(ffManager.m_pFeaLenInBatch_d, pFeaLenInBatch_h, sizeof(int) * feaBatch * numofSNode);

	for(int n = 0; n < numofSNode; n++)
	{
		for(int b = 0; b < feaBatch; b++)
		{
			int feaId = b + smallestFeaId;
			if(pStartPosEachFeaInBatch_h[b + n * feaBatch] - n * batchSize != plFeaStartPos[feaId])
			{
				cerr << "diff in start pos: " << pStartPosEachFeaInBatch_h[b + n * feaBatch] << " v.s. "
					 << plFeaStartPos[feaId] << "; feaId=" << feaId << endl;
			}
			if(pFeaLenInBatch_h[b + n * feaBatch] != pnKeyValue[feaId])
			{
				cerr << "diff in fea len: " << pFeaLenInBatch_h[b + n * feaBatch] << " v.s. " << pnKeyValue[feaId] << "; feaId=" << feaId << endl;
			}
		}
	}

	delete []pStartPosEachFeaInBatch_h;
	delete []pFeaLenInBatch_h;
#endif

	//compute prefix sum
	manager.MemcpyDeviceToDevice(ffManager.m_pGDOnEachFeaValue_d, ffManager.m_pGDPrefixSum_d, sizeof(float_point) * totalEleInWholeBatch);
	manager.MemcpyDeviceToDevice(ffManager.m_pHessOnEachFeaValue_d, ffManager.m_pHessPrefixSum_d, sizeof(float_point) * totalEleInWholeBatch);

	manager.MemcpyDeviceToHost(ffManager.m_pFeaLenInBatch_d, ffManager.m_pnEachFeaLen_h, sizeof(int) * feaBatch * numofSNode);
	PrefixSumForEachNode(feaBatch * numofSNode, ffManager.m_pGDPrefixSum_d, ffManager.m_pHessPrefixSum_d,
						 ffManager.m_pStartPosEachFeaInBatch_d, ffManager.m_pnEachFeaLen_h);

#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in PrefixSumForEachNode" << endl;
		exit(0);
	}

	float_point *pGDPrefixSumOnEachFeaValue_h = new float_point[totalEleInWholeBatch];
	float_point *pHessPrefixSumOnEachFeaValue_h = new float_point[totalEleInWholeBatch];
	manager.MemcpyDeviceToHost(ffManager.m_pGDPrefixSum_d, pGDPrefixSumOnEachFeaValue_h, sizeof(float_point) * totalEleInWholeBatch);
	manager.MemcpyDeviceToHost(ffManager.m_pHessPrefixSum_d, pHessPrefixSumOnEachFeaValue_h, sizeof(float_point) * totalEleInWholeBatch);

	float_point deltaTest = 0.01;
	for(int n = 0; n < numofSNode; n++)
	{
		e = 0;
		for(int f = 0; f < nNumofFeature; f++)
		{
			int numofKeyValue = pnKeyValue[f];
			float_point prefixSumGD = 0;
			float_point prefixSumHess = 0;
			for(int i = 0; i < numofKeyValue; i++)
			{
				prefixSumGD += pGDOnEachFeaVaue_h[e + n * batchSize];
				prefixSumHess += pHessOnEachFeaValue_h[e + n * batchSize];
				if(
				   abs(prefixSumGD - pGDPrefixSumOnEachFeaValue_h[e + n * batchSize]) > deltaTest ||
				   prefixSumHess != pHessPrefixSumOnEachFeaValue_h[e + n * batchSize])
					cerr << "hessian or gd diff: hess "<< prefixSumHess << " v.s. " << pHessPrefixSumOnEachFeaValue_h[e + n * batchSize]
						 << "; gd: " << prefixSumGD << " v.s. " << pGDPrefixSumOnEachFeaValue_h[e + n * batchSize] << endl;
				e++;
			}
		}
	}

	delete []pGDOnEachFeaVaue_h;
	delete []pHessOnEachFeaValue_h;
#endif

	//compute gain
	ComputeGain<<<dimGrid, dimBlock>>>(manager.m_pDNumofKeyValue, manager.m_pFeaStartPos, manager.m_pSNodeStat, smallestFeaId, feaBatch,
									   manager.m_pBuffIdVec, numofSNode, DeviceSplitter::m_lambda, ffManager.m_pGDPrefixSum_d,
									   ffManager.m_pHessPrefixSum_d, manager.m_pdDFeaValue, ffManager.m_pGainOnEachFeaValue_d);


#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in ComputeGain" << endl;
		exit(0);
	}
	nodeStat *pSNodeStat_h = new nodeStat[maxNumofSplittable];
	float_point *pGainOnEachFeaValue_h = new float_point[totalEleInWholeBatch];
	manager.MemcpyDeviceToHost(manager.m_pSNodeStat, pSNodeStat_h, sizeof(nodeStat) * maxNumofSplittable);
	manager.MemcpyDeviceToHost(ffManager.m_pGainOnEachFeaValue_d, pGainOnEachFeaValue_h, sizeof(float_point) * totalEleInWholeBatch);

	e = 0;
	float_point *pFeaBestSplit = new float_point[feaBatch * numofSNode];
	memset(pFeaBestSplit, 0, sizeof(float_point) * feaBatch * numofSNode);
	for(int f = smallestFeaId; f < feaBatch + smallestFeaId; f++)
	{
		int numofKeyValue = pnKeyValue[f];
		int init = e;
		float_point prefixSumGD = 0;
		float_point prefixSumHess = 0;
		for(int i = init; i < numofKeyValue + init; i++)
		{
			int insId = pnInsId[i];
			int nid = pInsToNodeId_h[insId];
			if(nid == -1)
			{
				e++;
				continue;
			}

			for(int n = 0; n < numofSNode; n++)
			{
				int hashValue_buffer = Hashing::HostGetBufferId(pSNIdToBuffId_h, nid, maxNumofSplittable);
				if(hashValue_buffer < 0)
					cerr << "hash value in host side is negative: " << hashValue_buffer << " for key " << nid << endl;
				int hashValue = pBuffIdVec_h[n];
				if(hashValue_buffer != hashValue)
					continue;

				float_point snGD = pSNodeStat_h[hashValue].sum_gd;
				float_point snHess = pSNodeStat_h[hashValue].sum_hess;
				float_point gainOnFeaValue;
				if(i == init)
				{
					gainOnFeaValue = 0;
				}
				else
				{
					int exclusiveSumPos = i + n * batchSize - 1;//follow xgboost using exlusive sum on gd and hess
					float_point tempGD = pGDPrefixSumOnEachFeaValue_h[exclusiveSumPos];
					float_point tempHess = pHessPrefixSumOnEachFeaValue_h[exclusiveSumPos];
					float_point fLChildGD = snGD - tempGD;
					float_point fLChildHess = snHess - tempHess;
					if(fLChildHess >= DeviceSplitter::min_child_weight && tempHess >= DeviceSplitter::min_child_weight)
					{
						gainOnFeaValue = (tempGD * tempGD)/(tempHess + DeviceSplitter::m_lambda) +
									 (fLChildGD * fLChildGD)/(fLChildHess + DeviceSplitter::m_lambda) -
									 (snGD * snGD)/(snHess + DeviceSplitter::m_lambda);
					}
					else
						gainOnFeaValue = 0;
				}
				if(abs(gainOnFeaValue - pGainOnEachFeaValue_h[i + n * batchSize]) > deltaTest)
				{
					cerr << "gain diff: "<< gainOnFeaValue << " v.s. " << pGainOnEachFeaValue_h[i + n * batchSize] << endl;
				}

				if(pFeaBestSplit[f + feaBatch * n] < gainOnFeaValue)
					pFeaBestSplit[f + feaBatch * n] = gainOnFeaValue;
			}
			e++;
		}
	}

	for(int n = 0; n < numofSNode; n++)
	{
		for(int f = 0; f < feaBatch; f++)
		{
//			cout << "local best before fixing for " << f << "th here is " << pFeaBestSplit[f + feaBatch * n] << endl;
		}
	}

	delete []pFeaBestSplit;

	delete []pInsToNodeId_h;
	delete []pnInsId;
	delete []pGDPrefixSumOnEachFeaValue_h;
	delete []pHessPrefixSumOnEachFeaValue_h;

	delete []pSNIdToBuffId_h;
#endif

	//remove invalid gains. The same value can only have one gain
	float_point *pLastBiggerValue_d;
	checkCudaErrors(hipMalloc((void**)&pLastBiggerValue_d, sizeof(float_point) * totalEleInWholeBatch));
	checkCudaErrors(hipMemset(pLastBiggerValue_d, 0, sizeof(float_point) * totalEleInWholeBatch));
	FixedGain<<<dimGrid, dimBlock>>>(manager.m_pDNumofKeyValue, manager.m_pFeaStartPos,  smallestFeaId, feaBatch, numofSNode,
									 ffManager.m_pHessOnEachFeaValue_d, manager.m_pdDFeaValue, ffManager.m_pGainOnEachFeaValue_d, pLastBiggerValue_d);

#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in FixedGain" << endl;
		exit(0);
	}
#endif
	//find the local best split in this batch of features
	int nBlockEachFea = dimGrid.x;
	int nElePerBlock = dimBlock.x;
	PickFeaLocalBestSplit<<<dimGrid, dimBlock>>>(manager.m_pDNumofKeyValue, manager.m_pFeaStartPos, ffManager.m_pGainOnEachFeaValue_d,
											  manager.m_pBuffIdVec, smallestFeaId, feaBatch,
											  numofSNode, maxNumofSplittable, ffManager.m_pfFeaLocalBestGain_d, ffManager.m_pnFeaLocalBestGainKey_d);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in ComputeGain" << endl;
		exit(0);
	}

	float_point *pfFeaLocalBestGain_h = new float_point[feaBatch * nBlockEachFea * numofSNode];
	int *pnFeaLocalBestGainKey_h = new int[feaBatch * nBlockEachFea * numofSNode];
	manager.MemcpyDeviceToHost(ffManager.m_pfFeaLocalBestGain_d, pfFeaLocalBestGain_h, sizeof(float_point) * feaBatch * nBlockEachFea * numofSNode);
	manager.MemcpyDeviceToHost(ffManager.m_pnFeaLocalBestGainKey_d, pnFeaLocalBestGainKey_h, sizeof(int) * feaBatch * nBlockEachFea * numofSNode);

	float_point *pGlobalBest = new float_point[numofSNode];
	float_point *pFeaBest = new float_point[numofSNode * feaBatch];
	for(int n = 0; n < numofSNode; n++)
	{
		pGlobalBest[n] = 100000;
		for(int f = 0; f < feaBatch; f++)
		{
			float_point localBest = 100000;
			for(int bl = 0; bl < nBlockEachFea; bl++)
			{
				float_point temp = pfFeaLocalBestGain_h[n * feaBatch * nBlockEachFea + f * nBlockEachFea + bl];
				if(localBest > temp)
					localBest = temp;
			}
//			cout << "local best for " << f << "th fea is " << localBest << " for node " << n << endl;
			if(pGlobalBest[n] > localBest)
				pGlobalBest[n] = localBest;
			pFeaBest[n * feaBatch + f] = localBest;
		}
	}

	//obtain the best for each node
	//use the fixed gain to compute the best gain
	manager.MemcpyDeviceToHost(ffManager.m_pGainOnEachFeaValue_d, pGainOnEachFeaValue_h, sizeof(float_point) * totalEleInWholeBatch);
	for(int n = 0; n < numofSNode; n++)
	{
		float_point bestGain = -1000000;
		int key = -1;
		for(int i = 0; i < batchSize; i++)
		{
			if(bestGain < pGainOnEachFeaValue_h[i + n * batchSize])
			{
				bestGain = pGainOnEachFeaValue_h[i + n * batchSize];
				key = i;
			}
		}

		//compute feature id
		int bestFeaId = -1;
		int valuePos = -1;
		for(int f = 0; f < feaBatch; f++)
		{
			int numofKeyValue = pnKeyValue[f];
			if(plFeaStartPos[f] + numofKeyValue < key)
				continue;
			else
			{
				bestFeaId = f;
				valuePos = key - plFeaStartPos[f];
				break;
			}
		}
//		cout << "snid=" << n << "; best gain = " << bestGain << "; key is " << key << "; f=" << bestFeaId << "; value pos=" << valuePos << endl;
		if(-bestGain != pGlobalBest[n])
			cerr << "best gain diff: " << bestGain << " v.s. " << pGlobalBest[n] << endl;
	}

	delete []plFeaStartPos;
	delete []pGainOnEachFeaValue_h;
	delete []pnKeyValue;

	for(int n = 0; n < numofSNode; n++)
	{
		float_point bestGain1 = 10000;
		int bestKey1 = -1;
		for(int i = 0; i < nBlockEachFea; i++)
		{
			if(bestGain1 > pfFeaLocalBestGain_h[i + n * nBlockEachFea])
			{
				bestGain1 = pfFeaLocalBestGain_h[i + n * nBlockEachFea];
				bestKey1 = pnFeaLocalBestGainKey_h[i + n * nBlockEachFea];
			}
		}
//		cout << "best gain = " << bestGain1 << "; best key = " << bestKey1 << endl;
	}

	delete []pfFeaLocalBestGain_h;
	delete []pnFeaLocalBestGainKey_h;
#endif

	//find the best split for each feature in the batch
	int nThreadFeaBestBlock = nBlockEachFea;
	if(nThreadFeaBestBlock > conf.m_maxBlockSize)
		nThreadFeaBestBlock = conf.m_maxBlockSize;

	dim3 dimBlockSizeFeaBest(nThreadFeaBestBlock, 1, 1);
	dim3 dimGridFeaBest(1, feaBatch, numofSNode);
	PickFeaGlobalBestSplit<<<dimGridFeaBest, dimBlockSizeFeaBest>>>(
							feaBatch, numofSNode, ffManager.m_pfFeaLocalBestGain_d, ffManager.m_pnFeaLocalBestGainKey_d,
							ffManager.m_pfFeaGlobalBestGain_d, ffManager.m_pnFeaGlobalBestGainKey_d, nBlockEachFea);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in ComputeGain" << endl;
		exit(0);
	}

	float_point *pfFeaGlobalBestGain_h = new float_point[feaBatch * numofSNode];
	int *pnFeaGlobalBestGainKey_h = new int[feaBatch * numofSNode];
	manager.MemcpyDeviceToHost(ffManager.m_pfFeaGlobalBestGain_d, pfFeaGlobalBestGain_h, sizeof(float_point) * feaBatch * numofSNode);
	manager.MemcpyDeviceToHost(ffManager.m_pnFeaGlobalBestGainKey_d, pnFeaGlobalBestGainKey_h, sizeof(int) * feaBatch * numofSNode);

	for(int n = 0; n < numofSNode; n++)
	{
		for(int f = 0; f < feaBatch; f++)
		{
			if(pfFeaGlobalBestGain_h[f + n * feaBatch] != pFeaBest[f + n * feaBatch])
				cerr << "fea best diff: " << pfFeaGlobalBestGain_h[f + n * feaBatch]
				     << " v.s. " << pFeaBest[f + n * feaBatch] << "; snid = " << n << "; fid=" << f << endl;
		}
	}

	delete []pFeaBest;
	delete []pfFeaGlobalBestGain_h;
	delete []pnFeaGlobalBestGainKey_h;
#endif

	//find the best feature
	//kernel configuration
	int blockSizeBestFeaBestSplit;
	dim3 tempNumofBlockBestFea;
	conf.ConfKernel(feaBatch, blockSizeBestFeaBestSplit, tempNumofBlockBestFea);
	int nBlockBestFea = tempNumofBlockBestFea.x;
	PROCESS_ERROR(tempNumofBlockBestFea.y == 1);
	dim3 dimBlockSizeBestFeaBestSplit(blockSizeBestFeaBestSplit, 1, 1);
	dim3 dimGridBestFeaBestSplit(nBlockBestFea, numofSNode, 1);

	PickLocalBestFeaBestSplit<<<dimGridBestFeaBestSplit, dimBlockSizeBestFeaBestSplit>>>
											(feaBatch, numofSNode, ffManager.m_pfFeaGlobalBestGain_d,
											 ffManager.m_pnFeaGlobalBestGainKey_d, ffManager.m_pfBlockBestFea_d, ffManager.m_pnBlockBestKey_d);

#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in PickBestFeaBestSplit" << endl;
		exit(0);
	}

	float_point *pfBlockBestFea_h = new float_point[nBlockBestFea * numofSNode];
	int *pnBlockBestKey_h = new int[nBlockBestFea * numofSNode];
	manager.MemcpyDeviceToHost(ffManager.m_pfBlockBestFea_d, pfBlockBestFea_h, sizeof(float_point) * nBlockBestFea * numofSNode);
	manager.MemcpyDeviceToHost(ffManager.m_pnBlockBestKey_d, pnBlockBestKey_h, sizeof(int) * nBlockBestFea * numofSNode);

	for(int n = 0; n < numofSNode; n++)
	{
		for(int f = 0; f < nBlockBestFea; f++)
		{
			if(pfBlockBestFea_h[n] != pGlobalBest[n] && nBlockBestFea == 1)
				cerr << "best gain diff: " << pfBlockBestFea_h[n] << " v.s. " << pGlobalBest[n] << endl;
		}
	}

	delete []pfBlockBestFea_h;
	delete []pnBlockBestKey_h;
#endif

	if(nBlockBestFea > 1)
	{
		int threadPerBlockBestFea;
		dim3 dimNumofBlockBestFea;
		conf.ConfKernel(nBlockBestFea, threadPerBlockBestFea, dimNumofBlockBestFea);
		PROCESS_ERROR(dimNumofBlockBestFea.y == 1 && dimNumofBlockBestFea.z == 1 && dimNumofBlockBestFea.x == 1);
		dimNumofBlockBestFea.x = numofSNode;
		PickGlobalBestFeaBestSplit<<<dimNumofBlockBestFea, threadPerBlockBestFea>>>(
													numofSNode, nBlockBestFea, ffManager.m_pfBlockBestFea_d,
													ffManager.m_pnBlockBestKey_d, ffManager.m_pfGlobalBestFea_d, ffManager.m_pnGlobalBestKey_d);
	}
	else
	{//local best fea is the global best fea
		manager.MemcpyDeviceToDevice(ffManager.m_pfBlockBestFea_d, ffManager.m_pfGlobalBestFea_d, sizeof(float_point) * numofSNode);
		manager.MemcpyDeviceToDevice(ffManager.m_pnBlockBestKey_d, ffManager.m_pnGlobalBestKey_d, sizeof(int) * numofSNode);
	}

#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in PickGlobalBestFeaBestSplit" << endl;
		exit(0);
	}

	float_point *pfGlobalBestFea_h = new float_point[numofSNode];
	int *pnGlobalBestKey_h = new int[numofSNode];
	manager.MemcpyDeviceToHost(ffManager.m_pfGlobalBestFea_d, pfGlobalBestFea_h, sizeof(float_point) * numofSNode);
	manager.MemcpyDeviceToHost(ffManager.m_pnGlobalBestKey_d, pnGlobalBestKey_h, sizeof(int) * numofSNode);

	for(int n = 0; n < numofSNode; n++)
	{

		if(pfGlobalBestFea_h[n] != pGlobalBest[n])
			cerr << "best gain diff: " << pfGlobalBestFea_h[n] << " v.s. " << pGlobalBest[n] << endl;
	}
	delete []pGlobalBest;
	delete []pnGlobalBestKey_h;
	delete []pfGlobalBestFea_h;
#endif

	//get split point info

	//Memory set for best split points; may not be necessary now.
	manager.MemcpyHostToDevice(manager.m_pBestPointHost, manager.m_pBestSplitPoint, sizeof(SplitPoint) * maxNumofSplittable);

	FindSplitInfo<<<1, numofSNode>>>(manager.m_pDNumofKeyValue, manager.m_pFeaStartPos, manager.m_pdDFeaValue,
								  feaBatch, smallestFeaId,
								  ffManager.m_pfGlobalBestFea_d, ffManager.m_pnGlobalBestKey_d, manager.m_pBuffIdVec,
								  manager.m_pSNodeStat, ffManager.m_pGDPrefixSum_d, ffManager.m_pHessPrefixSum_d,
								  manager.m_pBestSplitPoint, manager.m_pRChildStat, manager.m_pLChildStat,
								  manager.m_pLastValue, ffManager.m_pGainOnEachFeaValue_d);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cerr << "error in PickGlobalBestFeaBestSplit" << endl;
		exit(0);
	}
	SplitPoint *testBestSplitPoint1 = new SplitPoint[maxNumofSplittable];
	nodeStat *testpRChildStat = new nodeStat[maxNumofSplittable];
	nodeStat *testpLChildStat = new nodeStat[maxNumofSplittable];
	manager.MemcpyDeviceToHost(manager.m_pBestSplitPoint, testBestSplitPoint1, sizeof(SplitPoint) * maxNumofSplittable);
	manager.MemcpyDeviceToHost(manager.m_pRChildStat, testpRChildStat, sizeof(nodeStat) * maxNumofSplittable);
	manager.MemcpyDeviceToHost(manager.m_pLChildStat, testpLChildStat, sizeof(nodeStat) * maxNumofSplittable);

	for(int n = 0; n < numofSNode; n++)
	{
		int buffId = pBuffIdVec_h[n];
		if(testBestSplitPoint1[buffId].m_fGain != 0)
		{
			if(pSNodeStat_h[buffId].sum_hess != testpRChildStat[buffId].sum_hess + testpLChildStat[buffId].sum_hess ||
			   abs(pSNodeStat_h[buffId].sum_gd - testpRChildStat[buffId].sum_gd - testpLChildStat[buffId].sum_gd) > deltaTest)
				cerr << "parent node stat != child node stats: "<< pSNodeStat_h[buffId].sum_hess
					 << " != " << testpRChildStat[buffId].sum_hess << "+" << testpLChildStat[buffId].sum_hess
					 << "; " << pSNodeStat_h[buffId].sum_gd
					 << " != " << testpRChildStat[buffId].sum_gd << "+" << testpLChildStat[buffId].sum_gd << endl;
		}
	}

	delete []pSNodeStat_h;

	int threadPerBlock;
	dim3 dimNumofBlock;
	conf.ConfKernel(nNumofFeature, threadPerBlock, dimNumofBlock);

	clock_t begin_per_fea, begin_best;
	clock_t end_per_fea, end_best;
	hipDeviceSynchronize();
	begin_per_fea = clock();
	FindFeaSplitValue<<<dimNumofBlock, threadPerBlock>>>(
									  manager.m_pDNumofKeyValue, manager.m_pFeaStartPos, manager.m_pDInsId,
									  manager.m_pdDFeaValue, manager.m_pInsIdToNodeId,
									  manager.m_pGrad, manager.m_pHess,
									  manager.m_pTempRChildStatPerThread, manager.m_pLastValuePerThread,
									  manager.m_pSNodeStat, manager.m_pBestSplitPointPerThread,
									  manager.m_pRChildStatPerThread, manager.m_pLChildStatPerThread,
									  manager.m_pSNIdToBuffId, maxNumofSplittable, manager.m_pBuffIdVec, numofSNode,
									  DeviceSplitter::m_lambda, nNumofFeature);
	hipDeviceSynchronize();
	end_per_fea = clock();
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in FindFeaSplitValue" << endl;
		exit(0);
	}


	float_point *pfBestGain;
	int *pnBestGainKey;
	int threadPerBlockFindBest;
	dim3 dimNumofBlockFindBest;
	conf.ConfKernel(nNumofFeature, threadPerBlockFindBest, dimNumofBlockFindBest);
	PROCESS_ERROR(dimNumofBlockFindBest.y == 1);
	dimNumofBlockFindBest.y = numofSNode;
//	cout << "numof local best block is x=" << dimNumofBlockFindBest.x << " y=" << dimNumofBlockFindBest.y << endl;
	int numofBlockLocalBest = dimNumofBlockFindBest.x * dimNumofBlockFindBest.y;
	int numofBlockPerNode = dimNumofBlockFindBest.x;
	checkCudaErrors(hipMalloc((void**)&pfBestGain, sizeof(float_point) * maxNumofSplittable * numofBlockLocalBest));
	checkCudaErrors(hipMalloc((void**)&pnBestGainKey, sizeof(int) * maxNumofSplittable * numofBlockLocalBest));
	PickLocalBestFea<<<dimNumofBlockFindBest, threadPerBlockFindBest>>>(
					 manager.m_pBestSplitPointPerThread, manager.m_pBuffIdVec, numofSNode, nNumofFeature,
					 maxNumofSplittable, pfBestGain, pnBestGainKey);
	hipDeviceSynchronize();

	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in PickLocalBestFea" << endl;
		exit(0);
	}


	int blockSizeBestFea = numofBlockPerNode;
	if(blockSizeBestFea > conf.m_maxBlockSize)
		blockSizeBestFea = conf.m_maxBlockSize;

	PickGlobalBestFea<<<numofSNode, blockSizeBestFea>>>(manager.m_pLastValuePerThread,
					  manager.m_pBestSplitPointPerThread, manager.m_pRChildStatPerThread, manager.m_pLChildStatPerThread,
					  manager.m_pBuffIdVec, numofSNode, pfBestGain, pnBestGainKey, numofBlockPerNode);

	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in PickGlobalBestFea" << endl;
		exit(0);
	}


	//Memory set for best split points; may not be necessary now.
	manager.MemcpyHostToDevice(manager.m_pBestPointHost, manager.m_pBestSplitPoint, sizeof(SplitPoint) * maxNumofSplittable);
	manager.MemcpyDeviceToDevice(manager.m_pLastValuePerThread, manager.m_pLastValue, sizeof(float_point) * maxNumofSplittable);
	manager.MemcpyDeviceToDevice(manager.m_pRChildStatPerThread, manager.m_pRChildStat, sizeof(nodeStat) * maxNumofSplittable);
	manager.MemcpyDeviceToDevice(manager.m_pLChildStatPerThread, manager.m_pLChildStat, sizeof(nodeStat) * maxNumofSplittable);
	manager.MemcpyDeviceToDevice(manager.m_pBestSplitPointPerThread, manager.m_pBestSplitPoint, sizeof(SplitPoint) * maxNumofSplittable);


	//print best split points
	SplitPoint *testBestSplitPoint2 = new SplitPoint[maxNumofSplittable];
	nodeStat *testpRChildStat2 = new nodeStat[maxNumofSplittable];
	nodeStat *testpLChildStat2 = new nodeStat[maxNumofSplittable];
	manager.MemcpyDeviceToHost(manager.m_pBestSplitPoint, testBestSplitPoint2, sizeof(SplitPoint) * maxNumofSplittable);
	manager.MemcpyDeviceToHost(manager.m_pRChildStat, testpRChildStat2, sizeof(nodeStat) * maxNumofSplittable);
	manager.MemcpyDeviceToHost(manager.m_pLChildStat, testpLChildStat2, sizeof(nodeStat) * maxNumofSplittable);
	for(int sn = 0; sn < numofSNode; sn++)
	{
		int buffId = pBuffIdVec_h[sn];
		if(testBestSplitPoint1[buffId].m_nFeatureId != 0 && testBestSplitPoint2[buffId].m_nFeatureId != -1)
		{
		if(testBestSplitPoint1[buffId].m_nFeatureId != testBestSplitPoint2[buffId].m_nFeatureId ||
		   abs(testBestSplitPoint1[buffId].m_fGain - testBestSplitPoint2[buffId].m_fGain) > deltaTest ||
		   abs(testpRChildStat[buffId].sum_gd - testpRChildStat2[buffId].sum_gd) > deltaTest ||
		   testpRChildStat[buffId].sum_hess != testpRChildStat2[buffId].sum_hess ||
		   abs(testpLChildStat[buffId].sum_gd - testpLChildStat2[buffId].sum_gd) > deltaTest ||
		   testpLChildStat[buffId].sum_hess != testpLChildStat2[buffId].sum_hess)
			cerr << "final result diff: " << testBestSplitPoint1[buffId].m_nFeatureId << " v.s. " << testBestSplitPoint2[buffId].m_nFeatureId
				 << "; " << testBestSplitPoint1[buffId].m_fGain << " v.s. " << testBestSplitPoint2[buffId].m_fGain
				 << "; " << testBestSplitPoint1[buffId].m_fSplitValue << " v.s. " << testBestSplitPoint2[buffId].m_fSplitValue
				 << "; r gd: " << testpRChildStat[buffId].sum_gd << " v.s. " << testpRChildStat2[buffId].sum_gd
				 << "; r hess: " << testpRChildStat[buffId].sum_hess << " v.s. " << testpRChildStat2[buffId].sum_hess
				 << "; l gd: " << testpLChildStat[buffId].sum_gd << " v.s. " << testpLChildStat2[buffId].sum_gd
				 << "; l hess: " << testpLChildStat[buffId].sum_hess << " v.s. " << testpLChildStat2[buffId].sum_hess
				 << endl;
		}


//		cout << "nid=" << pTestBuffIdVect[sn] << "; snid=" << sn << "; gain=" << testBestSplitPoint[pTestBuffIdVect[sn]].m_fGain << "; fid="
//			 << testBestSplitPoint[pTestBuffIdVect[sn]].m_nFeatureId << "; sv=" << testBestSplitPoint[pTestBuffIdVect[sn]].m_fSplitValue << endl;
	}
	delete []testpRChildStat;
	delete []testpLChildStat;
	delete []testBestSplitPoint1;
	delete []testpRChildStat2;
	delete []testpLChildStat2;
	delete []testBestSplitPoint2;
	delete []pBuffIdVec_h;
#endif
}


