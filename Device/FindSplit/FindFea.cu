#include "hip/hip_runtime.h"
/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Hashing.h"
#include "../Bagging/BagManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/HostUtility.h"
#include "../../SharedUtility/powerOfTwo.h"
#include "../../SharedUtility/segmentedMax.h"

using std::cout;
using std::endl;
using std::make_pair;
using std::cerr;

template<class T>
__global__ void SetKey(uint *pSegStart, T *pSegLen, uint *pnKey){
	uint segmentId = blockIdx.x;//use one x covering multiple ys, because the maximum number of x-dimension is larger.
	__shared__ uint segmentLen, segmentStartPos;
	if(threadIdx.x == 0){//the first thread loads the segment length
		segmentLen = pSegLen[segmentId];
		segmentStartPos = pSegStart[segmentId];
	}
	__syncthreads();

	uint tid0 = blockIdx.y * blockDim.x;
	uint segmentThreadId = tid0 + threadIdx.x;
	if(tid0 >= segmentLen || segmentThreadId >= segmentLen)
		return;

	uint pos = segmentThreadId;
	while(pos < segmentLen){
		pnKey[pos + segmentStartPos] = segmentId;
		pos += blockDim.x;
	}
}

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	//reset memory for this bag
	{
		manager.MemsetAsync(bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);

		manager.MemsetAsync(bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(double) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = -1, maxNumFeaValueOneNode = -1;
	if(numofSNode > 1)
	{
		IndexComputer indexComp;
		indexComp.AllocMem(bagManager.m_numFea, numofSNode);
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
	
		clock_t start_gd = clock();
		//scatter operation
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		numofDenseValue = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_numIns, manager.m_pDInsId, manager.m_pdDFeaValue,
															   bagManager.m_pIndicesEachBag_d, numofDenseValue,
															   bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		indexComp.FreeMem();
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   	   	manager.m_pDInsId, manager.m_pdDFeaValue, bagManager.m_numFeaValue,
															   		bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		numofDenseValue = manager.m_numFeaValue;//for computing gain of each fvalue
		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
	}

//	cout << "prefix sum" << endl;
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	int totalNumArray = bagManager.m_numFea * numofSNode;
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled

	//construct keys for exclusive scan
	uint *pnKey_d;
	checkCudaErrors(hipMalloc((void**)&pnKey_d, bagManager.m_numFeaValue * sizeof(uint)));
	uint *pTempEachFeaStartEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;

	//set keys by GPU
	int maxSegLen = 0;
	int *pTempEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	int *pMaxLen = thrust::max_element(thrust::device, pTempEachFeaLenEachNode, pTempEachFeaLenEachNode + totalNumArray);
	checkCudaErrors(hipMemcpyAsync(&maxSegLen, pMaxLen, sizeof(int), hipMemcpyDeviceToHost, (*(hipStream_t*)pStream)));

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = totalNumArray;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<totalNumArray, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	double *pTempGDSum = bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	real *pTempHessSum = bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempGDSum, pTempGDSum);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempHessSum, pTempHessSum);


	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//default to left or right
	bool *pDefault2Right;
	checkCudaErrors(hipMalloc((void**)&pDefault2Right, sizeof(bool) * bagManager.m_numFeaValue));
	checkCudaErrors(hipMemset(pDefault2Right, 0, sizeof(bool) * bagManager.m_numFeaValue));

	//cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(numofDenseValue, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
											numofDenseValue, pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d, bagManager.m_numFea,
											bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
											pDefault2Right);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");
	
	//change the gain of the first feature value to 0
	int numFeaStartPos = bagManager.m_numFea * numofSNode;
//	printf("num fea start pos=%d (%d * %d)\n", numFeaStartPos, bagManager.m_numFea, numofSNode);
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numFeaStartPos, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
																bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
																numFeaStartPos, bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
																bagManager.m_numFeaValue);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after FirstFeaGain");

	clock_t end_comp_gain = clock();
	total_com_gain_t += (end_comp_gain - start_comp_gain);

//	cout << "searching" << endl;
	clock_t start_search = clock();
	real *pfGlobalBestGain_d;
	int *pnGlobalBestGainKey_d;
	checkCudaErrors(hipMalloc((void**)&pfGlobalBestGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pnGlobalBestGainKey_d, sizeof(int) * numofSNode));

	SegmentedMax(maxNumFeaValueOneNode, numofSNode, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
			bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
			bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue, pStream, pfGlobalBestGain_d, pnGlobalBestGainKey_d);

	hipStreamSynchronize((*(hipStream_t*)pStream));
	clock_t end_search = clock();
	total_search_t += end_search - start_search;

	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
									 bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
									 pfGlobalBestGain_d, pnGlobalBestGainKey_d,
				  	  	  	  	  	 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
				  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 pDefault2Right, pnKey_d,
				  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pnKey_d));
	checkCudaErrors(hipFree(pDefault2Right));
	checkCudaErrors(hipFree(pfGlobalBestGain_d));
	checkCudaErrors(hipFree(pnGlobalBestGainKey_d));
}

void CsrCompression(int numofSNode, uint &totalNumCsrFvalue, uint *eachCompressedFeaStartPos, uint *eachCompressedFeaLen,
		uint *eachNodeSizeInCsr, uint *eachCsrNodeStartPos, real *csrFvalue, double *csrGD_h, real *csrHess_h, uint *eachCsrLen){
	BagManager bagManager;
	real *fvalue_h = new real[bagManager.m_numFeaValue];
	uint *eachFeaLenEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	uint *eachFeaStartPosEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	checkCudaErrors(hipMemcpy(fvalue_h, bagManager.m_pDenseFValueEachBag, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(eachFeaLenEachNode_h, bagManager.m_pEachFeaLenEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(eachFeaStartPosEachNode_h, bagManager.m_pEachFeaStartPosEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));

	uint csrId = 0, curFvalueToCompress = 0;
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		eachCompressedFeaLen[i] = 0;
		uint feaLen = eachFeaLenEachNode_h[i];
		uint feaStart = eachFeaStartPosEachNode_h[i];
		if(feaLen == 0)continue;
		csrFvalue[csrId] = fvalue_h[feaStart];
		eachCsrLen[csrId] = 1;
		eachCompressedFeaLen[i] = 1;
		for(int l = 1; l < feaLen; l++){
			curFvalueToCompress++;
			if(fabs(fvalue_h[feaStart + l] - csrFvalue[csrId]) > DeviceSplitter::rt_eps){
				eachCompressedFeaLen[i]++;
				csrId++;
				csrFvalue[csrId] = fvalue_h[feaStart + l];
				eachCsrLen[csrId] = 1;
			}
			else
				eachCsrLen[csrId]++;
		}
		csrId++;
		curFvalueToCompress++;
	}
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		uint prefix = 0;
		for(int l = 0; l < i; l++)
			prefix += eachCompressedFeaLen[l];
		eachCompressedFeaStartPos[i] = prefix;
	}

	for(int i = 0; i < numofSNode; i++){
		int posOfLastFeaThisNode = (i + 1) * bagManager.m_numFea - 1;
		int posOfFirstFeaThisNode = i * bagManager.m_numFea;
		eachNodeSizeInCsr[i] = eachCompressedFeaStartPos[posOfLastFeaThisNode] - eachCompressedFeaStartPos[posOfFirstFeaThisNode];
		eachNodeSizeInCsr[i] += eachCompressedFeaLen[posOfLastFeaThisNode];
		eachCsrNodeStartPos[i] = eachCompressedFeaStartPos[posOfFirstFeaThisNode];
//		printf("node %d starts %u, len=%u\n", i, eachCsrNodeStartPos[i], eachNodeSizeInCsr[i]);
	}

	totalNumCsrFvalue = csrId;
//	printf("csrLen=%u, totalLen=%u, numofFeaValue=%u\n", csrId, totalLen, bagManager.m_numFeaValue);
	PROCESS_ERROR(totalNumCsrFvalue < bagManager.m_numFeaValue);
	//compute csr gd and hess
	double *gd_h = new double[bagManager.m_numFeaValue];
	real *hess_h = new real[bagManager.m_numFeaValue];
	checkCudaErrors(hipMemcpy(gd_h, bagManager.m_pdGDPrefixSumEachBag, sizeof(double) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hess_h, bagManager.m_pHessPrefixSumEachBag, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));

	uint globalPos = 0;
	for(int i = 0; i < csrId; i++){
		csrGD_h[i] = 0;
		csrHess_h[i] = 0;
		uint len = eachCsrLen[i];
		for(int v = 0; v < len; v++){
			csrGD_h[i] += gd_h[globalPos];
			csrHess_h[i] += hess_h[globalPos];
			globalPos++;
		}
	}

	printf("org=%u v.s. csr=%u\n", bagManager.m_numFeaValue, totalNumCsrFvalue);

	delete[] fvalue_h;
	delete[] eachFeaLenEachNode_h;
	delete[] eachFeaStartPosEachNode_h;
	delete[] gd_h;
	delete[] hess_h;
}

/**
 * @brief: efficient best feature finder
 */
__global__ void LoadFvalueInsId(const int *pOrgFvalueInsId, int *pNewFvalueInsId, const unsigned int *pDstIndexEachFeaValue, int numFeaValue)
{
	//one thread loads one value
	int gTid = GLOBAL_TID();

	if(gTid >= numFeaValue)//thread has nothing to load
		return;

	//index for scatter
	int idx = pDstIndexEachFeaValue[gTid];
	if(idx == -1)//instance is in a leaf node
		return;

	CONCHECKER(idx >= 0);
	CONCHECKER(idx < numFeaValue);

	//scatter: store GD, Hess and the feature value.
	pNewFvalueInsId[idx] = pOrgFvalueInsId[gTid];
}

__global__ void newCsrLenFvalue(const int *preFvalueInsId, int numFeaValue, const int *pInsId2Nid, int maxNid,
						  const uint *eachCsrStart, real *csrFvalue, uint numCsr, const uint *preRoundEachCsrFeaStartPos, const uint preRoundNumSN, int numFea,
						  real *eachCsrFvalueSparse, uint *csrNewLen, uint *eachCsrFeaLen, uint *eachNodeSizeInCsr){
	//one thread for one fvalue
	uint gTid = GLOBAL_TID();
	if(gTid >= numFeaValue)//thread has nothing to do
		return;

	int insId = preFvalueInsId[gTid];
	int pid = pInsId2Nid[insId] - maxNid - 1;//mapping to new node
	uint csrId = numCsr;
	RangeBinarySearch(gTid, eachCsrStart, numCsr, csrId);
	CONCHECKER(csrId < numCsr);
	uint segId = numFea * preRoundNumSN;
	RangeBinarySearch(csrId, preRoundEachCsrFeaStartPos, numFea * preRoundNumSN, segId);
	uint nodeId = segId / numFea;
	uint nodeStartPos = preRoundEachCsrFeaStartPos[nodeId * numFea];
	uint sizePreNodesAhead = nodeStartPos;
	uint curNodeSize;
	if(nodeId == preRoundNumSN - 1)
		curNodeSize = numCsr - nodeStartPos;
	else
		curNodeSize = preRoundEachCsrFeaStartPos[(nodeId + 1) * numFea] - nodeStartPos;
	uint localId = csrId - sizePreNodesAhead;
	uint orgValue;
	if(pid % 2 == 1){
		orgValue = atomicAdd(csrNewLen + sizePreNodesAhead * 2 + curNodeSize + localId, 1);
		if(orgValue == 0)
			eachCsrFvalueSparse[sizePreNodesAhead * 2 + curNodeSize + localId] = csrFvalue[csrId];
	}
	else{
		orgValue = atomicAdd(csrNewLen + sizePreNodesAhead * 2 + localId, 1);
		if(orgValue == 0)
			eachCsrFvalueSparse[sizePreNodesAhead * 2 + localId] = csrFvalue[csrId];
	}

	if(orgValue == 0){
		uint feaId = segId % numFea;
		CONCHECKER(feaId < numFea);
		atomicAdd(eachCsrFeaLen + pid * numFea + feaId, 1);
		atomicAdd(eachNodeSizeInCsr + pid, 1);
	}
}

__global__ void map2One(const uint *eachCsrFeaLen, uint numCsr, uint *csrMarker){
	uint gTid = GLOBAL_TID();
	if(gTid >= numCsr)
		return;
	if(eachCsrFeaLen[gTid] != 0)
		csrMarker[gTid] = 1;
	else
		csrMarker[gTid] = 0;
}

__global__ void loadDenseCsr(const real *eachCsrFvalueSparse, const uint *eachCsrFeaLen, uint numCsr, const uint *csrIdx, real *eachCsrFvalueDense, uint *eachCsrFeaLenDense){
	uint gTid = GLOBAL_TID();
	if(gTid >= numCsr)
		return;
	if(eachCsrFeaLen[gTid] != 0){
		uint idx = csrIdx[gTid] - 1;//inclusive scan is used to compute indices.
		eachCsrFeaLenDense[idx] = eachCsrFeaLen[gTid];
		eachCsrFvalueDense[idx] = eachCsrFvalueSparse[gTid];
	}
}

int *preFvalueInsId = NULL;
uint totalNumCsrFvalue_merge;
uint *eachCompressedFeaStartPos_merge;
uint *eachCompressedFeaLen_merge;
double *csrGD_h_merge;
real *csrHess_h_merge;
uint *eachNodeSizeInCsr_merge;
uint *eachCsrNodeStartPos_merge;
real *csrFvalue_merge;
uint *eachCsrLen_merge;
uint *eachNewCompressedFeaStart_merge;
void DeviceSplitter::FeaFinderAllNode2(void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	printf("preSN=%u, curSN=%u\n", bagManager.m_pPreNumSN_h[bagId], numofSNode);
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);
	//################
	int curNumofNode;
	manager.MemcpyDeviceToHostAsync(bagManager.m_pCurNumofNodeTreeOnTrainingEachBag_d + bagId, &curNumofNode, sizeof(int), pStream);
	vector<vector<real> > newCsrFvalue(numofSNode * bagManager.m_numFea, vector<real>());

	if(preFvalueInsId == NULL || curNumofNode == 1){
		eachNewCompressedFeaStart_merge = new uint[bagManager.m_numFea * bagManager.m_maxNumSplittable];
		eachCompressedFeaStartPos_merge = new uint[bagManager.m_numFea * bagManager.m_maxNumSplittable];
		eachCompressedFeaLen_merge = new uint[bagManager.m_numFea * bagManager.m_maxNumSplittable];
		csrGD_h_merge = new double[bagManager.m_numFeaValue];
		csrHess_h_merge = new real[bagManager.m_numFeaValue];
		eachCsrNodeStartPos_merge = new uint[bagManager.m_maxNumSplittable];
		eachCsrLen_merge = new uint[bagManager.m_numFeaValue];
		checkCudaErrors(hipHostMalloc((void**)&eachNodeSizeInCsr_merge, sizeof(uint) * bagManager.m_maxNumSplittable));
		checkCudaErrors(hipHostMalloc((void**)&csrFvalue_merge, sizeof(real) * bagManager.m_numFeaValue));
		checkCudaErrors(hipHostMalloc((void**)&preFvalueInsId, sizeof(int) * bagManager.m_numFeaValue));
		checkCudaErrors(hipMemcpy(preFvalueInsId, manager.m_pDInsId, sizeof(int) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	}
	//split nodes
	int *pInsId2Nid = new int[bagManager.m_numIns];//ins id to node id
	checkCudaErrors(hipMemcpy(pInsId2Nid, bagManager.m_pInsIdToNodeIdEachBag, sizeof(int) * bagManager.m_numIns, hipMemcpyDeviceToHost));
	//################3

	//reset memory for this bag
	{
		manager.MemsetAsync(bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(double) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = -1, maxNumFeaValueOneNode = -1;
	if(numofSNode > 1)
	{
		IndexComputer indexComp;
		indexComp.AllocMem(bagManager.m_numFea, numofSNode);
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;

		clock_t start_gd = clock();
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		numofDenseValue = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		indexComp.FreeMem();
		//###########
		printf("total csr fvalue=%u\n", totalNumCsrFvalue_merge);/**/
		PROCESS_ERROR(bagManager.m_numFeaValue >= totalNumCsrFvalue_merge);
		//split nodes
		uint *eachCsrStart;
		checkCudaErrors(hipHostMalloc((void**)&eachCsrStart, sizeof(uint) * totalNumCsrFvalue_merge));
		thrust::exclusive_scan(thrust::host, eachCsrLen_merge, eachCsrLen_merge + totalNumCsrFvalue_merge, eachCsrStart);
		uint *firstCsrLen;
		real *eachCsrFvalueSparse;
		uint *eachCsrFeaLen;
		uint *eachCsrFeaStartPos;
		checkCudaErrors(hipHostMalloc((void**)&firstCsrLen, sizeof(uint) * totalNumCsrFvalue_merge * 2));
		checkCudaErrors(hipHostMalloc((void**)&eachCsrFvalueSparse, sizeof(real) * totalNumCsrFvalue_merge * 2));
		checkCudaErrors(hipHostMalloc((void**)&eachCsrFeaLen, sizeof(uint) * bagManager.m_numFea * numofSNode));
		checkCudaErrors(hipHostMalloc((void**)&eachCsrFeaStartPos, sizeof(uint) * bagManager.m_numFea * bagManager.m_pPreNumSN_h[bagId]));
		checkCudaErrors(hipMemset(firstCsrLen, 0, sizeof(uint) * totalNumCsrFvalue_merge * 2));
		checkCudaErrors(hipMemset(eachCsrFeaLen, 0, sizeof(uint) * bagManager.m_numFea * numofSNode));
		checkCudaErrors(hipMemcpy(eachCsrFeaStartPos, eachCompressedFeaStartPos_merge, sizeof(uint) * bagManager.m_numFea * bagManager.m_pPreNumSN_h[bagId],
						hipMemcpyHostToDevice));
		checkCudaErrors(hipMemset(eachNodeSizeInCsr_merge, 0, sizeof(uint) * bagManager.m_maxNumSplittable));

		newCsrLenFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD>>>(preFvalueInsId, bagManager.m_numFeaValue,
											bagManager.m_pInsIdToNodeIdEachBag + bagId * bagManager.m_numIns,
											bagManager.m_pPreMaxNid_h[bagId], eachCsrStart,
											csrFvalue_merge, totalNumCsrFvalue_merge,
											eachCsrFeaStartPos, bagManager.m_pPreNumSN_h[bagId],
											bagManager.m_numFea, eachCsrFvalueSparse, firstCsrLen, eachCsrFeaLen,
											eachNodeSizeInCsr_merge);

		int blockSizeLoadCsrLen;
		dim3 dimNumofBlockToLoadCsrLen;
		conf.ConfKernel(totalNumCsrFvalue_merge * 2, blockSizeLoadCsrLen, dimNumofBlockToLoadCsrLen);
		uint *csrMarker;
		checkCudaErrors(hipHostMalloc((void**)&csrMarker, sizeof(uint) * totalNumCsrFvalue_merge * 2));
		checkCudaErrors(hipMemset(csrMarker, 0, sizeof(uint) * totalNumCsrFvalue_merge * 2));
		map2One<<<dimNumofBlockToLoadCsrLen, blockSizeLoadCsrLen>>>(firstCsrLen, totalNumCsrFvalue_merge * 2, csrMarker);
		thrust::inclusive_scan(thrust::device, csrMarker, csrMarker + totalNumCsrFvalue_merge * 2, csrMarker);
		hipDeviceSynchronize();
		uint totalNumCsrBest = csrMarker[totalNumCsrFvalue_merge * 2 - 1];
		printf("num csr=%u, dense csr=%u\n", totalNumCsrFvalue_merge * 2, totalNumCsrBest);
		uint *eachCsrFeaLenDense;
		real *eachCsrFvalueDense;
		checkCudaErrors(hipHostMalloc((void**)&eachCsrFeaLenDense, sizeof(uint) * totalNumCsrBest));
		checkCudaErrors(hipHostMalloc((void**)&eachCsrFvalueDense, sizeof(real) * totalNumCsrBest));
		checkCudaErrors(hipMemset(eachCsrFeaLenDense, -1, sizeof(uint) * totalNumCsrBest));
		loadDenseCsr<<<dimNumofBlockToLoadCsrLen, blockSizeLoadCsrLen>>>(eachCsrFvalueSparse, firstCsrLen, totalNumCsrFvalue_merge * 2, csrMarker, eachCsrFvalueDense, eachCsrFeaLenDense);

		hipDeviceSynchronize();

		printf("hello world org=%u v.s. csr=%u\n", bagManager.m_numFeaValue, totalNumCsrBest);
		thrust::exclusive_scan(thrust::host, eachCsrFeaLen, eachCsrFeaLen + numofSNode * bagManager.m_numFea, eachNewCompressedFeaStart_merge);
		delete[] pInsId2Nid;
		//###############################
		LoadFvalueInsId<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(
						manager.m_pDInsId, preFvalueInsId, bagManager.m_pIndicesEachBag_d, bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		thrust::exclusive_scan(thrust::host, eachNodeSizeInCsr_merge, eachNodeSizeInCsr_merge + numofSNode, eachCsrNodeStartPos_merge);//newly added#########
		totalNumCsrFvalue_merge = totalNumCsrBest;
		memcpy(eachCompressedFeaStartPos_merge, eachNewCompressedFeaStart_merge, sizeof(uint) * bagManager.m_numFea * numofSNode);
		memcpy(eachCompressedFeaLen_merge, eachCsrFeaLen, sizeof(uint) * bagManager.m_numFea * numofSNode);

		checkCudaErrors(hipMemcpy(csrFvalue_merge, eachCsrFvalueDense, sizeof(real) * totalNumCsrBest, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(eachCsrLen_merge, eachCsrFeaLenDense, sizeof(uint) * totalNumCsrBest, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		real *pInsGrad = new real[bagManager.m_numIns];
		real *pInsHess = new real[bagManager.m_numIns];
		checkCudaErrors(hipMemcpy(pInsGrad, bagManager.m_pInsGradEachBag, sizeof(real) * bagManager.m_numIns, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(pInsHess, bagManager.m_pInsHessEachBag, sizeof(real) * bagManager.m_numIns, hipMemcpyDeviceToHost));

		uint globalPos = 0;
		for(int i = 0; i < totalNumCsrFvalue_merge; i++){
			csrGD_h_merge[i] = 0;
			csrHess_h_merge[i] = 0;
			uint len = eachCsrLen_merge[i];
			for(int v = 0; v < len; v++){
				int insId = preFvalueInsId[globalPos];
				csrGD_h_merge[i] += pInsGrad[insId];
				csrHess_h_merge[i] += pInsHess[insId];
				globalPos++;
			}
		}
		delete[] pInsGrad;
		//##############
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   	   	manager.m_pDInsId, manager.m_pdDFeaValue, bagManager.m_numFeaValue,
															   		bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		numofDenseValue = manager.m_numFeaValue;//for computing gain of each fvalue
		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
		//###### compress
		CsrCompression(numofSNode, totalNumCsrFvalue_merge, eachCompressedFeaStartPos_merge, eachCompressedFeaLen_merge,
				   eachNodeSizeInCsr_merge, eachCsrNodeStartPos_merge, csrFvalue_merge, csrGD_h_merge, csrHess_h_merge, eachCsrLen_merge);
		printf("total csr fvalue=%u\n", totalNumCsrFvalue_merge);
	}

	//	cout << "prefix sum" << endl;
	int numSeg = bagManager.m_numFea * numofSNode;
	real *pCsrFvalue_d;
	uint *pEachCompressedFeaStartPos_d;
	uint *pEachCompressedFeaLen_d;
	double *pCsrGD_d;
	real *pCsrHess_d;
	uint *pEachCsrNodeSize_d;
	uint *pEachCsrNodeStart_d;
	checkCudaErrors(hipMalloc((void**)&pEachCompressedFeaStartPos_d, sizeof(uint) * numSeg));
	checkCudaErrors(hipMalloc((void**)&pEachCompressedFeaLen_d, sizeof(uint) * numSeg));
	checkCudaErrors(hipMalloc((void**)&pCsrFvalue_d, sizeof(real) * totalNumCsrFvalue_merge));
	checkCudaErrors(hipMalloc((void**)&pCsrGD_d, sizeof(double) * totalNumCsrFvalue_merge));
	checkCudaErrors(hipMalloc((void**)&pCsrHess_d, sizeof(real) * totalNumCsrFvalue_merge));
	checkCudaErrors(hipMalloc((void**)&pEachCsrNodeSize_d, sizeof(uint) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pEachCsrNodeStart_d, sizeof(uint) * numofSNode));

	checkCudaErrors(hipMemcpy(pEachCompressedFeaStartPos_d, eachCompressedFeaStartPos_merge, sizeof(uint) * numSeg, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pEachCompressedFeaLen_d, eachCompressedFeaLen_merge, sizeof(uint) * numSeg, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrFvalue_d, csrFvalue_merge, sizeof(real) * totalNumCsrFvalue_merge, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrHess_d, csrHess_h_merge, sizeof(real) * totalNumCsrFvalue_merge, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrGD_d, csrGD_h_merge, sizeof(double) * totalNumCsrFvalue_merge, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pEachCsrNodeSize_d, eachNodeSizeInCsr_merge, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pEachCsrNodeStart_d, eachCsrNodeStartPos_merge, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled

	//construct keys for exclusive scan
	uint *pnCsrKey_d;
	checkCudaErrors(hipMalloc((void**)&pnCsrKey_d, sizeof(uint) * totalNumCsrFvalue_merge));

	//set keys by GPU
	uint maxSegLen = 0;
	uint *pMaxLen = thrust::max_element(thrust::device, pEachCompressedFeaLen_d, pEachCompressedFeaLen_d + numSeg);
	checkCudaErrors(hipMemcpyAsync(&maxSegLen, pMaxLen, sizeof(uint), hipMemcpyDeviceToHost, (*(hipStream_t*)pStream)));

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = numSeg;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<numSeg, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(pEachCompressedFeaStartPos_d, pEachCompressedFeaLen_d, pnCsrKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	thrust::inclusive_scan_by_key(thrust::device, pnCsrKey_d, pnCsrKey_d + totalNumCsrFvalue_merge, pCsrGD_d, pCsrGD_d);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::device, pnCsrKey_d, pnCsrKey_d + totalNumCsrFvalue_merge, pCsrHess_d, pCsrHess_d);

	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//compute gain
	//default to left or right
	bool *pCsrDefault2Right_d;
	real *pGainEachCsrFvalue_d;
	checkCudaErrors(hipMalloc((void**)&pCsrDefault2Right_d, sizeof(bool) * totalNumCsrFvalue_merge));
	checkCudaErrors(hipMalloc((void**)&pGainEachCsrFvalue_d, sizeof(real) * totalNumCsrFvalue_merge));

	//cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(totalNumCsrFvalue_merge, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, pCsrGD_d, pCsrHess_d, pCsrFvalue_d,
											totalNumCsrFvalue_merge, pEachCompressedFeaStartPos_d, pEachCompressedFeaLen_d, pnCsrKey_d, bagManager.m_numFea,
											pGainEachCsrFvalue_d, pCsrDefault2Right_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");

	//change the gain of the first feature value to 0
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numSeg, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
										pEachCompressedFeaStartPos_d, numSeg, pGainEachCsrFvalue_d, totalNumCsrFvalue_merge);

	//	cout << "searching" << endl;
	clock_t start_search = clock();
	real *pMaxGain_d;
	uint *pMaxGainKey_d;
	checkCudaErrors(hipMalloc((void**)&pMaxGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pMaxGainKey_d, sizeof(uint) * numofSNode));
	//compute # of blocks for each node
	uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pEachCsrNodeSize_d, pEachCsrNodeSize_d + numofSNode);
	checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));

	SegmentedMax(maxNumFeaValueOneNode, numofSNode, pEachCsrNodeSize_d, pEachCsrNodeStart_d,
					  pGainEachCsrFvalue_d, pStream, pMaxGain_d, pMaxGainKey_d);

	hipStreamSynchronize((*(hipStream_t*)pStream));

	//find the split value and feature
	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
										 pEachCompressedFeaStartPos_d,
										 pEachCompressedFeaLen_d,
										 pCsrFvalue_d,
										 pMaxGain_d, pMaxGainKey_d,
										 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
					  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 pCsrGD_d,
					  	  	  	  	  	 pCsrHess_d,
					  	  	  	  	  	 pCsrDefault2Right_d, pnCsrKey_d,
					  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	checkCudaErrors(hipFree(pEachCsrNodeSize_d));
	checkCudaErrors(hipFree(pEachCsrNodeStart_d));
	checkCudaErrors(hipFree(pGainEachCsrFvalue_d));
	checkCudaErrors(hipFree(pMaxGain_d));
	checkCudaErrors(hipFree(pMaxGainKey_d));
	checkCudaErrors(hipFree(pEachCompressedFeaStartPos_d));
	checkCudaErrors(hipFree(pEachCompressedFeaLen_d));
	checkCudaErrors(hipFree(pCsrFvalue_d));
	checkCudaErrors(hipFree(pCsrGD_d));
	checkCudaErrors(hipFree(pCsrHess_d));
	checkCudaErrors(hipFree(pCsrDefault2Right_d));
	checkCudaErrors(hipFree(pnCsrKey_d));
}

