#include "hip/hip_runtime.h"
/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Hashing.h"
#include "../Bagging/BagManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/HostUtility.h"
#include "../../SharedUtility/powerOfTwo.h"

using std::cout;
using std::endl;
using std::make_pair;
using std::cerr;

__global__ void SetKey(uint *pSegStart, int *pSegLen, uint *pnKey){
	uint segmentId = blockIdx.x;//use one x covering multiple ys, because the maximum number of x-dimension is larger.
	__shared__ uint segmentLen, segmentStartPos;
	if(threadIdx.x == 0){//the first thread loads the segment length
		segmentLen = pSegLen[segmentId];
		segmentStartPos = pSegStart[segmentId];
	}
	__syncthreads();

	uint tid0 = blockIdx.y * blockDim.x;
	uint segmentThreadId = tid0 + threadIdx.x;
	if(tid0 >= segmentLen || segmentThreadId >= segmentLen)
		return;

	uint pos = segmentThreadId;
	while(pos < segmentLen){
		pnKey[pos + segmentStartPos] = segmentId;
		pos += blockDim.x;
	}
}

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	//reset memory for this bag
	{
		manager.MemsetAsync(bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);

		manager.MemsetAsync(bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(double) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = -1, maxNumFeaValueOneNode = -1;
	if(numofSNode > 1)
	{
		IndexComputer indexComp;
		indexComp.AllocMem(bagManager.m_numFea, numofSNode);
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
	
		clock_t start_gd = clock();
		//scatter operation
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		numofDenseValue = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_numIns, manager.m_pDInsId, manager.m_pdDFeaValue,
															   bagManager.m_pIndicesEachBag_d, numofDenseValue,
															   bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		indexComp.FreeMem();
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   	   	manager.m_pDInsId, manager.m_pdDFeaValue, bagManager.m_numFeaValue,
															   		bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		numofDenseValue = manager.m_numFeaValue;//for computing gain of each fvalue
		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
	}

//	cout << "prefix sum" << endl;
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	int totalNumArray = bagManager.m_numFea * numofSNode;
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled

	//construct keys for exclusive scan
	uint *pnKey_d;
	checkCudaErrors(hipMalloc((void**)&pnKey_d, bagManager.m_numFeaValue * sizeof(uint)));
	uint *pTempEachFeaStartEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;

	//set keys by GPU
	int maxSegLen = 0;
	int *pTempEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	int *pMaxLen = thrust::max_element(thrust::device, pTempEachFeaLenEachNode, pTempEachFeaLenEachNode + totalNumArray);
	checkCudaErrors(hipMemcpyAsync(&maxSegLen, pMaxLen, sizeof(int), hipMemcpyDeviceToHost, (*(hipStream_t*)pStream)));

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = totalNumArray;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<totalNumArray, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	double *pTempGDSum = bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	real *pTempHessSum = bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempGDSum, pTempGDSum);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempHessSum, pTempHessSum);


	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//default to left or right
	bool *pDefault2Right;
	checkCudaErrors(hipMalloc((void**)&pDefault2Right, sizeof(bool) * bagManager.m_numFeaValue));
	checkCudaErrors(hipMemset(pDefault2Right, 0, sizeof(bool) * bagManager.m_numFeaValue));

	//cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(numofDenseValue, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
											numofDenseValue, pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d, bagManager.m_numFea,
											bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
											pDefault2Right);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");
	
	//change the gain of the first feature value to 0
	int numFeaStartPos = bagManager.m_numFea * numofSNode;
//	printf("num fea start pos=%d (%d * %d)\n", numFeaStartPos, bagManager.m_numFea, numofSNode);
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numFeaStartPos, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
																bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
																numFeaStartPos, bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
																bagManager.m_numFeaValue);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after FirstFeaGain");

	clock_t end_comp_gain = clock();
	total_com_gain_t += (end_comp_gain - start_comp_gain);

//	cout << "searching" << endl;
	clock_t start_search = clock();
	real *pfLocalBestGain_d, *pfGlobalBestGain_d;
	int *pnLocalBestGainKey_d, *pnGlobalBestGainKey_d;
	//compute # of blocks for each node
	PROCESS_ERROR(maxNumFeaValueOneNode > 0);
	int blockSizeLocalBestGain;
	dim3 dimNumofBlockLocalBestGain;
	conf.ConfKernel(maxNumFeaValueOneNode, blockSizeLocalBestGain, dimNumofBlockLocalBestGain);
	PROCESS_ERROR(dimNumofBlockLocalBestGain.z == 1);
	dimNumofBlockLocalBestGain.z = numofSNode;//each node per super block
	int numBlockPerNode = dimNumofBlockLocalBestGain.x * dimNumofBlockLocalBestGain.y;

	checkCudaErrors(hipMalloc((void**)&pfLocalBestGain_d, sizeof(real) * numBlockPerNode * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pnLocalBestGainKey_d, sizeof(int) * numBlockPerNode * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pfGlobalBestGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pnGlobalBestGainKey_d, sizeof(int) * numofSNode));
	//find the block level best gain for each node
	PickLocalBestSplitEachNode<<<dimNumofBlockLocalBestGain, blockSizeLocalBestGain, 0, (*(hipStream_t*)pStream)>>>(
								bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
								pfLocalBestGain_d,
								pnLocalBestGainKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after PickLocalBestSplitEachNode");

	//find the global best gain for each node
	if(numBlockPerNode > 1){
		int blockSizeBestGain;
		dim3 dimNumofBlockDummy;
		conf.ConfKernel(numBlockPerNode, blockSizeBestGain, dimNumofBlockDummy);
		PickGlobalBestSplitEachNode<<<numofSNode, blockSizeBestGain, 0, (*(hipStream_t*)pStream)>>>(
									pfLocalBestGain_d,
									pnLocalBestGainKey_d,
									pfGlobalBestGain_d,
									pnGlobalBestGainKey_d,
								    numBlockPerNode, numofSNode);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		GETERROR("after PickGlobalBestSplitEachNode");
	}
	else{//local best fea is the global best fea
		manager.MemcpyDeviceToDeviceAsync(pfLocalBestGain_d, pfGlobalBestGain_d,
										sizeof(real) * numofSNode, pStream);
		manager.MemcpyDeviceToDeviceAsync(pnLocalBestGainKey_d, pnGlobalBestGainKey_d,
											sizeof(int) * numofSNode, pStream);
	}

	hipStreamSynchronize((*(hipStream_t*)pStream));
	clock_t end_search = clock();
	total_search_t += end_search - start_search;

	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
									 bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
									 pfGlobalBestGain_d, pnGlobalBestGainKey_d,
				  	  	  	  	  	 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
				  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 pDefault2Right, pnKey_d,
				  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pnKey_d));
	checkCudaErrors(hipFree(pDefault2Right));
	checkCudaErrors(hipFree(pfLocalBestGain_d));
	checkCudaErrors(hipFree(pfGlobalBestGain_d));
	checkCudaErrors(hipFree(pnLocalBestGainKey_d));
	checkCudaErrors(hipFree(pnGlobalBestGainKey_d));
}

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode2(void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	//reset memory for this bag
	{
		manager.MemsetAsync(bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);

		manager.MemsetAsync(bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(double) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = -1, maxNumFeaValueOneNode = -1;
	if(numofSNode > 1)
	{
		IndexComputer indexComp;
		indexComp.AllocMem(bagManager.m_numFea, numofSNode);
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;

		clock_t start_gd = clock();
		//scatter operation
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		numofDenseValue = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns,
															   bagManager.m_numIns, manager.m_pDInsId, manager.m_pdDFeaValue,
															   bagManager.m_pIndicesEachBag_d, numofDenseValue,
															   bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		indexComp.FreeMem();
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   	   	manager.m_pDInsId, manager.m_pdDFeaValue, bagManager.m_numFeaValue,
															   		bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		numofDenseValue = manager.m_numFeaValue;//for computing gain of each fvalue
		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
	}

	//compress fvalues ###########
	real *fvalue_h = new real[bagManager.m_numFeaValue];
	checkCudaErrors(hipMemcpy(fvalue_h, bagManager.m_pDenseFValueEachBag, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	real *csrFvalue = new real[bagManager.m_numFeaValue];
	uint *csrOrgFvalueStartPos = new uint[bagManager.m_numFeaValue];
	memset(csrOrgFvalueStartPos, -1, sizeof(uint) * bagManager.m_numFeaValue);
	uint *eachCsrLen = new uint[bagManager.m_numFeaValue];
	memset(eachCsrLen, -1, sizeof(uint) * bagManager.m_numFeaValue);
	uint *eachFeaLenEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	uint *eachFeaStartPosEachNode_h = new uint[bagManager.m_numFea * numofSNode];
	checkCudaErrors(hipMemcpy(eachFeaLenEachNode_h, bagManager.m_pEachFeaLenEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(eachFeaStartPosEachNode_h, bagManager.m_pEachFeaStartPosEachNodeEachBag_d, sizeof(uint) * bagManager.m_numFea * numofSNode, hipMemcpyDeviceToHost));
	uint *eachCompressedFeaLen = new uint[bagManager.m_numFea * numofSNode];
	memset(eachCompressedFeaLen, -1, sizeof(uint) * bagManager.m_numFea * numofSNode);
	uint *eachCompressedFeaStartPos = new uint[bagManager.m_numFea * numofSNode];
	memset(eachCompressedFeaStartPos, -1, sizeof(uint) * bagManager.m_numFea * numofSNode);
	uint csrId = 0, curFvalueToCompress = 0;
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		eachCompressedFeaLen[i] = 0;
		uint feaStart = eachFeaStartPosEachNode_h[i];
		uint feaLen = eachFeaLenEachNode_h[i];
		if(feaLen == 0)continue;
		csrFvalue[csrId] = fvalue_h[feaStart];
		csrOrgFvalueStartPos[csrId] = curFvalueToCompress;
		eachCsrLen[csrId] = 1;
		eachCompressedFeaLen[i] = 1;
		for(int l = 1; l < feaLen; l++){
			curFvalueToCompress++;
			if(fabs(fvalue_h[feaStart + l] - csrFvalue[csrId]) > DeviceSplitter::rt_eps){
				eachCompressedFeaLen[i]++;
				csrId++;
				csrFvalue[csrId] = fvalue_h[feaStart + l];
				csrOrgFvalueStartPos[csrId] = curFvalueToCompress;
				eachCsrLen[csrId] = 1;
			}
			else
				eachCsrLen[csrId]++;
		}
		csrId++;
		curFvalueToCompress++;
	}
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++){
		uint prefix = 0;
		for(int l = 0; l < i; l++)
			prefix += eachCompressedFeaLen[l];
		eachCompressedFeaStartPos[i] = prefix;
	}
	uint *eachNodeSizeInCsr = new uint[numofSNode];
	uint *eachCsrNodeStartPos = new uint[numofSNode];
	for(int i = 0; i < numofSNode; i++){
		int posOfLastFeaThisNode = (i + 1) * bagManager.m_numFea - 1;
		int posOfFirstFeaThisNode = i * bagManager.m_numFea;
		eachNodeSizeInCsr[i] = eachCompressedFeaStartPos[posOfLastFeaThisNode] - eachCompressedFeaStartPos[posOfFirstFeaThisNode];
		eachNodeSizeInCsr[i] += eachCompressedFeaLen[posOfLastFeaThisNode];
		eachCsrNodeStartPos[i] = eachCompressedFeaStartPos[posOfFirstFeaThisNode];
		printf("node %d starts %u, len=%u\n", i, eachCsrNodeStartPos[i], eachNodeSizeInCsr[i]);
	}

	uint totalLen = 0;
	for(int i = 0; i < csrId; i++){
		totalLen += eachCsrLen[i];
	}
	uint totalNumCsrFvalue = 0;
	for(int i = 0; i < bagManager.m_numFea * numofSNode; i++)
		totalNumCsrFvalue += eachCompressedFeaLen[i];
	printf("csrLen=%u, totalLen=%u, totalLen2=%u; numofFeaValue=%u\n", csrId, totalLen, totalNumCsrFvalue, bagManager.m_numFeaValue);
	PROCESS_ERROR(csrId == totalNumCsrFvalue);
	PROCESS_ERROR(totalNumCsrFvalue < bagManager.m_numFeaValue);
	//PROCESS_ERROR(totalLen == bagManager.m_numFeaValue);
	//update gd and hess
	double *gd_h = new double[bagManager.m_numFeaValue];
	real *hess_h = new real[bagManager.m_numFeaValue];
	checkCudaErrors(hipMemcpy(gd_h, bagManager.m_pdGDPrefixSumEachBag, sizeof(double) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hess_h, bagManager.m_pHessPrefixSumEachBag, sizeof(real) * bagManager.m_numFeaValue, hipMemcpyDeviceToHost));
	double *csrGD_h = new double[bagManager.m_numFeaValue];
	real *csrHess_h = new real[bagManager.m_numFeaValue];
	uint globalPos = 0;
	for(int i = 0; i < csrId; i++){
		csrGD_h[i] = 0;
		csrHess_h[i] = 0;
		uint len = eachCsrLen[i];
		for(int v = 0; v < len; v++){
			csrGD_h[i] += gd_h[globalPos];
			csrHess_h[i] += hess_h[globalPos];
			globalPos++;
		}
	}
	double totalGD = 0, totalHess = 0;
	for(int i = 0; i < csrId; i++){
		totalGD += csrGD_h[i];
		totalHess += csrHess_h[i];
	}
	double totalOrgGD = 0;
	for(int i = 0; i < bagManager.m_numFeaValue; i++){
		totalOrgGD += gd_h[i];
	}
	//printf("total gd=%f, total hess=%f, orgGD=%f\n", totalGD, totalHess, totalOrgGD);
	PROCESS_ERROR(fabs(totalGD - totalOrgGD) < 0.001);
	uint *pnKey_h = new uint[totalNumCsrFvalue];
	uint segStart = 0;
	for(int segId = 0; segId < bagManager.m_numFea * numofSNode; segId++){
		uint segLen = eachCompressedFeaLen[segId];
		for(int i = 0; i < segLen; i++){
			pnKey_h[i + segStart] = segId;
		}
		segStart += segLen;
	}
	thrust::inclusive_scan_by_key(thrust::host, pnKey_h, pnKey_h + totalNumCsrFvalue, csrGD_h, csrGD_h);
	thrust::inclusive_scan_by_key(thrust::host, pnKey_h, pnKey_h + totalNumCsrFvalue, csrHess_h, csrHess_h);
	//compute gain
	nodeStat *snNode_h = new nodeStat[bagManager.m_maxNumSplittable];
	checkCudaErrors(hipMemcpy(snNode_h, bagManager.m_pSNodeStatEachBag, sizeof(nodeStat) * bagManager.m_maxNumSplittable, hipMemcpyDeviceToHost));
	int *pid2snPos = new int[bagManager.m_maxNumSplittable];
	checkCudaErrors(hipMemcpy(pid2snPos, bagManager.m_pPartitionId2SNPosEachBag, sizeof(int) * bagManager.m_maxNumSplittable, hipMemcpyDeviceToHost));

	real *pGainOnEachFvalue_h = new real[totalNumCsrFvalue];
	bool *pDefault2Right_h = new bool[totalNumCsrFvalue];
	pGainOnEachFvalue_h[0] = 0;
	for(int i = 1; i < totalNumCsrFvalue; i++){
		//forward consideration (fvalues are sorted descendingly)
		double rChildGD = csrGD_h[i - 1];
		double rChildHess = csrHess_h[i - 1];
		uint segId = pnKey_h[i];
		uint pid = segId / bagManager.m_numFea;
		int snPos = pid2snPos[pid];
		PROCESS_ERROR(snPos >= 0 || snPos < bagManager.m_maxNumSplittable);
		double parentGD = snNode_h[snPos].sum_gd;
		double parentHess = snNode_h[snPos].sum_hess;
		PROCESS_ERROR(parentHess > 0);
		double tempGD = parentGD - rChildGD;
		double tempHess = parentHess - rChildHess;
		if(rChildHess >= 1 && tempHess >= 1)//need to compute the gain
		{
			double tempGain = (tempGD * tempGD)/(tempHess + DeviceSplitter::m_lambda) +
								   (rChildGD * rChildGD)/(rChildHess + DeviceSplitter::m_lambda) -
								   (parentGD * parentGD)/(parentHess + DeviceSplitter::m_lambda);
			pGainOnEachFvalue_h[i] = tempGain;
		}
		else{
			//assign gain to 0
			pGainOnEachFvalue_h[i] = 0;
		}

	    //backward consideration
	    int segLen = eachCompressedFeaLen[segId];
	    uint segStartPos = eachCompressedFeaStartPos[segId];
	    PROCESS_ERROR(segLen >= 0);
	    uint lastFvaluePos = segStartPos + segLen - 1;
	    PROCESS_ERROR(lastFvaluePos < totalNumCsrFvalue);
	    double totalMissingGD = parentGD - csrGD_h[lastFvaluePos];
	    double totalMissingHess = parentHess - csrHess_h[lastFvaluePos];
	    if(totalMissingHess < 1)//there is no instance with missing values
	    	continue;
	    //missing values to the right child
	    rChildGD += totalMissingGD;
	    rChildHess += totalMissingHess;
	    tempGD = parentGD - rChildGD;
	    tempHess = parentHess - rChildHess;
	    if(rChildHess >= 1 && tempHess >= 1){
	    	double tempGain = (tempGD * tempGD)/(tempHess + DeviceSplitter::m_lambda) +
				  	   	    (rChildGD * rChildGD)/(rChildHess + DeviceSplitter::m_lambda) -
				  	   	    (parentGD * parentGD)/(parentHess + DeviceSplitter::m_lambda);

	    	if(tempGain > 0 && tempGain - pGainOnEachFvalue_h[i] > 0.1){
	    		pGainOnEachFvalue_h[i] = tempGain;
	    		pDefault2Right_h[i] = true;
	    	}
	    }
	}

	//find best gain for each node
	int *pBestFeaId = new int[numofSNode];
	for(int i = 0; i < numofSNode; i++){
		for(int f = 0; f < bagManager.m_numFea; f++){
			uint segLen = eachCompressedFeaLen[i * bagManager.m_numFea + f];
			if(segLen == 0)continue;
			uint segStartPos = eachCompressedFeaStartPos[i * bagManager.m_numFea + f];
			pGainOnEachFvalue_h[segStartPos] = 0;//fix first gain
		}
	}

	//	cout << "searching" << endl;
	clock_t start_search = clock();
	uint *pEachCsrNodeSize_d;
	uint *pEachCsrNodeStart_d;
	real *pGainEachCsrFvalue_d;
	real *pMaxGain_d;
	uint *pMaxGainKey_d;
	checkCudaErrors(hipMalloc((void**)&pEachCsrNodeSize_d, sizeof(uint) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pEachCsrNodeStart_d, sizeof(uint) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pGainEachCsrFvalue_d, sizeof(real) * totalNumCsrFvalue));
	checkCudaErrors(hipMalloc((void**)&pMaxGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pMaxGainKey_d, sizeof(uint) * numofSNode));
	checkCudaErrors(hipMemcpy(pEachCsrNodeSize_d, eachNodeSizeInCsr, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pEachCsrNodeStart_d, eachCsrNodeStartPos, sizeof(uint) * numofSNode, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pGainEachCsrFvalue_d, pGainOnEachFvalue_h, sizeof(real) * totalNumCsrFvalue, hipMemcpyHostToDevice));
	real *pfLocalBestGain_d;
	uint *pnLocalBestGainKey_d;
	//compute # of blocks for each node
	uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pEachCsrNodeSize_d, pEachCsrNodeSize_d + numofSNode);
	checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
	PROCESS_ERROR(maxNumFeaValueOneNode > 0);
	int blockSizeLocalBestGain;
	dim3 dimNumofBlockLocalBestGain;
	conf.ConfKernel(maxNumFeaValueOneNode, blockSizeLocalBestGain, dimNumofBlockLocalBestGain);
	PROCESS_ERROR(dimNumofBlockLocalBestGain.z == 1);
	dimNumofBlockLocalBestGain.z = numofSNode;	//each node per super block
	int numBlockPerNode = dimNumofBlockLocalBestGain.x * dimNumofBlockLocalBestGain.y;

	checkCudaErrors(hipMalloc((void**)&pfLocalBestGain_d, sizeof(real) * numBlockPerNode * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pnLocalBestGainKey_d, sizeof(uint) * numBlockPerNode * numofSNode));
	//find the block level best gain for each node
	PickLocalBestSplitEachNode<<<dimNumofBlockLocalBestGain, blockSizeLocalBestGain, 0, (*(hipStream_t*)pStream)>>>(
									pEachCsrNodeSize_d,
									pEachCsrNodeStart_d,
									pGainEachCsrFvalue_d,
									pfLocalBestGain_d,
									pnLocalBestGainKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after PickLocalBestSplitEachNode");

	//find the global best gain for each node
	if(numBlockPerNode > 1){
		int blockSizeBestGain;
		dim3 dimNumofBlockDummy;
		conf.ConfKernel(numBlockPerNode, blockSizeBestGain, dimNumofBlockDummy);
		if(blockSizeBestGain < 64)//make sure the reduction is power of two
			blockSizeBestGain = 64;
		PickGlobalBestSplitEachNode<<<numofSNode, blockSizeBestGain, 0, (*(hipStream_t*)pStream)>>>(
										pfLocalBestGain_d,
										pnLocalBestGainKey_d,
										pMaxGain_d,
										pMaxGainKey_d,
									    numBlockPerNode, numofSNode);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		GETERROR("after PickGlobalBestSplitEachNode");
	}
	else{//local best fea is the global best fea
		manager.MemcpyDeviceToDeviceAsync(pfLocalBestGain_d, pMaxGain_d, sizeof(real) * numofSNode, pStream);
		manager.MemcpyDeviceToDeviceAsync(pnLocalBestGainKey_d, pMaxGainKey_d, sizeof(uint) * numofSNode, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pEachCsrNodeSize_d));
	checkCudaErrors(hipFree(pEachCsrNodeStart_d));
	checkCudaErrors(hipFree(pGainEachCsrFvalue_d));

	//find the split value and feature
	uint numofSeg = numofSNode * bagManager.m_numFea;
	uint *pEachCompressedFeaStartPos_d;
	uint *pEachCompressedFeaLen_d;
	real *pCsrFvalue_d;
	int *pPartId2SNPos_d;
	double *pCsrGD_d;
	real *pCsrHess_d;
	bool *pCsrDefault2Right_d;
	uint *pnCsrKey_d;

	checkCudaErrors(hipMalloc((void**)&pEachCompressedFeaStartPos_d, sizeof(uint) * numofSeg));
	checkCudaErrors(hipMalloc((void**)&pEachCompressedFeaLen_d, sizeof(uint) * numofSeg));
	checkCudaErrors(hipMalloc((void**)&pCsrFvalue_d, sizeof(real) * totalNumCsrFvalue));
	checkCudaErrors(hipMalloc((void**)&pPartId2SNPos_d, sizeof(int) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pCsrGD_d, sizeof(double) * totalNumCsrFvalue));
	checkCudaErrors(hipMalloc((void**)&pCsrHess_d, sizeof(real) * totalNumCsrFvalue));
	checkCudaErrors(hipMalloc((void**)&pCsrDefault2Right_d, sizeof(bool) * totalNumCsrFvalue));
	checkCudaErrors(hipMalloc((void**)&pnCsrKey_d, sizeof(uint) * totalNumCsrFvalue));

	checkCudaErrors(hipMemcpy(pEachCompressedFeaStartPos_d, eachCompressedFeaStartPos, sizeof(uint) * numofSeg, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pEachCompressedFeaLen_d, eachCompressedFeaLen, sizeof(uint) * numofSeg, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrFvalue_d, csrFvalue, sizeof(real) * totalNumCsrFvalue, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pPartId2SNPos_d, pid2snPos, sizeof(int) * numofSNode, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrGD_d, csrGD_h, sizeof(double) * totalNumCsrFvalue, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrHess_d, csrHess_h, sizeof(real) * totalNumCsrFvalue, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrDefault2Right_d, pDefault2Right_h, sizeof(bool) * totalNumCsrFvalue, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pnCsrKey_d, pnKey_h, sizeof(uint) * totalNumCsrFvalue, hipMemcpyHostToDevice));
	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
										 pEachCompressedFeaStartPos_d,
										 pEachCompressedFeaLen_d,
										 pCsrFvalue_d,
										 pMaxGain_d, pMaxGainKey_d,
										 pPartId2SNPos_d, nNumofFeature,
					  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 pCsrGD_d,
					  	  	  	  	  	 pCsrHess_d,
					  	  	  	  	  	 pCsrDefault2Right_d, pnCsrKey_d,
					  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
					  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pMaxGain_d));
	checkCudaErrors(hipFree(pMaxGainKey_d));
	checkCudaErrors(hipFree(pEachCompressedFeaStartPos_d));
	checkCudaErrors(hipFree(pEachCompressedFeaLen_d));
	checkCudaErrors(hipFree(pCsrFvalue_d));
	checkCudaErrors(hipFree(pPartId2SNPos_d));
	checkCudaErrors(hipFree(pCsrGD_d));
	checkCudaErrors(hipFree(pCsrHess_d));
	checkCudaErrors(hipFree(pCsrDefault2Right_d));
	checkCudaErrors(hipFree(pnCsrKey_d));
/*
//	cout << "prefix sum" << endl;
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	int totalNumArray = bagManager.m_numFea * numofSNode;
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled

	//construct keys for exclusive scan
	uint *pnKey_d;
	checkCudaErrors(hipMalloc((void**)&pnKey_d, bagManager.m_numFeaValue * sizeof(uint)));
	uint *pTempEachFeaStartEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;

	//set keys by GPU
	int maxSegLen = 0;
	int *pTempEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	int *pMaxLen = thrust::max_element(thrust::device, pTempEachFeaLenEachNode, pTempEachFeaLenEachNode + totalNumArray);
	checkCudaErrors(hipMemcpyAsync(&maxSegLen, pMaxLen, sizeof(int), hipMemcpyDeviceToHost, (*(hipStream_t*)pStream)));

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = totalNumArray;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<totalNumArray, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	double *pTempGDSum = bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	real *pTempHessSum = bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempGDSum, pTempGDSum);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempHessSum, pTempHessSum);


	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//default to left or right
	bool *pDefault2Right;
	checkCudaErrors(hipMalloc((void**)&pDefault2Right, sizeof(bool) * bagManager.m_numFeaValue));
	checkCudaErrors(hipMemset(pDefault2Right, 0, sizeof(bool) * bagManager.m_numFeaValue));

	//cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(numofDenseValue, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
											numofDenseValue, pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d, bagManager.m_numFea,
											bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
											pDefault2Right);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");

	//change the gain of the first feature value to 0
	int numFeaStartPos = bagManager.m_numFea * numofSNode;
//	printf("num fea start pos=%d (%d * %d)\n", numFeaStartPos, bagManager.m_numFea, numofSNode);
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numFeaStartPos, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
																bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
																numFeaStartPos, bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
																bagManager.m_numFeaValue);
*/
}

