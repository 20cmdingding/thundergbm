#include "hip/hip_runtime.h"
/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include "IndexComputer.h"
#include "FindFeaKernel.h"
#include "../Hashing.h"
#include "../Bagging/BagManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"
#include "../../SharedUtility/KernelConf.h"
#include "../../SharedUtility/HostUtility.h"

using std::cout;
using std::endl;
using std::make_pair;
using std::cerr;

__global__ void SetKey(uint *pSegStart, int *pSegLen, uint *pnKey){
	uint segmentId = blockIdx.x;//use one x covering multiple ys, because the maximum number of x-dimension is larger.
	__shared__ uint segmentLen, segmentStartPos;
	if(threadIdx.x == 0){//the first thread loads the segment length
		segmentLen = pSegLen[segmentId];
		segmentStartPos = pSegStart[segmentId];
	}
	__syncthreads();

	uint tid0 = blockIdx.y * blockDim.x;
	uint segmentThreadId = tid0 + threadIdx.x;
	if(tid0 >= segmentLen || segmentThreadId >= segmentLen)
		return;

	uint pos = segmentThreadId;
	while(pos < segmentLen){
		pnKey[pos + segmentStartPos] = segmentId;
		pos += blockDim.x;
	}
}

/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(vector<SplitPoint> &vBest, vector<nodeStat> &rchildStat, vector<nodeStat> &lchildStat, void *pStream, int bagId)
{
	GBDTGPUMemManager manager;
	BagManager bagManager;
	int numofSNode = bagManager.m_curNumofSplitableEachBag_h[bagId];
	int maxNumofSplittable = bagManager.m_maxNumSplittable;
//	cout << bagManager.m_maxNumSplittable << endl;
	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	//reset memory for this bag
	{
		manager.MemsetAsync(bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);

		manager.MemsetAsync(bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(double) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
		manager.MemsetAsync(bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
							0, sizeof(real) * bagManager.m_numFeaValue, pStream);
	}
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute index for each feature value
	KernelConf conf;
	int blockSizeLoadGD;
	dim3 dimNumofBlockToLoadGD;
	conf.ConfKernel(bagManager.m_numFeaValue, blockSizeLoadGD, dimNumofBlockToLoadGD);
	//# of feature values that need to compute gains; the code below cannot be replaced by indexComp.m_totalNumFeaValue, due to some nodes becoming leaves.
	int numofDenseValue = -1, maxNumFeaValueOneNode = -1;
	if(numofSNode > 1)
	{
		IndexComputer indexComp;
		indexComp.AllocMem(bagManager.m_numFea, numofSNode);
		PROCESS_ERROR(nNumofFeature == bagManager.m_numFea);
		clock_t comIdx_start = clock();
		//compute gather index via GPUs
		indexComp.ComputeIdxGPU(numofSNode, maxNumofSplittable, bagId);
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);

		//copy # of feature values of each node
		uint *pTempNumFvalueEachNode = bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable;
	
		clock_t start_gd = clock();
		//scatter operation
		//total fvalue to load may be smaller than m_totalFeaValue, due to some nodes becoming leaves.
		numofDenseValue = thrust::reduce(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		LoadGDHessFvalue<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, 
															   bagManager.m_numIns, manager.m_pDInsId, manager.m_pdDFeaValue,
															   bagManager.m_pIndicesEachBag_d, numofDenseValue,
															   bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);
		uint *pMaxNumFvalueOneNode = thrust::max_element(thrust::device, pTempNumFvalueEachNode, pTempNumFvalueEachNode + numofSNode);
		checkCudaErrors(hipMemcpy(&maxNumFeaValueOneNode, pMaxNumFvalueOneNode, sizeof(int), hipMemcpyDeviceToHost));
		indexComp.FreeMem();
	}
	else
	{
		clock_t start_gd = clock();
		LoadGDHessFvalueRoot<<<dimNumofBlockToLoadGD, blockSizeLoadGD, 0, (*(hipStream_t*)pStream)>>>(bagManager.m_pInsGradEachBag + bagId * bagManager.m_numIns,
															   	   	bagManager.m_pInsHessEachBag + bagId * bagManager.m_numIns, bagManager.m_numIns,
															   	   	manager.m_pDInsId, manager.m_pdDFeaValue, bagManager.m_numFeaValue,
															   		bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
															   	   	bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		clock_t end_gd = clock();
		total_fill_gd_t += (end_gd - start_gd);

		clock_t comIdx_start = clock();
		//copy # of feature values of a node
		manager.MemcpyHostToDeviceAsync(&manager.m_numFeaValue, bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
										sizeof(uint), pStream);
		//copy feature value start position of each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
									 	 sizeof(uint), pStream);
		//copy each feature start position in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pFeaStartPos, bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
										sizeof(uint) * nNumofFeature, pStream);
		//copy # of feature values of each feature in each node
		manager.MemcpyDeviceToDeviceAsync(manager.m_pDNumofKeyValue, bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									    sizeof(int) * nNumofFeature, pStream);

		numofDenseValue = manager.m_numFeaValue;//for computing gain of each fvalue
		maxNumFeaValueOneNode = manager.m_numFeaValue;
		clock_t comIdx_end = clock();
		total_com_idx_t += (comIdx_end - comIdx_start);
	}

//	cout << "prefix sum" << endl;
	clock_t start_scan = clock();
	//compute the feature with the maximum number of values
	int totalNumArray = bagManager.m_numFea * numofSNode;
	hipStreamSynchronize((*(hipStream_t*)pStream));//wait until the pinned memory (m_pEachFeaLenEachNodeEachBag_dh) is filled

	//construct keys for exclusive scan
	uint *pnKey_d;
	checkCudaErrors(hipMalloc((void**)&pnKey_d, bagManager.m_numFeaValue * sizeof(uint)));
	uint *pTempEachFeaStartEachNode = bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;

	//set keys by GPU
	int maxSegLen = 0;
	int *pTempEachFeaLenEachNode = bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea;
	int *pMaxLen = thrust::max_element(thrust::device, pTempEachFeaLenEachNode, pTempEachFeaLenEachNode + totalNumArray);
	checkCudaErrors(hipMemcpyAsync(&maxSegLen, pMaxLen, sizeof(int), hipMemcpyDeviceToHost, (*(hipStream_t*)pStream)));

	dim3 dimNumofBlockToSetKey;
	dimNumofBlockToSetKey.x = totalNumArray;
	uint blockSize = 128;
	dimNumofBlockToSetKey.y = (maxSegLen + blockSize - 1) / blockSize;
	SetKey<<<totalNumArray, blockSize, sizeof(uint) * 2, (*(hipStream_t*)pStream)>>>
			(pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));

	//compute prefix sum for gd and hess (more than one arrays)
	double *pTempGDSum = bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	real *pTempHessSum = bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue;
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempGDSum, pTempGDSum);//in place prefix sum
	thrust::inclusive_scan_by_key(thrust::system::cuda::par, pnKey_d, pnKey_d + bagManager.m_numFeaValue, pTempHessSum, pTempHessSum);


	clock_t end_scan = clock();
	total_scan_t += (end_scan - start_scan);

	//default to left or right
	bool *pDefault2Right;
	checkCudaErrors(hipMalloc((void**)&pDefault2Right, sizeof(bool) * bagManager.m_numFeaValue));
	checkCudaErrors(hipMemset(pDefault2Right, 0, sizeof(bool) * bagManager.m_numFeaValue));

	//cout << "compute gain" << endl;
	clock_t start_comp_gain = clock();
	int blockSizeComGain;
	dim3 dimNumofBlockToComGain;
	conf.ConfKernel(numofDenseValue, blockSizeComGain, dimNumofBlockToComGain);
	ComputeGainDense<<<dimNumofBlockToComGain, blockSizeComGain, 0, (*(hipStream_t*)pStream)>>>(
											bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
											bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable,
											DeviceSplitter::m_lambda, bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
											bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
											numofDenseValue, pTempEachFeaStartEachNode, pTempEachFeaLenEachNode, pnKey_d, bagManager.m_numFea,
											bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
											pDefault2Right);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after ComputeGainDense");
	
	//change the gain of the first feature value to 0
	int numFeaStartPos = bagManager.m_numFea * numofSNode;
//	printf("num fea start pos=%d (%d * %d)\n", numFeaStartPos, bagManager.m_numFea, numofSNode);
	int blockSizeFirstGain;
	dim3 dimNumofBlockFirstGain;
	conf.ConfKernel(numFeaStartPos, blockSizeFirstGain, dimNumofBlockFirstGain);
	FirstFeaGain<<<dimNumofBlockFirstGain, blockSizeFirstGain, 0, (*(hipStream_t*)pStream)>>>(
																bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
																numFeaStartPos, bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
																bagManager.m_numFeaValue);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after FirstFeaGain");

	clock_t end_comp_gain = clock();
	total_com_gain_t += (end_comp_gain - start_comp_gain);

//	cout << "searching" << endl;
	clock_t start_search = clock();
	real *pfLocalBestGain_d, *pfGlobalBestGain_d;
	int *pnLocalBestGainKey_d, *pnGlobalBestGainKey_d;
	//compute # of blocks for each node
	PROCESS_ERROR(maxNumFeaValueOneNode > 0);
	int blockSizeLocalBestGain;
	dim3 dimNumofBlockLocalBestGain;
	conf.ConfKernel(maxNumFeaValueOneNode, blockSizeLocalBestGain, dimNumofBlockLocalBestGain);
	PROCESS_ERROR(dimNumofBlockLocalBestGain.z == 1);
	dimNumofBlockLocalBestGain.z = numofSNode;//each node per super block
	int numBlockPerNode = dimNumofBlockLocalBestGain.x * dimNumofBlockLocalBestGain.y;

	checkCudaErrors(hipMalloc((void**)&pfLocalBestGain_d, sizeof(real) * numBlockPerNode * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pnLocalBestGainKey_d, sizeof(int) * numBlockPerNode * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pfGlobalBestGain_d, sizeof(real) * numofSNode));
	checkCudaErrors(hipMalloc((void**)&pnGlobalBestGainKey_d, sizeof(int) * numofSNode));
	//find the block level best gain for each node
	PickLocalBestSplitEachNode<<<dimNumofBlockLocalBestGain, blockSizeLocalBestGain, 0, (*(hipStream_t*)pStream)>>>(
								bagManager.m_pNumFvalueEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pFvalueStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable,
								bagManager.m_pGainEachFvalueEachBag + bagId * bagManager.m_numFeaValue,
								pfLocalBestGain_d,
								pnLocalBestGainKey_d);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	GETERROR("after PickLocalBestSplitEachNode");

	//find the global best gain for each node
	if(numBlockPerNode > 1){
		int blockSizeBestGain;
		dim3 dimNumofBlockDummy;
		conf.ConfKernel(numBlockPerNode, blockSizeBestGain, dimNumofBlockDummy);
		PickGlobalBestSplitEachNode<<<numofSNode, blockSizeBestGain, 0, (*(hipStream_t*)pStream)>>>(
									pfLocalBestGain_d,
									pnLocalBestGainKey_d,
									pfGlobalBestGain_d,
									pnGlobalBestGainKey_d,
								    numBlockPerNode, numofSNode);
		hipStreamSynchronize((*(hipStream_t*)pStream));
		GETERROR("after PickGlobalBestSplitEachNode");
	}
	else{//local best fea is the global best fea
		manager.MemcpyDeviceToDeviceAsync(pfLocalBestGain_d, pfGlobalBestGain_d,
										sizeof(real) * numofSNode, pStream);
		manager.MemcpyDeviceToDeviceAsync(pnLocalBestGainKey_d, pnGlobalBestGainKey_d,
											sizeof(int) * numofSNode, pStream);
	}

	hipStreamSynchronize((*(hipStream_t*)pStream));
	clock_t end_search = clock();
	total_search_t += end_search - start_search;

	FindSplitInfo<<<1, numofSNode, 0, (*(hipStream_t*)pStream)>>>(
									 bagManager.m_pEachFeaStartPosEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pEachFeaLenEachNodeEachBag_d + bagId * bagManager.m_maxNumSplittable * bagManager.m_numFea,
									 bagManager.m_pDenseFValueEachBag + bagId * bagManager.m_numFeaValue,
									 pfGlobalBestGain_d, pnGlobalBestGainKey_d,
				  	  	  	  	  	 bagManager.m_pPartitionId2SNPosEachBag + bagId * bagManager.m_maxNumSplittable, nNumofFeature,
				  	  	  	  	  	 bagManager.m_pSNodeStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pdGDPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 bagManager.m_pHessPrefixSumEachBag + bagId * bagManager.m_numFeaValue,
				  	  	  	  	  	 pDefault2Right, pnKey_d,
				  	  	  	  	  	 bagManager.m_pBestSplitPointEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pRChildStatEachBag + bagId * bagManager.m_maxNumSplittable,
				  	  	  	  	  	 bagManager.m_pLChildStatEachBag + bagId * bagManager.m_maxNumSplittable);
	hipStreamSynchronize((*(hipStream_t*)pStream));
	checkCudaErrors(hipFree(pnKey_d));
	checkCudaErrors(hipFree(pDefault2Right));
	checkCudaErrors(hipFree(pfLocalBestGain_d));
	checkCudaErrors(hipFree(pfGlobalBestGain_d));
	checkCudaErrors(hipFree(pnLocalBestGainKey_d));
	checkCudaErrors(hipFree(pnGlobalBestGainKey_d));
}
