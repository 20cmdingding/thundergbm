#include "hip/hip_runtime.h"
/*
 * CsrCompressor.cu
 *
 *  Created on: Jul 25, 2017
 *      Author: zeyi
 */
#include <vector>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "CsrCompressor.h"
#include "CsrSplit.h"
#include "BagCsrManager.h"
#include "../Bagging/BagManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"

using std::vector;

uint *CsrCompressor::eachFeaLenEachNode_h = NULL;
uint *CsrCompressor::eachFeaStartPosEachNode_h = NULL;
uint CsrCompressor::eachNodeSizeInCsr_h = 0;
int *CsrCompressor::insId_h = NULL;
uint CsrCompressor::totalOrgNumCsr = 0;

uint *CsrCompressor::pCsrFeaStartPos_d = NULL;
uint *CsrCompressor::pCsrFeaLen_d = NULL;
uint *CsrCompressor::pCsrLen_d = NULL;
real *CsrCompressor::pCsrFvalue_d = NULL;
uint *CsrCompressor::pCsrStart_d = NULL;
bool CsrCompressor::bUseCsr = false;
real *CsrCompressor::pOrgFvalue = NULL;

CsrCompressor::CsrCompressor(){
	if(pCsrFeaStartPos_d != NULL || bUseCsr == false)
		return;
	GBDTGPUMemManager manager;
	uint numFea = manager.m_numofFea;
	uint numFeaValue = manager.m_numFeaValue;

	uint *eachCsrFeaStartPos_h = new uint[numFea];
	uint *eachCompressedFeaLen_h = new uint[numFea];
	uint *eachCsrLen_h = new uint[numFeaValue];
	real *csrFvalue_h = new real[numFeaValue];

	eachNodeSizeInCsr_h = 0;
	uint csrId = 0, curFvalueToCompress = 0;
	for(int i = 0; i < numFea; i++){
		eachCompressedFeaLen_h[i] = 0;
		uint feaLen = eachFeaLenEachNode_h[i];
		uint feaStart = eachFeaStartPosEachNode_h[i];
		if(feaLen == 0)continue;
		csrFvalue_h[csrId] = pOrgFvalue[feaStart];
		eachCsrLen_h[csrId] = 1;
		eachCompressedFeaLen_h[i] = 1;
		for(int l = 1; l < feaLen; l++){
			curFvalueToCompress++;
			if(fabs(pOrgFvalue[feaStart + l] - csrFvalue_h[csrId]) > DeviceSplitter::rt_eps){
				eachCompressedFeaLen_h[i]++;
				csrId++;
				csrFvalue_h[csrId] = pOrgFvalue[feaStart + l];
				eachCsrLen_h[csrId] = 1;
			}
			else
				eachCsrLen_h[csrId]++;
		}
		csrId++;
		curFvalueToCompress++;
	}
	for(int i = 0; i < numFea; i++){
		uint prefix = 0;
		for(int l = 0; l < i; l++)
			prefix += eachCompressedFeaLen_h[l];
		eachCsrFeaStartPos_h[i] = prefix;
	}

	totalOrgNumCsr = csrId;
	eachNodeSizeInCsr_h = totalOrgNumCsr;
	printf("org=%u v.s. csr=%u\n", manager.m_numFeaValue, totalOrgNumCsr);
	PROCESS_ERROR(totalOrgNumCsr < manager.m_numFeaValue);

	checkCudaErrors(hipMalloc((void**)&pCsrFeaStartPos_d, sizeof(uint) * numFea));
	checkCudaErrors(hipMalloc((void**)&pCsrFeaLen_d, sizeof(uint) * numFea));
	checkCudaErrors(hipMalloc((void**)&pCsrLen_d, sizeof(uint) * totalOrgNumCsr));
	checkCudaErrors(hipMalloc((void**)&pCsrFvalue_d, sizeof(real) * totalOrgNumCsr));
	checkCudaErrors(hipMalloc((void**)&pCsrStart_d, sizeof(uint) * totalOrgNumCsr));
	checkCudaErrors(hipMemcpy(pCsrFeaStartPos_d, eachCsrFeaStartPos_h, sizeof(uint) * numFea, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrFeaLen_d, eachCompressedFeaLen_h, sizeof(uint) * numFea, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrFvalue_d, csrFvalue_h, sizeof(real) * totalOrgNumCsr, hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(pCsrLen_d, eachCsrLen_h, sizeof(uint) * totalOrgNumCsr, hipMemcpyDefault));
	thrust::exclusive_scan(thrust::device, pCsrLen_d, pCsrLen_d + totalOrgNumCsr, pCsrStart_d);

	delete[] eachCsrFeaStartPos_h;
	delete[] eachCompressedFeaLen_h;
	delete[] eachCsrLen_h;
	delete[] csrFvalue_h;
}

void CsrCompressor::CsrCompression(uint &totalNumCsrFvalue, uint *eachCompressedFeaStartPos_d, uint *eachCompressedFeaLen_d,
								   uint *eachNodeSizeInCsr_d, uint *eachCsrNodeStartPos_d){
	BagManager bagManager;
	GBDTGPUMemManager manager;
	BagCsrManager csrManager(manager.m_numofFea, bagManager.m_maxNumSplittable, manager.m_numFeaValue);
	totalNumCsrFvalue = totalOrgNumCsr;
	//compute csr gd and hess
	checkCudaErrors(hipMemcpy(eachCompressedFeaStartPos_d, pCsrFeaStartPos_d, sizeof(uint) * bagManager.m_numFea, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(eachCompressedFeaLen_d, pCsrFeaLen_d, sizeof(uint) * bagManager.m_numFea, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(csrManager.getMutableCsrFvalue(), pCsrFvalue_d, sizeof(real) * totalOrgNumCsr, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(csrManager.getMutableCsrLen(), pCsrLen_d, sizeof(uint) * totalOrgNumCsr, hipMemcpyDeviceToDevice));

	checkCudaErrors(hipMemset(eachCsrNodeStartPos_d, 0, sizeof(uint)));
	checkCudaErrors(hipMemcpy(eachNodeSizeInCsr_d, &eachNodeSizeInCsr_h, sizeof(uint), hipMemcpyHostToDevice));

	//need to compute for every new tree
	clock_t start = clock();
	dim3 dimNumofBlockForGD;
	dimNumofBlockForGD.x = totalOrgNumCsr;
	uint blockSize = 64;
	uint sharedMemSize = blockSize * (sizeof(double) + sizeof(real));
	ComputeGD<<<dimNumofBlockForGD, blockSize, sharedMemSize>>>(pCsrLen_d, pCsrStart_d, bagManager.m_pInsGradEachBag, bagManager.m_pInsHessEachBag,
			manager.m_pDInsId, csrManager.getMutableCsrGD(), csrManager.getMutableCsrHess());
	hipDeviceSynchronize();
	clock_t end = clock();
	printf("compute gd & hess time: %f\n", double(end - start) / CLOCKS_PER_SEC);
}
