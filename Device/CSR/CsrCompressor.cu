#include "hip/hip_runtime.h"
/*
 * CsrCompressor.cu
 *
 *  Created on: Jul 25, 2017
 *      Author: zeyi
 */
#include <vector>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "CsrCompressor.h"
#include "CsrSplit.h"
#include "BagCsrManager.h"
#include "../Bagging/BagManager.h"
#include "../Splitter/DeviceSplitter.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../../SharedUtility/CudaMacro.h"

using std::vector;

uint *CsrCompressor::eachFeaLenEachNode_h = NULL;
uint *CsrCompressor::eachFeaStartPosEachNode_h = NULL;
uint CsrCompressor::eachNodeSizeInCsr_h = 0;
int *CsrCompressor::insId_h = NULL;
uint CsrCompressor::totalOrgNumCsr = 0;

uint *CsrCompressor::pCsrFeaStartPos_d = NULL;
uint *CsrCompressor::pCsrFeaLen_d = NULL;
uint *CsrCompressor::pCsrLen_d = NULL;
real *CsrCompressor::pCsrFvalue_d = NULL;
uint *CsrCompressor::pCsrStart_d = NULL;
bool CsrCompressor::bUseCsr = false;
real *CsrCompressor::pOrgFvalue = NULL;

CsrCompressor::CsrCompressor(){
	if(pCsrFeaStartPos_d != NULL || bUseCsr == false)
		return;
	GBDTGPUMemManager manager;
	uint numFea = manager.m_numofFea;
	uint numFeaValue = manager.m_numFeaValue;

	uint *eachCsrFeaStartPos_h = new uint[numFea];
	uint *eachCompressedFeaLen_h = new uint[numFea];
	uint *eachCsrLen_h = new uint[numFeaValue];
	real *csrFvalue_h = new real[numFeaValue];

	eachNodeSizeInCsr_h = 0;
	uint csrId = 0, curFvalueToCompress = 0;
	for(int i = 0; i < numFea; i++){
		eachCompressedFeaLen_h[i] = 0;
		uint feaLen = eachFeaLenEachNode_h[i];
		uint feaStart = eachFeaStartPosEachNode_h[i];
		if(feaLen == 0)continue;
		csrFvalue_h[csrId] = pOrgFvalue[feaStart];
		eachCsrLen_h[csrId] = 1;
		eachCompressedFeaLen_h[i] = 1;
		for(int l = 1; l < feaLen; l++){
			curFvalueToCompress++;
			if(fabs(pOrgFvalue[feaStart + l] - csrFvalue_h[csrId]) > DeviceSplitter::rt_eps){
				eachCompressedFeaLen_h[i]++;
				csrId++;
				csrFvalue_h[csrId] = pOrgFvalue[feaStart + l];
				eachCsrLen_h[csrId] = 1;
			}
			else
				eachCsrLen_h[csrId]++;
		}
		csrId++;
		curFvalueToCompress++;
	}
	for(int i = 0; i < numFea; i++){
		uint prefix = 0;
		for(int l = 0; l < i; l++)
			prefix += eachCompressedFeaLen_h[l];
		eachCsrFeaStartPos_h[i] = prefix;
	}

	totalOrgNumCsr = csrId;
	eachNodeSizeInCsr_h = totalOrgNumCsr;
	printf("org=%u v.s. csr=%u\n", manager.m_numFeaValue, totalOrgNumCsr);
//	PROCESS_ERROR(totalOrgNumCsr < manager.m_numFeaValue);
	if(totalOrgNumCsr * 4 > manager.m_numFeaValue){
		bUseCsr = false;

		delete[] eachCsrFeaStartPos_h;
		delete[] eachCompressedFeaLen_h;
		delete[] eachCsrLen_h;
		delete[] csrFvalue_h;
		return;
	}

	checkCudaErrors(hipMalloc((void**)&pCsrFeaStartPos_d, sizeof(uint) * numFea));
	checkCudaErrors(hipMalloc((void**)&pCsrFeaLen_d, sizeof(uint) * numFea));
	checkCudaErrors(hipMalloc((void**)&pCsrLen_d, sizeof(uint) * totalOrgNumCsr));
	checkCudaErrors(hipMalloc((void**)&pCsrFvalue_d, sizeof(real) * totalOrgNumCsr));
	checkCudaErrors(hipMalloc((void**)&pCsrStart_d, sizeof(uint) * totalOrgNumCsr));
	checkCudaErrors(hipMemcpy(pCsrFeaStartPos_d, eachCsrFeaStartPos_h, sizeof(uint) * numFea, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrFeaLen_d, eachCompressedFeaLen_h, sizeof(uint) * numFea, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pCsrLen_d, eachCsrLen_h, sizeof(uint) * totalOrgNumCsr, hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(pCsrFvalue_d, csrFvalue_h, sizeof(real) * totalOrgNumCsr, hipMemcpyDefault));
	thrust::exclusive_scan(thrust::device, pCsrLen_d, pCsrLen_d + totalOrgNumCsr, pCsrStart_d);

	delete[] eachCsrFeaStartPos_h;
	delete[] eachCompressedFeaLen_h;
	delete[] eachCsrLen_h;
	delete[] csrFvalue_h;
}

void CsrCompressor::CsrCompression(uint &totalNumCsrFvalue, uint *eachCompressedFeaStartPos_d, uint *eachCompressedFeaLen_d,
								   uint *eachNodeSizeInCsr_d, uint *eachCsrNodeStartPos_d, real *pCsrFvalue, uint *pCsrLen){
	BagManager bagManager;
	GBDTGPUMemManager manager;
	BagCsrManager csrManager(manager.m_numofFea, bagManager.m_maxNumSplittable, manager.m_numFeaValue);
	totalNumCsrFvalue = totalOrgNumCsr;
	//compute csr gd and hess
	checkCudaErrors(hipMemcpy(eachCompressedFeaStartPos_d, pCsrFeaStartPos_d, sizeof(uint) * bagManager.m_numFea, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(eachCompressedFeaLen_d, pCsrFeaLen_d, sizeof(uint) * bagManager.m_numFea, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(pCsrFvalue, pCsrFvalue_d, sizeof(real) * totalOrgNumCsr, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(pCsrLen, pCsrLen_d, sizeof(uint) * totalOrgNumCsr, hipMemcpyDeviceToDevice));

	checkCudaErrors(hipMemset(eachCsrNodeStartPos_d, 0, sizeof(uint)));
	checkCudaErrors(hipMemcpy(eachNodeSizeInCsr_d, &eachNodeSizeInCsr_h, sizeof(uint), hipMemcpyHostToDevice));
}
