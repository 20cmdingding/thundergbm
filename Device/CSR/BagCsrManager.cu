#include "hip/hip_runtime.h"
/*
 * BagCsrManager.cu
 *
 *  Created on: Jul 23, 2017
 *      Author: zeyi
 */

#include <hip/hip_runtime_api.h>
#include "BagCsrManager.h"
#include "../../SharedUtility/CudaMacro.h"

uint *BagCsrManager::pEachCsrFeaStartPos = NULL;
uint *BagCsrManager::pEachCsrFeaLen = NULL;
uint *BagCsrManager::pEachCsrNodeStartPos = NULL;
uint *BagCsrManager::pEachNodeSizeInCsr = NULL;
int *BagCsrManager::preFvalueInsId = NULL;
uint BagCsrManager::curNumCsr = 0;
uint BagCsrManager::reservedMaxNumCsr = pow(2, 20);
MemVector BagCsrManager::csrLen;//shared with pCsrStart
MemVector BagCsrManager::csrMarker; //shared with old length
MemVector BagCsrManager::csrKey; //shared with pCsrStartCurRound
real *BagCsrManager::pCsrFvalue = NULL;
MemVector BagCsrManager::csrDefault2Right; //shared with csrId2Pid

BagCsrManager::BagCsrManager(int numFea, int maxNumSN, uint totalNumFeaValue){
	if(pCsrFvalue != NULL)//already reserved memory
		return;

	curNumCsr = 0;
	reservedMaxNumCsr = totalNumFeaValue/20;//10 times compression ratio

	checkCudaErrors(hipMalloc((void**)&pCsrFvalue, sizeof(real) * reservedMaxNumCsr));
	checkCudaErrors(hipMalloc((void**)&pEachCsrFeaStartPos, sizeof(uint) * numFea * maxNumSN));
	checkCudaErrors(hipMalloc((void**)&pEachCsrFeaLen, sizeof(uint) * numFea * maxNumSN));
	checkCudaErrors(hipMalloc((void**)&pEachCsrNodeStartPos, sizeof(uint) * maxNumSN));
	checkCudaErrors(hipMalloc((void**)&pEachNodeSizeInCsr, sizeof(uint) * maxNumSN));

	checkCudaErrors(hipMalloc((void**)&preFvalueInsId, sizeof(int) * totalNumFeaValue));
}

void BagCsrManager::reserveCsrSpace(){
	checkCudaErrors(hipFree(pCsrFvalue));
	//reserve larger memory
	printf("max num of csr is %u\n", reservedMaxNumCsr);
	checkCudaErrors(hipMalloc((void**) &pCsrFvalue, sizeof(real) * reservedMaxNumCsr));
}

/** operations on cross variable reused memory **/
uint *BagCsrManager::getMutableCsrLen(){
	PROCESS_ERROR(curNumCsr > 0);
	if(csrLen.reservedSize < curNumCsr)
		csrLen.reserveSpace(curNumCsr, sizeof(uint));
	PROCESS_ERROR(csrLen.addr != NULL);
	return (uint*)csrLen.addr;
}

uint *BagCsrManager::getMutableCsrKey(){
	PROCESS_ERROR(curNumCsr > 0);
	if(csrKey.reservedSize < curNumCsr)
		csrKey.reserveSpace(curNumCsr, sizeof(uint));
	PROCESS_ERROR(csrKey.addr != NULL);
	return (uint*)csrKey.addr;
}
bool *BagCsrManager::getMutableDefault2Right(){
	PROCESS_ERROR(curNumCsr > 0);
	if(csrDefault2Right.reservedSize < curNumCsr)
		csrDefault2Right.reserveSpace(curNumCsr, sizeof(bool));
	PROCESS_ERROR(csrDefault2Right.addr != NULL);
	return (bool*)csrDefault2Right.addr;
}

uint *BagCsrManager::getMutableCsrStartCurRound(){
	return getMutableCsrKey();
}
unsigned char *BagCsrManager::getMutableCsrId2Pid(){
	return (unsigned char*)getMutableDefault2Right();
}
uint *BagCsrManager::getMutableCsrMarker(){
	PROCESS_ERROR(curNumCsr > 0);
	if(csrMarker.reservedSize < curNumCsr * 2)
		csrMarker.reserveSpace(curNumCsr * 2, sizeof(uint));
	PROCESS_ERROR(csrMarker.addr != NULL);
	return (uint*)csrMarker.addr;
}

uint *BagCsrManager::getMutableCsrStart(){
	return getMutableCsrLen();
}

uint *BagCsrManager::getMutableCsrOldLen(){
	return (uint*)getMutableCsrMarker();
}

const uint *BagCsrManager::getCsrLen(){
	PROCESS_ERROR(csrLen.addr != NULL);
	return (uint*)csrLen.addr;
}

const uint *BagCsrManager::getCsrKey(){
	PROCESS_ERROR(csrKey.addr != NULL);
	return (uint*)csrKey.addr;
}

const uint *BagCsrManager::getCsrStart(){
	return getCsrLen();
}

const uint *BagCsrManager::getCsrMarker(){
	PROCESS_ERROR(csrMarker.addr != NULL);
	return (uint*)csrMarker.addr;
}

const uint *BagCsrManager::getCsrStartCurRound(){
	return getCsrKey();//reuse this memory
}
const unsigned char *BagCsrManager::getCsrId2Pid(){
	return (unsigned char*)getDefault2Right();
}
const uint *BagCsrManager::getCsrOldLen(){
	return (uint*)getCsrMarker();
}

/* operations on not cross variable reused memory */
real *BagCsrManager::getMutableCsrFvalue(){
	PROCESS_ERROR(curNumCsr > 0);
	if(reservedMaxNumCsr < curNumCsr){
		reservedMaxNumCsr = curNumCsr * 2;
		reserveCsrSpace();
	}
	PROCESS_ERROR(pCsrFvalue != NULL);
	return pCsrFvalue;
}
const real *BagCsrManager::getCsrFvalue(){
	PROCESS_ERROR(pCsrFvalue != NULL);
	return pCsrFvalue;
}
const bool *BagCsrManager::getDefault2Right(){
	PROCESS_ERROR(csrDefault2Right.addr != NULL);
	return (bool*)csrDefault2Right.addr;
}

