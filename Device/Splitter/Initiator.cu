#include "hip/hip_runtime.h"
/*
 * ComputeGD.cu
 *
 *  Created on: 21 Jun 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <stdio.h>
#include "Initiator.h"
#include "../ErrorChecker.h"
#include "../DeviceHashing.h"

__global__ void SaveToPredBuffer(const float_point *pfCurTreePredValue, int numPredIns, float_point *pfPreTreePredValue)
{

	int gTid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(gTid >= numPredIns)
		return;
	pfPreTreePredValue[gTid] += pfCurTreePredValue[gTid];//accumulate the current prediction to the buffer
}

__global__ void ComputeGDKernel(int numofIns, const float_point *pfPredValue, const float_point *pfTrueValue, float_point *pGrad, float_point *pHess)
{
	for(int i = 0; i < numofIns; i++)
	{
		pGrad[i] = pfPredValue[i] - pfTrueValue[i];
//		if(pGrad[i] < -2016 || pGrad[i] > -1920)
//			printf("pGrad is too small: %f\n", pGrad[i]);

		pHess[i] = 1;
	}

}

__global__ void InitNodeStat(int numofIns, const float_point *pGrad, const float_point *pHess,
							 nodeStat *pSNodeStat, int *pSNIdToBuffId, int maxNumofSplittable,
							 int *pBuffId, int *pNumofBuffId)
{
	float_point root_sum_gd = 0.0, root_sum_hess = 0.0;
	for(int i = 0; i < numofIns; i++)
	{
		root_sum_gd += pGrad[i];
		root_sum_hess += pHess[i];
	}

	int nid = 0;//id of root node is always 0.
	bool bIsNew = false;
	int buffId = AssignHashValue(pSNIdToBuffId, nid, maxNumofSplittable, bIsNew);
	if(buffId != 0)
		printf("buffId = %d\n", buffId);
	pSNodeStat[buffId].sum_gd = root_sum_gd;
	pSNodeStat[buffId].sum_hess = root_sum_hess;
	pBuffId[0] = buffId;//here we only initialise the root node
	pNumofBuffId[0] = 1;
}

/**
 * @brief: initialise the root node of a tree, and the current node of nodes in the tree.
 */
__global__ void InitRootNode(TreeNode *pAllTreeNode, int *pCurNumofNode)
{
	pAllTreeNode[0].nodeId = 0;
	pAllTreeNode[0].level = 0;
	*pCurNumofNode = 1;
}
