/*
 * DeviceSplitterSplitNode.cu
 *
 *  Created on: 12 May 2016
 *      Author: Zeyi Wen
 *		@brief: GPU version of splitAll function
 */

#include <iostream>
#include <algorithm>

#include "../../DeviceHost/MyAssert.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../Memory/SNMemManager.h"
#include "DeviceSplitter.h"
#include "../Preparator.h"
#include "../Hashing.h"
#include "DeviceSplitAllKernel.h"
#include "../KernelConf.h"

using std::cout;
using std::endl;
using std::pair;
using std::make_pair;
using std::sort;


/**
 * @brief: split all splittable nodes of the current level
 * @numofNode: for computing new children ids
 */
void DeviceSplitter::SplitAll(vector<TreeNode*> &splittableNode, const vector<SplitPoint> &vBest, RegTree &tree, int &m_nNumofNode,
		 	 	 	    const vector<nodeStat> &rchildStat, const vector<nodeStat> &lchildStat, bool bLastLevel)
{

	int preMaxNodeId = m_nNumofNode - 1;
	PROCESS_ERROR(preMaxNodeId >= 0);

	GBDTGPUMemManager manager;
	SNGPUManager snManager;//splittable node memory manager

	//compute the base_weight of tree node, also determines if a node is a leaf.
	KernelConf conf;
	int threadPerBlock;
	dim3 dimNumofBlock;
	conf.ConfKernel(manager.m_curNumofSplitable, threadPerBlock, dimNumofBlock);
	ComputeWeight<<<dimNumofBlock, threadPerBlock>>>(snManager.m_pTreeNode, manager.m_pSplittableNode, manager.m_pSNIdToBuffId,
			  	  	  	  	  manager.m_pBestSplitPoint, manager.m_pSNodeStat, rt_eps, LEAFNODE,
			  	  	  	  	  m_lambda, manager.m_curNumofSplitable, bLastLevel, manager.m_maxNumofSplittable);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in ComputeWeight" << endl;
		exit(0);
	}
#endif

	//copy the number of nodes in the tree to the GPU memory
	manager.Memset(snManager.m_pNumofNewNode, 0, sizeof(int));
	CreateNewNode<<<dimNumofBlock, threadPerBlock>>>(
							snManager.m_pTreeNode, manager.m_pSplittableNode, snManager.m_pNewSplittableNode,
							manager.m_pSNIdToBuffId, manager.m_pBestSplitPoint,
							snManager.m_pParentId, snManager.m_pLeftChildId, snManager.m_pRightChildId,
							manager.m_pLChildStat, manager.m_pRChildStat, snManager.m_pNewNodeStat,
							snManager.m_pCurNumofNode_d, snManager.m_pNumofNewNode, rt_eps,
							manager.m_curNumofSplitable, bLastLevel, manager.m_maxNumofSplittable);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in CreateNewNode" << endl;
		exit(0);
	}
#endif
	//find all used unique feature ids. We will use these features to organise instances into new nodes.
	manager.Memset(snManager.m_pFeaIdToBuffId, -1, sizeof(int) * snManager.m_maxNumofUsedFea);
	manager.Memset(snManager.m_pUniqueFeaIdVec, -1, sizeof(int) * snManager.m_maxNumofUsedFea);
	manager.Memset(snManager.m_pNumofUniqueFeaId, 0, sizeof(int));
	GetUniqueFid<<<dimNumofBlock, threadPerBlock>>>(snManager.m_pTreeNode, manager.m_pSplittableNode, manager.m_curNumofSplitable,
							 snManager.m_pFeaIdToBuffId, snManager.m_pUniqueFeaIdVec, snManager.m_pNumofUniqueFeaId,
			 	 	 	 	 snManager.m_maxNumofUsedFea, LEAFNODE, manager.m_nSNLock);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in GetUniqueFid" << endl;
		exit(0);
	}

	if(hipGetLastError() != hipSuccess)
	{
		cout << "error before InsToNewNode" << endl;
		exit(0);
	}
#endif

	//for each used feature to move instances to new nodes
	int numofUniqueFea = -1;
	manager.MemcpyDeviceToHost(snManager.m_pNumofUniqueFeaId, &numofUniqueFea, sizeof(int));

	if(numofUniqueFea == 0)
		PROCESS_ERROR(bLastLevel == true);
	if(numofUniqueFea > 0)//need to move instances to new nodes if there are new nodes.
	{
		dim3 dimGridThreadForEachUsedFea;
		conf.ComputeBlock(numofUniqueFea, dimGridThreadForEachUsedFea);
		int sharedMemSizeUsedFea = 1;
		InsToNewNode<<<dimGridThreadForEachUsedFea, sharedMemSizeUsedFea>>>(
								 snManager.m_pTreeNode, manager.m_pdDFeaValue, manager.m_pDInsId,
								 manager.m_pFeaStartPos, manager.m_pDNumofKeyValue,
								 manager.m_pInsIdToNodeId, manager.m_pSNIdToBuffId, manager.m_pBestSplitPoint,
								 snManager.m_pUniqueFeaIdVec, snManager.m_pNumofUniqueFeaId,
								 snManager.m_pParentId, snManager.m_pLeftChildId, snManager.m_pRightChildId,
								 preMaxNodeId, manager.m_numofFea, manager.m_numofIns, LEAFNODE);
	}
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in InsToNewNode" << endl;
		exit(0);
	}
#endif

	//for those instances of unknown feature values.
	int threadPerBlockEachIns;
	dim3 dimNumofBlockEachIns;
	conf.ConfKernel(manager.m_numofIns, threadPerBlockEachIns, dimNumofBlockEachIns);

	InsToNewNodeByDefault<<<dimNumofBlockEachIns, threadPerBlockEachIns>>>(
									snManager.m_pTreeNode, manager.m_pInsIdToNodeId, manager.m_pSNIdToBuffId,
									snManager.m_pParentId, snManager.m_pLeftChildId,
			   	   	   	   	   	   	preMaxNodeId, manager.m_numofIns, LEAFNODE);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in InsToNewNodeByDefault" << endl;
		exit(0);
	}
#endif

	//update new splittable nodes
	int numofNewSplittableNode = -1;
	manager.MemcpyDeviceToHost(snManager.m_pNumofNewNode, &numofNewSplittableNode, sizeof(int));
	if(numofNewSplittableNode == 0)
		PROCESS_ERROR(bLastLevel == true);
	if(numofNewSplittableNode > 0)//update splittable nodes when there are new splittable nodes
	{
		dim3 dimGridThreadForEachNewSN;
		conf.ComputeBlock(numofNewSplittableNode, dimGridThreadForEachNewSN);
		int sharedMemSizeNSN = 1;

		//reset nodeId to bufferId
		manager.Memset(manager.m_pSNIdToBuffId, -1, sizeof(int) * manager.m_maxNumofSplittable);
		manager.Memset(manager.m_pNumofBuffId, 0, sizeof(int));
		//reset nodeStat
		manager.Memset(manager.m_pSNodeStat, 0, sizeof(nodeStat) * manager.m_maxNumofSplittable);
		UpdateNewSplittable<<<dimGridThreadForEachNewSN, sharedMemSizeNSN>>>(
									  snManager.m_pNewSplittableNode, snManager.m_pNewNodeStat, manager.m_pSNIdToBuffId,
									  manager.m_pSNodeStat, snManager.m_pNumofNewNode, manager.m_pBuffIdVec, manager.m_pNumofBuffId,
									  manager.m_maxNumofSplittable, manager.m_nSNLock);
#if testing
	if(hipGetLastError() != hipSuccess)
	{
		cout << "error in UpdateNewSplittable" << endl;
		exit(0);
	}
#endif

		manager.MemcpyDeviceToDevice(snManager.m_pNewSplittableNode, manager.m_pSplittableNode, sizeof(TreeNode) * manager.m_maxNumofSplittable);
	}
}
