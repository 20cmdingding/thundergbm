#include "hip/hip_runtime.h"
/*
 * SplitAllKernel.cu
 *
 *  Created on: 15 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <string.h>
#include "DeviceSplitAllKernel.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../DeviceHashing.h"
#include "../ErrorChecker.h"

using std::string;

/**
 * @brief: compute the base_weight of tree node, also determines if a node is a leaf.
 */
__global__ void ComputeWeight(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, int *pSNIdToBufferId,
								  SplitPoint *pBestSplitPoint, nodeStat *pSNodeStat, float_point rt_eps, int flag_LEAFNODE,
								  float_point lambda, int numofSplittableNode, bool bLastLevel)
{
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(nGlobalThreadId < 0 || nGlobalThreadId >= numofSplittableNode)//one thread per splittable node
		printf("Error in computeWeight function, thread id=%d\n", nGlobalThreadId);

	int nid = pSplittableNode[nGlobalThreadId].nodeId;
	ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");

//		cout << "node " << nid << " needs to split..." << endl;
	int bufferPos = pSNIdToBufferId[nid];
	ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");

	//mark the node as a leaf node if (1) the gain is negative or (2) the tree reaches maximum depth.
	pAllTreeNode[nid].loss = pBestSplitPoint[bufferPos].m_fGain;
	ErrorChecker(pSNodeStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "pSNodeStat[bufferPos].sum_hess");

	float_point nodeWeight = (-pSNodeStat[bufferPos].sum_gd / (pSNodeStat[bufferPos].sum_hess + lambda));
	pAllTreeNode[nid].base_weight = nodeWeight;
	if(pBestSplitPoint[bufferPos].m_fGain <= rt_eps || bLastLevel == true)
	{
		//weight of a leaf node
		pAllTreeNode[nid].predValue = pAllTreeNode[nid].base_weight;
		pAllTreeNode[nid].rightChildId = flag_LEAFNODE;
	}
}

/**
 * @brief: create new nodes and associate new nodes with their parent id
 */
__global__ void CreateNewNode(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, TreeNode *pNewSplittableNode,
								 const int *pSNIdToBufferId, const SplitPoint *pBestSplitPoint,
								  int *pParentId, int *pLChildId, int *pRChildId,
								  const nodeStat *pLChildStat, const nodeStat *pRChildStat, nodeStat *pNewNodeStat,
								  int *pNumofNode, int *pNumofNewNode,
								  float_point rt_eps, int nNumofSplittableNode, bool bLastLevel)
{
	//for each splittable node, assign lchild and rchild ids
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(nGlobalThreadId < 0 || nGlobalThreadId >= nNumofSplittableNode)//one thread per splittable node
		printf("Error in CreateNewNode function, thread id=%d\n", nGlobalThreadId);

	ErrorChecker(*pNumofNewNode == 0, __PRETTY_FUNCTION__, "*pNumofNewNode == 0");

	int nid = pSplittableNode[nGlobalThreadId].nodeId;
	ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");
	int bufferPos = pSNIdToBufferId[nid];
	ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");

	if(!(pBestSplitPoint[bufferPos].m_fGain <= rt_eps || bLastLevel == true))
	{
		int childrenId = atomicAdd(pNumofNode, 2);

		int lchildId = childrenId;
		int rchildId = childrenId + 1;

		//parent id to child ids
		pParentId[bufferPos] = nid;
		pLChildId[bufferPos] = lchildId;
		pRChildId[bufferPos] = rchildId;
		ErrorChecker(pLChildStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "lchildStat[bufferPos].sum_hess");
		ErrorChecker(pRChildStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "rchildStat[bufferPos].sum_hess");

		//push left and right child statistics into a vector
		int newNodeId = atomicAdd(pNumofNewNode, 2);
		int leftNewNodeId = newNodeId;
		int rightNewNodeId = newNodeId + 1;
		pNewNodeStat[leftNewNodeId] = pLChildStat[bufferPos];
		pNewNodeStat[rightNewNodeId] = pRChildStat[bufferPos];


		//split into two nodes
		TreeNode &leftChild = pAllTreeNode[lchildId];
		TreeNode &rightChild = pAllTreeNode[rchildId];
		int nLevel = pAllTreeNode[nid].level;

		leftChild.nodeId = lchildId;
		leftChild.parentId = nid;
		leftChild.level = nLevel + 1;
		rightChild.nodeId = rchildId;
		rightChild.parentId = nid;
		rightChild.level = nLevel + 1;

		//they should just be pointers, not new content
		pNewSplittableNode[leftNewNodeId] = leftChild;
		pNewSplittableNode[rightNewNodeId] = rightChild;


		pAllTreeNode[nid].leftChildId = leftChild.nodeId;
		pAllTreeNode[nid].rightChildId = rightChild.nodeId;
		ErrorChecker(pBestSplitPoint[bufferPos].m_nFeatureId, __PRETTY_FUNCTION__, "pBestSplitPoint[bufferPos].m_nFeatureId");

		pAllTreeNode[nid].featureId = pBestSplitPoint[bufferPos].m_nFeatureId;
		pAllTreeNode[nid].fSplitValue = pBestSplitPoint[bufferPos].m_fSplitValue;

		//this is used in finding unique feature ids
		pSplittableNode[nGlobalThreadId].featureId = pBestSplitPoint[bufferPos].m_nFeatureId;

//			printf("cur # of node is %d\n", *pNumofNode);
	}

}

/**
 * @brief: get unique used feature ids of the splittable nodes
 */
__global__ void GetUniqueFid(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, int nNumofSplittableNode,
								 int *pFeaIdToBuffId, int *pUniqueFidVec, int *pNumofUniqueFid,
								 int maxNumofUsedFea, int flag_LEAFNODE, int *pnLock)
{
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(nGlobalThreadId < 0 || nGlobalThreadId >= nNumofSplittableNode)//one thread per splittable node
		printf("Error in GetUniqueFid function, thread id=%d\n", nGlobalThreadId);

	ErrorCond(*pNumofUniqueFid == 0, __PRETTY_FUNCTION__, "*pNumofUniqueFid == 0");

	int fid = pSplittableNode[nGlobalThreadId].featureId;
	int nid = pSplittableNode[nGlobalThreadId].nodeId;
	if(fid == -1 && pAllTreeNode[nid].rightChildId == flag_LEAFNODE)
	{//leaf node should satisfy two conditions at this step
		return;
	}
	ErrorChecker(fid, __PRETTY_FUNCTION__, "fid");

	bool bLeaveLoop = false;
	while(bLeaveLoop == false)
	{
		//critical region when assigning hash value
		if(atomicExch(pnLock, 1) == 0)
		{
			bool bIsNew = false;
			int hashValue = AssignHashValue(pFeaIdToBuffId, fid, maxNumofUsedFea, bIsNew);
			if(bIsNew == true)
			{
				int numofUniqueFid = atomicAdd(pNumofUniqueFid, 1);
				pUniqueFidVec[numofUniqueFid] = fid;
			}
			ErrorChecker(hashValue, __PRETTY_FUNCTION__, "hashValue");
			bLeaveLoop = true;
			atomicExch(pnLock, 0);
		}
	}
}

/**
 * @brief: assign instances (which have non-zero values on the feature of interest) to new nodes
 */
__global__ void InsToNewNode(TreeNode *pAllTreeNode, float_point *pdFeaValue, int *pInsId,
								 long long *pFeaStartPos, int *pNumofKeyValue,
								 int *pInsIdToNodeId, int *pSNIdToBuffId, SplitPoint *pBestSplitPoint,
								 int *pUniqueFidVec, int *pNumofUniqueFid,
								 int *pParentId, int *pLChildId, int *pRChildId,
								 int preMaxNodeId, int numofFea, int numofIns, int flag_LEAFNODE)
{
	int numofUniqueFid = *pNumofUniqueFid;

	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(nGlobalThreadId < 0 || nGlobalThreadId >= numofUniqueFid)//one thread per splittable node
		printf("Error in InsToNewNode function, thread id=%d\n", nGlobalThreadId);

	int ufid = pUniqueFidVec[nGlobalThreadId];
	ErrorChecker(ufid, __PRETTY_FUNCTION__, "ufid");
	ErrorChecker(numofFea - ufid, __PRETTY_FUNCTION__, "numofFea - ufid");

	//for each instance that has value on the feature
	long long curFeaStartPos = pFeaStartPos[ufid];
	float_point *pdCurFeaValue = pdFeaValue + curFeaStartPos;
	int *pCurFeaInsId = pInsId + curFeaStartPos;
	int nNumofPair = pNumofKeyValue[ufid];
	for(int i = 0; i < nNumofPair; i++)
	{
		int insId = pCurFeaInsId[i];
		ErrorChecker(insId, __PRETTY_FUNCTION__, "insId");
		ErrorChecker(numofIns - insId, __PRETTY_FUNCTION__, "numofIns - insId");
		int nid = pInsIdToNodeId[insId];

		if(nid < 0)//leaf node
			continue;

		if(nid > preMaxNodeId)//new node ids
			continue;

		ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");
		int bufferPos = pSNIdToBuffId[nid];
		ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");
		int fid = pBestSplitPoint[bufferPos].m_nFeatureId;
		if(fid != ufid)//this feature is not the splitting feature for the instance.
			continue;


		if(nid != pParentId[bufferPos])//node doesn't need to split (leaf node or new node)
		{
			if(pAllTreeNode[nid].rightChildId != flag_LEAFNODE)
			{
				ErrorChecker(preMaxNodeId - nid, __PRETTY_FUNCTION__, "preMaxNodeId - nid");
				continue;
			}
			ErrorCond(pAllTreeNode[nid].rightChildId == flag_LEAFNODE, __PRETTY_FUNCTION__, "pAllTreeNode[nid].rightChildId == flag_LEAFNODE");
			continue;
		}

		if(nid == pParentId[bufferPos])
		{//internal node (needs to split)
			ErrorCond(pRChildId[bufferPos] == pLChildId[bufferPos] + 1, __PRETTY_FUNCTION__, "rChild=lChild+1");//right child id > than left child id

			double fPivot = pBestSplitPoint[bufferPos].m_fSplitValue;
			double fvalue = pdCurFeaValue[i];
			if(fvalue >= fPivot)
			{
				pInsIdToNodeId[insId] = pRChildId[bufferPos];//right child id
			}
			else
				pInsIdToNodeId[insId] = pLChildId[bufferPos];//left child id
		}
	}

}

__global__ void InsToNewNodeByDefault(TreeNode *pAllTreeNode, int *pInsIdToNodeId, int *pSNIdToBuffId,
										   int *pParentId, int *pLChildId,
										   int preMaxNodeId, int numofIns, int flag_LEAFNODE)
{
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(nGlobalThreadId >= numofIns)//not used threads
		return;

	ErrorChecker(preMaxNodeId, __PRETTY_FUNCTION__, "maxId shoud >= 0");

	int nid = pInsIdToNodeId[nGlobalThreadId];
	if(nid == -1 || nid > preMaxNodeId)//processed node (i.e. leaf node or new node)
		return;
	//newly constructed leaf node
	if(pAllTreeNode[nid].rightChildId == flag_LEAFNODE)
	{
		pInsIdToNodeId[nGlobalThreadId] = -1;
	}
	else
	{
		int bufferPos = pSNIdToBuffId[nid];
		pInsIdToNodeId[nGlobalThreadId] = pLChildId[bufferPos];//by default the instance with unknown feature value going to left child
		ErrorCond(bufferPos != -1, __PRETTY_FUNCTION__, "rChild=lChild+1");
	}

}

__global__ void UpdateNewSplittable(TreeNode *pNewSplittableNode, nodeStat *pNewNodeStat, int *pSNIdToBuffId,
								   	    nodeStat *pSNodeStat, int *pNumofNewNode, int *pBuffIdVec, int *pBuffIdCounter,
								   	    int maxNumofSplittable, int *pnLock)
{
	int numofNewNode = *pNumofNewNode;
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(nGlobalThreadId < 0 || nGlobalThreadId >= numofNewNode)//one thread per splittable node
		printf("Error in InsToNewNode function, thread id=%d\n", nGlobalThreadId);

	ErrorChecker(*pBuffIdCounter == 0, __PRETTY_FUNCTION__, "*pBuffIdCounter == 0");

	int nid = pNewSplittableNode[nGlobalThreadId].nodeId;
	ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");


	bool bLeaveLoop = false;
	while(bLeaveLoop == false)
	{
		//critical region when assigning hash value
		if(atomicExch(pnLock, 1) == 0)
		{
			bool bIsNew = false;
			int bufferPos = AssignHashValue(pSNIdToBuffId, nid, maxNumofSplittable, bIsNew);

			ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");
			pSNodeStat[bufferPos] = pNewNodeStat[nGlobalThreadId];
			if(bIsNew == true)
			{
				int counter = atomicAdd(pBuffIdCounter, 1);
				pBuffIdVec[counter] = bufferPos;
			}
			bLeaveLoop = true;
			atomicExch(pnLock, 0);
		}
	}

}
