#include "hip/hip_runtime.h"
/*
 * DeviceSplitterKernel.cu
 *
 *  Created on: 10 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <stdio.h>

#include "DeviceFindFeaKernel.h"
#include "DeviceSplitter.h"
#include "../DeviceHashing.h"

const float rt_2eps = 2.0 * DeviceSplitter::rt_eps;

__global__ void FindFeaSplitValue(const int *pnNumofKeyValues, const long long *pnFeaStartPos, const int *pInsId, const float_point *pFeaValue,
								  const int *pInsIdToNodeId, const float_point *pGD, const float_point *pHess,
								  nodeStat *pTempRChildStatPerThread, float_point *pLastValuePerThread,
								  const nodeStat *pSNodeStatPerThread, SplitPoint *pBestSplitPointPerThread,
								  nodeStat *pRChildStatPerThread, nodeStat *pLChildStatPerThread,
								  const int *pSNIdToBuffId, int maxNumofSplittable, const int *pBuffId, int numofSNode,
								  float_point lambda, int numofFea)
{
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	int feaId = nGlobalThreadId;
	if(feaId > numofFea)
	{
		printf("should not happened!\n");
		return;
	}

	//addresses of instance ids and key-value pairs
		//compute start position key-value pairs of the current feature
	long long startPosOfPrevFea = 0;
	int numofPreFeaKeyValues = 0;
	if(feaId > 0)
	{
		//number of key values of the previous feature
		numofPreFeaKeyValues = pnNumofKeyValues[feaId - 1];
		//copy value of the start position of the previous feature
		startPosOfPrevFea = pnFeaStartPos[feaId - 1];
	}
	long long startPosOfCurFea = startPosOfPrevFea + numofPreFeaKeyValues;
	const int *InsIdStartAddress = pInsId + startPosOfCurFea;
	const float_point *pInsValueStartAddress = pFeaValue + startPosOfCurFea;

    for(int i = 0; i < pnNumofKeyValues[nGlobalThreadId]; i++)
    {
    	int insId = InsIdStartAddress[i];
    	int nid = pInsIdToNodeId[insId];
		if(nid < -1)
		{
			printf("Error: nid=%d\n", nid);
			return;
		}
		if(nid == -1)
			continue;

		// start working
		double fvalue = pInsValueStartAddress[i];

		// get the buffer id of node nid
		int hashValue = GetBufferId(pSNIdToBuffId, nid, maxNumofSplittable);
		int bufferPos = hashValue + feaId * maxNumofSplittable;

		if(pTempRChildStatPerThread[bufferPos].sum_hess == 0.0)//equivalent to IsEmpty()
		{
			pTempRChildStatPerThread[bufferPos].sum_gd += pGD[insId];
			pTempRChildStatPerThread[bufferPos].sum_hess += pHess[insId];
			pLastValuePerThread[bufferPos] = fvalue;
		}
		else
		{
			// try to find a split
			if(fabs(fvalue - pLastValuePerThread[bufferPos]) > rt_2eps)
			{
				float_point tempGD = pSNodeStatPerThread[bufferPos].sum_gd - pTempRChildStatPerThread[bufferPos].sum_gd;
				float_point tempHess = pSNodeStatPerThread[bufferPos].sum_hess - pTempRChildStatPerThread[bufferPos].sum_hess;
				bool needUpdate = NeedUpdate(pTempRChildStatPerThread[bufferPos].sum_hess, tempHess);
				if(needUpdate == true)
				{
					double sv = (fvalue + pLastValuePerThread[bufferPos]) * 0.5f;
					if(hashValue == 1)
					{
//						double loss_chg = CalGain(pSNodeStatPerThread[bufferPos], pTempRChildStatPerThread[bufferPos], tempGD, tempHess, lambda);
//						printf("nid=%d, sv=%f, gain=%f\n", nid, sv, loss_chg);
					}

		            UpdateSplitInfo(pSNodeStatPerThread[bufferPos], pBestSplitPointPerThread[bufferPos], pRChildStatPerThread[bufferPos],
		            							  pLChildStatPerThread[bufferPos], pTempRChildStatPerThread[bufferPos], tempGD, tempHess,
		            							  lambda, sv, feaId);
				}
			}
			//update the statistics
			pTempRChildStatPerThread[bufferPos].sum_gd += pGD[insId];
			pTempRChildStatPerThread[bufferPos].sum_hess += pHess[insId];
			pLastValuePerThread[bufferPos] = fvalue;
		}
	}

    // finish updating all statistics, check if it is possible to include all sum statistics
    for(int i = 0; i < numofSNode; i++)
    {
    	if(pBuffId[i] < 0)
    		printf("Error in buffer id %d, i=%d, numofSN=%d\n", pBuffId[i], i, numofSNode);

    	int buffId = pBuffId[i] + feaId * maxNumofSplittable;//an id in the buffer
    	float_point tempGD = pSNodeStatPerThread[buffId].sum_gd - pTempRChildStatPerThread[buffId].sum_gd;
    	float_point tempHess = pSNodeStatPerThread[buffId].sum_hess - pTempRChildStatPerThread[buffId].sum_hess;
    	bool needUpdate = NeedUpdate(pTempRChildStatPerThread[buffId].sum_hess, tempHess);
        if(needUpdate == true)
        {
            const float delta = fabs(pLastValuePerThread[buffId]) + DeviceSplitter::rt_eps;
            float_point sv = pLastValuePerThread[buffId] + delta;

            UpdateSplitInfo(pSNodeStatPerThread[buffId], pBestSplitPointPerThread[buffId], pRChildStatPerThread[buffId], pLChildStatPerThread[buffId],
            							  pTempRChildStatPerThread[buffId], tempGD, tempHess, lambda, sv, feaId);
        }
    }
}

__device__ void CopyNodeStat(nodeStat *pDest, const nodeStat *pSrc)
{
	pDest[0].sum_gd = pSrc[0].sum_gd;
	pDest[0].sum_hess = pSrc[0].sum_hess;
}

__global__ void PickBestFea(float_point *pLastValuePerThread,
							SplitPoint *pBestSplitPointPerThread, nodeStat *pRChildStatPerThread, nodeStat *pLChildStatPerThread,
							const int *pBuffId, int numofSNode, int numofFea, int maxNumofSplittable)
{
	//the best splittable node is stored in the first numofSNode nodes.
	for(int f = 0; f < numofFea; f++)
	{//Caution when to do in parallel
		for(int n = 0; n < numofSNode; n++)
		{
			if(pBuffId[n] < 0 || pBuffId[n] >= maxNumofSplittable)
				printf("Error in PickBestFea\n");
			int nodePos = f * maxNumofSplittable + pBuffId[n];//compute splittable node position in buffer
			int firstPosOfSplitPoint = pBuffId[n];
			if(pBestSplitPointPerThread[nodePos].m_fGain > pBestSplitPointPerThread[firstPosOfSplitPoint].m_fGain)
			{
				pLastValuePerThread[firstPosOfSplitPoint] = pLastValuePerThread[nodePos];

				pBestSplitPointPerThread[firstPosOfSplitPoint].m_fGain = pBestSplitPointPerThread[nodePos].m_fGain;
				pBestSplitPointPerThread[firstPosOfSplitPoint].m_fSplitValue = pBestSplitPointPerThread[nodePos].m_fSplitValue;
				pBestSplitPointPerThread[firstPosOfSplitPoint].m_nFeatureId = pBestSplitPointPerThread[nodePos].m_nFeatureId;

				CopyNodeStat(pRChildStatPerThread + firstPosOfSplitPoint, pRChildStatPerThread + nodePos);
				CopyNodeStat(pLChildStatPerThread + firstPosOfSplitPoint, pLChildStatPerThread + nodePos);
			}
		}
	}
}

__device__ double CalGain(const nodeStat &parent, const nodeStat &r_child,
						  const float_point &l_child_GD, const float_point &l_child_Hess,
						  const float_point &lambda)
{
	PROCESS_ERROR(abs(parent.sum_gd - l_child_GD - r_child.sum_gd) < 0.0001);
	PROCESS_ERROR(parent.sum_hess == l_child_Hess + r_child.sum_hess);

//	printf("lgd=%f, lhe=%f, rgd=%f, rhe=%f, pgd=%f, phe=%f, lamb=%f\n", l_child_GD, l_child_Hess,
//			r_child.sum_gd, r_child.sum_hess, parent.sum_gd, parent.sum_hess, lambda);

	//compute the gain
	double fGain = (l_child_GD * l_child_GD)/(l_child_Hess + lambda) +
				   (r_child.sum_gd * r_child.sum_gd)/(r_child.sum_hess + lambda) -
				   (parent.sum_gd * parent.sum_gd)/(parent.sum_hess + lambda);
//	if(fGain > -10)
//	{
//		printf("gain=%f, lgd=%f, lhe=%f, rgd=%f, rhe=%f, pgd=%f, phe=%f, lamb=%f\n", fGain, l_child_GD, l_child_Hess,
//				r_child.sum_gd, r_child.sum_hess, parent.sum_gd, parent.sum_hess, lambda);
//	}


	return fGain;
}


 __device__ bool UpdateSplitPoint(SplitPoint &curBest, double fGain, double fSplitValue, int nFeatureId)
{
	if(fGain > curBest.m_fGain )//|| (fGain == m_fGain && nFeatureId == m_nFeatureId) NOT USE (second condition is for updating to a new split value)
	{
		curBest.m_fGain = fGain;
		curBest.m_fSplitValue = fSplitValue;
		curBest.m_nFeatureId = nFeatureId;
		return true;
	}
	return false;
}

__device__ void UpdateLRStat(nodeStat &RChildStat, nodeStat &LChildStat,
							 const nodeStat &TempRChildStat, const float_point &grad, const float_point &hess)
{
	LChildStat.sum_gd = grad;
	LChildStat.sum_hess = hess;
	RChildStat = TempRChildStat;
}

__device__ bool NeedUpdate(float_point &RChildHess, float_point &LChildHess)
{
	if(LChildHess >= DeviceSplitter::min_child_weight && RChildHess >= DeviceSplitter::min_child_weight)
		return true;
	return false;
}

__device__ void UpdateSplitInfo(const nodeStat &snStat, SplitPoint &bestSP, nodeStat &RChildStat, nodeStat &LChildStat,
								const nodeStat &TempRChildStat, const float_point &tempGD, const float_point &tempHess,
								const float_point &lambda, const float_point &sv, const int &featureId)
{
	double loss_chg = CalGain(snStat, TempRChildStat, tempGD, tempHess, lambda);
    bool bUpdated = UpdateSplitPoint(bestSP, loss_chg, sv, featureId);
	if(bUpdated == true)
	{
		UpdateLRStat(RChildStat, LChildStat, TempRChildStat, tempGD, tempHess);
	}
}

