/*
 * initCuda.cpp
 *
 *  Created on: 10/12/2014
 *      Author: Zeyi Wen
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::cerr;
using std::endl;

/**
 * @brief: initialize CUDA device
 */

bool InitCUDA(char gpuType, hipCtx_t &context)
{
    int count;

    checkCudaErrors(hipGetDeviceCount(&count));
    if(count == 0)
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipDevice_t device;

    int i;
    bool bUseTesla = false;
    for(i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        checkCudaErrors(hipGetDeviceProperties(&prop, i));
        if(hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
        	cout << prop.name << endl;
        	if(prop.name[0] == gpuType && prop.name[1] == 'e')
        	{
        		cout << "Using " << prop.name << endl;
       			bUseTesla = true;
//        		checkCudaErrors(hipSetDevice(i));
        		hipDeviceGet(&device, i);
        		hipCtxCreate(&context, hipDeviceScheduleAuto, device);
        		break;
        	}
            if(prop.major >= 1)
            {
            	cout << count << " device(s) with compute capability " << prop.major << endl;
            }
        }
    }

    if(i == count)
    {
        cerr << "There is no device of " << gpuType << " series. Please reset the parameter of \""
        	 << __PRETTY_FUNCTION__ << "\"" <<endl;
        return false;
    }

    if(!bUseTesla)
    {
    	checkCudaErrors(hipSetDevice(0));
    }

    return true;
}

bool ReleaseCuda(hipCtx_t &context)
{
	cuCtxDetach(context);
	return true;
}
