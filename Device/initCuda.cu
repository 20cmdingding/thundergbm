/*
 * initCuda.cpp
 *
 *  Created on: 10/12/2014
 *      Author: Zeyi Wen
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::cerr;
using std::endl;

/**
 * @brief: set the device to use
 */
void UseDevice(int deviceId, hipCtx_t &context)
{
    hipDevice_t device;
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, deviceId));
    cout << "Using " << prop.name << "; device id is " << deviceId << endl;
    checkCudaErrors(hipSetDevice(deviceId));
    hipDeviceGet(&device, deviceId);
    hipCtxCreate(&context, hipDeviceMapHost, device);
    if(!prop.canMapHostMemory)
		fprintf(stderr, "Device %d cannot map host memory!\n", deviceId);
}


/**
 * @brief: initialize CUDA device
 */

bool InitCUDA(hipCtx_t &context, char gpuType = 'T')
{
    int count;

    checkCudaErrors(hipGetDeviceCount(&count));
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        checkCudaErrors(hipGetDeviceProperties(&prop, i));
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
        	cout << prop.name << endl;
        	if(prop.name[0] == gpuType)
        	{//choose the prefer device
                UseDevice(i, context);
       			break;
        	}
        }
    }

    cout << i << " v.s. " << count << endl;
    if(i == count)
    {
        cout << "There is no device of \"" << gpuType << "\" series" << endl;
        UseDevice(0, context);
    }

    return true;
}

bool ReleaseCuda(hipCtx_t &context)
{
	cuCtxDetach(context);
	return true;
}
