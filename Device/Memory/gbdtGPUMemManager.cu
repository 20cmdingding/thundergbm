#include "hip/hip_runtime.h"
/*
 * gbdtGPUMemManager.cu
 *
 *  Created on: 4 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <hip/hip_runtime_api.h>

#include "gbdtGPUMemManager.h"
#include "../../DeviceHost/MyAssert.h"

//memory for instances (key on feature id)
int *GBDTGPUMemManager::m_pDInsId = NULL;				//all the instance ids for each key-value pair
float_point *GBDTGPUMemManager::m_pdDFeaValue = NULL; 	//all the feature values
int *GBDTGPUMemManager::m_pDNumofKeyValue = NULL;		//the number of key-value pairs of each feature
long long *GBDTGPUMemManager::m_pFeaStartPos = NULL;	//start key-value position of each feature
//memory for instances (key on instance id)
int *GBDTGPUMemManager::m_pDFeaId = NULL;				//all the feature ids for every instance
float_point *GBDTGPUMemManager::m_pdDInsValue = NULL;	//all the feature values for every instance
int *GBDTGPUMemManager::m_pDNumofFea = NULL;			//the number of features for each instance
long long *GBDTGPUMemManager::m_pInsStartPos = NULL;	//the start position of each instance

//memory for prediction
float_point *GBDTGPUMemManager::m_pPredBuffer = NULL;
float_point *GBDTGPUMemManager::m_pdTrueTargetValue = NULL;
float_point *GBDTGPUMemManager::m_pdDenseIns = NULL;
float_point *GBDTGPUMemManager::m_pTargetValue = NULL;		//will support prediction in parallel
int GBDTGPUMemManager::maxNumofDenseIns = -1;
int *GBDTGPUMemManager::m_pHashFeaIdToDenseInsPos = NULL;	//hash map for used feature ids of all trees to the dense instance position
int *GBDTGPUMemManager::m_pSortedUsedFeaId = NULL;			//sorted used feature ids
int GBDTGPUMemManager::m_maxUsedFeaInTrees = -1;		//maximum number of used features in all the trees

int *GBDTGPUMemManager::m_pInsIdToNodeId = NULL; 		//map instance id to node id
long long GBDTGPUMemManager::m_totalNumofValues = -1;
int GBDTGPUMemManager::m_numofIns = -1;
int GBDTGPUMemManager::m_numofFea = -1;

//memory for gradient and hessian
float_point *GBDTGPUMemManager::m_pGrad = NULL;
float_point *GBDTGPUMemManager::m_pHess = NULL;

//memory for splittable nodes
int GBDTGPUMemManager::m_maxNumofSplittable = -1;
int GBDTGPUMemManager::m_curNumofSplitable = -1;
TreeNode *GBDTGPUMemManager::m_pSplittableNode = NULL;
SplitPoint *GBDTGPUMemManager::m_pBestSplitPoint = NULL;//(require memset!) store the best split points
nodeStat *GBDTGPUMemManager::m_pSNodeStat = NULL;	//splittable node statistics
nodeStat *GBDTGPUMemManager::m_pRChildStat = NULL;
nodeStat *GBDTGPUMemManager::m_pLChildStat = NULL;
nodeStat *GBDTGPUMemManager::m_pTempRChildStat = NULL;//(require memset!) store temporary statistics of right child
float_point *GBDTGPUMemManager::m_pLastValue = NULL;//store the last processed value (for computing split point)
int *GBDTGPUMemManager::m_nSNLock = NULL;

int *GBDTGPUMemManager::m_pSNIdToBuffId = NULL;	//(require memset!) map splittable node id to buffer position
int *GBDTGPUMemManager::m_pBuffIdVec = NULL;	//store all the buffer ids for splittable nodes
int *GBDTGPUMemManager::m_pNumofBuffId = NULL;	//the total number of buffer ids in the current round.

//host memory for GPU memory reset
SplitPoint *GBDTGPUMemManager::m_pBestPointHost = NULL;//best split points

/**
 * @brief: allocate memory for instances
 */
void GBDTGPUMemManager::allocMemForIns(int nTotalNumofValue, int numofIns, int numofFeature)
{
	PROCESS_ERROR(nTotalNumofValue > 0);
	PROCESS_ERROR(numofFeature > 0);
	PROCESS_ERROR(numofIns > 0);
	m_totalNumofValues = nTotalNumofValue;
	m_numofIns = numofIns;
	m_numofFea = numofFeature;

	//memory for instances (key on feature id)
	checkCudaErrors(hipMalloc((void**)&m_pDInsId, sizeof(int) * m_totalNumofValues));
	checkCudaErrors(hipMalloc((void**)&m_pdDFeaValue, sizeof(float_point) * m_totalNumofValues));
	checkCudaErrors(hipMalloc((void**)&m_pDNumofKeyValue, sizeof(int) * m_numofFea));
	checkCudaErrors(hipMalloc((void**)&m_pFeaStartPos, sizeof(long long) * m_numofFea));
	//memory for instances (key on instance id)
	checkCudaErrors(hipMalloc((void**)&m_pDFeaId, sizeof(int) * m_totalNumofValues));
	checkCudaErrors(hipMalloc((void**)&m_pdDInsValue, sizeof(float_point) * m_totalNumofValues));
	checkCudaErrors(hipMalloc((void**)&m_pDNumofFea, sizeof(int) * m_numofIns));
	checkCudaErrors(hipMalloc((void**)&m_pInsStartPos, sizeof(long long) * m_numofIns));

	//memory for prediction. Buffering previous predicted values
	checkCudaErrors(hipMalloc((void**)&m_pPredBuffer, sizeof(float_point) * m_numofIns));
	checkCudaErrors(hipMemset(m_pPredBuffer, 0, sizeof(float_point) * m_numofIns));
	checkCudaErrors(hipMalloc((void**)&m_pdTrueTargetValue, sizeof(float_point) * m_numofIns));
	checkCudaErrors(hipMalloc((void**)&m_pdDenseIns, sizeof(float_point) * m_numofFea * maxNumofDenseIns));//######### whill have bugs when (numofFea < usedFea)
	checkCudaErrors(hipMalloc((void**)&m_pTargetValue, sizeof(float_point) * m_numofIns));
	checkCudaErrors(hipMalloc((void**)&m_pHashFeaIdToDenseInsPos, sizeof(int) * m_maxUsedFeaInTrees));
	checkCudaErrors(hipMemset(m_pHashFeaIdToDenseInsPos, -1, sizeof(int) * m_maxUsedFeaInTrees));
	checkCudaErrors(hipMalloc((void**)&m_pSortedUsedFeaId, sizeof(int) * m_maxUsedFeaInTrees));

	checkCudaErrors(hipMalloc((void**)&m_pInsIdToNodeId, sizeof(int) * m_numofIns));

	//gradient and hessian
	checkCudaErrors(hipMalloc((void**)&m_pGrad, sizeof(float_point) * m_numofIns));
	checkCudaErrors(hipMalloc((void**)&m_pHess, sizeof(float_point) * m_numofIns));
}

/**
 * @brief: allocate memory for splittable nodes
 */
void GBDTGPUMemManager::allocMemForSplittableNode(int nMaxNumofSplittableNode)
{
	PROCESS_ERROR(nMaxNumofSplittableNode > 0);
	PROCESS_ERROR(sizeof(TreeNode) > sizeof(int) * 9);
	PROCESS_ERROR(m_maxNumofSplittable == -1);

	m_maxNumofSplittable = nMaxNumofSplittableNode;

	checkCudaErrors(hipMalloc((void**)&m_pSplittableNode, sizeof(TreeNode) * m_maxNumofSplittable));
	checkCudaErrors(hipMalloc((void**)&m_pBestSplitPoint, sizeof(SplitPoint) * m_maxNumofSplittable));

	checkCudaErrors(hipMalloc((void**)&m_pSNodeStat, sizeof(nodeStat) * m_maxNumofSplittable));
	checkCudaErrors(hipMalloc((void**)&m_pRChildStat, sizeof(nodeStat) * m_maxNumofSplittable));
	checkCudaErrors(hipMalloc((void**)&m_pLChildStat, sizeof(nodeStat) * m_maxNumofSplittable));

	//temporary space for splittable nodes
	checkCudaErrors(hipMalloc((void**)&m_pTempRChildStat, sizeof(nodeStat) * m_maxNumofSplittable));
	checkCudaErrors(hipMalloc((void**)&m_pLastValue, sizeof(float_point) * m_maxNumofSplittable));
	checkCudaErrors(hipMemset(m_pLastValue, 0, sizeof(float_point) * m_maxNumofSplittable));

	checkCudaErrors(hipMalloc((void**)&m_nSNLock, sizeof(int)));//a lock for critical region
	checkCudaErrors(hipMemset(m_nSNLock, 0, sizeof(int)));


	//map splittable node to buffer id
	checkCudaErrors(hipMalloc((void**)&m_pSNIdToBuffId, sizeof(int) * m_maxNumofSplittable));
	checkCudaErrors(hipMalloc((void**)&m_pBuffIdVec, sizeof(int) * m_maxNumofSplittable));
	checkCudaErrors(hipMalloc((void**)&m_pNumofBuffId, sizeof(int)));
}

/**
 * @brief: allocate some host memory for GPU memory reset
 */
void GBDTGPUMemManager::allocHostMemory()
{
	m_pBestPointHost = new SplitPoint[m_maxNumofSplittable];
}

/**
 * @brief: release host memory
 */
void GBDTGPUMemManager::releaseHostMemory()
{
	delete []m_pBestPointHost;
}
