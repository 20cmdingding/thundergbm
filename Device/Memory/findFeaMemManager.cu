#include "hip/hip_runtime.h"
/*
 * findFeaMemManager.cu
 *
 *  Created on: 16 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <hip/hip_runtime_api.h>
#include <iostream>

#include "findFeaMemManager.h"
#include "../../DeviceHost/MyAssert.h"
#include "../KernelConf.h"
#include "../../DeviceHost/svm-shared/MemInfo.h"
#include "../../DeviceHost/svm-shared/HostUtility.h"

#ifdef testing
#undef testing
#endif

using std::cout;
using std::endl;

float_point *FFMemManager::m_pGDOnEachFeaValue_d = NULL;	//gradient of each feature list (same size of each node)
float_point *FFMemManager::m_pHessOnEachFeaValue_d = NULL;	//hessian of each feature list (same size of each node)
float_point *FFMemManager::m_pValueOnEachFeaValue_d = NULL;	//fea value of each item in a feature list
float_point *FFMemManager::m_pGainOnEachFeaValue_d = NULL;	//gain of each fea value in the feature list

int *FFMemManager::m_pFeaLenInBatch_d = NULL;	//length of each feature list
int *FFMemManager::m_pnEachFeaLen_h = NULL;	//length of each feature list in host
int *FFMemManager::m_pStartPosEachFeaInBatch_d = NULL; //start position of each feature list in the feature batch

float_point *FFMemManager::m_pGDPrefixSum_d = NULL;		//prefix sum of gradient of each feature list (same size of each node)
float_point *FFMemManager::m_pHessPrefixSum_d = NULL;	//prefix sum of hessian of each feature list (same size of each node)

float_point *FFMemManager::m_pfFeaLocalBestGain_d = NULL;	//feature best gain in block
int *FFMemManager::m_pnFeaLocalBestGainKey_d = NULL;		//feature key of best gain in block
float_point *FFMemManager::m_pfFeaGlobalBestGain_d = NULL;	//feature global best gain
int *FFMemManager::m_pnFeaGlobalBestGainKey_d = NULL; //feature key of global best gain

float_point *FFMemManager::m_pfBlockBestFea_d = NULL;	//block level feature with best split
int *FFMemManager::m_pnBlockBestKey_d = NULL;			//block level feature key with best split
float_point *FFMemManager::m_pfGlobalBestFea_d = NULL;	//global level feature with best split
int *FFMemManager::m_pnGlobalBestKey_d = NULL;			//global level feature key with best split

float_point *FFMemManager::m_pLastBiggerValue_d = NULL;	//unused variable

long long FFMemManager::m_totalEleInWholeBatch = -1; //a private variable
int FFMemManager::maxNumofSNodeInFF = -1;	//maximum number of splittable nodes in each round of find fea, due to the GPU memory constraint.

//for dense array
float_point *FFMemManager::pGDEachFeaValue = NULL;
float_point *FFMemManager::pHessEachFeaValue = NULL;
float_point *FFMemManager::pDenseFeaValue = NULL;	//feature values of consideration
float_point *FFMemManager::pGDPrefixSum = NULL;
float_point *FFMemManager::pHessPrefixSum = NULL;
float_point *FFMemManager::pGainEachFeaValue = NULL;
int FFMemManager::m_totalNumFeaValue = -1;
float_point *FFMemManager::pfLocalBestGain_d = NULL;
int *FFMemManager::pnLocalBestGainKey_d = NULL;
float_point *FFMemManager::pfGlobalBestGain_d = NULL;
int *FFMemManager::pnGlobalBestGainKey_d = NULL;
//corresponding to pinned memory
int *FFMemManager::m_pIndices_d = NULL;
long long *FFMemManager::m_pFeaValueStartPosEachNode_d = NULL;
long long *FFMemManager::m_pNumFeaValueEachNode_d = NULL;
long long *FFMemManager::m_pEachFeaStartPosEachNode_d = NULL;
int *FFMemManager::m_pEachFeaLenEachNode_d = NULL;

/**
 * @brief: get the maximum number of splittable nodes that can be processed in each round of findFea
 */
int FFMemManager::getMaxNumofSN(int numofValuesInABatch, int maxNumofNode)
{
	long long nFloatPoint = MemInfo::GetFreeGPUMem();

	int tempMaxNumofSN = nFloatPoint / (numofValuesInABatch * 8);//7 such batches for find fea function, using 8 to reserve extra memory for other usage.
	PROCESS_ERROR(tempMaxNumofSN > 0);
	if(tempMaxNumofSN > maxNumofNode)
		tempMaxNumofSN = maxNumofNode;

	int round = Ceil(maxNumofNode, tempMaxNumofSN);
	cout << "find fea requires " << round << " round(s) for the last level of " << maxNumofNode << " nodes" << endl;
	maxNumofSNodeInFF = Ceil(maxNumofNode, round);//take the average number of nodes

	return maxNumofSNodeInFF;
}

/**
 * @brief: allocate memory for finding best feature
 */
void FFMemManager::allocMemForFindFea(int numofValuesInABatch, int maxNumofValuePerFea, int maxNumofFea, int maxNumofSN)
{
	PROCESS_ERROR(numofValuesInABatch > 0);
	int maxNumofNode = maxNumofSNodeInFF;
	PROCESS_ERROR(maxNumofNode > 0);
	long long totalEleInWholeBatch = numofValuesInABatch * maxNumofNode;
	m_totalEleInWholeBatch = totalEleInWholeBatch;
	checkCudaErrors(hipMalloc((void**)&m_pGDPrefixSum_d, sizeof(float_point) * totalEleInWholeBatch));
	checkCudaErrors(hipMalloc((void**)&m_pHessPrefixSum_d, sizeof(float_point) * totalEleInWholeBatch));

	checkCudaErrors(hipMalloc((void**)&m_pGDOnEachFeaValue_d, sizeof(float_point) * totalEleInWholeBatch));
	checkCudaErrors(hipMalloc((void**)&m_pHessOnEachFeaValue_d, sizeof(float_point) * totalEleInWholeBatch));
	checkCudaErrors(hipMalloc((void**)&m_pValueOnEachFeaValue_d, sizeof(float_point) * totalEleInWholeBatch));

	checkCudaErrors(hipMalloc((void**)&m_pStartPosEachFeaInBatch_d, sizeof(int) * maxNumofFea * maxNumofNode));
	checkCudaErrors(hipMalloc((void**)&m_pFeaLenInBatch_d, sizeof(int) * maxNumofFea * maxNumofNode));

	checkCudaErrors(hipMalloc((void**)&m_pGainOnEachFeaValue_d, sizeof(float_point) * totalEleInWholeBatch));
	checkCudaErrors(hipMalloc((void**)&m_pLastBiggerValue_d, sizeof(float_point) * totalEleInWholeBatch));

	int blockSizeFillGD;
	dim3 dimNumofBlockToFillGD;
	KernelConf conf;
	conf.ConfKernel(maxNumofValuePerFea, blockSizeFillGD, dimNumofBlockToFillGD);
	int maxNumofBlockEachFea = dimNumofBlockToFillGD.x;

	checkCudaErrors(hipMalloc((void**)&m_pfFeaLocalBestGain_d, sizeof(float_point) * maxNumofFea * maxNumofBlockEachFea * maxNumofNode));
	checkCudaErrors(hipMalloc((void**)&m_pnFeaLocalBestGainKey_d, sizeof(int) * maxNumofFea * maxNumofBlockEachFea * maxNumofNode));

	checkCudaErrors(hipMalloc((void**)&m_pfFeaGlobalBestGain_d, sizeof(float_point) * maxNumofFea * maxNumofNode));
	checkCudaErrors(hipMalloc((void**)&m_pnFeaGlobalBestGainKey_d, sizeof(int) * maxNumofFea * maxNumofNode));

	int blockSizeBestFeaBestSplit;
	dim3 tempNumofBlockBestFea;
	conf.ConfKernel(maxNumofFea, blockSizeBestFeaBestSplit, tempNumofBlockBestFea);
	int nBlockBestFea = tempNumofBlockBestFea.x;

	checkCudaErrors(hipMalloc((void**)&m_pfBlockBestFea_d, sizeof(float_point) * nBlockBestFea * maxNumofNode));
	checkCudaErrors(hipMalloc((void**)&m_pnBlockBestKey_d, sizeof(int) * nBlockBestFea * maxNumofNode));

	checkCudaErrors(hipMalloc((void**)&m_pfGlobalBestFea_d, sizeof(float_point) * maxNumofNode));
	checkCudaErrors(hipMalloc((void**)&m_pnGlobalBestKey_d, sizeof(int) * maxNumofNode));


	checkCudaErrors(hipMemset(m_pGDOnEachFeaValue_d, 0, sizeof(float_point) * totalEleInWholeBatch));
	checkCudaErrors(hipMemset(m_pHessOnEachFeaValue_d, 0, sizeof(float_point) * totalEleInWholeBatch));
	checkCudaErrors(hipMemset(m_pValueOnEachFeaValue_d, 0, sizeof(float_point) * totalEleInWholeBatch));

	checkCudaErrors(hipMemset(m_pLastBiggerValue_d, 0, sizeof(float_point) * totalEleInWholeBatch));

	m_pnEachFeaLen_h = new int[maxNumofFea * maxNumofNode];

	//for dense array
	PROCESS_ERROR(m_totalNumFeaValue > 0);
	checkCudaErrors(hipMalloc((void**)&pGDEachFeaValue, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMalloc((void**)&pHessEachFeaValue, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMalloc((void**)&pDenseFeaValue, sizeof(float_point) * m_totalNumFeaValue));

	checkCudaErrors(hipMalloc((void**)&pGDPrefixSum, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMalloc((void**)&pHessPrefixSum, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMalloc((void**)&pGainEachFeaValue, sizeof(float_point) * m_totalNumFeaValue));
	int blockSizeLocalBest;
	dim3 tempNumofBlockLocalBest;
	conf.ConfKernel(m_totalNumFeaValue, blockSizeLocalBest, tempNumofBlockLocalBest);
	int maxNumofBlockPerNode = tempNumofBlockLocalBest.x * tempNumofBlockLocalBest.y;
	checkCudaErrors(hipMalloc((void**)&pfLocalBestGain_d, sizeof(float_point) * maxNumofBlockPerNode * maxNumofSN));
	checkCudaErrors(hipMalloc((void**)&pnLocalBestGainKey_d, sizeof(int) * maxNumofBlockPerNode * maxNumofSN));
	checkCudaErrors(hipMalloc((void**)&pfGlobalBestGain_d, sizeof(float_point) * maxNumofSN));
	checkCudaErrors(hipMalloc((void**)&pnGlobalBestGainKey_d, sizeof(int) * maxNumofSN));
	//corresponding to pinned memory
	checkCudaErrors(hipMalloc((void**)&m_pIndices_d, sizeof(int) * m_totalNumFeaValue));
	checkCudaErrors(hipMalloc((void**)&m_pNumFeaValueEachNode_d, sizeof(long long) * maxNumofSN));
	checkCudaErrors(hipMalloc((void**)&m_pFeaValueStartPosEachNode_d, sizeof(long long) * maxNumofSN));
	checkCudaErrors(hipMalloc((void**)&m_pEachFeaStartPosEachNode_d, sizeof(long long) * maxNumofSN * maxNumofFea));
	checkCudaErrors(hipMalloc((void**)&m_pEachFeaLenEachNode_d, sizeof(int) * maxNumofSN * maxNumofFea));
}

/**
 * @brief: reset memory
 */
void FFMemManager::resetMemForFindFea()
{
#if true
	checkCudaErrors(hipMemset(m_pGDOnEachFeaValue_d, 0, sizeof(float_point) * m_totalEleInWholeBatch));
	checkCudaErrors(hipMemset(m_pHessOnEachFeaValue_d, 0, sizeof(float_point) * m_totalEleInWholeBatch));
	checkCudaErrors(hipMemset(m_pValueOnEachFeaValue_d, 0, sizeof(float_point) * m_totalEleInWholeBatch));

	checkCudaErrors(hipMemset(m_pLastBiggerValue_d, 0, sizeof(float_point) * m_totalEleInWholeBatch));
#endif

	//for dense array
	checkCudaErrors(hipMemset(pGDEachFeaValue, 0, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMemset(pHessEachFeaValue, 0, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMemset(pDenseFeaValue, 0, sizeof(float_point) * m_totalNumFeaValue));

	checkCudaErrors(hipMemset(pGDPrefixSum, 0, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMemset(pHessPrefixSum, 0, sizeof(float_point) * m_totalNumFeaValue));
	checkCudaErrors(hipMemset(pGainEachFeaValue, 0, sizeof(float_point) * m_totalNumFeaValue));
}

/**
 * @brief: free memory
 */
void FFMemManager::freeMemForFindFea()
{
	checkCudaErrors(hipFree(m_pGDPrefixSum_d));
	checkCudaErrors(hipFree(m_pHessPrefixSum_d));
	checkCudaErrors(hipFree(m_pfGlobalBestFea_d));
	checkCudaErrors(hipFree(m_pnGlobalBestKey_d));
	checkCudaErrors(hipFree(m_pfBlockBestFea_d));
	checkCudaErrors(hipFree(m_pnBlockBestKey_d));
	checkCudaErrors(hipFree(m_pGDOnEachFeaValue_d));
	checkCudaErrors(hipFree(m_pHessOnEachFeaValue_d));
	checkCudaErrors(hipFree(m_pValueOnEachFeaValue_d));
	checkCudaErrors(hipFree(m_pStartPosEachFeaInBatch_d));
	checkCudaErrors(hipFree(m_pFeaLenInBatch_d));
	checkCudaErrors(hipFree(m_pGainOnEachFeaValue_d));
	checkCudaErrors(hipFree(m_pfFeaLocalBestGain_d));
	checkCudaErrors(hipFree(m_pnFeaLocalBestGainKey_d));
	checkCudaErrors(hipFree(m_pfFeaGlobalBestGain_d));
	checkCudaErrors(hipFree(m_pnFeaGlobalBestGainKey_d));
	delete[] m_pnEachFeaLen_h;
}
