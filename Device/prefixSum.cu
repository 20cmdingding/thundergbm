#include "hip/hip_runtime.h"
/*
 * prefixSum.cu
 *
 *  Created on: 6 Jul 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <iomanip>

#include "prefixSum.h"

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

#define NUM_BLOCKS 511
#define BLOCK_SIZE 512

using std::cout;
using std::endl;

/**
 * @brief: compute prefix sum for in_array with the number of elements equals to "size".
 */
__global__ void cuda_prefixsum(T *in_array, T *out_array, int size)
{
	// this is still inefficient because 1/2 threads are unused
	// shared should be sized to blockDim.x
	extern __shared__ T shared[];

	unsigned int tid = threadIdx.x;
	unsigned int b_offset = blockIdx.x * blockDim.x;
	unsigned int offset = 1;

	int i = tid;
	int j = tid + blockDim.x / 2;
	int offset_i = CONFLICT_FREE_OFFSET(i);
	int offset_j = CONFLICT_FREE_OFFSET(j);

	if(j >= size)//skip threads with large indices
		return;
	shared[i + offset_i] = in_array[i + b_offset];
	shared[j + offset_j] = in_array[j + b_offset];

	// scan up
	for (int s = (blockDim.x >> 1); s > 0; s >>= 1) {
		__syncthreads();

		if (tid < s) {
			int i = offset * (2 * tid + 1) - 1;
			int j = offset * (2 * tid + 2) - 1;
			i += CONFLICT_FREE_OFFSET(i);
			j += CONFLICT_FREE_OFFSET(j);
			shared[j] += shared[i];
		}
		offset <<= 1;
	}

	if (tid == 0) {
		shared[blockDim.x - 1 + CONFLICT_FREE_OFFSET(blockDim.x - 1)] =
		    0;
	}
	// scan down
	for (int s = 1; s < blockDim.x; s <<= 1) {
		offset >>= 1;
		__syncthreads();

		if (tid < s) {
			int i = offset * (2 * tid + 1) - 1;
			int j = offset * (2 * tid + 2) - 1;
			i += CONFLICT_FREE_OFFSET(i);
			j += CONFLICT_FREE_OFFSET(j);
			T tmp = shared[i];
			shared[i] = shared[j];
			shared[j] += tmp;
		}
	}
	__syncthreads();
	// copy data back to main memory
	// scan is exclusive, make it inclusive by left shifting elements
	if (tid < blockDim.x / 2) {
		if (tid > 0) {
			in_array[b_offset + i - 1] = shared[i + offset_i];
		} else {
			// re-calc the last element, drop it in out array
			in_array[b_offset + blockDim.x - 1] +=
			    shared[blockDim.x - 1 +
				   CONFLICT_FREE_OFFSET(blockDim.x - 1)];
			out_array[blockIdx.x] =
			    in_array[b_offset + blockDim.x - 1];
		}
		in_array[b_offset + j - 1] = shared[j + offset_j];
	}
}


/**
 * @brief: post processing of prefix sum for large array
 */
__global__ void cuda_updatesum(T *array, T *update_array, int size)
{
	extern __shared__ T shared[];

	unsigned int tid = threadIdx.x;
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	int op = 0;

	if (blockIdx.x > 0) {
		op = update_array[blockIdx.x - 1];
	}

	shared[tid] = array[id] + op;
	array[id] = shared[tid];
}

/**
 * @brief: prefix sum for an array in device memory
 */
void prefixsumForDeviceArray(int blocks, int threads, T *array_d, int size)
{
	T *out_array_d;
	T *tmp_d;

	dim3 dim_grid(blocks, 1, 1);
	dim3 dim_block(threads, 1, 1);

	// allocate temp, block sum, and device arrays
	hipMalloc((void **)&tmp_d, blocks * sizeof(T));
	hipMalloc((void **)&out_array_d, blocks * sizeof(T));

	// do prefix sum for each block
	cuda_prefixsum <<< dim_grid, dim_block, threads * sizeof(T) >>> (array_d, out_array_d, size);
	// do prefix sum for block sum
	cuda_prefixsum <<< dim_grid, dim_block, threads * sizeof(T) >>> (out_array_d, tmp_d, blocks);
	// update original array using block sum
	cuda_updatesum <<< dim_grid, dim_block, threads * sizeof(T) >>> (array_d, out_array_d, size);

	hipFree(out_array_d);
	hipFree(tmp_d);
}

/**
 * @brief: prefix sum for an array in host memory
 */
void prefixsumForHostArray(int blocks, int threads, T *array_h, int size)
{
	T *array_d;

	dim3 dim_grid(blocks, 1, 1);
	dim3 dim_block(threads, 1, 1);

	// allocate temp, block sum, and device arrays
	hipMalloc((void **)&array_d, size * sizeof(T));
	hipMemcpy(array_d, array_h, size * sizeof(T), hipMemcpyHostToDevice);

	prefixsumForDeviceArray(blocks, threads, array_d, size);

	// copy resulting array back to host
	hipMemcpy(array_h, array_d, size * sizeof(T), hipMemcpyDeviceToHost);

	hipFree(array_d);
}

///////////////// for testing
void prefixsum_host(T *array_h, int size)
{
	for (int i = 0; i < size; i++) {
		if (i > 0) {
			array_h[i] += array_h[i - 1];
		}
	}
}

void usage(int which)
{
	switch (which) {
	default:
		printf("usage: prefixsum [-h|-b blocks|-t threads] max\n");
		break;
	case 1:
		printf("prefixsum requires numbers <= threads*blocks\n");
		break;
	}
}

void print_array(T *array, int count)
{
	for (int i = 0; i < count; i++) {
		cout << array[i] << endl;
	}
}

void prepare_numbers(T **array, int count)
{
	T *numbers = new T[count];

	// load array
	for (int i = 0; i < count; i++) {
		numbers[i] = i + 1.0;
	}

	*array = numbers;
}



int TestPrefixSum(int argc, char *argv[])
{
	int opt, host_mode, blocks, threads, max;
	T *array;

	// set options
	host_mode = 0;
	blocks = 1;
	threads = 64;
	while ((opt = getopt(argc, argv, "hb:t:")) != -1) {
		switch (opt) {
		case 'h':
			host_mode = 1;
			break;
		case 'b':
			blocks = atoi(optarg);
			break;
		case 't':
			threads = atoi(optarg);
			break;
		default:
			usage(0);
			return 0;
		}
	}

	// check to make sure we are feeding in correct number of args
	if (argc == optind + 1) {
		max = atoi(argv[optind]);
	} else {
		usage(0);
		return 0;
	}
	// pre-init numbers
	array = NULL;
	prepare_numbers(&array, max);

	if (host_mode) {
		printf("prefix sum using host\n");
		prefixsum_host(array, max);
	} else {
		printf("prefix sum using CUDA\n");
		prefixsumForHostArray(blocks, threads, array, max);
	}

	// print array
	print_array(array, max);

	free(array);

	return 0;
}
