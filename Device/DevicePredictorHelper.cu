#include "hip/hip_runtime.h"
/*
 * DevicePredictorHelper.cu
 *
 *  Created on: 27 Jun 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include "DevicePredictorHelper.h"
#include "ErrorChecker.h"
#include "DeviceHashing.h"

__device__ int GetNext(TreeNode *pNode, float_point feaValue)
{
    if(feaValue < pNode->fSplitValue)
    {
      return pNode->leftChildId;
    }
    else
    {
      return pNode->rightChildId;
    }
}

__global__ void PredTarget(TreeNode *pAllTreeNode, int totalNode, float_point *pDenseIns, int nNumofFea,
									   int *pnHashFeaIdToPos, float_point *pdTargetValue, int maxDepth)
{
	int pid = 0; //node id
	TreeNode *curNode = pAllTreeNode + pid;
	if(curNode->nodeId != 0)
	{
		printf("id of root node is %d should be 0\n", curNode->nodeId);
		return;
	}
	int counter = 0;
	while(curNode->featureId != -1)//!curNode->isLeaf()
	{
		int fid = curNode->featureId;
		ErrorChecker(fid, __PRETTY_FUNCTION__, "fid < 0");

		int maxNumofUsedFea = nNumofFea;
		int pos = GetBufferId(pnHashFeaIdToPos, fid, maxNumofUsedFea);
//		printf("%d hash to %d: fea v=%f\n", fid, pos, pDenseIns[pos]);

		if(pos < nNumofFea)//feature value is available in the dense vector
			pid = GetNext(curNode, pDenseIns[pos]);
		else//feature value is stored in the dense vector (due to truncating)
			pid = GetNext(curNode, 0);
		curNode = pAllTreeNode + pid;

		counter++;
		if(counter > maxDepth)//for skipping from deadlock
		{
			printf("%s has bugs\n", __PRETTY_FUNCTION__);
			break;
		}
	}

	pdTargetValue[0] += pAllTreeNode[pid].predValue;
}

__global__ void FillDense(const float_point *pdSparseInsValue, const int *pnSpareInsFeaId, int numofFeaValue,
						  float_point *pdDenseIns, const int *pSortedUsedFea, const int *pHashFeaIdToDenseInsPos, int totalUsedFea)
{
	//for each value in the sparse instance
	ErrorChecker(numofFeaValue - 1, __PRETTY_FUNCTION__, "numofFeaValue <= 0");
	int curDenseTop = 0;
	for(int i = 0; i < numofFeaValue; i++)
	{
		int feaId = pnSpareInsFeaId[i];

		while(feaId > pSortedUsedFea[curDenseTop])
		{
			int pos = GetBufferId(pHashFeaIdToDenseInsPos, pSortedUsedFea[curDenseTop], totalUsedFea);
			pdDenseIns[pos] = 0;
			curDenseTop++;
		}

		if(feaId == pSortedUsedFea[curDenseTop])
		{//this is a feature needed to be stored in dense instance
			int pos = GetBufferId(pHashFeaIdToDenseInsPos, pSortedUsedFea[curDenseTop], totalUsedFea);
			pdDenseIns[pos] = pdSparseInsValue[i];
			curDenseTop++;
		}
	}

}

__global__ void FillMultiDense(const float_point *pdSparseInsValue, const long long *pInsStartPos, const int *pnSpareInsFeaId,
							   const int *pNumofFeaValue, float_point *pdDenseIns, const int *pSortedUsedFea,
							   const int *pHashFeaIdToDenseInsPos, int numofUsedFea,
						  	   int startInsId, int numofInsToFill)
{
	int nGlobalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(nGlobalThreadId >= numofInsToFill)
		return;

	int insId = startInsId + nGlobalThreadId;
	long long startPos = pInsStartPos[insId];
	int numofFeaValue = pNumofFeaValue[insId];
	int denseInsStartPos = nGlobalThreadId * numofUsedFea;

	//for each value in the sparse instance
	int curDenseTop = 0;
	for(int i = 0; i < numofFeaValue; i++)
	{
		int feaId = pnSpareInsFeaId[startPos + i];

		while(feaId > pSortedUsedFea[curDenseTop])
		{
			int pos = GetBufferId(pHashFeaIdToDenseInsPos, pSortedUsedFea[curDenseTop], numofUsedFea);
			pdDenseIns[denseInsStartPos + pos] = 0;
			curDenseTop++;
		}

		if(feaId == pSortedUsedFea[curDenseTop])
		{//this is a feature needed to be stored in dense instance
			int pos = GetBufferId(pHashFeaIdToDenseInsPos, pSortedUsedFea[curDenseTop], numofUsedFea);
			pdDenseIns[denseInsStartPos + pos] = pdSparseInsValue[startPos + i];
			curDenseTop++;
		}
	}

}

