#include "hip/hip_runtime.h"
/*
 * FillDenseInsKernel.cu
 *
 *  Created on: 21 Jun 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <stdio.h>
#include "DevicePredKernel.h"
#include "../DeviceHashing.h"
#include "../ErrorChecker.h"

__global__ void FillDense(float_point *pdSparseInsValue, int *pnSpareInsFeaId, int numofFeaValue,
						  float_point *pdDenseIns, int *pSortedUsedFea, int *pHashFeaIdToDenseInsPos, int totalUsedFea)
{
	//for each value in the sparse instance
	ErrorChecker(numofFeaValue - 1, __PRETTY_FUNCTION__, "numofFeaValue <= 0");
	int curDenseTop = 0;
	for(int i = 0; i < numofFeaValue; i++)
	{
		int feaId = pnSpareInsFeaId[i];

		while(feaId > pSortedUsedFea[curDenseTop])
		{
			int pos = GetBufferId(pHashFeaIdToDenseInsPos, pSortedUsedFea[curDenseTop], totalUsedFea);
			pdDenseIns[pos] = 0;
			curDenseTop++;
		}

		if(feaId == pSortedUsedFea[curDenseTop])
		{//this is a feature needed to be stored in dense instance
			int pos = GetBufferId(pHashFeaIdToDenseInsPos, pSortedUsedFea[curDenseTop], totalUsedFea);
			pdDenseIns[pos] = pdSparseInsValue[i];
			curDenseTop++;
		}
	}

}


