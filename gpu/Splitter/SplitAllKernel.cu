#include "hip/hip_runtime.h"
/*
 * SplitAllKernel.cu
 *
 *  Created on: 15 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <string.h>
#include "DeviceSplitAllKernel.h"
#include "../Memory/gbdtGPUMemManager.h"

using std::string;

__device__ void ErrorChecker(int value, const char* functionName, const char* temp)
{
	if(value < 0)
	{
		printf("Error in %s: %s=%d\n", functionName, temp, value);
	}
}

/**
 * @brief: compute the base_weight of tree node, also determines if a node is a leaf.
 */
__global__ void ComputeWeight(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, int *pSNIdToBufferId,
								  SplitPoint *pBestSplitPoint, nodeStat *pSNodeStat, float_point rt_eps, int flag_LEAFNODE,
								  float_point lambda, int numofSplittableNode, bool bLastLevel)
{
	for(int n = 0; n < numofSplittableNode; n++)
	{
		int nid = pSplittableNode[n].nodeId;
		ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");

//		cout << "node " << nid << " needs to split..." << endl;
		int bufferPos = pSNIdToBufferId[nid];
		ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");

		//mark the node as a leaf node if (1) the gain is negative or (2) the tree reaches maximum depth.
		pAllTreeNode[nid].loss = pBestSplitPoint[bufferPos].m_fGain;
		ErrorChecker(pSNodeStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "pSNodeStat[bufferPos].sum_hess");

		float_point nodeWeight = (-pSNodeStat[bufferPos].sum_gd / (pSNodeStat[bufferPos].sum_hess + lambda));
		pAllTreeNode[nid].base_weight = nodeWeight;
		if(pBestSplitPoint[bufferPos].m_fGain <= rt_eps || bLastLevel == true)
		{
			//weight of a leaf node
			pAllTreeNode[nid].predValue = pAllTreeNode[nid].base_weight;
			pAllTreeNode[nid].rightChildId = flag_LEAFNODE;
		}
	}

}

/**
 * @brief: create new nodes and associate new nodes with their parent id
 */
__global__ void CreateNewNode(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, TreeNode *pNewSplittableNode,
								  int *pSNIdToBufferId, SplitPoint *pBestSplitPoint,
								  int *pParentId, int *pLChildId, int *pRChildId,
								  nodeStat *pLChildStat, nodeStat *pRChildStat, nodeStat *pNewNodeStat,
								  int *m_nNumofNode,
								  float_point rt_eps, int nNumofSplittableNode, bool bLastLevel)
{
	//for each splittable node, assign lchild and rchild ids
//	vector<TreeNode*> newSplittableNode;

	int numofNewNode = 0;
	for(int n = 0; n < nNumofSplittableNode; n++)
	{
		int nid = pSplittableNode[n].nodeId;
		ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");
		int bufferPos = pSNIdToBufferId[nid];
		ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");

		if(!(pBestSplitPoint[bufferPos].m_fGain <= rt_eps || bLastLevel == true))
		{
			int lchildId = *m_nNumofNode;
			int rchildId = *m_nNumofNode + 1;

			//parent id to child ids
			pParentId[bufferPos] = nid;
			pLChildId[bufferPos] = lchildId;
			pRChildId[bufferPos] = rchildId;
			ErrorChecker(pLChildStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "lchildStat[bufferPos].sum_hess");
			ErrorChecker(pRChildStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "rchildStat[bufferPos].sum_hess");

			//push left and right child statistics into a vector
			int leftNewNodeId = numofNewNode;
			int rightNewNodeId = numofNewNode + 1;
			pNewNodeStat[leftNewNodeId] = pLChildStat[bufferPos];
			pNewNodeStat[rightNewNodeId] = pRChildStat[bufferPos];
			numofNewNode += 2;

			//split into two nodes
			TreeNode &leftChild = pAllTreeNode[lchildId];
			TreeNode &rightChild = pAllTreeNode[rchildId];
			int nLevel = pAllTreeNode[nid].level;

			leftChild.nodeId = lchildId;
			leftChild.parentId = nid;
			leftChild.level = nLevel + 1;
			rightChild.nodeId = rchildId;
			rightChild.parentId = nid;
			rightChild.level = nLevel + 1;

			//they should just be pointers, not new content
			pNewSplittableNode[leftNewNodeId] = leftChild;
			pNewSplittableNode[rightNewNodeId] = rightChild;


			pAllTreeNode[nid].leftChildId = leftChild.nodeId;
			pAllTreeNode[nid].rightChildId = rightChild.nodeId;
			ErrorChecker(pBestSplitPoint[bufferPos].m_nFeatureId + 1, __PRETTY_FUNCTION__, "pBestSplitPoint[bufferPos].m_nFeatureId");

			pAllTreeNode[nid].featureId = pBestSplitPoint[bufferPos].m_nFeatureId;
			pAllTreeNode[nid].fSplitValue = pBestSplitPoint[bufferPos].m_fSplitValue;

			m_nNumofNode += 2;
		}
	}

}
