#include "hip/hip_runtime.h"
/*
 * SplitAllKernel.cu
 *
 *  Created on: 15 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <string.h>
#include "DeviceSplitAllKernel.h"
#include "../Memory/gbdtGPUMemManager.h"

using std::string;

__device__ void ErrorCond(bool bCon, const char* functionName, const char* temp)
{
	if(bCon == false)
	{
		printf("Error in %s: %s=%d\n", functionName, temp);
	}
}

__device__ void ErrorChecker(int value, const char* functionName, const char* temp)
{
	if(value < 0)
	{
		printf("Error in %s: %s=%d\n", functionName, temp, value);
	}
}

/**
 * @brief: has an identical version in host
 */
__device__ int AssignHashValue(int *pEntryToHashValue, int snid, int m_maxNumofSplittable, bool &bIsNew)
{
	bIsNew = false;//
	int buffId = -1;

	int remain = snid % m_maxNumofSplittable;//use mode operation as Hash function to find the buffer position

	//the entry has been seen before, and is found without hash conflict
	if(pEntryToHashValue[remain] == snid)
	{
		return remain;
	}

	//the entry hasn't been seen before, and its hash value is found without hash conflict
	if(pEntryToHashValue[remain] == -1)
	{
		bIsNew = true;
		buffId = remain;
		pEntryToHashValue[remain] = snid;
	}
	else//the hash value is used for other entry
	{
		//Hash conflict
		for(int i = m_maxNumofSplittable - 1; i > 0; i--)
		{
			bool hashValueFound = false;
			if(pEntryToHashValue[i] == -1)//the entry hasn't been seen before, and now is assigned a hash value.
			{
				hashValueFound = true;
				bIsNew = true;
			}
			else if(pEntryToHashValue[i] == snid)//the entry has been seen before, and now its hash value is found.
				hashValueFound = true;

			if(hashValueFound == true)
			{
				buffId = i;
				break;
			}
		}
	}

	ErrorChecker(buffId, __PRETTY_FUNCTION__, "buffId");
	return buffId;
}

/**
 * @brief: compute the base_weight of tree node, also determines if a node is a leaf.
 */
__global__ void ComputeWeight(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, int *pSNIdToBufferId,
								  SplitPoint *pBestSplitPoint, nodeStat *pSNodeStat, float_point rt_eps, int flag_LEAFNODE,
								  float_point lambda, int numofSplittableNode, bool bLastLevel)
{
	for(int n = 0; n < numofSplittableNode; n++)
	{
		int nid = pSplittableNode[n].nodeId;
		ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");

//		cout << "node " << nid << " needs to split..." << endl;
		int bufferPos = pSNIdToBufferId[nid];
		ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");

		//mark the node as a leaf node if (1) the gain is negative or (2) the tree reaches maximum depth.
		pAllTreeNode[nid].loss = pBestSplitPoint[bufferPos].m_fGain;
		ErrorChecker(pSNodeStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "pSNodeStat[bufferPos].sum_hess");

		float_point nodeWeight = (-pSNodeStat[bufferPos].sum_gd / (pSNodeStat[bufferPos].sum_hess + lambda));
		pAllTreeNode[nid].base_weight = nodeWeight;
		if(pBestSplitPoint[bufferPos].m_fGain <= rt_eps || bLastLevel == true)
		{
			//weight of a leaf node
			pAllTreeNode[nid].predValue = pAllTreeNode[nid].base_weight;
			pAllTreeNode[nid].rightChildId = flag_LEAFNODE;
		}
	}

}

/**
 * @brief: create new nodes and associate new nodes with their parent id
 */
__global__ void CreateNewNode(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, TreeNode *pNewSplittableNode,
								  int *pSNIdToBufferId, SplitPoint *pBestSplitPoint,
								  int *pParentId, int *pLChildId, int *pRChildId,
								  nodeStat *pLChildStat, nodeStat *pRChildStat, nodeStat *pNewNodeStat,
								  int *pNumofNode,
								  float_point rt_eps, int nNumofSplittableNode, bool bLastLevel)
{
	//for each splittable node, assign lchild and rchild ids
//	vector<TreeNode*> newSplittableNode;

	int numofNewNode = 0;
	for(int n = 0; n < nNumofSplittableNode; n++)
	{
		int nid = pSplittableNode[n].nodeId;
		ErrorChecker(nid, __PRETTY_FUNCTION__, "nid");
		int bufferPos = pSNIdToBufferId[nid];
		ErrorChecker(bufferPos, __PRETTY_FUNCTION__, "bufferPos");

		if(!(pBestSplitPoint[bufferPos].m_fGain <= rt_eps || bLastLevel == true))
		{
			int lchildId = *pNumofNode;
			int rchildId = *pNumofNode + 1;

			//parent id to child ids
			pParentId[bufferPos] = nid;
			pLChildId[bufferPos] = lchildId;
			pRChildId[bufferPos] = rchildId;
			ErrorChecker(pLChildStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "lchildStat[bufferPos].sum_hess");
			ErrorChecker(pRChildStat[bufferPos].sum_hess, __PRETTY_FUNCTION__, "rchildStat[bufferPos].sum_hess");

			//push left and right child statistics into a vector
			int leftNewNodeId = numofNewNode;
			int rightNewNodeId = numofNewNode + 1;
			pNewNodeStat[leftNewNodeId] = pLChildStat[bufferPos];
			pNewNodeStat[rightNewNodeId] = pRChildStat[bufferPos];
			numofNewNode += 2;

			//split into two nodes
			TreeNode &leftChild = pAllTreeNode[lchildId];
			TreeNode &rightChild = pAllTreeNode[rchildId];
			int nLevel = pAllTreeNode[nid].level;

			leftChild.nodeId = lchildId;
			leftChild.parentId = nid;
			leftChild.level = nLevel + 1;
			rightChild.nodeId = rchildId;
			rightChild.parentId = nid;
			rightChild.level = nLevel + 1;

			//they should just be pointers, not new content
			pNewSplittableNode[leftNewNodeId] = leftChild;
			pNewSplittableNode[rightNewNodeId] = rightChild;


			pAllTreeNode[nid].leftChildId = leftChild.nodeId;
			pAllTreeNode[nid].rightChildId = rightChild.nodeId;
			ErrorChecker(pBestSplitPoint[bufferPos].m_nFeatureId, __PRETTY_FUNCTION__, "pBestSplitPoint[bufferPos].m_nFeatureId");

			pAllTreeNode[nid].featureId = pBestSplitPoint[bufferPos].m_nFeatureId;
			pAllTreeNode[nid].fSplitValue = pBestSplitPoint[bufferPos].m_fSplitValue;

			//this is used in finding unique feature ids
			pSplittableNode[n].featureId = pBestSplitPoint[bufferPos].m_nFeatureId;

			*pNumofNode = (*pNumofNode + 2);
		}
	}

}

/**
 * @brief: get unique used feature ids of the splittable nodes
 */
__global__ void GetUniqueFid(TreeNode *pAllTreeNode, TreeNode *pSplittableNode, int nNumofSplittableNode,
								 int *pFeaIdToBuffId, int *pUniqueFidVec,int *pNumofUniqueFid,
								 int maxNumofUsedFea, int flag_LEAFNODE)
{
	ErrorCond(*pNumofUniqueFid == 0, __PRETTY_FUNCTION__, "*pNumofUniqueFid == 0");
	for(int n = 0; n < nNumofSplittableNode; n++)
	{
		int fid = pSplittableNode[n].featureId;
		int nid = pSplittableNode[n].nodeId;
		if(fid == -1 && pAllTreeNode[nid].rightChildId == flag_LEAFNODE)
		{//leaf node should satisfy two conditions at this step
			continue;
		}
		ErrorChecker(fid, __PRETTY_FUNCTION__, "fid");
		bool bIsNew = false;
		int hashValue = AssignHashValue(pFeaIdToBuffId, fid, maxNumofUsedFea, bIsNew);
		if(bIsNew == true)
		{
			pUniqueFidVec[*pNumofUniqueFid] = fid;
			*pNumofUniqueFid = *pNumofUniqueFid + 1;
		}
	}

	ErrorChecker(nNumofSplittableNode - *pNumofUniqueFid, __PRETTY_FUNCTION__, "nNumofSplittableNode - pNumofUniqueFid");
}
