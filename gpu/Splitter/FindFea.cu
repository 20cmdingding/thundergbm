/*
 * DeviceSplitter.cu
 *
 *  Created on: 5 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <iostream>

#include "../../pureHost/MyAssert.h"
#include "../../pureHost/SparsePred/DenseInstance.h"
#include "../Memory/gbdtGPUMemManager.h"
#include "../Memory/SplitNodeMemManager.h"
#include "DeviceSplitter.h"
#include "DeviceFindFeaKernel.h"
#include "../Preparator.h"
#include "DevicePredKernel.h"
#include "../Hashing.h"
#include "../DevicePrediction.h"

using std::cout;
using std::endl;
using std::make_pair;


/**
 * @brief: efficient best feature finder
 */
void DeviceSplitter::FeaFinderAllNode(vector<SplitPoint> &vBest, vector<nodeStat> &rchildStat, vector<nodeStat> &lchildStat)
{
	int numofSNode = vBest.size();

	GBDTGPUMemManager manager;
	//allocate reusable host memory
	manager.allocHostMemory();

	int nNumofFeature = manager.m_numofFea;
	PROCESS_ERROR(nNumofFeature > 0);

	DataPreparator preparator;
	//copy gd and hess to GPU memory
	preparator.PrepareGDHess(m_vGDPair_fixedPos);

	float_point *pGD = manager.m_pGrad;
	float_point *pHess = manager.m_pHess;

	//copy instance id to node id infomation
	PROCESS_ERROR(manager.m_numofIns == m_nodeIds.size());
	preparator.VecToArray(m_nodeIds, manager.m_pInsToNodeIdHost);
	manager.MemcpyHostToDevice(manager.m_pInsToNodeIdHost, manager.m_pInsIdToNodeId, sizeof(int) * manager.m_numofIns);

	//copy splittable node information and buffer ids to GPU memory
	preparator.PrepareSNodeInfo(mapNodeIdToBufferPos, m_nodeStat);
	nodeStat *pSNodeState = manager.m_pSNodeStat;

	//use short names for temporary info
	nodeStat *pTempRChildStat = manager.m_pTempRChildStat;
	float_point *pLastValue = manager.m_pLastValue;

	//use short names for instance info
	int *pInsId = manager.m_pDInsId;
	float_point *pFeaValue = manager.m_pdDFeaValue;
	int *pNumofKeyValue = manager.m_pDNumofKeyValue;

	//reset the best splittable points
	int maxNumofSplittable = manager.m_maxNumofSplittable;
	manager.MemcpyHostToDevice(manager.m_pBestPointHost, manager.m_pBestSplitPoint, sizeof(SplitPoint) * maxNumofSplittable);

	for(int f = 0; f < nNumofFeature; f++)
	{
		//the number of key values of the f{th} feature
		int numofCurFeaKeyValues = 0;
		manager.MemcpyDeviceToHost(pNumofKeyValue + f, &numofCurFeaKeyValues, sizeof(int));
		PROCESS_ERROR(numofCurFeaKeyValues > 0);

		long long startPosOfPrevFea = 0;
		int numofPreFeaKeyValues = 0;
		if(f > 0)
		{
			//number of key values of the previous feature
			manager.MemcpyDeviceToHost(pNumofKeyValue + (f - 1), &numofPreFeaKeyValues, sizeof(int));
			PROCESS_ERROR(numofPreFeaKeyValues > 0);
			//copy value of the start position of the previous feature
			manager.MemcpyDeviceToHost(manager.m_pFeaStartPos + (f - 1), &startPosOfPrevFea, sizeof(long long));
		}
		PROCESS_ERROR(startPosOfPrevFea >= 0);
		long long startPosOfCurFea = startPosOfPrevFea + numofPreFeaKeyValues;
		//copy the value of the start position of the current feature
//		manager.MemcpyHostToDevice(&startPosOfCurFea, manager.m_pFeaStartPos + f, sizeof(long long));

		//reset the temporary right child statistics
		checkCudaErrors(hipMemset(pTempRChildStat, 0, sizeof(nodeStat) * maxNumofSplittable));


		//find the split value for this feature
		int *idStartAddress = pInsId + startPosOfCurFea;
		float_point *pValueStartAddress = pFeaValue + startPosOfCurFea;

		FindFeaSplitValue<<<1, 1>>>(numofCurFeaKeyValues, idStartAddress, pValueStartAddress, manager.m_pInsIdToNodeId,
									pTempRChildStat, pGD, pHess, pLastValue, pSNodeState, manager.m_pBestSplitPoint,
									manager.m_pRChildStat, manager.m_pLChildStat, manager.m_pSNIdToBuffId,
									manager.m_maxNumofSplittable, f, manager.m_pBuffIdVec, numofSNode, DeviceSplitter::m_labda);
		hipDeviceSynchronize();


		//copy back the best split points to vectors
		preparator.CopyBestSplitPoint(mapNodeIdToBufferPos, vBest, rchildStat, lchildStat);
	}

	preparator.ReleaseMem();
}

/**
 * @brief: prediction and compute gradient descent
 */
void DeviceSplitter::ComputeGD(vector<RegTree> &vTree)
{
	GBDTGPUMemManager manager;
	SNGPUManager snManager;

	vector<double> v_fPredValue;

//	pred.PredictSparseIns(m_vvInsSparse, vTree, v_fPredValue, m_vPredBuffer);

	//get features and store the feature ids in a way that the access is efficient
	DenseInsConverter denseInsConverter(vTree);

	//hash feature id to position id
	int numofUsedFea = denseInsConverter.usedFeaSet.size();
	int *pHashUsedFea = NULL;
	int *pSortedUsedFea = NULL;
	if(numofUsedFea > 0)
	{
		pHashUsedFea = new int[numofUsedFea];
		memset(pHashUsedFea, -1, sizeof(int) * numofUsedFea);
		for(int uf = 0; uf < numofUsedFea; uf++)
		{
			bool bIsNewHashValue = false;
			int hashValue = Hashing::HostAssignHashValue(pHashUsedFea, denseInsConverter.usedFeaSet[uf], numofUsedFea, bIsNewHashValue);
			#ifdef _DEBUG
//			cout << "hash value of " << denseInsConverter.usedFeaSet[uf] << " is " << hashValue << endl;
			#endif
		}

		pSortedUsedFea = new int[numofUsedFea];
		for(int uf = 0; uf < numofUsedFea; uf++)
			pSortedUsedFea[uf] = denseInsConverter.usedFeaSet[uf];

		//copy hash map to gpu memory
		manager.MemcpyHostToDevice(pHashUsedFea, manager.m_pHashFeaIdToDenseInsPos, sizeof(int) * numofUsedFea);
		manager.MemcpyHostToDevice(pSortedUsedFea, manager.m_pSortedUsedFeaId, sizeof(int) * numofUsedFea);
	}

	//for each tree
	int nNumofTree = vTree.size();
	int nNumofIns = manager.m_numofIns;
	PROCESS_ERROR(nNumofIns > 0);

	//copy tree from GPU memory
	if(nNumofTree - 1 >= 0)
	{
		int numofNode = 0;
		manager.MemcpyDeviceToHost(snManager.m_pCurNumofNode, &numofNode, sizeof(int));
		TreeNode *pAllNode = new TreeNode[numofNode];
		manager.MemcpyDeviceToHost(snManager.m_pTreeNode, pAllNode, sizeof(TreeNode) * numofNode);

		#ifdef _DEBUG
		cout << numofNode << " v.s. " << vTree[nNumofTree - 1].nodes.size() << endl;
		//compare each node
		for(int n = 0; n < numofNode; n++)
		{
			if(!(pAllNode[n].nodeId == vTree[nNumofTree - 1].nodes[n]->nodeId
			   && pAllNode[n].featureId == vTree[nNumofTree - 1].nodes[n]->featureId
			   && pAllNode[n].fSplitValue == vTree[nNumofTree - 1].nodes[n]->fSplitValue))
			{
				cout << "node id: " << pAllNode[n].nodeId << " v.s. " << vTree[nNumofTree - 1].nodes[n]->nodeId
					 <<	"; feat id: " << pAllNode[n].featureId << " v.s. " << vTree[nNumofTree - 1].nodes[n]->featureId
					 << "; sp value: " << pAllNode[n].fSplitValue << " v.s. " << vTree[nNumofTree - 1].nodes[n]->fSplitValue
					 << "; rc id: " << pAllNode[n].rightChildId << " v.s. " << vTree[nNumofTree - 1].nodes[n]->rightChildId << endl;
			}
		}
		#endif
	}

	for(int i = 0; i < nNumofIns; i++)
	{
		double fValue = 0;
		checkCudaErrors(hipMemset(manager.m_pTargetValue, 0, sizeof(float_point)));
		manager.MemcpyDeviceToHost(manager.m_pPredBuffer + i, &fValue, sizeof(float_point));

		//start prediction ###############

		vector<double> vDense;
		if(nNumofTree > 0)
		{
			long long startPos = -1;
			long long *pInsStartPos = manager.m_pInsStartPos + (long long)i;
			manager.MemcpyDeviceToHost(pInsStartPos, &startPos, sizeof(long long));
//			cout << "start pos ins" << i << "=" << startPos << endl;
			float_point *pDevInsValue = manager.m_pdDInsValue + startPos;
			int *pDevFeaId = manager.m_pDFeaId + startPos;
			int numofFeaValue = -1;
			int *pNumofFea = manager.m_pDNumofFea + i;
			manager.MemcpyDeviceToHost(pNumofFea, &numofFeaValue, sizeof(int));

			checkCudaErrors(hipMemset(manager.m_pdDenseIns, 0, sizeof(float_point) * numofUsedFea));
			FillDense<<<1, 1>>>(pDevInsValue, pDevFeaId, numofFeaValue, manager.m_pdDenseIns,
								manager.m_pSortedUsedFeaId, manager.m_pHashFeaIdToDenseInsPos, numofUsedFea);

			//construct dense instance #### now for testing
			denseInsConverter.SparseToDense(vv_insDebug[i], vDense);
			//denseInsConverter.PrintDenseVec(vDense);

			//copy the dense instance to vector for testing
			float_point *pDense = new float_point[numofUsedFea];
			manager.MemcpyDeviceToHost(manager.m_pdDenseIns, pDense, sizeof(float_point) * numofUsedFea);

			bool bDiff = false;
			for(int i = 0; i < numofUsedFea; i++)
			{

				int pos = Hashing::HostGetBufferId(pHashUsedFea, pSortedUsedFea[i], numofUsedFea);
				if(vDense[i] != pDense[pos])
				{
					cout << "different: " << vDense[i] << " v.s. " << pDense[pos] << "\t";
					bDiff = true;
				}

				if(bDiff == true && (i == manager.m_numofFea - 1 || i == vDense.size() - 1))
					cout << endl;

				//vDense.push_back(pDense[i]);
			}
			////////end for testing
		}

		//prediction using the last tree
		if(nNumofTree - 1 >= 0)
		{
			int numofNode = 0;
			manager.MemcpyDeviceToHost(snManager.m_pCurNumofNode, &numofNode, sizeof(int));
			PredTarget<<<1, 1>>>(snManager.m_pTreeNode, numofNode, manager.m_pdDenseIns, numofUsedFea,
								 manager.m_pHashFeaIdToDenseInsPos, manager.m_pTargetValue);

			#ifdef _DEBUG
			//host prediction
			for(int t = nNumofTree - 1; t >= 0 && t < nNumofTree; t++)
			{
				int nodeId = vTree[t].GetLeafIdSparseInstance(vDense, denseInsConverter.fidToDensePos);
				fValue += vTree[t][nodeId]->predValue;
			}

			float_point fTarget = 0;
			manager.MemcpyDeviceToHost(manager.m_pTargetValue, &fTarget, sizeof(float_point));
			if(fValue != fTarget)
				cout << "Target value diff " << fValue << " v.s. " << fTarget << endl;
			#endif

		}

		v_fPredValue.push_back(fValue);
		manager.MemcpyDeviceToDevice(manager.m_pTargetValue, manager.m_pPredBuffer + i, sizeof(float_point));
	}

	if(pHashUsedFea == NULL)
		delete []pHashUsedFea;
	if(pSortedUsedFea == NULL)
		delete []pSortedUsedFea;

//	ComputeGDSparse(v_fPredValue, m_vTrueValue);
	//compute GD
	int nTotal = nNumofIns;
	for(int i = 0; i < nTotal; i++)
	{
		float_point fTrueValue = 0;
		manager.MemcpyDeviceToHost(manager.m_pdTrueTargetValue + i, &fTrueValue, sizeof(float_point));
		m_vGDPair_fixedPos[i].grad = v_fPredValue[i] - fTrueValue;
		m_vGDPair_fixedPos[i].hess = 1;
	}

	nodeStat rootStat;
	for(int i = 0; i < nTotal; i++)
	{
		rootStat.sum_gd += m_vGDPair_fixedPos[i].grad;
		rootStat.sum_hess += m_vGDPair_fixedPos[i].hess;
	}


	m_nodeStat.clear();
	m_nodeStat.push_back(rootStat);
	mapNodeIdToBufferPos.insert(make_pair(0,0));//node0 in pos0 of buffer
}

