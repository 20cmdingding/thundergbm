/*
 * gbdtGPUMemManager.cu
 *
 *  Created on: 4 May 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <hip/hip_runtime_api.h>

#include "gbdtGPUMemManager.h"
#include "../pureHost/MyAssert.h"

/**
 * @brief: constructor
 */
GBDTGPUMemManager::GBDTGPUMemManager()
{
	totalNumofValues = -1;
	pDInsId = NULL;			//all the instance ids
	pdDFeaValue = NULL;		//all the feature values
	pDNumofKeyValue = NULL;	//the number of key-value pairs of each feature
}

/**
 * @brief: allocate memory for instances
 */
void GBDTGPUMemManager::allocMemForIns(int nTotalNumofValue, int numofFeature)
{
	PROCESS_ERROR(nTotalNumofValue > 0);
	PROCESS_ERROR(numofFeature > 0);
	totalNumofValues = nTotalNumofValue;
	checkCudaErrors(hipMalloc((void**)&pDInsId, sizeof(int) * totalNumofValues));
	checkCudaErrors(hipMalloc((void**)&pdDFeaValue, sizeof(float_point) * totalNumofValues));
	checkCudaErrors(hipMalloc((void**)&pDNumofKeyValue, sizeof(int) * numofFeature));
}

