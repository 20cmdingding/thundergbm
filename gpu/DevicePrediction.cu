#include "hip/hip_runtime.h"
/*
 * DevicePrediction.cu
 *
 *  Created on: 23 Jun 2016
 *      Author: Zeyi Wen
 *		@brief: 
 */

#include <stdio.h>
#include "../DeviceHost/DefineConst.h"
#include "ErrorChecker.h"
#include "DeviceHashing.h"
#include "../DeviceHost/TreeNode.h"

__device__ int GetNext(TreeNode *pNode, float_point feaValue)
{
    if(feaValue < pNode->fSplitValue)
    {
      return pNode->leftChildId;
    }
    else
    {
      return pNode->rightChildId;
    }
}

__global__ void PredTarget(TreeNode *pAllTreeNode, int totalNode, float_point *pDenseIns, int nNumofFea,
									   int *pnHashFeaIdToPos, float_point *pdTargetValue)
{
	int pid = 0; //node id
	TreeNode *curNode = &pAllTreeNode[pid];
	while(curNode->featureId != -1)//!curNode->isLeaf()
	{
		int fid = curNode->featureId;
		ErrorChecker(fid, __PRETTY_FUNCTION__, "fid < 0");

		int maxNumofUsedFea = nNumofFea;
		int pos = GetBufferId(pnHashFeaIdToPos, fid, maxNumofUsedFea);
//		printf("%d hash to %d: fea v=%f\n", fid, pos, pDenseIns[pos]);

		if(pos < nNumofFea)//feature value is available in the dense vector
			pid = GetNext(curNode, pDenseIns[pos]);
		else//feature value is stored in the dense vector (due to truncating)
			pid = GetNext(curNode, 0);
		curNode = &pAllTreeNode[pid];
	}

	pdTargetValue[0] = pAllTreeNode[pid].predValue;
}



