#include "hip/hip_runtime.h"
/*
 * GBDTMain.cpp
 *
 *  Created on: 6 Jan 2016
 *      Author: Zeyi Wen
 *		@brief: project main function
 */

#include <math.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "Host/DataReader/LibsvmReaderSparse.h"
#include "Host/HostTrainer.h"
#include "Host/Evaluation/RMSE.h"
#include "DeviceHost/MyAssert.h"

#include "Device/Memory/gbdtGPUMemManager.h"
#include "Device/Memory/SplitNodeMemManager.h"
#include "Device/Memory/dtMemManager.h"
#include "Device/DeviceTrainer.h"
#include "Device/DevicePredictor.h"
#include "Device/initCuda.h"
#include "Host/PureHostGBDTMain.h"


#include "Device/prefix-sum/prefixSum.h"

int main(int argc, char *argv[])
{
	TestPrefixSum(argc, argv);
	return 1;
	string strFileName = "data/abalone.txt";

//	mainPureHost(strFileName);
//	return 1;

	hipCtx_t context;
	if(!InitCUDA('T', context))
	{
		cerr << "cannot initialise GPU" << endl;
		return 0;
	}

	clock_t begin_whole, end_whole;
	/********* read training instances from a file **************/
	int maxNumofSplittableNode = 100;
	int maxNumofUsedFeature = 1000;
	int maxNumofDenseIns = 1;//###### is later set to the number of instances
	int maxUsedFeaInTrees = 1000;

	//for training
	int nNumofTree = 2;
	int nMaxDepth = 5;
	double fLabda = 1;//this one is constant in xgboost
	double fGamma = 1;//minimum loss

	DevicePredictor pred;

	DeviceSplitter splitter;
	DeviceTrainer trainer(&splitter);

	cout << "reading data..." << endl;
	LibSVMDataReader dataReader;
	int nNumofFeatures;
	int nNumofExamples;
	long long nNumofValue;
	dataReader.GetDataInfo(strFileName, nNumofFeatures, nNumofExamples, nNumofValue);

	vector<float_point> v_fLabel;
	vector<vector<KeyValue> > v_vInsSparse;
	dataReader.ReadLibSVMFormatSparse(v_vInsSparse, v_fLabel, strFileName, nNumofFeatures, nNumofExamples);
	cout << "data has " << nNumofFeatures << " features and " << nNumofExamples << " instances" << endl;

	//allocate memory for trees
	DTGPUMemManager treeMemManager;
	int maxNumofNodePerTree = pow(2, nMaxDepth + 1) - 1;
	treeMemManager.allocMemForTrees(nNumofTree, maxNumofNodePerTree, nMaxDepth);


	//initialise gpu memory allocator
	GBDTGPUMemManager memAllocator;
	PROCESS_ERROR(nNumofValue > 0);
	memAllocator.m_totalNumofValues = nNumofValue;
	memAllocator.maxNumofDenseIns = nNumofExamples;
	memAllocator.m_maxUsedFeaInTrees = maxUsedFeaInTrees;

	//allocate memory for instances
	memAllocator.allocMemForIns(nNumofValue, nNumofExamples, nNumofFeatures);
	memAllocator.allocMemForSplittableNode(maxNumofSplittableNode);//use in find features (i.e. best split points) process
	memAllocator.allocHostMemory();//allocate reusable host memory
	//allocate numofFeature*numofSplittabeNode
	memAllocator.allocMemForSNForEachThread(nNumofFeatures, maxNumofSplittableNode);

	SNGPUManager snManger;
	snManger.allocMemForTree(maxNumofNodePerTree);//reserve memory for the tree
	snManger.allocMemForParenChildIdMapping(maxNumofSplittableNode);
	snManger.allocMemForNewNode(maxNumofSplittableNode);
	snManger.allocMemForUsedFea(maxNumofUsedFeature);//use in splitting all nodes process

	begin_whole = clock();
	cout << "start training..." << endl;
	/********* run the GBDT learning process ******************/

	trainer.m_vvInsSparse = v_vInsSparse;
	trainer.m_vTrueValue = v_fLabel;

	Pruner::min_loss = fGamma;

	clock_t start_init = clock();
	trainer.InitTrainer(nNumofTree, nMaxDepth, fLabda, fGamma, nNumofFeatures);
	clock_t end_init = clock();

	//store feature key-value into array
	int *pInsId = new int[memAllocator.m_totalNumofValues];
	float_point *pdValue = new float_point[memAllocator.m_totalNumofValues];
	int *pNumofKeyValue = new int[nNumofFeatures];
	long long *plFeaStartPos = new long long[nNumofFeatures];//get start position of each feature

	KeyValue::VecToArray(trainer.splitter->m_vvFeaInxPair, pInsId, pdValue, pNumofKeyValue, plFeaStartPos);
	KeyValue::TestVecToArray(trainer.splitter->m_vvFeaInxPair, pInsId, pdValue, pNumofKeyValue);

	//copy feature key-value to device memory
	memAllocator.MemcpyHostToDevice(pInsId, memAllocator.m_pDInsId, nNumofValue * sizeof(int));
	memAllocator.MemcpyHostToDevice(pdValue, memAllocator.m_pdDFeaValue, nNumofValue * sizeof(float_point));
	memAllocator.MemcpyHostToDevice(pNumofKeyValue, memAllocator.m_pDNumofKeyValue, nNumofFeatures * sizeof(int));
	memAllocator.MemcpyHostToDevice(plFeaStartPos, memAllocator.m_pFeaStartPos, nNumofFeatures * sizeof(long long));

	memAllocator.TestMemcpyDeviceToHost();
	memAllocator.TestMemcpyDeviceToDevice();
	memAllocator.TestMemcpyHostToDevice(pInsId, memAllocator.m_pDInsId, nNumofValue * sizeof(int));
	memAllocator.TestMemcpyHostToDevice(pdValue, memAllocator.m_pdDFeaValue, nNumofValue * sizeof(float_point));
	memAllocator.TestMemcpyHostToDevice(pNumofKeyValue, memAllocator.m_pDNumofKeyValue, nNumofFeatures * sizeof(int));

	//store sparse instances to GPU memory
	int *pFeaId = new int[nNumofValue];
	float_point *pdFeaValue = new float_point[nNumofValue];
	int *pNumofFea = new int[nNumofExamples];
	long long *plInsStartPos = new long long[nNumofExamples];
	KeyValue::VecToArray(trainer.m_vvInsSparse, pFeaId, pdFeaValue, pNumofFea, plInsStartPos);
	KeyValue::TestVecToArray(trainer.m_vvInsSparse, pFeaId, pdFeaValue, pNumofFea);

	//copy instance key-value to device memory
	memAllocator.MemcpyHostToDevice(pFeaId, memAllocator.m_pDFeaId, nNumofValue * sizeof(int));
	memAllocator.MemcpyHostToDevice(pdFeaValue, memAllocator.m_pdDInsValue, nNumofValue * sizeof(float_point));
	memAllocator.MemcpyHostToDevice(pNumofFea, memAllocator.m_pDNumofFea, nNumofExamples * sizeof(int));
	memAllocator.MemcpyHostToDevice(plInsStartPos, memAllocator.m_pInsStartPos, nNumofExamples * sizeof(long long));

	//free host memory
	delete []pInsId;
	delete []pdValue;
	delete []pNumofKeyValue;
	delete []plFeaStartPos;
	delete []pFeaId;
	delete []pdFeaValue;
	delete []pNumofFea;
	delete []plInsStartPos;

	float_point *pTrueLabel = new float_point[nNumofExamples];
	for(int i = 0; i < nNumofExamples; i++)
	{
		pTrueLabel[i] = v_fLabel[i];
	}
	//copy true labels to gpu memory
	memAllocator.MemcpyHostToDevice(pTrueLabel, memAllocator.m_pdTrueTargetValue, nNumofExamples * sizeof(float_point));
	delete[] pTrueLabel;

	//training trees
	vector<RegTree> v_Tree;
	clock_t start_train_time = clock();
	trainer.TrainGBDT(v_Tree);
	clock_t end_train_time = clock();

	//save the trees to a file
	end_whole = clock();
	cout << "saved to file" << endl;
	trainer.SaveModel("tree.txt", v_Tree);

	double total_init = (double(end_init - start_init) / CLOCKS_PER_SEC);
	cout << "total init time = " << total_init << endl;
	double total_train = (double(end_train_time - start_train_time) / CLOCKS_PER_SEC);
	cout << "total training time = " << total_train << endl;
	double total_all = (double(end_whole - begin_whole) / CLOCKS_PER_SEC);
	cout << "all sec = " << total_all << endl;

	//read testing instances from a file


	//run the GBDT prediction process
	clock_t begin_pre, end_pre;
	vector<float_point> v_fPredValue;

	begin_pre = clock();
	pred.PredictSparseIns(v_vInsSparse, v_Tree, v_fPredValue);
	end_pre = clock();
	double prediction_time = (double(end_pre - begin_pre) / CLOCKS_PER_SEC);
	cout << "prediction sec = " << prediction_time << endl;

	EvalRMSE rmse;
	float fRMSE = rmse.Eval(v_fPredValue, v_fLabel);
	cout << "rmse=" << fRMSE << endl;

	trainer.ReleaseTree(v_Tree);
	memAllocator.releaseHostMemory();
	memAllocator.freeMemForSNForEachThread();

	ReleaseCuda(context);

	return 0;
}


