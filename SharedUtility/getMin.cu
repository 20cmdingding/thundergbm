#include "hip/hip_runtime.h"
#include <stdio.h>
#include "getMin.h"


__device__ int getBlockMin(const float *values, int *index) {
	CONCHECKER(blockDim.x % 32 == 0);
	int tid = threadIdx.x;
	index[tid] = tid;
	__syncthreads();
	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if (tid < offset) {
			if (values[index[tid + offset]] < values[index[tid]]) {
				index[tid] = index[tid + offset];
			}
		}
		__syncthreads();
	}
	return index[0];
}

__device__ void GetMinValueOriginal(real *pfValues)
{
	CONCHECKER(blockDim.x % 32 == 0);
	//Reduce by a factor of 2, and minimize step size
	for (int i = blockDim.x / 2; i > 0 ; i >>= 1) {
		int tid = threadIdx.x;
		if (tid < i)
			if (pfValues[tid + i] < pfValues[tid])
				pfValues[tid] = pfValues[tid + i];
        __syncthreads();
	}
}
