/*
 * memVector.cu
 *
 *  Created on: Aug 11, 2017
 *      Author: zeyi
 */

#include <hip/hip_runtime_api.h>
#include "memVector.h"

void MemVector::reserveSpace(uint newSize, uint numByteEachValue){
	if(addr != NULL){
		checkCudaErrors(hipFree(addr));
	}
	size = newSize;
	reservedSize = newSize * 1.5;
	checkCudaErrors(hipMalloc((void**)&addr, numByteEachValue * reservedSize));
}
