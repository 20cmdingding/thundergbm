/*
 * memVector.cu
 *
 *  Created on: Aug 11, 2017
 *      Author: zeyi
 */

#include <hip/hip_runtime_api.h>
#include "memVector.h"

void MemVector::reserveSpace(uint newSize, uint numByteEachValue){
	if(addr != NULL){
		checkCudaErrors(hipFree(addr));
	}
	size = newSize;
	if(newSize * numByteEachValue > 4 * 1024 * 1024 * 1024)//larger than 1GB
		reservedSize = newSize;
	else
		reservedSize = newSize * 2;
	checkCudaErrors(hipMalloc((void**)&addr, numByteEachValue * reservedSize));
}
