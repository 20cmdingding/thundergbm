/*
 * memVector.cu
 *
 *  Created on: Aug 11, 2017
 *      Author: zeyi
 */

#include <hip/hip_runtime_api.h>
#include "memVector.h"

void MemVector::reserveSpace(uint newSize, uint numByteEachValue){
	if(addr != NULL){
		checkCudaErrors(hipFree(addr));
	}
	size = newSize;
	if(newSize * numByteEachValue > (uint)8 * 1024 * 1024 * (uint)1024)//larger than 2GB
		reservedSize = newSize;
	else
		reservedSize = newSize * 1;
	checkCudaErrors(hipMalloc((void**)&addr, numByteEachValue * reservedSize));
}
