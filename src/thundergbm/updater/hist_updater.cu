#include "hip/hip_runtime.h"

#include <thundergbm/updater/hist_updater.h>

#include "thundergbm/updater/hist_updater.h"
#include "thundergbm/util/cub_wrapper.h"

void HistUpdater::init_cut(const vector<std::shared_ptr<SparseColumns>> &v_columns, InsStat &stats, int n_instances) {
    LOG(TRACE) << "init cut";
    if (!do_cut) {
        v_cut.resize(n_devices);
        for (int i = 0; i < n_devices; i++)
            v_cut[i].get_cut_points(*v_columns[i], stats, max_num_bin, n_instances, i);
        bin_id.resize(n_devices);
        cub_seg_sort_by_key(v_columns[0]->csc_row_idx, v_columns[0]->csc_val, v_columns[0]->csc_col_ptr, true);
        DO_ON_MULTI_DEVICES(n_devices, [&](int device_id) {
            get_bin_ids(*v_columns[device_id]);
        });
    }
//    LOG(INFO) << v_cut[0].cut_points;
//    LOG(INFO) << v_cut[0].cut_points_val;
//    LOG(INFO) << v_cut[0].cut_row_ptr;
//    LOG(INFO) << v_columns[0]->csc_val;
//    LOG(INFO) << *bin_id[0];
    do_cut = 1;
}

void HistUpdater::get_bin_ids(const SparseColumns &columns) {
    using namespace thrust;
    int cur_device;
    hipGetDevice(&cur_device);
    int n_column = columns.n_column;
    int nnz = columns.nnz;
    auto cut_row_ptr = v_cut[cur_device].cut_row_ptr.device_data();
    auto cut_points_ptr = v_cut[cur_device].cut_points_val.device_data();
    auto csc_val_data = columns.csc_val.device_data();
    auto csc_col_data = columns.csc_col_ptr.device_data();
    bin_id[cur_device].reset(new SyncArray<int>(nnz));
    auto bin_id_ptr = (*bin_id[cur_device]).device_data();
    device_loop(n_column, [=]__device__(int cid) {
        auto cutbegin = cut_points_ptr + cut_row_ptr[cid];
        auto cutend = cut_points_ptr + cut_row_ptr[cid + 1];
        auto valbeign = csc_val_data + csc_col_data[cid];
        auto valend = csc_val_data + csc_col_data[cid + 1];
        lower_bound(cuda::par, cutbegin, cutend, valbeign, valend,
                    bin_id_ptr + csc_col_data[cid], thrust::greater<float_type>());
//        for_each(cuda::par, bin_id_ptr + csc_col_data[cid],
//                 bin_id_ptr + csc_col_data[cid + 1], thrust::placeholders::_1 += cut_row_ptr[cid]);
    });
}

__global__ void
hist_kernel(GHPair *hist_data, int fea_offset, const int *bid, int bin_id_len, int n_fea_bin, int n_bins,
            const int *iid, const GHPair *gh, const int *nid, int nid_offset, int n_nodes_in_level) {
    //n_nodes_in_level * n_fea_bin
    extern __shared__ GHPair local_hist[];
    for (int i = threadIdx.x; i < n_fea_bin * n_nodes_in_level; i += blockDim.x) {
        local_hist[i] = 0;
    }
    __syncthreads();

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < bin_id_len; i += blockDim.x * gridDim.x) {
        int ins_id = iid[i];
        int node_id0 = nid[ins_id] - nid_offset;
        if (node_id0 < 0) return;
        int bin_id = bid[i];
        GHPair &dest = local_hist[node_id0 * n_fea_bin + bin_id];
        const GHPair &src = gh[ins_id];
        atomicAdd(&dest.g, src.g);
        atomicAdd(&dest.h, src.h);
    }
    __syncthreads();
    for (int i = threadIdx.x; i < n_fea_bin * n_nodes_in_level; i += blockDim.x) {
        int node_id0 = i / n_fea_bin;
        int bin_id = i % n_fea_bin;
        GHPair &dest = hist_data[node_id0 * n_bins + fea_offset + bin_id];
        GHPair &src = local_hist[i];
        atomicAdd(&dest.g, src.g);
        atomicAdd(&dest.h, src.h);
    }
}

void HistUpdater::find_split(int level, const SparseColumns &columns, const Tree &tree, const InsStat &stats,
                             const HistCut &cut,
                             SyncArray<SplitPoint> &sp) {
    int n_nodes_in_level = static_cast<int>(pow(2, level));
    int nid_offset = static_cast<int>(pow(2, level) - 1);
    int n_column = columns.n_column;
    int n_partition = n_column * n_nodes_in_level;
    int nnz = columns.nnz;
    int n_bins = cut.cut_points.size();
    int n_block = std::min((nnz / n_column - 1) / 256 + 1, 32 * 56);
    int n_max_nodes = 2 << this->depth;
    int n_max_splits = n_max_nodes * n_bins;
    int n_split = n_nodes_in_level * n_bins;

    LOG(TRACE) << "start finding split";

    //find the best split locally
    {
        using namespace thrust;

        //calculate split information for each split
        SyncArray<GHPair> hist(n_max_splits);
        SyncArray<GHPair> missing_gh(n_partition);
        auto cut_fid_data = cut.cut_fid.device_data();
        auto i2fid = [=] __device__(int i) {
            return cut_fid_data[i % n_bins];
        };
        auto hist_fid = make_transform_iterator(counting_iterator<int>(0), i2fid);
        {
            {
                TIMED_SCOPE(timerObj, "histogram");
                //input
                auto *nid_data = stats.nid.device_data();
                auto hist_data = hist.device_data();
                auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
                auto iid_data = columns.csc_row_idx.device_data();
                auto gh_data = stats.gh_pair.device_data();
                auto bin_id_data = bin_id[0]->device_data();

                {
//                    TIMED_SCOPE(timerOBj, "hist");
//                    device_loop_2d(n_column, columns.csc_col_ptr.device_data(), [=]__device__(int fid, int i) {
//                        int iid = iid_data[i];
//                        int nid0 = nid_data[iid] - nid_offset;
//                        if (nid0 < 0) return;
//                        int hist_offset = nid0 * n_bins;
//                        int feature_offset = cut_row_ptr_data[fid];
//                        int bin_id = bin_id_data[i];
//                        GHPair &dest = hist_data[hist_offset + feature_offset + bin_id];
//                        const GHPair &src = gh_data[iid];
//                        //TODO use shared memory
//                        atomicAdd(&dest.g, src.g);
//                        atomicAdd(&dest.h, src.h);
//                    }, n_block);
                }
                SyncArray<int> node_idx(stats.n_instances);
                SyncArray<int> node_ptr(n_nodes_in_level + 1);
                {
                    TIMED_SCOPE(timerObj, "gather node idx");
                    SyncArray<unsigned char> nid4sort(stats.n_instances);
                    nid4sort.copy_from(stats.nid);
                    sequence(cuda::par, node_idx.device_data(), node_idx.device_end(), 0);
                    cub_sort_by_key(nid4sort, node_idx);
                    auto counting_iter = make_counting_iterator < int > (nid_offset);
                    node_ptr.host_data()[0] = lower_bound(cuda::par, nid4sort.device_data(), nid4sort.device_end(), nid_offset) - nid4sort.device_data();
                    upper_bound(cuda::par, nid4sort.device_data(), nid4sort.device_end(), counting_iter,
                                counting_iter + n_nodes_in_level, node_ptr.device_data() + 1);
//                    LOG(INFO)<<nid4sort;
                }
//                LOG(INFO)<<node_idx;
//                LOG(INFO)<<node_ptr;
                {
                    TIMED_SCOPE(timerObj, "hist3");
                    for (int nid0 = 0; nid0 < n_nodes_in_level; ++nid0) {
                        auto idx_begin = node_ptr.host_data()[nid0];
                        auto idx_end = node_ptr.host_data()[nid0 + 1];
                        auto node_idx_data = node_idx.device_data();

                        auto hist_data = hist.device_data() + nid0 * n_bins;
                        auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
                        auto gh_data = stats.gh_pair.device_data();
                        auto dense_bin_id_data = dense_bin_id.device_data();
                        auto max_num_bin = this->max_num_bin;

                        device_loop((idx_end - idx_begin) * n_column, [=]__device__(int i) {
                            int iid = node_idx_data[i / n_column + idx_begin];
                            int fid = i % n_column;
                            unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                            if (bid != max_num_bin) {
                                int feature_offset = cut_row_ptr_data[fid];
                                const GHPair src = gh_data[iid];
                                GHPair &dest = hist_data[feature_offset + bid];
                                //TODO use shared memory
                                atomicAdd(&dest.g, src.g);
                                atomicAdd(&dest.h, src.h);
                            }
                        });
                        PERFORMANCE_CHECKPOINT(timerObj);
                    }
                }
                {
//                    SyncArray<GHPair> hist(n_max_splits);
//                    TIMED_SCOPE(timerObj, "hist2");
//                    auto nid_data = stats.nid.device_data();
//                    auto hist_data = hist.device_data();
//                    auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
//                    auto gh_data = stats.gh_pair.device_data();
//                    auto dense_bin_id_data = dense_bin_id.device_data();
//                    auto max_num_bin = this->max_num_bin;
//                    device_loop(stats.n_instances * n_column, [=]__device__(int i) {
//                        unsigned char bid = dense_bin_id_data[i];
//                        if (bid != max_num_bin) {
//                            int iid = i / n_column;
//                            int fid = i % n_column;
//                            int nid0 = nid_data[iid] - nid_offset;
//                            if (nid0 < 0) return;
//                            int hist_offset = nid0 * n_bins;
//                            int feature_offset = cut_row_ptr_data[fid];
//                            GHPair &dest = hist_data[hist_offset + feature_offset + bid];
//                            const GHPair &src = gh_data[iid];
//                            //TODO use shared memory
//                            atomicAdd(&dest.g, src.g);
//                            atomicAdd(&dest.h, src.h);
//                        }
//                    });
//                    LOG(INFO)<<hist;
//                    LOG(INFO)<<hist2;
//                    for (int i = 0; i < n_max_splits; ++i) {
//                        GHPair gh1 = hist.host_data()[i];
//                        GHPair gh2 = hist2.host_data()[i];
//                        CHECK_EQ(gh1.g, gh2.g);
//                        CHECK_EQ(gh1.h, gh2.h);
//                    }
                }
                LOG(DEBUG) << "feature offset = " << cut.cut_row_ptr;
                LOG(DEBUG) << "hist old = " << hist;
//                {
//                    TIMED_SCOPE(timerOBj, "hist");
//                    //for each feature
//                    //construct hist[node][bin]
//                    //multi block, each block has a local histogram
//                    //shared memory size = (4+4)Bytes * #node * #bin
//                    //syncthreads
//                    //sum local histogram in thread0 to global memory
//                    for (int fid = 0; fid < n_column; ++fid) {
//                        auto feature_start = columns.csc_col_ptr.host_data()[fid];
//                        auto feature_len = columns.csc_col_ptr.host_data()[fid + 1] - feature_start;
//                        const int *iid_data = columns.csc_row_idx.device_data() + feature_start;
//                        const int *bin_id_data = bin_id[0]->device_data() + feature_start;
//                        int fea_offset = cut.cut_row_ptr.host_data()[fid];
//                        int n_fea_bin = cut.cut_row_ptr.host_data()[fid + 1] - cut.cut_row_ptr.host_data()[fid];
//                        int shared_mem_size = sizeof(GHPair) * n_nodes_in_level * n_fea_bin;
//                        LOG(DEBUG)<<"smem size = " << shared_mem_size / 1024.0 << "KB";
//                        auto hist_data = hist.device_data();
//                        hist_kernel << < 2 * 56, 256, shared_mem_size >> >
//                                                      (hist_data, fea_offset, bin_id_data, feature_len, n_fea_bin, n_bins, iid_data,
//                                                              stats.gh_pair.device_data(), stats.nid.device_data(), nid_offset, n_nodes_in_level);
//                        CUDA_CHECK(hipGetLastError());
//                    }
//                }
//                for (int i = 0; i < hist.size(); ++i) {
//                    CHECK_EQ(hist.host_data()[i].g, hist2.host_data()[i].g);
//                    CHECK_EQ(hist.host_data()[i].h, hist2.host_data()[i].h);
//                }
                LOG(DEBUG) << "hist new = " << hist;
                //calculate missing value for each partition
                int temp = reduce_by_key(cuda::par, hist_fid, hist_fid + n_split, hist.device_data(),
                                         make_discard_iterator(), missing_gh.device_data()).second -
                           missing_gh.device_data();
//                LOG(INFO)<<temp;
                CHECK_EQ(temp, n_partition);
                LOG(DEBUG) << missing_gh;
                auto nodes_data = tree.nodes.device_data();
                auto missing_gh_data = missing_gh.device_data();
                device_loop(n_partition, [=]__device__(int pid) {
                    int nid0 = pid / n_column;
                    int nid = nid0 + nid_offset;
                    missing_gh_data[pid] = nodes_data[nid].sum_gh_pair - missing_gh_data[pid];
                });
                LOG(DEBUG) << missing_gh;
                inclusive_scan_by_key(cuda::par, hist_fid, hist_fid + n_split,
                                      hist.device_data(), hist.device_data());
                LOG(DEBUG) << hist;
            }
        }
        //calculate gain of each split
        SyncArray<float_type> gain(n_max_splits);
        {
            TIMED_SCOPE(timerObj, "calculate gain");
            auto compute_gain = []__device__(GHPair father, GHPair lch, GHPair rch, float_type min_child_weight,
                                             float_type lambda) -> float_type {
                if (lch.h >= min_child_weight && rch.h >= min_child_weight)
                    return (lch.g * lch.g) / (lch.h + lambda) + (rch.g * rch.g) / (rch.h + lambda) -
                           (father.g * father.g) / (father.h + lambda);
                else
                    return 0;
            };

            const Tree::TreeNode *nodes_data = tree.nodes.device_data();
            GHPair *gh_prefix_sum_data = hist.device_data();
            float_type *gain_data = gain.device_data();
            const auto missing_gh_data = missing_gh.device_data();
            //for lambda expression
            float_type mcw = min_child_weight;
            float_type l = lambda;
            device_loop(n_split, [=]__device__(int i) {
                int nid0 = i / n_bins;
                int nid = nid0 + nid_offset;
                if (nodes_data[nid].is_valid) {
                    int pid = nid0 * n_bins + hist_fid[i];
                    GHPair father_gh = nodes_data[nid].sum_gh_pair;
                    GHPair p_missing_gh = missing_gh_data[pid];
                    GHPair rch_gh = gh_prefix_sum_data[i];
                    float_type max_gain = max(0., compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l));
                    if (p_missing_gh.h > 1) {
                        rch_gh = rch_gh + p_missing_gh;
                        float_type temp_gain = compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l);
                        if (temp_gain > 0 && temp_gain - max_gain > 0.1) {//FIXME 0.1?
                            max_gain = -temp_gain;//negative means default split to right
                        }
                    }
                    gain_data[i] = max_gain;
                } else gain_data[i] = 0;
            });
            LOG(DEBUG) << "gain = " << gain;
        }

        SyncArray<int_float> best_idx_gain(n_nodes_in_level);
        {
            TIMED_SCOPE(timerObj, "get best gain");
            auto arg_abs_max = []__device__(const int_float &a, const int_float &b) {
                if (fabsf(get<1>(a)) == fabsf(get<1>(b)))
                    return get<0>(a) < get<0>(b) ? a : b;
                else
                    return fabsf(get<1>(a)) > fabsf(get<1>(b)) ? a : b;
            };

            auto nid_iterator = make_transform_iterator(counting_iterator<int>(0), placeholders::_1 / n_bins);

            reduce_by_key(
                    cuda::par,
                    nid_iterator, nid_iterator + n_split,
                    make_zip_iterator(make_tuple(counting_iterator<int>(0), gain.device_data())),
                    make_discard_iterator(),
                    best_idx_gain.device_data(),
                    thrust::equal_to<int>(),
                    arg_abs_max
            );
            LOG(DEBUG) << n_split;
            LOG(DEBUG) << "best idx & gain = " << best_idx_gain;
        }

        //get split points
        {
            const int_float *best_idx_gain_data = best_idx_gain.device_data();
            auto hist_data = hist.device_data();
            const auto missing_gh_data = missing_gh.device_data();
            auto cut_val_data = cut.cut_points_val.device_data();

            sp.resize(n_nodes_in_level);
            auto sp_data = sp.device_data();
            auto nodes_data = tree.nodes.device_data();

            int column_offset = columns.column_offset;
            LOG(DEBUG) << cut.cut_points_val;
            device_loop(n_nodes_in_level, [=]__device__(int i) {
                int_float bst = best_idx_gain_data[i];
                float_type best_split_gain = get<1>(bst);
                int split_index = get<0>(bst);
                if (!nodes_data[i + nid_offset].is_valid) {
                    sp_data[i].split_fea_id = -1;
                    sp_data[i].nid = -1;
                    return;
                }
                sp_data[i].split_fea_id = hist_fid[split_index] + column_offset;
                sp_data[i].nid = i + nid_offset;
                sp_data[i].gain = fabsf(best_split_gain);
                sp_data[i].fval = cut_val_data[split_index % n_bins];
                sp_data[i].fea_missing_gh = missing_gh_data[i * n_column + hist_fid[split_index]];
                sp_data[i].default_right = best_split_gain < 0;
                sp_data[i].rch_sum_gh = hist_data[split_index];
            });
        }
    }

    LOG(DEBUG) << "split points (gain/fea_id/nid): " << sp;
}

void HistUpdater::grow(Tree &tree, const vector<std::shared_ptr<SparseColumns>> &v_columns, InsStat &stats) {
    TIMED_SCOPE(timerObj, "grow tree");

    int n_instances = stats.n_instances;
    int cur_device = 0;

    LOG(TRACE) << "broadcast tree and stats";
    v_stats.resize(n_devices);
    v_trees.resize(n_devices);
    init_tree(tree, stats);
    DO_ON_MULTI_DEVICES(n_devices, [&](int device_id) {
        //copy stats and tree from host (stats, tree) to multi-device (v_stats, v_trees)

        //stats
        int n_instances = stats.n_instances;
        v_stats[device_id].reset(new InsStat());
        InsStat &gpu_stats = *v_stats[device_id];
        gpu_stats.resize(n_instances);
        gpu_stats.gh_pair.copy_from(stats.gh_pair.host_data(), n_instances);
        gpu_stats.nid.copy_from(stats.nid.host_data(), n_instances);
        //        gpu_stats.y.copy_from(stats.y.host_data(), n_instances);
//        gpu_stats.y_predict.copy_from(stats.y_predict.host_data(), n_instances);

        //tree
        v_trees[device_id].reset(new Tree());
        Tree &gpu_tree = *v_trees[device_id];
        gpu_tree.nodes.resize(tree.nodes.size());
        gpu_tree.nodes.copy_from(tree.nodes.host_data(), tree.nodes.size());
    });

    for (int i = 0; i < depth; ++i) {
        LOG(TRACE) << "growing tree at depth " << i;
        vector<SyncArray<SplitPoint>> local_sp(n_devices);
        {
            TIMED_SCOPE(timerObj, "find split");
            DO_ON_MULTI_DEVICES(n_devices, [&](int device_id) {
                LOG(TRACE) << string_format("finding split on device %d", device_id);
                find_split(i, *v_columns[device_id], *v_trees[device_id], *v_stats[device_id], v_cut[device_id],
                           local_sp[device_id]);
            });
        }

        int n_max_nodes_in_level = 1 << i;//2^i
        int nid_offset = (1 << i) - 1;//2^i - 1
        SyncArray<SplitPoint> global_sp(n_max_nodes_in_level);
        {
            TIMED_SCOPE(timerObj, "split point all reduce");
            if (n_devices > 1)
                split_point_all_reduce(local_sp, global_sp, i);
            else
                global_sp.copy_from(local_sp[0].device_data(), local_sp[0].size());
//            if (n_executor > 1) {
//                if (rank == 0) {
//                    SyncArray<SplitPoint> global_sp2(n_max_nodes_in_level);
//                    MPI_Recv(global_sp2.host_data(), global_sp2.mem_size(), MPI_CHAR, 1, 0, MPI_COMM_WORLD,
//                             MPI_STATUS_IGNORE);
//                    auto global_sp_data = global_sp.host_data();
//                    auto global_sp2_data = global_sp2.host_data();
//                    for (int j = 0; j < global_sp.size(); ++j) {
//                        if (global_sp2_data[j].gain > global_sp_data[j].gain)
//                            global_sp_data[j] = global_sp2_data[j];
//                    }
//                } else if (rank == 1) {
//                    MPI_Send(global_sp.host_data(), global_sp.mem_size(), MPI_CHAR, 0, 0, MPI_COMM_WORLD);
//                }
//                if (rank == 0) {
//                    MPI_Send(global_sp.host_data(), global_sp.mem_size(), MPI_CHAR, 1, 0, MPI_COMM_WORLD);
//                } else {
//                    MPI_Recv(global_sp.host_data(), global_sp.mem_size(), MPI_CHAR, 0, 0, MPI_COMM_WORLD,
//                             MPI_STATUS_IGNORE);
//                }
//            }
        }
//        LOG(DEBUG) << "rank " << rank << " sp" << global_sp;

        //do split
        {
            TIMED_SCOPE(timerObj, "update tree");
            update_tree(*v_trees[0], global_sp);
        }

        //broadcast tree
        if (n_devices > 1) {
            LOG(TRACE) << "broadcasting updated tree";
            //copy tree on gpu 0 to host, prepare to broadcast
            v_trees[0]->nodes.to_host();
            DO_ON_MULTI_DEVICES(n_devices, [&](int device_id) {
                v_trees[device_id]->nodes.copy_from(v_trees[0]->nodes.host_data(), v_trees[0]->nodes.size());
            });
        }

        {
            vector<bool> v_has_split(n_devices);
            LOG(TRACE) << "reset ins2node id";
            DO_ON_MULTI_DEVICES(n_devices, [&](int device_id) {
                v_has_split[device_id] = reset_ins2node_id(*v_stats[device_id], *v_trees[device_id],
                                                           *v_columns[device_id]);
            });

            LOG(TRACE) << "gathering ins2node id";
            //get final result of the reset instance id to node id
//            if (n_executor == 1) {
            bool has_split = false;
            for (int d = 0; d < n_devices; d++) {
                has_split |= v_has_split[d];
            }
            if (!has_split) {
                LOG(INFO) << "no splittable nodes, stop";
                break;
            }
//            } else {
//                todo early stop
//            }
        }

        //get global ins2node id
        {
            TIMED_SCOPE(timerObj, "global ins2node id");
            SyncArray<int> local_ins2node_id(n_instances);
            auto local_ins2node_id_data = local_ins2node_id.device_data();
            auto global_ins2node_id_data = v_stats[0]->nid.device_data();
            for (int d = 1; d < n_devices; d++) {
                CUDA_CHECK(hipMemcpyPeerAsync(local_ins2node_id_data, cur_device,
                                               v_stats[d]->nid.device_data(), d,
                                               sizeof(int) * n_instances));
                hipDeviceSynchronize();
                device_loop(n_instances, [=]__device__(int i) {
                    global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
                                                 global_ins2node_id_data[i] : local_ins2node_id_data[i];
                });
            }
//            if (n_executor > 1) {
//                if (rank == 0) {
//                    MPI_Recv(local_ins2node_id.host_data(), local_ins2node_id.mem_size(), MPI_CHAR, 1, 0,
//                             MPI_COMM_WORLD,
//                             MPI_STATUS_IGNORE);
//                    auto local_ins2node_id_data = local_ins2node_id.device_data();
//                    auto global_ins2node_id_data = stats.nid.device_data();
//                    device_loop(n_instances, [=]__device__(int i) {
//                        global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
//                                                     global_ins2node_id_data[i] : local_ins2node_id_data[i];
//                    });
//                } else {
//                    MPI_Send(stats.nid.host_data(), stats.nid.mem_size(), MPI_CHAR, 0, 0, MPI_COMM_WORLD);
//                }
//                if (rank == 0) {
//                    MPI_Send(stats.nid.host_data(), stats.nid.mem_size(), MPI_CHAR, 1, 0, MPI_COMM_WORLD);
//                } else {
//                    MPI_Recv(stats.nid.host_data(), stats.nid.mem_size(), MPI_CHAR, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
//                }
//            }
        }
        //processing missing value
        {
            TIMED_SCOPE(timerObj, "process missing value");
            LOG(TRACE) << "update ins2node id for each missing fval";
            auto global_ins2node_id_data = v_stats[0]->nid.device_data();//essential
            auto nodes_data = v_trees[0]->nodes.device_data();//already broadcast above
            device_loop(n_instances, [=]__device__(int iid) {
                int nid = global_ins2node_id_data[iid];
                //if the instance is not on leaf node and not goes down
                if (nodes_data[nid].splittable() && nid < nid_offset + n_max_nodes_in_level) {
                    //let the instance goes down
                    const Tree::TreeNode &node = nodes_data[nid];
                    if (node.default_right)
                        global_ins2node_id_data[iid] = node.rch_index;
                    else
                        global_ins2node_id_data[iid] = node.lch_index;
                }
            });
            LOG(DEBUG) << "new nid = " << stats.nid;
            //broadcast ins2node id
            v_stats[0]->nid.to_host();
            DO_ON_MULTI_DEVICES(n_devices, [&](int device_id) {
//                v_stats[device_id]->nid.copy_from(stats.nid.host_data(), stats.nid.size());
                v_stats[device_id]->nid.copy_from(v_stats[0]->nid.host_data(), stats.nid.size());
            });
        }
    }
    tree.nodes.copy_from(v_trees[0]->nodes);
    stats.nid.copy_from(v_stats[0]->nid);
}

void HistUpdater::init_dense_data(const SparseColumns &columns, int n_instances) {
    LOG(TRACE) << "init dense data";
    int n_column = columns.n_column;
    int nnz = columns.nnz;
    int n_block = std::min((nnz / n_column - 1) / 256 + 1, 32 * 56);
    dense_bin_id.resize(n_instances * n_column);
    auto bin_id_data = bin_id[0]->device_data();
    auto csc_row_idx_data = columns.csc_row_idx.device_data();
    auto dense_bin_id_data = dense_bin_id.device_data();
    auto max_num_bin = this->max_num_bin;
    device_loop(n_instances * n_column, [=]__device__(int i) {
        dense_bin_id_data[i] = max_num_bin;
    });
    device_loop_2d(n_column, columns.csc_col_ptr.device_data(), [=]__device__(int fid, int i) {
        int row = csc_row_idx_data[i];
        unsigned char bid = bin_id_data[i];
        dense_bin_id_data[row * n_column + fid] = bid;
    }, n_block);
}
