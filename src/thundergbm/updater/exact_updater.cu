#include "hip/hip_runtime.h"
//
// Created by shijiashuai on 5/7/18.
//
#include <thundergbm/updater/exact_updater.h>

#include "thundergbm/updater/exact_updater.h"
#include "thundergbm/util/cub_wrapper.h"


void ExactUpdater::grow(Tree &tree) {
    for_each_shard([&](Shard &shard) {
        shard.tree.init(shard.stats, param);
    });
    for (int level = 0; level < param.depth; ++level) {
        for_each_shard([&](Shard &shard) {
            shard.find_split(level);
        });
        split_point_all_reduce(level);
        for_each_shard([&](Shard &shard) {
            shard.update_tree();
            shard.reset_ins2node_id();
        });
        {
            LOG(TRACE) << "gathering ins2node id";
            //get final result of the reset instance id to node id
            bool has_split = false;
            for (int d = 0; d < param.n_device; d++) {
                has_split |= shards[d]->has_split;
            }
            if (!has_split) {
                LOG(INFO) << "no splittable nodes, stop";
                break;
            }
        }
        ins2node_id_all_reduce(level);
    }



    for_each_shard([&](Shard &shard) {
        shard.tree.prune_self(param.gamma);
        shard.predict_in_training();
        shard.stats.updateGH();
    });
    tree.nodes.resize(shards.front()->tree.nodes.size());
    tree.nodes.copy_from(shards.front()->tree.nodes);
}

void ExactUpdater::init(const DataSet &dataset) {
    shards.resize(param.n_device);
    for (int i = 0; i < param.n_device; ++i) {
        shards[i].reset(new Shard());
    }
    SparseColumns columns;
    columns.from_dataset(dataset);
    //todo refactor v_columns
    vector<SparseColumns *> v_columns(param.n_device);
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i] = &shards[i]->columns;
    }
    for_each_shard([&](Shard &shard) {
        shard.param = param;
        int n_instances = dataset.n_instances();
        shard.stats.resize(n_instances);
        shard.stats.y.copy_from(dataset.y.data(), n_instances);
        shard.stats.updateGH();
    });
    columns.to_multi_devices(v_columns);

}

void ExactUpdater::split_point_all_reduce(int depth) {
    //get global best split of each node
    int n_nodes_in_level = 1 << depth;//2^i
    int nid_offset = (1 << depth) - 1;//2^i - 1
    auto global_sp_data = shards.front()->sp.host_data();
    vector<bool> active_sp(n_nodes_in_level);

    for (int device_id = 0; device_id < param.n_device; device_id++) {
        auto local_sp_data = shards[device_id]->sp.host_data();
        for (int j = 0; j < shards[device_id]->sp.size(); j++) {
            int sp_nid = local_sp_data[j].nid;
            if (sp_nid == -1) continue;
            int global_pos = sp_nid - nid_offset;
            if (!active_sp[global_pos])
                global_sp_data[global_pos] = local_sp_data[j];
            else
                global_sp_data[global_pos] = (global_sp_data[global_pos].gain >= local_sp_data[j].gain)
                                             ?
                                             global_sp_data[global_pos] : local_sp_data[j];
            active_sp[global_pos] = true;
        }
    }
    //set inactive sp
    for (int n = 0; n < n_nodes_in_level; n++) {
        if (!active_sp[n])
            global_sp_data[n].nid = -1;
    }
    for_each_shard([&](Shard &shard) {
        shard.sp.copy_from(shards.front()->sp);
    });
    LOG(DEBUG) << "global best split point = " << shards.front()->sp;
}

void ExactUpdater::ins2node_id_all_reduce(int depth) {
    //get global ins2node id
    {
        SyncArray<int> local_ins2node_id(shards.front()->stats.n_instances);
        auto local_ins2node_id_data = local_ins2node_id.device_data();
        auto global_ins2node_id_data = shards.front()->stats.nid.device_data();
        for (int d = 1; d < param.n_device; d++) {
            local_ins2node_id.copy_from(shards[d]->stats.nid);
            device_loop(shards.front()->stats.n_instances, [=]__device__(int i) {
                global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
                                             global_ins2node_id_data[i] : local_ins2node_id_data[i];
            });
        }
    }
    for_each_shard([&](Shard &shard) {
        shard.stats.nid.copy_from(shards.front()->stats.nid);
    });
    //processing missing value
    {
        int n_nodes_in_level = 1 << depth;//2^i
        int nid_offset = (1 << depth) - 1;//2^i - 1
        TIMED_SCOPE(timerObj, "process missing value");
        LOG(TRACE) << "update ins2node id for each missing fval";
        auto global_ins2node_id_data = shards.front()->stats.nid.device_data();//essential
        auto nodes_data = shards.front()->tree.nodes.device_data();//already broadcast above
        device_loop(shards.front()->stats.n_instances, [=]__device__(int iid) {
            int nid = global_ins2node_id_data[iid];
            //if the instance is not on leaf node and not goes down
            if (nodes_data[nid].splittable() && nid < nid_offset + n_nodes_in_level) {
                //let the instance goes down
                const Tree::TreeNode &node = nodes_data[nid];
                if (node.default_right)
                    global_ins2node_id_data[iid] = node.rch_index;
                else
                    global_ins2node_id_data[iid] = node.lch_index;
            }
        });
        LOG(DEBUG) << "new nid = " << shards.front()->stats.nid;
        //broadcast ins2node id
    }
}

std::ostream &operator<<(std::ostream &os, const int_float &rhs) {
    os << string_format("%d/%f", thrust::get<0>(rhs), thrust::get<1>(rhs));
    return os;
}

void ExactUpdater::Shard::predict_in_training() {
    auto y_predict_data = stats.y_predict.device_data();
    auto nid_data = stats.nid.device_data();
    const Tree::TreeNode *nodes_data = tree.nodes.device_data();
    device_loop(stats.n_instances, [=]__device__(int i) {
        int nid = nid_data[i];
        while (nid != -1 && (nodes_data[nid].is_pruned)) nid = nodes_data[nid].parent_index;
        y_predict_data[i] += nodes_data[nid].base_weight;
    });
}

void ExactUpdater::Shard::find_split(int level) {
    int n_max_nodes_in_level = static_cast<int>(pow(2, level));
    int nid_offset = static_cast<int>(pow(2, level) - 1);
    int n_column = columns.n_column;
    int n_partition = n_column * n_max_nodes_in_level;
    int nnz = columns.nnz;
    int n_block = std::min((nnz / n_column - 1) / 256 + 1, 32 * 56);

    LOG(TRACE) << "start finding split";

    //find the best split locally
    {
        using namespace thrust;

        //calculate split information for each split
        int n_split;
        SyncArray<GHPair> gh_prefix_sum(nnz);
        SyncArray<GHPair> missing_gh(n_partition);
        SyncArray<int_float> rle_key(nnz);
        if (nnz * 4 > 1.5 * (1 << 30)) rle_key.resize(int(nnz * 0.1));
        auto rle_pid_data = make_transform_iterator(rle_key.device_data(),
                                                    [=]__device__(int_float key) { return get<0>(key); });
        auto rle_fval_data = make_transform_iterator(rle_key.device_data(),
                                                     [=]__device__(int_float key) { return get<1>(key); });
        {
            SyncArray<int> fvid2pid(nnz);
            {
                TIMED_SCOPE(timerObj, "fvid2pid");
                //input
                auto *nid_data = stats.nid.device_data();
                const int *iid_data = columns.csc_row_idx.device_data();

                LOG(TRACE) << "after using v_stats and columns";
                //output
                int *fvid2pid_data = fvid2pid.device_data();
                device_loop_2d(
                        n_column, columns.csc_col_ptr.device_data(),
                        [=]__device__(int col_id, int fvid) {
                            //feature value id -> instance id -> node id
                            int nid = nid_data[iid_data[fvid]];
                            int pid;
                            //if this node is leaf node, move it to the end
                            if (nid < nid_offset) pid = INT_MAX;//todo negative
                            else pid = col_id * n_max_nodes_in_level + nid - nid_offset;
                            fvid2pid_data[fvid] = pid;
                        },
                        n_block);
                hipDeviceSynchronize();
                LOG(DEBUG) << "fvid2pid " << fvid2pid;
            }

            //gather g/h pairs and do prefix sum
            {
                //get feature value id mapping for partition, new -> old
                SyncArray<int> fvid_new2old(nnz);
                {
                    TIMED_SCOPE(timerObj, "fvid_new2old");
                    sequence(cuda::par, fvid_new2old.device_data(), fvid_new2old.device_end(), 0);

                    //using prefix sum memory for temporary storage
                    cub_sort_by_key(fvid2pid, fvid_new2old, -1, true, (void *) gh_prefix_sum.device_data());
                    LOG(DEBUG) << "sorted fvid2pid " << fvid2pid;
                    LOG(DEBUG) << "fvid_new2old " << fvid_new2old;
                    hipDeviceSynchronize();
                }

                //do prefix sum
                {
                    TIMED_SCOPE(timerObj, "do prefix sum");
                    //same feature value in the same part has the same key.
                    auto key_iter = make_zip_iterator(
                            make_tuple(
                                    fvid2pid.device_data(),
                                    make_permutation_iterator(
                                            columns.csc_val.device_data(),
                                            fvid_new2old.device_data())));//use fvid_new2old to access csc_val
                    n_split = reduce_by_key(
                            cuda::par,
                            key_iter, key_iter + nnz,
                            make_permutation_iterator(                   //ins id -> gh pair
                                    stats.gh_pair.device_data(),
                                    make_permutation_iterator(                 //old fvid -> ins id
                                            columns.csc_row_idx.device_data(),
                                            fvid_new2old.device_data())),             //new fvid -> old fvid
                            rle_key.device_data(),
                            gh_prefix_sum.device_data()
                    ).first - rle_key.device_data();
                    CHECK_LE(n_split, rle_key.size());
                    LOG(INFO) << "RLE ratio = " << (float) n_split / nnz;

                    //prefix sum
                    inclusive_scan_by_key(
                            cuda::par,
                            rle_pid_data, rle_pid_data + n_split,
                            gh_prefix_sum.device_data(),
                            gh_prefix_sum.device_data());
                    LOG(DEBUG) << "gh prefix sum = " << gh_prefix_sum;
                    hipDeviceSynchronize();
                }
            }
        }

        //calculate missing value for each partition
        {
            TIMED_SCOPE(timerObj, "calculate missing value");
            SyncArray<int> pid_ptr(n_partition + 1);
            counting_iterator<int> search_begin(0);
            upper_bound(cuda::par, rle_pid_data, rle_pid_data + n_split, search_begin,
                        search_begin + n_partition, pid_ptr.device_data() + 1);
            LOG(DEBUG) << "pid_ptr = " << pid_ptr;

            auto pid_ptr_data = pid_ptr.device_data();
            auto rle_key_data = rle_key.device_data();
            float_type rt_eps = param.rt_eps;
            device_loop(n_split, [=]__device__(int i) {
                int pid = rle_pid_data[i];
                if (pid == INT_MAX) return;
                float_type f = rle_fval_data[i];
                if ((pid_ptr_data[pid + 1] - 1) == i)//the last RLE
                    //using "get" to get a modifiable lvalue
                    get<1>(rle_key_data[i]) = (f - fabsf(rle_fval_data[pid_ptr_data[pid]]) - rt_eps);
                else
                    //FIXME read/write collision
                    get<1>(rle_key_data[i]) = (f + rle_fval_data[i + 1]) * 0.5f;
            });

            const auto gh_prefix_sum_data = gh_prefix_sum.device_data();
            const auto node_data = tree.nodes.device_data();
            auto missing_gh_data = missing_gh.device_data();
            device_loop(n_partition, [=]__device__(int pid) {
                int nid = pid % n_max_nodes_in_level + nid_offset;
                if (pid_ptr_data[pid + 1] != pid_ptr_data[pid])
                    missing_gh_data[pid] =
                            node_data[nid].sum_gh_pair - gh_prefix_sum_data[pid_ptr_data[pid + 1] - 1];
            });
            LOG(DEBUG) << "missing gh = " << missing_gh;
            hipDeviceSynchronize();
        }

        //calculate gain of each split
        SyncArray<float_type> gain(nnz);
        {
            TIMED_SCOPE(timerObj, "calculate gain");
            auto compute_gain = []__device__(GHPair father, GHPair lch, GHPair rch, float_type min_child_weight,
                                             float_type lambda) -> float_type {
                if (lch.h >= min_child_weight && rch.h >= min_child_weight)
                    return (lch.g * lch.g) / (lch.h + lambda) + (rch.g * rch.g) / (rch.h + lambda) -
                           (father.g * father.g) / (father.h + lambda);
                else
                    return 0;
            };
            const Tree::TreeNode *nodes_data = tree.nodes.device_data();
            GHPair *gh_prefix_sum_data = gh_prefix_sum.device_data();
            float_type *gain_data = gain.device_data();
            const auto missing_gh_data = missing_gh.device_data();
            //for lambda expression
            float_type mcw = param.min_child_weight;
            float_type l = param.lambda;
            device_loop(n_split, [=]__device__(int i) {
                int pid = rle_pid_data[i];
                int nid0 = pid % n_max_nodes_in_level;
                int nid = nid0 + nid_offset;
                if (pid == INT_MAX) return;
                GHPair father_gh = nodes_data[nid].sum_gh_pair;
                GHPair p_missing_gh = missing_gh_data[pid];
                GHPair rch_gh = gh_prefix_sum_data[i];
                float_type max_gain = max(0., compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l));
                if (p_missing_gh.h > 1) {
                    rch_gh = rch_gh + p_missing_gh;
                    float_type temp_gain = compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l);
                    if (temp_gain > 0 && temp_gain - max_gain > 0.1) {//FIXME 0.1?
                        max_gain = -temp_gain;//negative means default split to right
                    }
                }
                gain_data[i] = max_gain;
            });
            LOG(DEBUG) << "gain = " << gain;
            hipDeviceSynchronize();
        }

        //get best gain and the index of best gain for each feature and each node
        SyncArray<int_float> best_idx_gain(n_partition);
        int n_nodes_in_level;
        {
            TIMED_SCOPE(timerObj, "get best gain");
            auto arg_abs_max = []__device__(const int_float &a, const int_float &b) {
                if (fabsf(get<1>(a)) == fabsf(get<1>(b)))
                    return get<0>(a) < get<0>(b) ? a : b;
                else
                    return fabsf(get<1>(a)) > fabsf(get<1>(b)) ? a : b;
            };

            //reduce to get best split of each node for this feature
            SyncArray<int> feature_nodes_pid(n_partition);
            int n_feature_with_nodes = reduce_by_key(
                    cuda::par,
                    rle_pid_data, rle_pid_data + n_split,
                    make_zip_iterator(make_tuple(counting_iterator<int>(0), gain.device_data())),
                    feature_nodes_pid.device_data(),
                    best_idx_gain.device_data(),
                    thrust::equal_to<int>(),
                    arg_abs_max).second - best_idx_gain.device_data();

            LOG(DEBUG) << "aaa = " << n_feature_with_nodes;
            LOG(DEBUG) << "f n pid" << feature_nodes_pid;
            LOG(DEBUG) << "best idx & gain = " << best_idx_gain;

            auto feature_nodes_pid_data = feature_nodes_pid.device_data();
            device_loop(n_feature_with_nodes, [=]__device__(int i) {
                feature_nodes_pid_data[i] = feature_nodes_pid_data[i] % n_max_nodes_in_level;
            });
            LOG(DEBUG) << "f n pid" << feature_nodes_pid;
            cub_sort_by_key(feature_nodes_pid, best_idx_gain, n_feature_with_nodes);
            LOG(DEBUG) << "f n pid" << feature_nodes_pid;
            LOG(DEBUG) << "best idx & gain = " << best_idx_gain;
            n_nodes_in_level = reduce_by_key(
                    cuda::par,
                    feature_nodes_pid.device_data(), feature_nodes_pid.device_data() + n_feature_with_nodes,
                    best_idx_gain.device_data(),
                    make_discard_iterator(),
                    best_idx_gain.device_data(),
                    thrust::equal_to<int>(),
                    arg_abs_max
            ).second - best_idx_gain.device_data();
            LOG(DEBUG) << "#nodes in level = " << n_nodes_in_level;
            LOG(DEBUG) << "best idx & gain = " << best_idx_gain;
            hipDeviceSynchronize();
        }

        //get split points
        const int_float *best_idx_gain_data = best_idx_gain.device_data();
        GHPair *gh_prefix_sum_data = gh_prefix_sum.device_data();
        const auto missing_gh_data = missing_gh.device_data();

        sp.resize(n_max_nodes_in_level);
        auto sp_data = sp.device_data();

        int column_offset = columns.column_offset;
        device_loop(n_max_nodes_in_level, [=]__device__(int i){
           sp_data[i].nid = -1;
        });
        device_loop(n_nodes_in_level, [=]__device__(int i) {
            int_float bst = best_idx_gain_data[i];
            float_type best_split_gain = get<1>(bst);
            int split_index = get<0>(bst);
            int pid = rle_pid_data[split_index];
            if (pid != INT_MAX){
                int nid0 = pid % n_max_nodes_in_level;
                sp_data[nid0].nid = nid0 + nid_offset;
                sp_data[nid0].split_fea_id = pid / n_max_nodes_in_level + column_offset;
                sp_data[nid0].gain = fabsf(best_split_gain);
                sp_data[nid0].fval = rle_fval_data[split_index];
                sp_data[nid0].fea_missing_gh = missing_gh_data[pid];
                sp_data[nid0].default_right = best_split_gain < 0;
                sp_data[nid0].rch_sum_gh = gh_prefix_sum_data[split_index];
            }
        });
    }

    LOG(DEBUG) << "split points (gain/fea_id/nid): " << sp;
}

void ExactUpdater::Shard::update_tree() {
    auto sp_data = sp.device_data();
    LOG(DEBUG) << sp;
    int n_nodes_in_level = sp.size();

    Tree::TreeNode *nodes_data = tree.nodes.device_data();
    float_type rt_eps = param.rt_eps;
    float_type lambda = param.lambda;

    LOG(DEBUG) << n_nodes_in_level;
    device_loop(n_nodes_in_level, [=]__device__(int i) {
        float_type best_split_gain = sp_data[i].gain;
        if (best_split_gain > rt_eps) {
            //do split
            if (sp_data[i].nid == -1) return;
            int nid = sp_data[i].nid;
            Tree::TreeNode &node = nodes_data[nid];
            node.gain = best_split_gain;

            Tree::TreeNode &lch = nodes_data[node.lch_index];//left child
            Tree::TreeNode &rch = nodes_data[node.rch_index];//right child
            lch.is_valid = true;
            rch.is_valid = true;
            node.split_feature_id = sp_data[i].split_fea_id;
            GHPair p_missing_gh = sp_data[i].fea_missing_gh;
            //todo process begin
            node.split_value = sp_data[i].fval;
            node.split_bid = sp_data[i].split_bid;
            rch.sum_gh_pair = sp_data[i].rch_sum_gh;
            if (sp_data[i].default_right) {
                rch.sum_gh_pair = rch.sum_gh_pair + p_missing_gh;
                node.default_right = true;
            }
            lch.sum_gh_pair = node.sum_gh_pair - rch.sum_gh_pair;
            lch.calc_weight(lambda);
            rch.calc_weight(lambda);
        } else {
            //set leaf
            if (sp_data[i].nid == -1) return;
            int nid = sp_data[i].nid;
            Tree::TreeNode &node = nodes_data[nid];
            node.is_leaf = true;
            nodes_data[node.lch_index].is_valid = false;
            nodes_data[node.rch_index].is_valid = false;
        }
//    }
    });
    LOG(DEBUG) << tree.nodes;
}

void ExactUpdater::Shard::reset_ins2node_id() {
    SyncArray<bool> has_splittable(1);
    //set new node id for each instance
    {
        TIMED_SCOPE(timerObj, "get new node id");
        auto nid_data = stats.nid.device_data();
        const int *iid_data = columns.csc_row_idx.device_data();
        const Tree::TreeNode *nodes_data = tree.nodes.device_data();
        const int *col_ptr_data = columns.csc_col_ptr.device_data();
        const float_type *f_val_data = columns.csc_val.device_data();
        has_splittable.host_data()[0] = false;
        bool *h_s_data = has_splittable.device_data();
        int column_offset = columns.column_offset;

        int n_column = columns.n_column;
        int nnz = columns.nnz;
        int n_block = std::min((nnz / n_column - 1) / 256 + 1, 32 * 56);

        LOG(TRACE) << "update ins2node id for each fval";
        device_loop_2d(n_column, col_ptr_data,
                       [=]__device__(int col_id, int fvid) {
                           //feature value id -> instance id
                           int iid = iid_data[fvid];
                           //instance id -> node id
                           int nid = nid_data[iid];
                           //node id -> node
                           const Tree::TreeNode &node = nodes_data[nid];
                           //if the node splits on this feature
                           if (node.splittable() && node.split_feature_id == col_id + column_offset) {
                               h_s_data[0] = true;
                               if (f_val_data[fvid] < node.split_value)
                                   //goes to left child
                                   nid_data[iid] = node.lch_index;
                               else
                                   //right child
                                   nid_data[iid] = node.rch_index;
                           }
                       }, n_block);

    }
    LOG(DEBUG) << "new tree_id = " << stats.nid;
    has_split = has_splittable.host_data()[0];
}
