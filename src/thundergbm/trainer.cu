#include "hip/hip_runtime.h"
//
// Created by zeyi on 1/9/19.
//
#include <fstream>
#include <thundergbm/tree.h>
#include <thundergbm/updater/exact_updater.h>
#include <thundergbm/updater/hist_updater.h>
#include <thundergbm/trainer.h>
#include <thundergbm/metric/metric.h>
#include "thundergbm/util/device_lambda.cuh"
#include "thrust/reduce.h"

float_type TreeTrainer::compute_rmse(const InsStat &stats) {
    TIMED_FUNC(timerObj);
    SyncArray<float_type> sq_err(stats.n_instances);
    auto sq_err_data = sq_err.device_data();
    const float_type *y_data = stats.y.device_data();
    const float_type *y_predict_data = stats.y_predict.device_data();
    device_loop(stats.n_instances, [=] __device__(int i) {
        float_type e = y_predict_data[i] - y_data[i];
        sq_err_data[i] = e * e;
    });
    float_type rmse =
            sqrt(thrust::reduce(thrust::cuda::par, sq_err.device_data(), sq_err.device_end()) / stats.n_instances);
    return rmse;
}

void TreeTrainer::save_trees(GBMParam &param, vector<Tree> &trees) {
    std::ofstream out(param.out_model_name);
    int round = 0;
    for (Tree &tree:trees) {
        string str_tree = string_format("booster[%d]:", round) + tree.dump(param.depth);
        //LOG(INFO) << "\n" << str_tree;
        out << str_tree;
        round++;
    }
    out.close();
}

float_type TreeTrainer::train(GBMParam &param) {
    dataSet.load_from_file(param.path, param);
    float_type rmse;
    if (param.tree_method.compare("exact") == 0)
        rmse = train_exact(param);
    else if (param.tree_method.compare("hist") == 0)
        rmse = train_hist(param);
    else {
        bool exact_sp_producer = false;
        if (dataSet.n_features() > 20000)//#TODO: use data set density ratio
            exact_sp_producer = true;
        if (exact_sp_producer == true)
            rmse = train_exact(param);
        else
            rmse = train_hist(param);
    }
    return rmse;
}

float_type TreeTrainer::train_exact(GBMParam &param) {
    LOG(INFO) << "using exact split to train the trees";
    int n_instances = dataSet.n_instances();
    vector<Tree> trees;
    trees.resize(param.n_trees);

    ExactUpdater updater(param);
    updater.init(dataSet);
    int round = 0;
    float_type rmse = 0;
    SyncMem::clear_cache();
    {
        TIMED_SCOPE(timerObj, "construct tree");
        for (Tree &tree:trees) {
            updater.grow(tree);
            //next round
            round++;
            rmse = compute_rmse(updater.shards.front()->stats);
            LOG(INFO) << "rmse = " << rmse;
        }
        save_trees(param, trees);
    }
    return rmse;
}

///// upgrading
//float_type TreeTrainer::train_exact(GBMParam &param) {
//    DataSet dataSet;
//    dataSet.load_from_file(param.path, param);
//    int n_instances = dataSet.n_instances();
//    vector<Tree> trees;
//    trees.resize(param.n_trees);
//
//    ExactUpdater updater(param);
//    updater.init(dataSet);
//    int round = 0;
//    float_type rmse = 0;
//    SyncMem::clear_cache();
//    {
//        TIMED_SCOPE(timerObj, "construct tree");
//        for (Tree &tree:trees) {
//            updater.grow(tree);
//            //next round
//            round++;
//            rmse = compute_rmse(updater.shards.front()->stats);
//            LOG(INFO) << "rmse = " << rmse;
//        }
//        save_trees(param, trees);
//    }
//    return rmse;
//}

float_type TreeTrainer::train_hist(GBMParam &param) {
    LOG(INFO) << "using histogram based approach to find split";
    SyncMem::clear_cache();

    vector<vector<Tree>> trees;
    vector<HistUpdater::ShardT> shards(param.n_device);

    //TODO refactor these
    SparseColumns columns;
    columns.from_dataset(dataSet);
    vector<std::unique_ptr<SparseColumns>> v_columns(param.n_device);
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].reset(&shards[i].columns);
    }
    columns.to_multi_devices(v_columns);

    HistUpdater updater(param);
    HistUpdater::for_each_shard(shards, [&](Shard &shard) {
        int n_instances = shard.columns.n_row;
        shard.stats.resize(n_instances);
        shard.stats.y.copy_from(dataSet.y.data(), n_instances);
        shard.stats.obj.reset(ObjectiveFunction::create(param.objective));
        shard.stats.obj->configure(param, dataSet);
        shard.param = param;
        shard.param.learning_rate /= param.n_parallel_trees;//average trees in one iteration
    });
    updater.init(shards);

    SyncMem::clear_cache();

    std::unique_ptr<Metric> metric;
    metric.reset(Metric::create(shards.front().stats.obj->default_metric()));
    metric->configure(param, dataSet);

    int round = 0;
    float_type score = 0;
    {
        TIMED_SCOPE(timerObj, "construct tree");
        int n_instances = shards.front().stats.n_instances;
        SyncArray<GHPair> all_gh_pair(n_instances * param.num_class);
        SyncArray<float_type> all_y(n_instances * param.num_class);
        for (int iter = 0; iter < param.n_trees; iter++) {
            //one boosting iteration

            trees.emplace_back();
            vector<Tree> &tree = trees.back();
            tree.resize(param.n_parallel_trees);
            if (param.num_class == 1) {
                //update gradient
                HistUpdater::for_each_shard(shards, [&](Shard &shard) {
                    shard.stats.update_gradient();
                    LOG(DEBUG) << "gh = " << shard.stats.gh_pair;
                    if (updater.param.bagging) {
                        shard.stats.gh_pair_backup.resize(shard.stats.n_instances);
                        shard.stats.gh_pair_backup.copy_from(shard.stats.gh_pair);
                    }
                });
                updater.grow(tree, shards);

                //next round
                round++;
                score = metric->get_score(shards.front().stats.y_predict);
            } else {
                shards.front().stats.obj->get_gradient(shards.front().stats.y, all_y, all_gh_pair);
                for (int i = 0; i < param.num_class; ++i) {
                    trees.emplace_back();
                    vector<Tree> &tree = trees.back();
                    tree.resize(param.n_parallel_trees);
                    HistUpdater::for_each_shard(shards, [&](Shard &shard) {
                        shard.stats.gh_pair.copy_from(all_gh_pair.device_data() + i * n_instances, n_instances);
                        shard.stats.y_predict.copy_from(all_y.device_data() + i * n_instances, n_instances);
                    });
                    updater.grow(tree, shards);
                    CUDA_CHECK(hipMemcpy(all_y.device_data() + i * n_instances,
                                          shards.front().stats.y_predict.device_data(),
                                          sizeof(float_type) * n_instances, hipMemcpyDefault));
                }
                score = metric->get_score(all_y);
            }
            LOG(INFO) << metric->get_name() << " = " << score;
        }
//        LOG(INFO) << trees.back().back().dump(param.depth);
    }
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].release();
    }
    return score;
}